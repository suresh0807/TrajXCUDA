#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void SDreduce(float *a, float *b, int SD_store, int nElt1)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  if(rowid < SD_store )
  {
    for(int i=0;i<nElt1;i++)
    {
      b[rowid]+=a[i+rowid*nElt1];
    }
    b[rowid] = b[rowid]/float(nElt1);
  }
}

__global__ void SDreduce(float *a, float *b, int SD_store, int nElt1,int fairy)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  if(rowid < SD_store )
  {
    for(int i=0;i<nElt1;i++)
    {
      b[rowid]+=a[i+rowid*nElt1];
    }
  }
}

