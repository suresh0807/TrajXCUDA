#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"


void read_inputs(void)
{
 int lnElt1,lnElt2;
 nstruct=0;
 lnElt2=0;
 lnElt1=0;
 
// Read the number of structures and atoms per structure in the trajectory
 //command = "grep -c -E 'Atoms|generated' input.xyz > nstruct";
 string cmd("grep -c -E 'Atoms|generated' ");
 cmd += file_path;
 cmd += " > nstruct";
 system (cmd.c_str());
 ifstream nstructure;
 nstructure.open("nstruct");
 nstructure >> nstruct;
 nstructure.close();
 system("rm nstruct");
 ifstream traj;
 if(max_frames!=0){nstruct=max_frames;}
 nstruct=nstruct-start_frame;
 //if(unwrap == "yes") {traj.open("input.xyz_unwrapped",ios::in);}
 //else {traj.open("input.xyz",ios::in);}
 cout <<"using file in "<< file_path<<endl;
 traj.open(file_path.c_str());
//count number of atoms in each frame
 traj>>natoms;
 traj.seekg(0);
 cout << natoms <<" atoms found in "<< nstruct <<" frames"<<endl; 

//Count the number of atoms per chosen element from the trajectory's first frame
 float *maxmin;
 maxmin= (float *) malloc (sizeof(float)*natoms*3);
 count_metal=0;
  for(int i = 0; i< 1; i++)
  {
    do
    traj.get(c);
    while ( c != '\n');
    do
    traj.get(c);
    while ( c != '\n');
    cout<<"counting atoms of chosen element"<<endl;
    for(int j=0; j<natoms; j++)
    {
      traj >> felt >> maxmin[j*3+i*natoms*3] >> maxmin[j*3+i*natoms*3+1]
           >> maxmin[j*3+i*natoms*3+2];
      if(felt == Elt1)
      {
        lnElt1+=1;
      }
      else if(felt == Elt2)
      {
        lnElt2+=1;
      }
      else if(felt == rdf_metal_exclude)
      {
	count_metal+=1;
      }
      else if (Elt1 == "water")
      {
	if(felt =="O" || felt =="H")
	{
	  lnElt1+=1;
	}
      }
      if(felt == metal_species)
      {
	count_metal+=1;
      }
      do
      traj.get(c);
      while ( c != '\n');
    }
  }

  traj.seekg(0);
  if(Elt1==Elt2)
  {
    lnElt2=lnElt1;
    cout <<Elt1<<" = "<<lnElt1<<endl;
  }
  else
  {
    cout <<Elt1 <<" = "<< lnElt1<<" ; "<<Elt2<<" = "<<lnElt2<<endl;    
  }
  if(Elt1=="All" && Elt2=="All") {lnElt1=natoms; lnElt2=natoms;}
  if(type=="rdf") count_metal/=2;
//Now to map the volume spanned by the selected atom species,
//the minimum and maximum values of x,y,z coordinates are
//computed and stored - useful for the density/velocity plots.
int stride=0;
    minx = minimum(maxmin,natoms,stride);
    miny = minimum(maxmin,natoms,stride+1);
    minz = minimum(maxmin,natoms,stride+2);
    maxx = maximum(maxmin,natoms,stride);
    maxy = maximum(maxmin,natoms,stride+1);
    maxz = maximum(maxmin,natoms,stride+2);
    free(maxmin);

    
    
//Allocate matrices for the element coordinates as well as the 
//distance and histogram components
//Matrix A contains positions of species 1 and in case of RDF, B matrix
//holds the positions of the pair element.
//In case of same species distribution function, A and B are identical.

  if(average=="yes")
  {
   Aavg= (float *) malloc (sizeof(float)*lnElt1*3);
   for(int i=0; i<lnElt1*3;i++)
   {
    Aavg[i]=0.0;
   }
  }

      //experimental ///mainly for the diffusion density of int/bulk water
if(scope =="bulk-interface")  
  {
    float *first_frame, *metal_frame;
    first_frame=(float*) malloc(sizeof(float)*natoms*3);
    metal_frame=(float*) malloc(sizeof(float)*count_metal*3);
    int *species_order;
    species_order=(int *) malloc (sizeof(int)*natoms);
    for(int i=0;i<natoms;i++)
    {
      species_order[i]=0;
    }
    //Read coordinates of respective elements into the matrices
printf("Reading the positions of first frame in to data structures\n");
float Dum1,Dum2,Dum3;
//int *globalindexA, *globalindexB, *globalindexM;
//globalindexA = (int *) malloc (sizeof(int) * lnElt1);
//globalindexB = (int *) malloc (sizeof(int) * lnElt2);
//globalindexm = (int *) malloc (sizeof(int) * count_metal);
 for(int i=0;i<(start_frame*(natoms+2));i++)
 {
    getline(traj,Dummy);
 }
 for(int i=0; i< 1;i=i+1)
  {
    //int counta=0, countb=0, 
    int countc=0;
    getline(traj,Dummy);
    getline(traj,Dummy);
    for(int j=0; j< natoms; j++)
    {
      traj >> felt >> Dum1 >> Dum2 >> Dum3;
      first_frame[j*3]=Dum1;
      first_frame[j*3+1]=Dum2;
      first_frame[j*3+2]=Dum3;
      if(felt==Elt1){species_order[j]=1;}
      else if(felt==Elt2){species_order[j]=2;}
      /*if(felt==Elt1)
      {
        A[counta*3+i*lnElt1*3]= Dum1;
        A[counta*3+i*lnElt1*3+1]= Dum2;
        A[counta*3+i*lnElt1*3+2]= Dum3;
	globalindexA[counta]=j;
	counta=counta+1;
      }
      else if(felt==Elt2)
      {
        B[countb*3+i*lnElt2*3] = Dum1;
        B[countb*3+i*lnElt2*3+1] = Dum2;
        B[countb*3+i*lnElt2*3+2] = Dum3;
	globalindexB[countb]=j;
        countb=countb+1;
      }*/
      if(felt==metal_species)
      {
        metal_frame[countc*3] = Dum1;
        metal_frame[countc*3+1] = Dum2;
        metal_frame[countc*3+2] = Dum3;
	//globalindexM[countc]=j;
        countc=countc+1;
      }
    }
    getline(traj,Dummy);
    for (int m = 0 ; m < skip-1; m++)
    {
      for(int j=0;j<natoms;j++)
      {
	getline(traj,Dummy);
      }
      getline(traj,Dummy);
      getline(traj,Dummy);
    }
  }
  traj.seekg(0);
 // read the lattice information from the last line of the trajectory
 string cmd("tail -n 1 ");
 cmd += file_path;
 cmd += " > cellsize";
 system (cmd.c_str());
 ifstream cellsize;
 cellsize.open("cellsize",ios::in);
 yxvec =0; 
 if(cell_type == "orthorhombic") cellsize>> xvec >>yvec>> zvec;
 else if(cell_type == "monoclinic") cellsize >>xvec>>yvec>>zvec>>yxvec;
 cellsize.close();
 system("rm cellsize");
 cout <<"File read complete"<<endl;
 cout <<"Lattice length (X Y Z) = "<< xvec << " "<< yvec << " "<< zvec << " " << yxvec <<endl;
 
    
    nA_int=0;
    nB_int=0;
    nA_bulk=0;
    nB_bulk=0;
    int *local_distmat;
    local_distmat = (int *) malloc (sizeof(int)*natoms);
      for(int j=0; j<natoms; j++)
      {
	local_distmat[j]=0;
      }

//int *localindexA, *localindexB, *localindexM;
//localindexA = (int *) malloc (sizeof(int) * lnElt1);
//localindexB = (int *) malloc (sizeof(int) * lnElt2);
//localindexm = (int *) malloc (sizeof(int) * count_metal); 

     for(int i=0; i<count_metal; i++)
    {
      for(int j=0; j<natoms; j++)
      {
	float chkx,chky,chkz;
	chkx=fabs(metal_frame[i*3]-first_frame[j*3]);
        chky=fabs(metal_frame[i*3+1]-first_frame[j*3+1]);
        chkz=fabs(metal_frame[i*3+2]-first_frame[j*3+2]);
	if(cell_type=="monoclinic")
	{
        if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
        if(chkx > xvec/2) {chkx = chkx - xvec;}
        if(chkz > zvec/2) {chkz = chkz - zvec;}
	if (sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz)) < bondist ){local_distmat[j]+=1;}
	}
	else if(cell_type == "orthorhombic")
	{
        if(chky > yvec/2) {chky = chky - yvec;}
        if(chkx > xvec/2) {chkx = chkx - xvec;}
        if(chkz > zvec/2) {chkz = chkz - zvec;}
	if (sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz)) < bondist ){local_distmat[j]+=1;}
	}
      }
    }
      for(int j=0; j<natoms; j++)
      {
	if(local_distmat[j] >=1 && species_order[j]==1){nA_int++;}
	else if(local_distmat[j] >=1 && species_order[j]==2){nB_int++;}
	if(local_distmat[j] <1 && species_order[j]==1){nA_bulk++;}
	else if(local_distmat[j] <1 && species_order[j]==2){nB_bulk++;}
	
      }


A_int=(float*)malloc(sizeof(float)*nA_int*(nstruct/skip)*3);
B_int=(float*)malloc(sizeof(float)*nB_int*(nstruct/skip)*3);
A_bulk=(float*)malloc(sizeof(float)*nA_bulk*(nstruct/skip)*3);
B_bulk=(float*)malloc(sizeof(float)*nB_bulk*(nstruct/skip)*3);

printf("Reading the positions in to data structures\n");
cout << "for every frame "<< skip-1 <<" frames will be skipped"<<endl;
cout << nstruct/skip <<" frames will be used for the analysis"<<endl;

 for(int i=0;i<(start_frame*(natoms+2));i++)
 {
    getline(traj,Dummy);
 }

 for(int i=0; i< nstruct/skip;i=i+1)
  {
    int counta=0, countb=0, counta1=0, countb1=0;
    getline(traj,Dummy);
    getline(traj,Dummy);
    for(int j=0; j< natoms; j++)
    {
      traj >> felt >> Dum1 >> Dum2 >> Dum3;
      if(felt==Elt1 && local_distmat[j] >=1)
      {
        A_int[counta*3+i*nA_int*3] = Dum1;
        A_int[counta*3+i*nA_int*3+1] = Dum2;
        A_int[counta*3+i*nA_int*3+2]= Dum3;
	counta=counta+1;
      }
      else if(felt==Elt2 && local_distmat[j] >=1)
      {
        B_int[countb*3+i*nB_int*3] = Dum1;
        B_int[countb*3+i*nB_int*3+1] = Dum2;
        B_int[countb*3+i*nB_int*3+2] = Dum3;
        countb=countb+1;
      }
      else if(felt==Elt1 && local_distmat[j] <1)
      {
        A_bulk[counta1*3+i*nA_bulk*3] = Dum1;
        A_bulk[counta1*3+i*nA_bulk*3+1] = Dum2;
        A_bulk[counta1*3+i*nA_bulk*3+2]= Dum3;
	counta1=counta1+1;
      }
      else if(felt==Elt2 && local_distmat[j] <1)
      {
        B_bulk[countb1*3+i*nB_bulk*3] = Dum1;
        B_bulk[countb1*3+i*nB_bulk*3+1] = Dum2;
        B_bulk[countb1*3+i*nB_bulk*3+2] = Dum3;
        countb1=countb1+1;
      }
    }
    getline(traj,Dummy); 
    for (int m = 0 ; m < skip-1; m++)
    {
      for(int j=0;j<natoms;j++)
      {
	getline(traj,Dummy);
      }
      getline(traj,Dummy);
      getline(traj,Dummy);
    }
  }
   traj.close();  
   

   
}



     //for everything else...
else if (scope =="bulk")
{
    //
  A= (float *) malloc (sizeof(float)*lnElt1*(nstruct/skip)*3);

  if (Elt2 != "")
  {
  B= (float *) malloc (sizeof(float)*lnElt2*(nstruct/skip)*3);
  }
  if (metal_species != "")
  {
    METAL= (float *) malloc (sizeof(float)*count_metal*(nstruct/skip)*3);
  }
  if (unwrap == "yes")
  {
      Aint= (float *) malloc (sizeof(float)*lnElt1*(nstruct/skip)*3);
  }
  
  if(Elt1=="All" && Elt2=="All")
  {
  A_all= (float *) malloc (sizeof(float)*lnElt1*(nstruct/skip)*4);
  B_all= (float *) malloc (sizeof(float)*lnElt2*(nstruct/skip)*4);
  }
//Read coordinates of respective elements into the matrices

 //cout << (nstruct/skip)<<endl;
  lattice = (float *) malloc (sizeof(float)*6*(nstruct/skip));
  for(int i =0; i<(nstruct/skip); i++)
  {
      for(int j=0; j<6;j++)
      {
      lattice[j+i*6]=0.0;
      }
  }


printf("Reading the positions in to data structures\n");
cout << "for every frame "<< skip-1 <<" frames will be skipped"<<endl;
cout << nstruct/skip <<" frames will be used for the analysis"<<endl;
float Dum1,Dum2,Dum3;

 for(int i=0;i<(start_frame*(natoms+2));i++)
 {
    getline(traj,Dummy);
 }
float Dum4;
 for(int i=0; i<(nstruct/skip);i=i+1)
  {// cout<<i<<endl; 
    int counta=0, countb=0, countc=0;
    getline(traj,Dummy);
    //getline(traj,Dummy);
    traj >> lattice[i*6] >> lattice[1+i*6] >> lattice[2+i*6]>> lattice[3+i*6]>>lattice[4+i*6]>>lattice[5+i*6];
    //cout <<" "<< lattice[i*6] <<" "<< lattice[1+i*6] <<" "<< lattice[2+i*6]<<endl;
    //getline(traj,Dummy);
    for(int j=0; j< natoms; j++)
    {
      
      if(ext_charge=="yes"){traj >> felt >> Dum1 >> Dum2 >> Dum3 >> Dum4;}
      else {traj >> felt >> Dum1 >> Dum2 >> Dum3;}
      if(felt==Elt1)
      {
        A[counta*3+i*lnElt1*3] = Dum1;
        A[counta*3+i*lnElt1*3+1] = Dum2;
        A[counta*3+i*lnElt1*3+2]= Dum3;
	counta=counta+1;
      }
      else if(felt==Elt2)
      {
        B[countb*3+i*lnElt2*3] = Dum1;
        B[countb*3+i*lnElt2*3+1] = Dum2;
        B[countb*3+i*lnElt2*3+2] = Dum3;
        countb=countb+1;
      }
      
      else if(Elt1=="water")
      {
	if(felt=="O" || felt=="H")
	{
	A[counta*3+i*lnElt1*3] = Dum1;
        A[counta*3+i*lnElt1*3+1] = Dum2;
        A[counta*3+i*lnElt1*3+2]= Dum3;
	counta=counta+1;
	}
      }
      if(felt==metal_species)
      {
        METAL[countc*3+i*count_metal*3] = Dum1;
        METAL[countc*3+i*count_metal*3+1] = Dum2;
        METAL[countc*3+i*count_metal*3+2] = Dum3;
        countc=countc+1;
      }
      if(Elt1=="All" && Elt2=="All")
      {
	A_all[counta*4+i*lnElt1*4] = Dum1;
        A_all[counta*4+i*lnElt1*4+1] = Dum2;
        A_all[counta*4+i*lnElt1*4+2]= Dum3;
	if(ext_charge=="yes"){A_all[counta*4+i*lnElt1*4+3] = Dum4;}
	else{
	if(felt=="O") {A_all[counta*4+i*lnElt1*4+3]= -2.0;}
	else if(felt=="H") {A_all[counta*4+i*lnElt1*4+3]= 1.0;}
	else if(felt=="Cu") {A_all[counta*4+i*lnElt1*4+3]= 0.0;}
	else if(felt=="Zn") {A_all[counta*4+i*lnElt1*4+3]= 2.0;}
	}
	counta=counta+1;
      }
      
    }
    getline(traj,Dummy);
    for (int m = 0 ; m < skip-1; m++)
    {
      for(int j=0;j<natoms;j++)
      {
	getline(traj,Dummy);
      }
      getline(traj,Dummy);
      getline(traj,Dummy);
    }
  }
  if(Elt1=="All" && Elt2=="All") {B_all = A_all;}
   traj.close();  
   //cout <<"HI"<<endl;
// read the lattice information from the last line of the trajectory
 string cmd("tail -n 1 ");
 cmd += file_path;
 cmd += " > cellsize";
 system (cmd.c_str());
 ifstream cellsize;
 cellsize.open("cellsize",ios::in);
 yxvec =0; 
 if(cell_type == "orthorhombic") cellsize>> xvec >>yvec>> zvec;
 else if(cell_type == "monoclinic") cellsize >>xvec>>yvec>>zvec>>yxvec;
 cellsize.close();
 system("rm cellsize");
 cout <<"File read complete"<<endl;
 cout <<"Lattice length (X Y Z) = "<< xvec << " "<< yvec << " "<< zvec << " " << yxvec <<endl;  
}
//velocity information is also read and stored incase of velocity plot
  if(veloc=="yes")
  {
    float Dum1,Dum2,Dum3;
  printf("Reading the velocities in to data structures\n");
  if(Elt1=="all")
  {
  VEL= (float *) malloc (sizeof(float)*natoms*(nstruct/skip)*3);
  }
  else
  {
  VEL= (float *) malloc (sizeof(float)*lnElt1*(nstruct/skip)*3);  
  }
  ifstream trajvel;
  trajvel.open("inputvel.xyz",ios::in);
  
   for(int i=0;i<(start_frame*(natoms+9));i++)
   {
    getline(trajvel,Dummy);
   }
  
  for(int i=0; i<(nstruct/skip);i=i+1)
  {
    for(int a=0; a<9; a++) //UGLY- careful special for the file from lammps with 10 empty lines - needs fix
    {
      getline(trajvel,Dummy);
    }
    if(Elt1=="all")
    {
    for(int j=0; j< natoms; j++)
    {
      trajvel >> felt >> Dum1 >>Dum2>>Dum3;
        VEL[j*3+i*natoms*3] = Dum1;
        VEL[j*3+i*natoms*3+1] = Dum2;
        VEL[j*3+i*natoms*3+2]= Dum3;
    }
    }
    else if(Elt1=="water")
    {
      int counta=0;
    for(int j=0; j< natoms; j++)
    {
      trajvel >> felt >> Dum1 >>Dum2>>Dum3;
      if(felt=="O"||felt=="H")
      {
        VEL[counta*3+i*lnElt1*3] = Dum1;
        VEL[counta*3+i*lnElt1*3+1] = Dum2;
        VEL[counta*3+i*lnElt1*3+2]= Dum3;
	counta++;
      }
    }      
    }
    else
    {
      int counta=0;
    for(int j=0; j< natoms; j++)
    {
      trajvel >> felt >> Dum1 >>Dum2>>Dum3;
      if(felt==Elt1)
      {
        VEL[counta*3+i*lnElt1*3] = Dum1;
        VEL[counta*3+i*lnElt1*3+1] = Dum2;
        VEL[counta*3+i*lnElt1*3+2]= Dum3;
	counta++;
      }
    }      
    }
    getline(trajvel,Dummy);
    for (int m = 0 ; m < skip-1; m++)
    {
     for(int a=0; a<9; a++) //UGLY- careful special for the file from lammps with 10 empty lines - needs fix
     {
       getline(trajvel,Dummy);
     }      
     for(int j=0;j<natoms;j++)
     {
       getline(trajvel,Dummy);
     }
    }
  }
  trajvel.close();
  }

 
//########################## UNWRAPPER
/*
if( unwrap == "yes")
{
  
  cout << "Unwrapping the trajectory begins"<<endl;
  float lr[3];

for(int i =0; i<(nstruct/skip); i++)
{
 for(int j =0 ; j<lnElt1; j++)
 {
  for(int k =0; k<3;k++)
  {
   Aint[j*3+i*lnElt1*3+k]=0.0;  
  }
  if(i==0)
  {
   for(int k =0; k<3;k++)
   {
    Aint[j*3+i*lnElt1*3+k]=A[j*3+i*lnElt1*3+k];
   }
  }
 }
}

if(cell_type == "orthorhombic")
 {
  for(int i=1; i<(nstruct/skip); i++)
  {//cout<<lnElt1<<endl;
  //cout<<"hi"<<endl;
      for(int j=0;j<lnElt1;j++)
      {//cout<<Elt1<<" ";
          for(int k=0 ;k<3;k++)
          {
          lr[k] = A[j*3+i*lnElt1*3+k] - A[j*3+(i-1)*lnElt1*3+k];
          if(abs(lr[k]) > lattice[k+i*6]/2.0)
          {
               //cout<<"I am working"<<endl;
              if(lr[k] > 0)
              {
              lr[k] = abs(lr[k]) - lattice[k+i*6];
              Aint[j*3+i*lnElt1*3+k] = Aint[j*3+(i-1)*lnElt1*3+k] - abs(lr[k]); 
              }
              else
              {
              lr[k] = abs(lr[k]) - lattice[k+i*6];
              Aint[j*3+i*lnElt1*3+k] = Aint[j*3+(i-1)*lnElt1*3+k] + abs(lr[k]); 
              }     
          }
           else
           {
              Aint[j*3+i*lnElt1*3+k] = Aint[j*3+(i-1)*lnElt1*3+k] + lr[k]; 
           } //cout<<Aint[j*3+i*lnElt1*3+k]<<" ";
          }//cout<<endl;
      }
  }
 }
 else if(cell_type == "monoclinic")
 {
  for(int i=1; i<(nstruct/skip); i++)
  {
      for(int j=0;j<lnElt1;j++)
      {        
        lr[0]=A[j*3+i*lnElt1*3]-A[j*3+(i-1)*lnElt1*3];
        lr[1]=A[j*3+i*lnElt1*3+1]-A[j*3+(i-1)*lnElt1*3+1];
        lr[2]=A[j*3+i*lnElt1*3+2]-A[j*3+(i-1)*lnElt1*3+2];
	
	if(abs(lr[1]) > lattice[1+i*6]/2.0)
        {
        if(lr[1] > 0)
        {
            lr[1] = abs(lr[1]) - lattice[1+i*6];
            Aint[j*3+i*lnElt1*3+1] = Aint[j*3+(i-1)*lnElt1*3+1] - abs(lr[1]);
            lr[0] = abs(lr[0]) - lattice[3+i*6];
            Aint[j*3+i*lnElt1*3+0] = Aint[j*3+(i-1)*lnElt1*3+0] - abs(lr[0]);
	   // Aint[j*3+i*lnElt1*3+0] = Aint[j*3+i*lnElt1*3+0] - lattice[3+i*6];
        }
        else
        {
            lr[1] = abs(lr[1]) - lattice[1+i*6];
            Aint[j*3+i*lnElt1*3+1] = Aint[j*3+(i-1)*lnElt1*3+1] + abs(lr[1]);
            lr[0] = abs(lr[0]) - lattice[3+i*6];
            Aint[j*3+i*lnElt1*3+0] = Aint[j*3+(i-1)*lnElt1*3+0] + abs(lr[0]);
	    //Aint[j*3+i*lnElt1*3+0] = Aint[j*3+i*lnElt1*3+0] + lattice[3+i*6];
        }
        }
        else
        {
        Aint[j*3+i*lnElt1*3+1] = Aint[j*3+(i-1)*lnElt1*3+1] + lr[1];
        }
	
	
        for(int k=0;k<3;k=k+2)
        {
        if(abs(lr[k]) > lattice[k+i*6]/2.0)
        {
            if(lr[k] > 0)
            {
                lr[k]= abs(lr[k]) - lattice[k+i*6];
                Aint[j*3+i*lnElt1*3+k] = Aint[j*3+(i-1)*lnElt1*3+k] - abs(lr[k]);
            }
            else
            {
                lr[k]= abs(lr[k]) - lattice[k+i*6];
                Aint[j*3+i*lnElt1*3+k] = Aint[j*3+(i-1)*lnElt1*3+k] + abs(lr[k]);
            }
        }
        else
        {
            Aint[j*3+i*lnElt1*3+k] = Aint[j*3+(i-1)*lnElt1*3+k] + lr[k];
        }
        }
      }
  }
 }


for(int i =0; i<(nstruct/skip); i++)
{
for(int j =0 ; j<lnElt1; j++)
{
for(int k =0; k<3;k++)
{
A[j*3+i*lnElt1*3+k]=Aint[j*3+i*lnElt1*3+k];
}
}
}

if(unwrapout=="yes")
{

ofstream unwrapped;
unwrapped.open("input_unwrapped.xyz");

for(int i =0; i<(nstruct/skip); i++)
{
unwrapped<<lnElt1<<endl;
unwrapped<<endl;
for(int j =0 ; j<lnElt1; j++)
{unwrapped<<Elt1 <<" ";
for(int k =0; k<3;k++)
{
unwrapped<<Aint[j*3+i*lnElt1*3+k]<<" ";
}
unwrapped<<endl;
}
}
unwrapped.close();
}

free(Aint);
}


//############################# UNWRAPPER FINISH
  */
//in case of same element RDF

  if(Elt1==Elt2)
  {
    B=A;
  }  
  nElt1=lnElt1;
  nElt2=lnElt2;
}




