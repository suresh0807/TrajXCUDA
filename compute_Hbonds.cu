#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void compute_Hbonds(void)
{

  
  //lines for exchange matrix------NO implication whatsoever for the Hbonds computation-------------------------------
  //------------------------------------------------------------------------------------------------------------------
cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;

  int *exch1;
  int *dev_exch1;
  exch1= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  // Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch1[i+l*nElt1]=0;
        }
        Aintnum[l]=0;
        Abulknum[l]=0;
      }
      
      float GLOBALAminz=A[dircover];
      float GLOBALAmaxz=A[dircover];
      float MIDAz=0.0, MINAz=0.0, MAXAz=0.0;
      
  for(int i=0; i<1; i++)
  {
    for(int j=1; j<nElt1; j++)
    {
    if(A[j*3+i*nElt1*3+dircover] > GLOBALAmaxz)    GLOBALAmaxz=A[j*3+i*nElt1*3+dircover];
    else if(A[j*3+i*nElt1*3+dircover] < GLOBALAminz)    GLOBALAminz=A[j*3+i*nElt1*3+dircover];
    }
  }
  
  MIDAz=(GLOBALAmaxz+GLOBALAminz)/2.0;
  
  MAXAz=MIDAz+(bondist/2.0);
  MINAz=MIDAz-(bondist/2.0);
  float AVGintnum=0.0, AVGbulknum=0.0;
  
  if(HB_for=="int")
  {
//Allocate memory in GPU device

float *dev_A1;

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A1,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));


//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A1,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    


//dim3 struct to define elements of the execution configuration

    dim3 dimBlockc(10,10,10);
    dim3 dimGridc((nElt1+dimBlockc.x-1)/dimBlockc.x,(count_metal+dimBlockc.y-1)/dimBlockc.y,((nstruct/skip)+dimBlockc.z-1)/dimBlockc.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

  
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
  
     hipFree(dev_A1);
     hipFree(dev_B);
     
   /*  ofstream exchplot;
    exchplot.open("exchange1.data");

   for(int l=0; l<(nstruct/skip); l++)
    {
      exchplot <<l<<" ";
      for(int j=0; j<nElt1; j++)
      {

          exchplot << exch1[j+l*nElt1]<<" ";
      }
      exchplot <<endl;
    }
    exchplot.close();*/

   
   // 1 in exch matrix meand a HBOND is present in interface, 0 means it is in bulk
   
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch1[i+l*nElt1]==0) Abulknum[l]++;
        }
        AVGintnum+=Aintnum[l];
        AVGbulknum+=Abulknum[l];
    }
AVGintnum/=float(nstruct/skip);
AVGbulknum/=float(nstruct/skip);
cout <<AVGintnum<<" "<<AVGbulknum<<" "<<AVGintnum+AVGbulknum<<endl;
  }
  
  else if(HB_for=="bulk")
  {
 float *dev_A1;

    hipMalloc((void **)&dev_A1,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));


//Copy data from host to device

    hipMemcpy(dev_A1,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    


//dim3 struct to define elements of the execution configuration

    dim3 dimBlockc(32,1,32);
    dim3 dimGridc((nElt1+dimBlockc.x-1)/dimBlockc.x,1,((nstruct/skip)+dimBlockc.z-1)/dimBlockc.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

  
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
  
     hipFree(dev_A1);
     
   /*  ofstream exchplot;
    exchplot.open("exchange1.data");

   for(int l=0; l<(nstruct/skip); l++)
    {
      exchplot <<l<<" ";
      for(int j=0; j<nElt1; j++)
      {

          exchplot << exch1[j+l*nElt1]<<" ";
      }
      exchplot <<endl;
    }
    exchplot.close();*/

   
   // 1 in exch matrix meand a HBOND is present in interface, 0 means it is in bulk
   
   
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Abulknum[l]++;}
          else if (exch1[i+l*nElt1]==0) Aintnum[l]++;
        }
        AVGintnum+=Aintnum[l];
        AVGbulknum+=Abulknum[l];
    }
AVGintnum/=float(nstruct/skip);
AVGbulknum/=float(nstruct/skip);
cout <<AVGintnum<<" "<<AVGbulknum<<" "<<AVGintnum+AVGbulknum<<endl;   
  }
  
  else if (HB_for=="all")
  {
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
  }
//exchange matrix created--------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------
  
  
  
  int *Onum; //number of O atoms within a specific distance (must be greater than 2 if used 3.2) of the H atom.
  int *Hbondneigh; // Ids of the (O) neighbours of the H atom. 
  float *Hbondneighdist; // OH distance for all neighbours.
  float *Hbondxyz; //storage.
  
  float *Hdistmat, *dev_Hdistmat;// Distance matrix to capture the O within x \AA of H atom.
  
  int OHsamples=20; /// maximum number of OH bonds considered within bonddist sphere
  
    Onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    Hbondneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    Hbondneighdist = (float *) malloc (sizeof(float)*OHsamples*nElt1*(nstruct/skip));
    Hbondxyz = (float *) malloc (sizeof(float)*9*nElt1*(nstruct/skip));  //X,Y and Z of Hatom followed by angle (depends on criterium set), OO distance, neighbour O id, and O-H distance,OH distance,O-HO/OO-H
    Hdistmat = (float *) malloc (sizeof(float)*nElt1*nElt2);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      Onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k]=0.0;
      }
      for(int k =0 ;k<9;k++)
      {
      Hbondxyz[j*9+i*nElt1*9+k]=0.0;
      }
    }
  }   
      
 for(int i=0;i<nElt1;i++)
  {
    for(int j=0; j<nElt2; j++)
    {
      Hdistmat[j+i*nElt2]=0.0;
    }
  }
      
cout <<"gpu begins OH distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_Hdistmat,sizeof(float)*nElt1*nElt2);
    
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_Hdistmat,Hdistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
    
   //dim3 struct to define elements of the execution configuration


    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,1);
for(int i = 0; i<(nstruct/skip); i++) /// go into each frame
{
   for(int i1=0;i1<nElt1;i1++)///initialize Hdistmat for 1 frame
  {
    for(int j1=0; j1<nElt2; j1++)
    {
      Hdistmat[j1+i1*nElt2]=0.0;
    }
  }
  hipMemcpy(dev_Hdistmat,Hdistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_Hdistmat,bondist,nElt1,nElt2,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_Hdistmat,bondist,nElt1,nElt2,dev_lattice,i);      
    }

hipMemcpy(Hdistmat,dev_Hdistmat,sizeof(float)*nElt1*nElt2,hipMemcpyDeviceToHost);
   
  for(int i1=0;i1<nElt1;i1++) //hydrogen
  {
    int k=0;
    for(int j1=0; j1<nElt2; j1++) //oxygen 
    {
      if(Hdistmat[j1+i1*nElt2] !=0.0) // check if an axygen atom is within the 'bonddist' distance from hydrogen atom
      {
	Onum[i1+i*nElt1]++;
	Hbondneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	Hbondneighdist[i1*OHsamples+i*nElt1*OHsamples+k] = Hdistmat[j1+i1*nElt2];
	k++;
      }
    }
  }

}   
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_Hdistmat);
    hipFree(dev_lattice);
    


//Hbondneigh contain the IDs of four possible oxygen neighbour atoms if the hydrogen chosen is participating in a hydrogen bond.
//Onum has the number of oxygen atoms within a specified distance (see input file).

cout <<"gpu ends"<<endl; 

      
//swapping the indices of nearest oxygens to get the two shortest OH bonds up front, 
//since it usually describes the H bond

cout<<"Swapping of OH bonds based on distance "<<endl;

float swapa,swapb;
    for(int i=0;i<(nstruct/skip);i++)
     {
      for(int j=0; j<nElt1; j++)
       {
	 
	for(int swf=0; swf<Onum[j+i*nElt1]-1;swf++)
	{
	for(int sws=swf+1; sws<Onum[j+i*nElt1];sws++)
	{
	if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf] > Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws] && sws !=swf && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws] !=0.0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf] !=0.0) 
	{
	   swapa = Hbondneigh[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    Hbondneigh[j*OHsamples+i*nElt1*OHsamples+swf]=Hbondneigh[j*OHsamples+i*nElt1*OHsamples+sws];
	    Hbondneigh[j*OHsamples+i*nElt1*OHsamples+sws]=swapa;
	   swapb = Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf]=Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws];
	    Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws]=swapb;
	}
	}
	}	
       }
     }	
   
cout<<" done !!!"<<endl;
   
   
cout<<"All angles and distance computation begins"<<endl;
   
 float  *distOOsamples, *anglesOHOsamples, *anglesHOOsamples;
 
 distOOsamples = (float*) malloc (sizeof(float)*(OHsamples)*nElt1*(nstruct/skip));
 anglesOHOsamples = (float*) malloc (sizeof(float)*(OHsamples)*nElt1*(nstruct/skip));
 anglesHOOsamples = (float*) malloc (sizeof(float)*(OHsamples)*nElt1*(nstruct/skip)); 
 
 for(int i=0; i<(nstruct/skip); i++)
 {
   for(int j=0; j<nElt1; j++)
   {
     for(int k=0; k<(OHsamples); k++)
     {
       distOOsamples[k+j*(OHsamples)+i*nElt1*(OHsamples)]=0.0;
       anglesOHOsamples[k+j*(OHsamples)+i*nElt1*(OHsamples)]=0.0;
       anglesHOOsamples[k+j*(OHsamples)+i*nElt1*(OHsamples)]=0.0;
     }
   }
 }
 float vec1[3],vec2[3];
 for(int i=0;i<(nstruct/skip);i++) // into each frame one at a time
     {
      for(int j=0; j<nElt1; j++) // for every hydrogen atom
       {
	 for(int k =0 ;k<3;k++)
         {
	 vec1[k] = B[i*nElt2*3+Hbondneigh[j*OHsamples+i*nElt1*OHsamples]*3+k];
	 }
	 
	 //for(int l=0; l<OHsamples; l++)
	 for(int l=0; l<8; l++)//only the first eight O neighbours enough
	 {
	   
	 if(cell_type == "orthorhombic")
	 {
	for(int k =0 ;k<3;k++)
	{
	vec2[k] = B[i*nElt2*3+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+l]*3+k];
	if(k==0) 
	{
	  if(fabs(A[j*3+i*nElt1*3+k]-vec1[k]) > xvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec1[k] > 0) {vec1[k] = vec1[k] +xvec;}
		   else {vec1[k] = vec1[k] -xvec;}}
	  if(fabs(A[j*3+i*nElt1*3+k]-vec2[k]) > xvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec2[k] > 0) {vec2[k] = vec2[k] +xvec;}
		   else {vec2[k] = vec2[k] -xvec;}}	   
	}
		   
	else if(k==1) 
	{
	  if(fabs(A[j*3+i*nElt1*3+k]-vec1[k]) > yvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec1[k] > 0) {vec1[k] = vec1[k] +yvec;}
		   else {vec1[k] = vec1[k] -yvec;}}
	  if(fabs(A[j*3+i*nElt1*3+k]-vec2[k]) > yvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec2[k] > 0) {vec2[k] = vec2[k] +yvec;}
		   else {vec2[k] = vec2[k] -yvec;}}   
	}
		   
	else if(k==2) 
	{
	  if(fabs(A[j*3+i*nElt1*3+k]-vec1[k]) > zvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec1[k] > 0) {vec1[k] = vec1[k] +zvec;}
		   else {vec1[k] = vec1[k] -zvec;}}
	  if(fabs(A[j*3+i*nElt1*3+k]-vec2[k]) > zvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec2[k] > 0) {vec2[k] = vec2[k] +zvec;}
		   else {vec2[k] = vec2[k] -zvec;}}	   
	}
	}
	 }
	 else if(cell_type =="monoclinic")
	 {
	int k=0;
	vec2[0] = B[i*nElt2*3+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+l]*3];
	vec2[1] = B[i*nElt2*3+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+l]*3+1];
	vec2[2] = B[i*nElt2*3+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+l]*3+2];
	k=1;
	if(fabs(A[j*3+i*nElt1*3+k]-vec1[k]) > yvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec1[k] > 0) {vec1[k] = vec1[k] +yvec; vec1[0]=vec1[0]+yxvec;}
		   else {vec1[k] = vec1[k] -yvec; vec1[0]=vec1[0]-yxvec;}}
	if(fabs(A[j*3+i*nElt1*3+k]-vec2[k]) > yvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec2[k] > 0) {vec2[k] = vec2[k] +yvec; vec2[0]=vec2[0]+yxvec;}
		   else {vec2[k] = vec2[k] -yvec; vec2[0]=vec2[0]-yxvec;}}	   
	k=0;
	if(fabs(A[j*3+i*nElt1*3+k]-vec1[k]) > xvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec1[k] > 0) {vec1[k] = vec1[k] +xvec;}
		   else {vec1[k] = vec1[k] -xvec;}}
	if(fabs(A[j*3+i*nElt1*3+k]-vec2[k]) > xvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec2[k] > 0) {vec2[k] = vec2[k] +xvec;}
		   else {vec2[k] = vec2[k] -xvec;}} 	 
	k=2;
	if(fabs(A[j*3+i*nElt1*3+k]-vec1[k]) > zvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec1[k] > 0) {vec1[k] = vec1[k] +zvec;}
		   else {vec1[k] = vec1[k] -zvec;}}
	if(fabs(A[j*3+i*nElt1*3+k]-vec2[k]) > zvec/2) 
	          {if (A[j*3+i*nElt1*3+k]-vec2[k] > 0) {vec2[k] = vec2[k] +zvec;}
		   else {vec2[k] = vec2[k] -zvec;}}
	
	 }
	 //angle computed with the central element at first and the others following
	 anglesOHOsamples[l+j*(OHsamples)+i*nElt1*(OHsamples)]= angle(A[j*3+i*nElt1*3],A[j*3+i*nElt1*3+1],A[j*3+i*nElt1*3+2],vec1[0], vec1[1], vec1[2], vec2[0], vec2[1], vec2[2]);
	 anglesHOOsamples[l+j*(OHsamples)+i*nElt1*(OHsamples)]= angle(vec1[0], vec1[1], vec1[2],vec2[0], vec2[1], vec2[2],A[j*3+i*nElt1*3],A[j*3+i*nElt1*3+1],A[j*3+i*nElt1*3+2]);
	 distOOsamples[l+j*(OHsamples)+i*nElt1*(OHsamples)]= sqrt(square(vec2[0]-vec1[0])+square(vec2[1]-vec1[1])+square(vec2[2]-vec1[2]));

	   
	}
	 
       }
     }
 

   ofstream Hneigh;
Hneigh.open("Hneigh.data");
Hneigh<<"S H N F FD      S   SD      OHO     ALEX<   MATTI   OO"<<endl;
  for(int i=0;i<nstruct/skip;i++)
  {      
    for(int j=0; j<nElt1; j++)
    {    //Hneigh<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" ";
      for(int k=0; k<OHsamples;k++)
      {
	if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] !=0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != Hbondneighdist[j*OHsamples+i*nElt1*OHsamples])
	{
      Hneigh<<i<<" "<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k]<<" "<<anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)]*(180.0/3.14159)<<" " << 1.37+((-1.71)*(cos(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)])) )<<" "<< anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)]*(180.0/3.14159)<<" "<<distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)]<<endl;
	}
      //Hneigh<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+2]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+2]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+3]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+3]<<endl;
      }
    } 
  }  
Hneigh.close();
  
cout<<"Done!!!"<<endl;

/*
   ofstream Hneigh;
Hneigh.open("Hneigh.data");

  for(int i=0;i<nstruct/skip;i++)
  {      
    for(int j=0; j<nElt1; j++)
    {    
      Hneigh<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+1]<<" "<< Hbondxyz[j*5+i*nElt1*5+3]*(180.0/3.14159)<<" "<< angHoo[j+i*nElt1]*(180.0/3.14159)<<" "<<Hbondxyz[j*5+i*nElt1*5+4]<<endl;
      Hneigh<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+2]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+2]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+3]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+3]<<endl;
      
    } 
  }  
Hneigh.close();
*/

 ///Histograms here- yippie - I can derive the fit for Hbond criterium from here.
 
  int whichwater;
      if(HB_for=="int"){whichwater=1;}
      else if(HB_for=="bulk"){whichwater=1;}
      else if(HB_for=="all"){whichwater=2;} 
      
 if(HB_histograms=="yes")
 {
 
 cout<<"Histograms for angles and distance combinations "<<endl;
 
 float  OHint=0, OHmax=2;
 float  O_Hint=0, O_Hmax=bondist;
 float  OOint=0, OOmax=5.0;
 float  OHOint=0, OHOmax=180;
 float  HOOint=0, HOOmax=180;
 int num_binsOH=500;
 int num_binsO_H=500;
 int num_binsOO=500;
 int num_binsOHO=500;
 int num_binsHOO=500;
  OHint=OHmax/float(num_binsOH);
  O_Hint=O_Hmax/float(num_binsO_H);
  OOint=OOmax/float(num_binsOO);
  OHOint=OHOmax/float(num_binsOHO);
  HOOint=HOOmax/float(num_binsHOO);
    
  float *OHtics, *O_Htics, *OOtics, *OHOtics, *HOOtics;
  
  
  OHtics = (float *) malloc (sizeof(float)*num_binsOH*2);
  O_Htics = (float *) malloc (sizeof(float)*num_binsO_H*2);
  OOtics = (float *) malloc (sizeof(float)*num_binsOH*2);
  OHOtics = (float *) malloc (sizeof(float)*num_binsO_H*2);
  HOOtics = (float *) malloc (sizeof(float)*num_binsO_H*2);
  
  int *OH_O_H, *OH_OO, *OH_OHO, *OH_HOO;
  
  OH_O_H = (int *) malloc (sizeof(int)*num_binsOH*num_binsO_H);
  OH_OO = (int *) malloc (sizeof(int)*num_binsOH*num_binsOO);
  OH_OHO = (int *) malloc (sizeof(int)*num_binsOH*num_binsOHO);
  OH_HOO = (int *) malloc (sizeof(int)*num_binsOH*num_binsHOO);
  
  int *O_H_OO, *O_H_OHO, *O_H_HOO;
  
  O_H_OO = (int *) malloc (sizeof(int)*num_binsO_H*num_binsOO);
  O_H_OHO = (int *) malloc (sizeof(int)*num_binsO_H*num_binsOHO);
  O_H_HOO = (int *) malloc (sizeof(int)*num_binsO_H*num_binsHOO);
  
  int *OO_OHO, *OO_HOO;
  
  OO_OHO = (int *) malloc (sizeof(int)*num_binsOO*num_binsOHO);
  OO_HOO = (int *) malloc (sizeof(int)*num_binsOO*num_binsHOO);
  
  int *OHO_HOO;
  
  OHO_HOO = (int *) malloc (sizeof(int)*num_binsOHO*num_binsHOO);
  
  for(int i=0; i<num_binsOH; i++)
  {
   for(int j=0; j<num_binsO_H; j++)
   {
     OH_O_H[j+i*num_binsO_H]=0;
   }
   for(int j=0; j<num_binsOO; j++)
   {
     OH_OO[j+i*num_binsOO]=0;
   }
   for(int j=0; j<num_binsOHO; j++)
   {
     OH_OHO[j+i*num_binsOHO]=0;
   }
   for(int j=0; j<num_binsHOO; j++)
   {
     OH_HOO[j+i*num_binsHOO]=0;
   }
  }
  
  for(int i=0; i<num_binsO_H; i++)
  {
   for(int j=0; j<num_binsOO; j++)
   {
     O_H_OO[j+i*num_binsOO]=0;
   }
   for(int j=0; j<num_binsOHO; j++)
   {
     O_H_OHO[j+i*num_binsOHO]=0;
   }
   for(int j=0; j<num_binsHOO; j++)
   {
     O_H_HOO[j+i*num_binsHOO]=0;
   }
  }
  
  for(int i=0; i<num_binsOO; i++)
  {
   for(int j=0; j<num_binsOHO; j++)
   {
     OO_OHO[j+i*num_binsOHO]=0;
   }
   for(int j=0; j<num_binsHOO; j++)
   {
     OO_HOO[j+i*num_binsHOO]=0;
   }
  }
  
  for(int i=0; i<num_binsOHO; i++)
  {
   for(int j=0; j<num_binsHOO; j++)
   {
     OHO_HOO[j+i*num_binsHOO]=0;
   }
  }

  
for(int i=0; i<num_binsOH; i++)
{
 OHtics[i*2]=i*OHint;
 OHtics[i*2+1]=(i+1)*OHint;
}  
for(int i=0; i<num_binsO_H; i++)
{
 O_Htics[i*2]=i*O_Hint;
 O_Htics[i*2+1]=(i+1)*O_Hint;
} 
for(int i=0; i<num_binsOO; i++)
{
 OOtics[i*2]=i*OOint;
 OOtics[i*2+1]=(i+1)*OOint;
} 
for(int i=0; i<num_binsOHO; i++)
{
 OHOtics[i*2]=i*OHOint;
 OHOtics[i*2+1]=(i+1)*OHOint;
} 
for(int i=0; i<num_binsHOO; i++)
{
 HOOtics[i*2]=i*HOOint;
 HOOtics[i*2+1]=(i+1)*HOOint;
} 

/*
for(int i=0; i<num_binsOH; i++)
{
cout<< OHtics[i*2]<<" "<< OHtics[i*2+1]<<endl;
} 
for(int i=0; i<num_binsO_H; i++)
{
cout<< O_Htics[i*2]<<" "<< O_Htics[i*2+1]<<endl;
} 
*/
if(whichwater == 2)
{  
for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      //for(int k=0; k<OHsamples; k++)
      for(int k=0; k<5; k++) //only the four neighbouring water molecules
      {
	if(Onum[j+i*nElt1] >1 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] > 0.0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != Hbondneighdist[j*OHsamples+i*nElt1*OHsamples])
	{
         
	  for(int i1=0; i1<num_binsOH; i1++)
          {
	   if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples] > OHtics[i1*2] && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples] < OHtics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsO_H; i2++)
            {
             if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] > O_Htics[i2*2] && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] < O_Htics[i2*2+1])
	     {
	      OH_O_H[i2+i1*num_binsO_H]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsOO; i2++)
            {
             if(distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] > OOtics[i2*2] && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] < OOtics[i2*2+1])
	     {
	      OH_OO[i2+i1*num_binsOO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsOHO; i2++)
            {
             if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i2*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i2*2+1])
	     {
	      OH_OHO[i2+i1*num_binsOHO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      OH_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	   break;
	   }//if for il ends
         }//i1 ends
         
         
         for(int i1=0; i1<num_binsO_H; i1++)
          {
	   if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] > O_Htics[i1*2] && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] < O_Htics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsOO; i2++)
            {
             if(distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] > OOtics[i2*2] && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] < OOtics[i2*2+1])
	     {
	      O_H_OO[i2+i1*num_binsOO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsOHO; i2++)
            {
             if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i2*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i2*2+1])
	     {
	      O_H_OHO[i2+i1*num_binsOHO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      O_H_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	    break;
	   }//if for il ends
         }//i1 ends
         
         for(int i1=0; i1<num_binsOO; i1++)
          {
	   if(distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] > OOtics[i1*2] && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] < OOtics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsOHO; i2++)
            {
             if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i2*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i2*2+1])
	     {
	      OO_OHO[i2+i1*num_binsOHO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      OO_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	    break;
	   }//if for il ends
         }//i1 ends
         
         for(int i1=0; i1<num_binsOHO; i1++)
          {
	   if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i1*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      OHO_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	   break;
	   }//if for il ends
         }//i1 ends
      }//if for k ends
     }//k ends
   }//j ends
  }//i ends
}  

else if(whichwater != 2)
{  
for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
     if(exch1[j+i*nElt1] == whichwater)
     {
      //for(int k=0; k<OHsamples; k++)
       for(int k=0; k<5; k++) ///only the four neighbouring water molecules
      {
	if(Onum[j+i*nElt1] >1 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] >0.0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != Hbondneighdist[j*OHsamples+i*nElt1*OHsamples])
	{
         
	  for(int i1=0; i1<num_binsOH; i1++)
          {
	   if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples] > OHtics[i1*2] && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples] < OHtics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsO_H; i2++)
            {
             if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] > O_Htics[i2*2] && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] < O_Htics[i2*2+1])
	     {
	      OH_O_H[i2+i1*num_binsO_H]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsOO; i2++)
            {
             if(distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] > OOtics[i2*2] && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] < OOtics[i2*2+1])
	     {
	      OH_OO[i2+i1*num_binsOO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsOHO; i2++)
            {
             if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i2*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i2*2+1])
	     {
	      OH_OHO[i2+i1*num_binsOHO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      OH_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	   break;
	   }//if for il ends
         }//i1 ends
         
         
         for(int i1=0; i1<num_binsO_H; i1++)
          {
	   if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] > O_Htics[i1*2] && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] < O_Htics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsOO; i2++)
            {
             if(distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] > OOtics[i2*2] && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] < OOtics[i2*2+1])
	     {
	      O_H_OO[i2+i1*num_binsOO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsOHO; i2++)
            {
             if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i2*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i2*2+1])
	     {
	      O_H_OHO[i2+i1*num_binsOHO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      O_H_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	  break;
	   }//if for il ends
         }//i1 ends
         
         for(int i1=0; i1<num_binsOO; i1++)
          {
	   if(distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] > OOtics[i1*2] && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k] < OOtics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsOHO; i2++)
            {
             if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i2*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i2*2+1])
	     {
	      OO_OHO[i2+i1*num_binsOHO]+=1;
	      break;
	     }
	    }
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      OO_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	  break;
	   }//if for il ends
         }//i1 ends
         
         for(int i1=0; i1<num_binsOHO; i1++)
          {
	   if(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > OHOtics[i1*2] && anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < OHOtics[i1*2+1])
	   {
	   
	    for(int i2=0; i2<num_binsHOO; i2++)
            {
             if(anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) > HOOtics[i2*2] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k]*(180.0/3.14159) < HOOtics[i2*2+1])
	     {
	      OHO_HOO[i2+i1*num_binsHOO]+=1;
	      break;
	     }
	    }
	    break;
	   }//if for il ends
         }//i1 ends
      }//if for k ends
     }//k ends
     }
   }//j ends
  }//i ends
}
  ofstream DOH_AOH, DOH_OO, DOH_OHO, DOH_HOO, AOH_OO, AOH_OHO, AOH_HOO, pOO_HOO, pOO_OHO, pOHO_HOO;
  
  DOH_AOH.open("DOH_AOH.data");
  DOH_OO.open("DOH_OO.data");
  DOH_OHO.open("DOH_OHO.data");
  DOH_HOO.open("DOH_HOO.data");
  AOH_OO.open("AOH_OO.data");
  AOH_OHO.open("AOH_OHO.data");
  AOH_HOO.open("AOH_HOO.data");
  pOO_OHO.open("OO_OHO.data");
  pOO_HOO.open("OO_HOO.data");
  pOHO_HOO.open("OHO_HOO.data");
  
  
  float normnum;
  
      if(HB_for=="int"){normnum = AVGintnum;}
      //if(HB_for=="int"){normnum = nElt1;}
      else if(HB_for=="bulk"){normnum = AVGbulknum;}
      else if(HB_for=="all"){normnum = AVGintnum+AVGbulknum;} 
      
      
  for(int i=0; i<num_binsOH; i++)
  {
   for(int j=0; j<num_binsO_H; j++)
   {
     DOH_AOH<<(OHtics[i*2]+OHtics[i*2+1])/2<<" "<<(O_Htics[j*2]+O_Htics[j*2+1])/2<<" "<<OH_O_H[j+i*num_binsO_H]/normnum<<endl;
   }DOH_AOH<<endl;
   for(int j=0; j<num_binsOO; j++)
   {
     DOH_OO<<(OHtics[i*2]+OHtics[i*2+1])/2<<" "<<(OOtics[j*2]+OOtics[j*2+1])/2<<" "<<OH_OO[j+i*num_binsOO]/normnum<<endl;
   } DOH_OO<<endl;
   for(int j=0; j<num_binsOHO; j++)
   {
     DOH_OHO<<(OHtics[i*2]+OHtics[i*2+1])/2<<" "<<(OHOtics[j*2]+OHOtics[j*2+1])/2<<" "<<OH_OHO[j+i*num_binsOHO]/normnum<<endl;
   }DOH_OHO<<endl;
   for(int j=0; j<num_binsHOO; j++)
   {
     DOH_HOO<<(OHtics[i*2]+OHtics[i*2+1])/2<<" "<<(HOOtics[j*2]+HOOtics[j*2+1])/2<<" "<<OH_HOO[j+i*num_binsHOO]/normnum<<endl;
   }DOH_HOO<<endl;
   
  }
  
  for(int i=0; i<num_binsO_H; i++)
  {
   for(int j=0; j<num_binsOO; j++)
   {
     AOH_OO<<(O_Htics[i*2]+O_Htics[i*2+1])/2<<" "<<(OOtics[j*2]+OOtics[j*2+1])/2<<" "<<O_H_OO[j+i*num_binsOO]/normnum<<endl;
   }AOH_OO<<endl;
   for(int j=0; j<num_binsOHO; j++)
   {
     AOH_OHO<<(O_Htics[i*2]+O_Htics[i*2+1])/2<<" "<<(OHOtics[j*2]+OHOtics[j*2+1])/2<<" "<<O_H_OHO[j+i*num_binsOHO]/normnum<<endl;
   }AOH_OHO<<endl;
   for(int j=0; j<num_binsHOO; j++)
   {
     AOH_HOO<<(O_Htics[i*2]+O_Htics[i*2+1])/2<<" "<<(HOOtics[j*2]+HOOtics[j*2+1])/2<<" "<<O_H_HOO[j+i*num_binsHOO]/normnum<<endl;
   }AOH_HOO<<endl;
  }
  
  for(int i=0; i<num_binsOO; i++)
  {
   for(int j=0; j<num_binsOHO; j++)
   {
     pOO_OHO<<(OOtics[i*2]+OOtics[i*2+1])/2<<" "<<(OHOtics[j*2]+OHOtics[j*2+1])/2<<" "<<OO_OHO[j+i*num_binsOHO]/normnum<<endl;
   }pOO_OHO<<endl;
   for(int j=0; j<num_binsHOO; j++)
   {
     pOO_HOO<<(OOtics[i*2]+OOtics[i*2+1])/2<<" "<<(HOOtics[j*2]+HOOtics[j*2+1])/2<<" "<<OO_HOO[j+i*num_binsHOO]/normnum<<endl;
   }pOO_HOO<<endl;
  }
  
  for(int i=0; i<num_binsOHO; i++)
  {
   for(int j=0; j<num_binsHOO; j++)
   {
     pOHO_HOO<<(OHOtics[i*2]+OHOtics[i*2+1])/2<<" "<<(HOOtics[j*2]+HOOtics[j*2+1])/2<<" "<<OHO_HOO[j+i*num_binsHOO]/normnum<<endl;
   }pOHO_HOO<<endl;
  }
  
  
  DOH_AOH.close();
  DOH_OO.close();
  DOH_OHO.close();
  DOH_HOO.close();
  AOH_OO.close();
  AOH_OHO.close();
  AOH_HOO.close();
  pOO_OHO.close();
  pOO_HOO.close();
  pOHO_HOO.close();

//Now, getting the Hbond per oxygen atom.

cout<<"Done !!!"<<endl;
}// histogram yes condition ends

ofstream Hbonds; //writes down the ids of H that are H bonded
Hbonds.open("Hbonds.data");
cout<<"Hydrogen bond identification based on the criterium set "<<endl;  
  
if(HB_criterium_set =="OOH-OO")  //matti
{
    for(int i=0;i<(nstruct/skip);i++)
     {Hbonds<<i<<endl;
      for(int j=0; j<nElt1; j++)
       {
	 for(int k=0; k<OHsamples; k++)
	 {
	   //actual H bond criterion
	   if(Onum[j+i*nElt1] >1 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] !=0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != Hbondneighdist[j*OHsamples+i*nElt1*OHsamples] && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)] <  Hbond_angle_dev && anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)] !=  0.0&& distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)]<  max_O_O && distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)] != 0)
	   {
	    Hbondxyz[j*9+i*nElt1*9]=A[j*3+i*nElt1*3];
	    Hbondxyz[j*9+i*nElt1*9+1]=A[j*3+i*nElt1*3+1];
	    Hbondxyz[j*9+i*nElt1*9+2]=A[j*3+i*nElt1*3+2]; 
	    Hbondxyz[j*9+i*nElt1*9+3]= anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k];
	    Hbondxyz[j*9+i*nElt1*9+4]= distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k];
	    Hbondxyz[j*9+i*nElt1*9+5]= Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k];
	    Hbondxyz[j*9+i*nElt1*9+6]= Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k];
	    Hbondxyz[j*9+i*nElt1*9+7]= Hbondneighdist[j*OHsamples+i*nElt1*OHsamples];
	    Hbondxyz[j*9+i*nElt1*9+8]= anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k];
	    Hbonds<<j<<" ";
	    break;
	     
	  }
	 }
       }
       Hbonds<<endl;
     }
}

Hbonds.close();

if(HB_criterium_set=="OHO-OH")   //alex
{
    for(int i=0;i<(nstruct/skip);i++)
     {
      for(int j=0; j<nElt1; j++)
       {
	 for(int k=0; k<OHsamples; k++)
	 {
	   if(Onum[j+i*nElt1] >1 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] !=0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != Hbondneighdist[j*OHsamples+i*nElt1*OHsamples] &&  Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != 0.0 &&  Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] < 1.37+((-1.71)*(cos(anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+(k)]))) )
	   {
	    Hbondxyz[j*9+i*nElt1*9]=A[j*3+i*nElt1*3];
	    Hbondxyz[j*9+i*nElt1*9+1]=A[j*3+i*nElt1*3+1];
	    Hbondxyz[j*9+i*nElt1*9+2]=A[j*3+i*nElt1*3+2];  
	    Hbondxyz[j*9+i*nElt1*9+3]= anglesOHOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k];
	    Hbondxyz[j*9+i*nElt1*9+4]= distOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k];
	    Hbondxyz[j*9+i*nElt1*9+5]= Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k];
	    Hbondxyz[j*9+i*nElt1*9+6]= Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k];
	    Hbondxyz[j*9+i*nElt1*9+7]= Hbondneighdist[j*OHsamples+i*nElt1*OHsamples];
	    Hbondxyz[j*9+i*nElt1*9+8]= anglesHOOsamples[j*(OHsamples)+i*nElt1*(OHsamples)+k];
	    break;
	   }
	 }
       }
     }  
}
     




cout<<"Done !!!"<<endl;

cout<<"Mapping Hbonds to the donor acceptopr matrix for lifetime calculation"<<endl;
  
  
  ofstream Hexchplot, Hexchplotac;
 
 
 
 //copy coordinates for retrieval information
 int *initHBnum; // number of H bonds in each frame of the trajectory
 

//counting the number of Hbonds in each structure
    initHBnum = (int *) malloc (sizeof(int)*nstruct/skip);
    if(whichwater != 2)
    {
     for(int i=0;i<nstruct/skip;i++) //for each frame
     {
      initHBnum[i]=0;
      for(int j=0; j<nElt1; j++) // for each H atom
       {
	if(Hbondxyz[j*9+i*nElt1*9+4] !=0 && exch1[j+i*nElt1] == whichwater) 
        {
	   initHBnum[i]+=1;
	}
       }
     }
    }
    else if(whichwater ==2)
    {
     for(int i=0;i<nstruct/skip;i++)
     {
      initHBnum[i]=0;
      for(int j=0; j<nElt1; j++)
       { 
	if(Hbondxyz[j*9+i*nElt1*9+4] !=0) 
        {
	   initHBnum[i]+=1;
	}
       }
     }      
    }
    
//storing the coordinates of the Hbonds in nElt2*nElt2*nstruct/skip matrix

//hexch1 stores the acceptor - H matrix - for the H bond lifetime

//only H bond lifetime and no pair lifetimes!!!
if(HBlifetime == "yes")
   {

    float *Hexch, *Hexch1;
    Hexch = (float *) malloc (sizeof(float)*nElt2*nElt1*(nstruct/skip));
    Hexch1 = (float *) malloc (sizeof(float)*nElt2*nElt1*(nstruct/skip));
    
         for(int i=0;i<(nstruct/skip);i++)
         {
           for(int j=0; j<nElt2; j++)//acceptor oxygen
           {
	     for(int k=0; k<nElt1; k++)//shared hydrogen
             {
	       Hexch[k+j*nElt1+i*nElt2*nElt1]=0.0;
	       Hexch1[k+j*nElt1+i*nElt2*nElt1]=0.0;
	     }
	   }
	 }
cout <<"exch matrix"<<endl;	 

int **initHBcol1;// storing the location of the H bond in the donor/acceptor , acceptor-H matrix 

initHBcol1 = (int **) malloc (sizeof(int *)*nstruct/skip);
     for(int i=0; i<nstruct/skip; i++)// for each frame
     {
     initHBcol1[i] = (int *) malloc (sizeof(int)*initHBnum[i]);
     for(int j=0; j<initHBnum[i]; j++)// for number of H bonds this frame
     {
      initHBcol1[i][j]=0;
     }
     }
     
    if(whichwater != 2)
    {
    for(int i=0;i<(nstruct/skip);i++)
     {int chker=0;
      for(int j=0; j<nElt1; j++)
       {
	if( Hbondxyz[j*9+i*nElt1*9+4] !=0 && exch1[j+i*nElt1] == whichwater ) 
        {
	  Hexch1[j+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]*nElt1+i*nElt2*nElt1]=1.0; //acceptor check
	  initHBcol1[i][chker] = (j+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]*nElt1); chker+=1;
	}
	if( Hbondxyz[j*9+i*nElt1*9+4] !=0 ) 
        {
	  Hexch[j+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]*nElt1+i*nElt2*nElt1]=1.0; //acceptor check
	}
	 
      }
     }
    }
    else if(whichwater == 2)
    {
     for(int i=0;i<(nstruct/skip);i++)
     {int chker=0;
      for(int j=0; j<nElt1; j++)
       {
	if(Hbondxyz[j*9+i*nElt1*9+4] !=0) 
        {
	  Hexch1[j+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]*nElt1+i*nElt2*nElt1]=1.0;
	  Hexch[j+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]*nElt1+i*nElt2*nElt1]=1.0;
	  initHBcol1[i][chker] = (j+Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]*nElt1); chker+=1;
	}
       }
     }  
    }
  /*
  lifetime
    Hexchplot.open("Hexch.data");

    for(int i=0;i<(nstruct/skip);i++)
         {
           for(int j=0; j<nElt2; j++)//donor
           {
	     for(int k=0; k<nElt2; k++)//acceptor
             {
	       Hexchplot << Hexch[k+j*nElt2+i*nElt2*nElt2]<<" ";
	     }
	     Hexchplot <<endl;
	   }
	   Hexchplot <<endl<<endl;
	 }
    
    Hexchplot.close(); */
cout<<" Done !!!"<<endl;     
  
  //HB lifetime check - timesaver
//---------------------------------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------------------------------  
////////////////////////////////////LIFETIME-PART///////////////////////////////////////////
  
  
  
  
    total_time = (nstruct/skip)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    num_bins =  int ((nstruct/skip)/((diffuse_time*1000)/timestep));
    num_bins -= 1; 
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    cout <<num_bins <<" number of bins are chosen for the Hydrogen bond analysis"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    origins = SD_store;
    cout <<"This will correlate for "<<SD_store<<" frames in the input trajectory "<<endl;
    int restarts = origins/skips;
    cout <<restarts<<" restarts from each bin will be taken"<<endl;
  
    
    
       SDsum1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
	   SDsum1[ia]=0.0;
	}
	
	float *FDsum1;
	FDsum1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=1;ia<SD_store-1;ia++)
        {
	   FDsum1[ia]=0.0;
	}
 //########################################################################################################################################
//########################################################################################################################################
//BINNING AND AVERAGING THE HBAF
//#########################################################################################################################################
//######################################################################################################################################### 
	 
    for(int i=0; i<num_bins;i++)  //start bin
    {//int avgcounter=0; 
        printf("Cuda start: All lifetimes  %d \n",i);
     for(int j=0; j<SD_store; j=j+skips) //go through restart points
     // for(int j=0; j<1; j++)
     {      
       
       //cout<<initHBnum[j+i*SD_store]<<endl;
       float *Hexch_sized;//O-pair correlation
       float *Hexch_sized1;//H-bond correlation
       Hexch_sized = (float *) malloc (sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
       Hexch_sized1 = (float *) malloc (sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
         for(int ia=0;ia<SD_store;ia++)
         {
           for(int ja=0; ja<initHBnum[j+i*SD_store]; ja++)////only those H that are hydrogen bonded in the first frame in the restart bin
           { //if(i==0) cout << initHBcol[j]<<endl;
	     //cout<<ja<<" "<<initHBcol[j+i*SD_store][ja]<<endl;
	       Hexch_sized1[ja+ia*initHBnum[j+i*SD_store]] = Hexch1[initHBcol1[j+i*SD_store][ja]+(j+i*SD_store)*nElt2*nElt1+ia*nElt2*nElt1];
	       Hexch_sized[ja+ia*initHBnum[j+i*SD_store]]  = Hexch[initHBcol1[j+i*SD_store][ja]+(j+i*SD_store)*nElt2*nElt1+ia*nElt2*nElt1];
	   }
	 }
	 
     
     
 
      
       //SD=(float *) malloc (sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
       SD1=(float *) malloc (sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
	 for(int ja=0; ja<initHBnum[j+i*SD_store]; ja++)
	 {
           //SD[ja+ia*initHBnum[j+i*SD_store]]=0.0;
	   SD1[ja+ia*initHBnum[j+i*SD_store]]=0.0;
	 }
	}
       
      
      dim3 dimBlocka(32,1,32);
      dim3 dimGrida((initHBnum[j+i*SD_store]+dimBlocka.x-1)/dimBlocka.x,1,(SD_store+dimBlocka.z-1)/dimBlocka.z);

       
      
      
      float *dev_A1;
            
      hipMalloc((void **)&dev_A,sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
      hipMalloc((void **)&dev_A1,sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
      hipMalloc((void **)&dev_SD1,sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
      
      hipMemcpy(dev_A,Hexch_sized1,sizeof(float)*initHBnum[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
      hipMemcpy(dev_A1,Hexch_sized,sizeof(float)*initHBnum[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
      hipMemcpy(dev_SD1,SD1,sizeof(float)*initHBnum[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
   
      HBAF_calc<<<dimGrida,dimBlocka>>>(dev_A,dev_A1,dev_SD1,SD_store,initHBnum[j+i*SD_store],i,j,origins,skips);
      hipMemcpy(SD1,dev_SD1,sizeof(float)*initHBnum[j+i*SD_store]*SD_store,hipMemcpyDeviceToHost);
      hipFree(dev_A);
      hipFree(dev_A1);
      
      //SDavg=(float *) malloc (sizeof(float)*SD_store);
      SDavg1=(float *) malloc (sizeof(float)*SD_store);
      for(int ja =0; ja<SD_store;ja++)
      {
      //SDavg[ja]=0.0;
      SDavg1[ja]=0.0;
      }
      int fairy=0;
      //hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
      //hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

      dim3 dimBlocks(1024,1,1);
      dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);
      //SDreduce<<<dimGrids,dimBlocks>>>(dev_SD,dev_SDavg,SD_store,initHBnum[j+i*SD_store],fairy);
      //hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
       
      //hipFree(dev_SD);
      //hipFree(dev_SDavg);
      
      hipMalloc((void **)&dev_SDavg1,sizeof(float)*SD_store);
      hipMemcpy(dev_SDavg1,SDavg1,sizeof(float)*SD_store,hipMemcpyHostToDevice);
      SDreduce<<<dimGrids,dimBlocks>>>(dev_SD1,dev_SDavg1,SD_store,initHBnum[j+i*SD_store],fairy);
      hipMemcpy(SDavg1,dev_SDavg1,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
       
      hipFree(dev_SD1);
      hipFree(dev_SDavg1);
      
           
      
      for(int ja =0; ja<SD_store;ja++)
      {
      //SDsum[ja]+=SDavg[ja];
      SDsum1[ja]+=SDavg1[ja];
      }
  
      for(int ja=1; ja<SD_store -1;ja++)
      {
      //FDsum[ja]+=-((SDavg[ja+1]-SDavg[ja-1]) / (((ja+1)*(timestep/1000))-((ja-1)*(timestep/1000))));
      FDsum1[ja]+=-((SDavg1[ja+1]-SDavg1[ja-1]) / (((ja+1)*(timestep/1000))-((ja-1)*(timestep/1000))));      
      }
      

      
      
      free(Hexch_sized);
      //free(SDavg);
      //free(SD);
      free(Hexch_sized1);
      free(SDavg1);
      free(SD1);
      //avgcounter++;
    }
  
   // for(int ja =1; ja<SD_store-1;ja++)
   //   {
   //    FDsum[ja]/=avgcounter;
   //   }

    
   }
     //for(int ja =1; ja<SD_store;ja++)
     //{
      // SDsum[ja]/=SDsum[0];
      // SDsum1[ja]/=SDsum1[0];
     // } 
     // SDsum[0]=1.0;
     // SDsum1[0]=1.0;
      //Hexchplot.open("ct-pair.data");
      Hexchplotac.open("ct.data");
     for(int ja =0; ja<SD_store;ja++)
     {
       //SDsum[ja]/=float(num_bins*restarts);
       SDsum1[ja]/=float(num_bins*restarts);
       //Hexchplot <<ja*timestep/1000<<" "<<SDsum[ja]/SDsum[0]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<SDsum1[ja]/SDsum1[0]<<endl;
       //Hexchplot <<ja*timestep/1000<<" "<<SDsum[ja]<<endl;
      } 
      //Hexchplot.close();
      Hexchplotac.close();
      
      //averaging for a proper plot
    
      /*
      for(int ja =0; ja<SD_store;ja++)
     {
       SDsum[ja]-=SDsum[SD_store-1];
       SDsum1[ja]-=SDsum1[SD_store-1];
      }*/
      
/*   
     cout << "Gnuplot fitting begins"<< endl;
  FILE *pipe_gp = popen("gnuplot", "w");
   fputs("set fit logfile 'fit-pair.data'\n",pipe_gp);
   fputs("set fit quiet\n",pipe_gp);
  fputs("f(x)=A*exp(-x/i)+B*exp(-x/j)+C*exp(-x/k)\n", pipe_gp);
  fputs("A               = 0.339959\n", pipe_gp);
  fputs("i               = 2.14969\n", pipe_gp);
  fputs("B               = 0.303315\n", pipe_gp);
  fputs("j               = 0.0457113\n", pipe_gp);
  fputs("C               = 0.259671\n", pipe_gp);
  fputs("k               = 7.19704\n", pipe_gp);
  fputs("fit f(x) 'ct-pair.data' u 1:2 via A, i, B, j, C, k\n", pipe_gp);
  fputs("exit\n", pipe_gp);
  pclose(pipe_gp);
  
  FILE *pipe_gpa = popen("gnuplot", "w");
fputs("set fit logfile 'fit.data'\n",pipe_gpa);
fputs("set fit quiet\n",pipe_gpa);
  fputs("f(x)=A*exp(-x/i)+B*exp(-x/j)+C*exp(-x/k)\n", pipe_gpa);
  fputs("A               = 0.339959\n", pipe_gpa);
  fputs("i               = 2.14969\n", pipe_gpa);
  fputs("B               = 0.303315\n", pipe_gpa);
  fputs("j               = 0.0457113\n", pipe_gpa);
  fputs("C               = 0.259671\n", pipe_gpa);
  fputs("k               = 7.19704\n", pipe_gpa);
  fputs("fit f(x) 'ct.data' u 1:2 via A, i, B, j, C, k\n", pipe_gpa);
  fputs("exit\n", pipe_gpa);
  pclose(pipe_gpa);
  
  FILE *pipe_gp2 = popen("gnuplot", "w");
fputs("set fit logfile 'fit2.data'\n",pipe_gp2);
fputs("set fit quiet\n",pipe_gp2);
  fputs("f(x)=A*exp(-x/i)+B*exp(-x/j)\n", pipe_gp2);
  fputs("A               = 0.326531\n", pipe_gp2);
  fputs("i               = 0.514234\n", pipe_gp2);
  fputs("B               = 0.592641\n", pipe_gp2);
  fputs("j               = 4.81101\n", pipe_gp2); 
  fputs("fit f(x) 'ct.data' u 1:2 via A, i, B, j\n", pipe_gp2);
  fputs("exit\n", pipe_gp2);
  pclose(pipe_gp2);
     
  FILE *pipe_gp2a = popen("gnuplot", "w");
fputs("set fit logfile 'fit2-pair.data'\n",pipe_gp2a);
fputs("set fit quiet\n",pipe_gp2a);
  fputs("f(x)=A*exp(-x/i)+B*exp(-x/j)\n", pipe_gp2a);
  fputs("A               = 0.326531\n", pipe_gp2a);
  fputs("i               = 0.514234\n", pipe_gp2a);
  fputs("B               = 0.592641\n", pipe_gp2a);
  fputs("j               = 4.81101\n", pipe_gp2a); 
  fputs("fit f(x) 'ct-pair.data' u 1:2 via A, i, B, j\n", pipe_gp2a);
  fputs("exit\n", pipe_gp2a);
  pclose(pipe_gp2a);
  
cout<<"Gnuplot fits done"<<endl;  
*/
      //Hexchplot.open("ft-pair.data");
      Hexchplotac.open("ft.data");
     for(int ja =1; ja<SD_store-1;ja++)
      {
       //FDsum[ja]/=num_bins*restarts;
       FDsum1[ja]/=num_bins*restarts;
       //Hexchplot <<ja*timestep/1000<<" "<<FDsum[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<FDsum1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     
     int avg_every = 100;
     float *FD_avg1;
     //FD_avg=(float *) malloc (sizeof(float)*SD_store);
     FD_avg1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
          // FD_avg[ia]=0.0;
	   FD_avg1[ia]=0.0;
	}
     
      for(int ja=(avg_every/2)+1; ja<SD_store -(avg_every/2)-1;ja++)
      {
	for(int ka=ja-(avg_every/2);ka<=ja+(avg_every/2)-1;ka++)
	{
	 // FD_avg[ja]+= FDsum[ka];
	  FD_avg1[ja]+= FDsum1[ka];
	}
	//FD_avg[ja]/=avg_every;
	FD_avg1[ja]/=avg_every;
      }
      
      Hexchplotac.open("ft-avg.data");
      //Hexchplot.open("ft-avg-pair.data");
     for(int ja =(avg_every/2)+1; ja<SD_store -(avg_every/2)-1;ja++)
      {
       //Hexchplot <<ja*timestep/1000<<" "<<FD_avg[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<FD_avg1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     
      
      
      
     float lifetime1;
     float *int_SDsum1;
     float *cum_SDsum1;
     //int_SDsum = (float*) malloc (sizeof(float)*SD_store);
     //cum_SDsum = (float*) malloc (sizeof(float)*SD_store);
     int_SDsum1 = (float*) malloc (sizeof(float)*SD_store);
     cum_SDsum1 = (float*) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
           //int_SDsum[ia]=0.0;
	   //cum_SDsum[ia]=0.0;
	   int_SDsum1[ia]=0.0;
	   cum_SDsum1[ia]=0.0;
	}
     for(int ja =1; ja<SD_store;ja++)
      {
       //int_SDsum[ja]= (((ja*timestep*0.001) - ((ja-1)*timestep*0.001)) * ((SDsum[ja] + SDsum[ja-1])/2.0));
       //cum_SDsum[ja]=cum_SDsum[ja-1]+int_SDsum[ja];
       int_SDsum1[ja]= (((ja*timestep*0.001) - ((ja-1)*timestep*0.001)) * ((SDsum1[ja] + SDsum1[ja-1])/2.0));
       cum_SDsum1[ja]=cum_SDsum1[ja-1]+int_SDsum1[ja];
       if(ja == SD_store-1) {//lifetime = cum_SDsum[ja];
	 lifetime1 = cum_SDsum1[ja];}
      }  
     
     //Hexchplot.open("ct-integrate-pair.data");
     Hexchplotac.open("ct-integrate.data");
     for(int ja =0; ja<SD_store;ja++)
      {
       //Hexchplot <<ja*timestep/1000<<" "<<cum_SDsum[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<cum_SDsum1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     //Hexchplot.open("Hbond-lifetime-pair.data");
     //Hexchplot <<"Lifetime from the integral of c(t) is : "<<lifetime<<" ps"<<endl;
     //Hexchplot.close();
     Hexchplot.open("Hbond-lifetime.data");
     Hexchplot <<"Lifetime from the integral of c(t) is : "<<lifetime1<<" ps"<<endl;
     Hexchplot.close();
     
 
 
 
    float *Hbondnum;
    
    Hbondnum =(float*) malloc (sizeof(float)*(nstruct/skip));
    for(int i=0; i<(nstruct/skip); i++)
    {
      Hbondnum[i]=0;
    }
    for(int i=0; i<(nstruct/skip); i++)
    {
      for(int j=0; j<nElt2; j++)//acceptor
      {
	for(int k=0; k<nElt1; k++)//shared hydrogen
	{
	  Hbondnum[i]+=Hexch1[k+j*nElt1+i*nElt2*nElt1];
	}
      }
      //cout << Hbondnum[i]<<endl;
      //Hbondnum[i]/=(0.5*nElt2*(nElt2-1));
    }
    float avg_Hbondnum=0.0;
    
        Hexchplot.open("Hbondnum.data");

         for(int i=0;i<(nstruct/skip);i++)
         {
	   avg_Hbondnum+=Hbondnum[i];
	   Hexchplot <<i<<" "<<Hbondnum[i]<<endl;
	 }
	 avg_Hbondnum /= (nstruct/skip);
        Hexchplot.close();
	Hexchplot.open("Avg_Hbondnum.data");
	Hexchplot <<"Average number of Hbonds per structure = "<<avg_Hbondnum<<endl;
        Hexchplot.close();
 
  free(Hexch);
  free(Hexch1);
  free(Hbondnum);
  free(initHBcol1);
}  
  free(A);
  free(B);
  free(Onum);
  free(Hbondneigh);
  free(Hbondneighdist);
 
  /////////////////Histogramming part - trivial/////////////////////////////////////////////////////////
  
cout<<"Hbonds and angles: " << nElt1 <<" "<< Elt1 << " atoms are there in each structure"<<endl;

int Dirn,split;
float min,max;

if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
  Dirn = 2;
  min=minz;
  max=maxz;
  split=zsplit;
  
}

else if(set_max_y == "yes")
{
  minz = set_miny;
  maxz = set_maxy;
  Dirn=1;
  min=miny;
  max=maxy;
  split=ysplit;
}
else if(set_max_x == "yes")
{
  minz = set_minx;
  maxz = set_maxx;
  Dirn=0;
  min=minx;
  max=maxx;
  split=xsplit;
}

printf("%f %f\n",min,max);

    zrange=max-min;

    zint = zrange/split;

    ztick=(float*) malloc (sizeof(float)*split*2);
    
    for(int i=0;i<split;i++)
    {
      ztick[i*2]=min+(i*zint);
      ztick[i*2+1]=min+((i+1)*zint);
    }
    

FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<split;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
fclose(xyztick);

   float *Hbond_density, *dev_Hbond_density;
   float *angle_density, *dev_angle_density;
   float *distance_density, *dev_distance_density;
   float *OH_density, *dev_OH_density;
   float *O_H_density, *dev_O_H_density;
   float *ang_density, *dev_ang_density;

    Hbond_density=(float *) malloc (sizeof(float)*split);
    angle_density=(float *) malloc (sizeof(float)*split);
    distance_density=(float *) malloc (sizeof(float)*split);
    OH_density=(float *) malloc (sizeof(float)*split);
    O_H_density=(float *) malloc (sizeof(float)*split);
    ang_density=(float *) malloc (sizeof(float)*split);

        for(int k=0;k<split;k++)
        {
          Hbond_density[k]=0.0;
          angle_density[k]=0.0;
          distance_density[k]=0.0;
	  OH_density[k]=0.0;
          ang_density[k]=0.0;
          O_H_density[k]=0.0;
        }


    printf("Start of cuda calculation\n");
    
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*9);
    hipMalloc((void **)&dev_Hbond_density,sizeof(float)*split);
    hipMalloc((void **)&dev_angle_density,sizeof(float)*split);
    hipMalloc((void **)&dev_distance_density,sizeof(float)*split);
    hipMalloc((void **)&dev_OH_density,sizeof(float)*split);
    hipMalloc((void **)&dev_O_H_density,sizeof(float)*split);
    hipMalloc((void **)&dev_ang_density,sizeof(float)*split);
    hipMalloc((void **)&dev_ztick, sizeof(float)*split*2);
    
    hipMemcpy(dev_A,Hbondxyz,sizeof(float)*(nstruct/skip)*nElt1*9,hipMemcpyHostToDevice);
    hipMemcpy(dev_Hbond_density,Hbond_density,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_angle_density,angle_density,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_distance_density,distance_density,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_OH_density,OH_density,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_ang_density,ang_density,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_O_H_density,O_H_density,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*split*2,hipMemcpyHostToDevice);
    
    cout << "Memory for storing structure data: "<<((nstruct/skip)*nElt1*9)*4/float(1000000000)<< " Gbs" << endl;
    cout << "Memory for local storage: "<<((split*6)+(split*2))*4/float(1000000000)<< " Gbs" << endl;   
    
    dim3 dimBlockz(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGridz(((nstruct/skip)+dimBlockz.x-1)/dimBlockz.x,(nElt1+dimBlockz.y-1)/dimBlockz.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    Hbond_calc<<<dimGridz,dimBlockz>>>(dev_A,dev_Hbond_density,dev_angle_density,dev_distance_density,dev_OH_density,dev_O_H_density,dev_ang_density,(nstruct/skip),nElt1,split,dev_ztick,Hbond_angle_dev,max_O_O,Dirn);
    hipMemcpy(Hbond_density,dev_Hbond_density,sizeof(float)*split,hipMemcpyDeviceToHost);
    hipMemcpy(angle_density,dev_angle_density,sizeof(float)*split,hipMemcpyDeviceToHost);
    hipMemcpy(distance_density,dev_distance_density,sizeof(float)*split,hipMemcpyDeviceToHost);
    hipMemcpy(OH_density,dev_OH_density,sizeof(float)*split,hipMemcpyDeviceToHost);
    hipMemcpy(O_H_density,dev_O_H_density,sizeof(float)*split,hipMemcpyDeviceToHost);
    hipMemcpy(ang_density,dev_ang_density,sizeof(float)*split,hipMemcpyDeviceToHost);
  
    float cellvolume;
    
    if(cell_type=="orthorhombic")
    {
    if(set_max_z == "yes") cellvolume = xvec*yvec*zint;
      if(set_max_x == "yes") cellvolume = yvec*zvec*zint;
      if(set_max_y == "yes") cellvolume = xvec*zvec*zint;
    }
    else if(cell_type=="monoclinic")
    {
    if(set_max_z == "yes") cellvolume= xvec*xvec*zint*0.866025403; //*sin(60) for rhombohedral 
    if(set_max_x == "yes") cellvolume= yvec*zvec*zint*0.866025403;
    if(set_max_y == "yes") cellvolume= xvec*zvec*zint*0.866025403;
      
    }
    
FILE *Zdistfile=fopen("Hbonddist.data","wt");

int zstart = 0;
int zend = split;

for(int m=zstart;m<zend;m++)
{
fprintf(Zdistfile,"%f %f %f %f %f %f %f \n ",ztick[m*2+1], Hbond_density[m]/(cellvolume*(nstruct/skip)),
        (angle_density[m]/Hbond_density[m])*(180.0/3.14159),distance_density[m]/Hbond_density[m], OH_density[m]/Hbond_density[m], O_H_density[m]/Hbond_density[m], (ang_density[m]/Hbond_density[m])*(180.0/3.14159));
}
fclose(Zdistfile);

hipFree(dev_Hbond_density);
hipFree(dev_A);
hipFree(dev_ztick);
hipFree(dev_distance_density);
hipFree(dev_angle_density);
hipFree(dev_OH_density);
hipFree(dev_ang_density);
hipFree(dev_O_H_density);

free(Hbondxyz);
free(Hbond_density);
free(angle_density);
free(distance_density);
free(OH_density);
free(ang_density);
free(O_H_density);
free(ztick);
  
  
  
  
}
