#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"
//float minx,maxx,minz,maxz,miny,maxy;
float xrange,yrange,zrange;
float xint,yint,zint;
float *xtick,*ytick,*ztick,*dev_xtick,*dev_ytick,*dev_ztick;
int *density, *dev_density;
void compute_densitygrid()
{
  
  
  ////////////////////Initial setup////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  
    if(plot=="int")        { nElt1 = nA_int;}
    else if(plot=="bulk")  { nElt1 = nA_bulk;}
    
cout << type <<" is chosen"<<endl;
cout<<"densitygrid: " << nElt1 <<" "<< Elt1 << " atoms are there in each structure"<<endl;
    printf("%f %f %f %f %f %f\n",minx,maxx,miny,maxy,minz,maxz);
    
    if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
}
    if(set_max_x == "yes")
{
  minx = set_minx;
  maxx = set_maxx;
}
    if(set_max_y == "yes")
{
  miny = set_miny;
  maxy = set_maxy;
}
    xrange=maxx-minx;
    yrange=maxy-miny;
    zrange=maxz-minz;

    xint = xrange/xsplit;
    yint = yrange/ysplit;
    zint = zrange/zsplit;

    xtick=(float*) malloc (sizeof(float)*xsplit*2);
    ytick=(float*) malloc (sizeof(float)*ysplit*2);
    ztick=(float*) malloc (sizeof(float)*zsplit*2);
    for(int i=0;i<xsplit;i++)
    {
      xtick[i*2]=minx+(i*xint);
      xtick[i*2+1]=minx+((i+1)*xint);
    }
    for(int i=0;i<ysplit;i++)
    {
      ytick[i*2]=miny+(i*yint);
      ytick[i*2+1]=miny+((i+1)*yint);
    }
    for(int i=0;i<zsplit;i++)
    {
      ztick[i*2]=minz+(i*zint);
      ztick[i*2+1]=minz+((i+1)*zint);
    }
    
    ofstream plotgnu;
    ofstream plotgnu1;
    plotgnu.open("plot.gnu");
    plotgnu1.open("plot1.gnu");
    
  if(type=="density") 
  {
    
    
     plotgnu1.precision(4);    
     plotgnu1<<"set view map"<<endl\
     <<"r(x) = (x==0 ? 1 : abs(2*x - 0.5)) "<<endl\
     <<"g(x) = (x==0 ? 1 : sin(pi*x))"<<endl\
     <<"b(x) = (x==0 ? 1 : cos( pi*x/2))"<<endl\
     <<"set palette functions r(gray),g(gray),b(gray)"<<endl\
     //<<"set palette defined (0  0.0 0.0 0.5, 1  0.0 0.0 1.0, \
                     2  0.0 0.5 1.0, \
                     3  0.0 1.0 1.0, \
                     4  0.5 1.0 0.5, \
                     5  1.0 1.0 0.0, \
                     6  1.0 0.5 0.0, \
                     7  1.0 0.0 0.0, \
                     8  0.5 0.0 0.0 )"<<endl\
                     
     <<"set pm3d interpolate 20,20"<<endl\
     <<"unset key"<<endl;
     
    plotgnu.precision(4);
    plotgnu<<"unset key"<<endl;
  }
  if(type=="density-top") 
  {
    if(order=="YZX") {
      ztick[0]=mintop;
      ztick[1]=maxtop;      
    }
     if(order=="XYZ") {
      ytick[0]=mintop;
      ytick[1]=maxtop;      
    }
    if(order=="ZXY") {
      xtick[0]=mintop;
      xtick[1]=maxtop;      
    }
    if(choose_atoms != "yes" &&  choose_atoms_from != "yes")
    {
      maxbondist=maxtop;
      minbondist=mintop;
    }
    plotgnu.precision(4);    
    
    plotgnu<<"set view map"<<endl\

     
     <<"r(x) = ( abs(2*x - 0.5)) "<<endl\
     <<"g(x) = ( sin(pi*x))"<<endl\
     <<"b(x) = ( cos( pi*x/2))"<<endl\
     <<"set palette functions r(gray),g(gray),b(gray)"<<endl\

<<"set pm3d interpolate 20,20"<<endl\
//<<"set size ratio @@ratio@@"<<endl\

<<"unset key"<<endl\
<<"#set contour base"<<endl\
<<"#set cntrparam bspline"<<endl\
<<"#set cntrparam levels 5"<<endl;
  }

  FILE *xyztick = fopen("xyztick.dat","wt");
  fprintf(xyztick,"Xsplit");
    for(int i=0;i<xsplit;i++)
    {
      fprintf(xyztick,"%d %f %f \n",i,xtick[i*2],xtick[i*2+1]);
    }
      fprintf(xyztick," \n");
      fprintf(xyztick,"Ysplit");
    for(int i=0;i<ysplit;i++)
    {
      fprintf(xyztick,"%d %f %f \n",i,ytick[i*2],ytick[i*2+1]);    
    }
      fprintf(xyztick," \n");
      fprintf(xyztick,"Zsplit");
    for(int i=0;i<zsplit;i++)
    {
      fprintf(xyztick,"%d %f %f \n",i,ztick[i*2],ztick[i*2+1]);
    }
    fprintf(xyztick," \n");
    int x1= xsplit ;
    int x2= (xsplit+(xsplit*2))/2;
    int x3= (xsplit*2);
    int y1=ysplit;
    int y2=(ysplit*2);
    if(order=="YZX")  {fprintf(xyztick,"%f %f %f %f %f\n",xtick[0],xtick[(xsplit-1)*2+1],ytick[0],ytick[(ysplit-1)*2+1],xtick[(xsplit-1)*2+1]/ytick[(ysplit-1)*2+1]); 
                       plotgnu<<"set size ratio "<<xtick[(xsplit-1)*2+1]/ytick[(ysplit-1)*2+1]<<endl;
		       plotgnu1<<"set size ratio "<<xtick[(xsplit-1)*2+1]/ytick[(ysplit-1)*2+1]<<endl;
                       if(type=="density-top")
                       {
                plotgnu<<"set xrange ["<< xsplit-(xsplit/10)<<":"<< xsplit*2+(xsplit/10)<<"]"<<endl \
                       <<"set yrange ["<< ysplit-(ysplit/10)<<":"<< ysplit*2+(ysplit/10)<<"]"<<endl \
                       <<"set encoding iso_8859_1"<<endl \
		       <<"set xlabel \"Y-direction [\305]\" font 'Verdana,22' offset 0,-1" <<endl \
                       <<"set ylabel \"X-direction [\305]\" font 'Verdana,22' offset -2,0" <<endl \
                       <<"set xtic  font 'Verdana,22' " <<endl \
                       <<"set ytic  font 'Verdana,22' " <<endl \
                        <<"set cbtic  font 'Verdana,22' " <<endl \
                       <<"set cblabel \"Atom density\" font 'Verdana,22' offset 2,0" <<endl \
                       <<"set format x \"%.0f\";set ytics (\"0\" "<< x1<<", \""<<xtick[(xsplit-1)*2+1]/2<<"\" "<<x2<<", \""<< xtick[(xsplit-1)*2+1]<<"\" "<< x3<<" )"<<endl\
                       <<"set format y \"%.0f\";set xtics (\"0\" "<< y1<<", \""<< ytick[(ysplit-1)*2+1]<<"\" "<< y2<<" )"<<endl;}
                       
                       if(type=="density")
                       {
		plotgnu<<"set xrange [ 0 :"<< xtick[(xsplit*2)-1] <<"]"<<endl;
                plotgnu<<"set yrange [ 0 :"<< ytick[(ysplit*2)-1] <<"]"<<endl; 
                plotgnu<<"set xlabel \"Z-direction\" font 'Verdana,22' offset 0,-1" <<endl\
                       <<"set ylabel \"X-direction\" font 'Verdana,22' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,20' " <<endl\
                       <<"set ytic  font 'Verdana,20' " <<endl\
                       <<"set xtics 5 "<<endl\
                       <<"set ytics 5"<<endl;
	       plotgnu1<<"set xrange ["<< xsplit-(xsplit/10)<<":"<< xsplit*2+(xsplit/10)<<"]"<<endl\
		       <<"set yrange ["<< ysplit<<":"<< ysplit*2+(ysplit/10)<<"]"<<endl\
                       <<"set xlabel \"Z-direction\" font 'Verdana,22' offset 0,-1" <<endl\
                       <<"set ylabel \"X-direction\" font 'Verdana,22' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,22' " <<endl\
                       <<"set ytic  font 'Verdana,22' " <<endl\
                       <<"set cbtic  font 'Verdana,22' " <<endl\
                       <<"set cblabel \"Atom density\" font 'Verdana,22' offset 2,0" <<endl\
                       <<"set format x \"%.0f\";set ytics (\"0\" "<< x1<<", \""<<xtick[(xsplit-1)*2+1]/2<<"\" "<<x2<<", \""<< xtick[(xsplit-1)*2+1]<<"\" "<< x3<<" )"<<endl\
                       <<"set format y \"%.0f\";set xtics (\"0\" "<< y1<<",  \""<< ytick[(ysplit-1)*2+1]<<"\" "<< y2<<" )"<<endl\
		       <<"unset cblabel; unset colorbox"<<endl;
		      }
                      }
                       
    else if(order=="XYZ") { fprintf(xyztick,"%f %f \n",ztick[0],ztick[(zsplit-1)*2+1],xtick[0],xtick[(xsplit-1)*2+1],ztick[(zsplit-1)*2+1]/xtick[(xsplit-1)*2+1]);
                       plotgnu<<"set size ratio "<<ztick[(zsplit-1)*2+1]/xtick[(xsplit-1)*2+1]<<endl;
		       plotgnu1<<"set size ratio "<<ztick[(zsplit-1)*2+1]/xtick[(xsplit-1)*2+1]<<endl;
                       if(type=="density-top")
                       {
                plotgnu<<"set xrange ["<< xsplit-(xsplit/10)<<":"<< xsplit*2+(xsplit/10)<<"]"<<endl\
                       <<"set yrange ["<< zsplit-(zsplit/10)<<":"<< zsplit*2+(zsplit/10)<<"]"<<endl\
                       <<"set encoding iso_8859_1"<<endl \
		       <<"set xlabel \"Y-direction [\305]\" font 'Verdana,22' offset 0,-1" <<endl \
                       <<"set ylabel \"X-direction [\305]\" font 'Verdana,22' offset -2,0" <<endl \
                       <<"set xtic  font 'Verdana,22' " <<endl\
                       <<"set ytic  font 'Verdana,22' " <<endl\
                        <<"set cbtic  font 'Verdana,22' " <<endl\
                       <<"set cblabel \"Atom density\" font 'Verdana,22' offset 2,0" <<endl\
                       <<"set format x \"%.0f\";set ytics (\"0\" "<< zsplit<<", \""<<ztick[(zsplit-1)*2+1]/2<<"\" "<<(zsplit+(zsplit*2))/2<<", \""<< ztick[(zsplit-1)*2+1]<<"\" "<< (zsplit*2)<<" )"<<endl\
                       <<"set format y \"%.0f\";set xtics (\"0\" "<< x1<<",  \""<< xtick[(xsplit-1)*2+1]<<"\" "<< (xsplit*2)<<" )"<<endl;
			 
		      }
                       
                       
                       if(type=="density")
                       {
		plotgnu<<"set xrange [0:"<< xtick[(xsplit*2)-1] <<"]"<<endl\
                       <<"set yrange [0:"<< ztick[(zsplit*2)-1] <<"]"<<endl\
                       <<"set xlabel \"X-direction\" font 'Verdana,22' offset 0,-1" <<endl\
                       <<"set ylabel \"Z-direction\" font 'Verdana,22' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,20' " <<endl\
                       <<"set ytic  font 'Verdana,20' " <<endl\
                       <<"set xtics 5 "<<endl\
                       <<"set ytics 5"<<endl;
		plotgnu1<<"set xrange ["<< xsplit-(xsplit/10)<<":"<< xsplit*2+(xsplit/10)<<"]"<<endl\
		<<"set yrange ["<< zsplit<<":"<< zsplit*2+(zsplit/10)<<"]"<<endl\
                       <<"set xlabel \"X-direction\" font 'Verdana,22' offset 0,-1" <<endl\
                       <<"set ylabel \"Z-direction\" font 'Verdana,22' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,20'" <<endl\
                       <<"set ytic  font 'Verdana,20' " <<endl\
                        <<"set cbtic  font 'Verdana,20' " <<endl\
                       <<"set cblabel \"Atom density\" font 'Verdana,20' offset 2,0" <<endl\
                       <<"set format x \"%.0f\";set ytics (\"0\" "<< zsplit<<", \""<<ztick[(zsplit-1)*2+1]/2<<"\" "<<(zsplit+(zsplit*2))/2<<", \""<< ztick[(zsplit-1)*2+1]<<"\" "<< (zsplit*2)<<" )"<<endl\
                       <<"set format y \"%.0f\";set xtics (\"0\" "<< x1<<",  \""<< xtick[(xsplit-1)*2+1]<<"\" "<< (xsplit*2)<<" )"<<endl\
		       <<"unset cblabel; unset colorbox"<<endl; 
		      }
                       }
    
    else if(order=="ZXY") { fprintf(xyztick,"%f %f \n",ytick[0],ytick[(ysplit-1)*2+1],ztick[0],ztick[(zsplit-1)*2+1],ytick[(ysplit-1)*2+1]/ztick[(zsplit-1)*2+1]);
                       plotgnu<<"set size ratio "<<ytick[(ysplit-1)*2+1]/ztick[(zsplit-1)*2+1]<<endl;
		       plotgnu1<<"set size ratio "<<ytick[(ysplit-1)*2+1]/ztick[(zsplit-1)*2+1]<<endl;
                       if(type=="density-top")
                       {
                       plotgnu<<"set xrange ["<< zsplit-(zsplit/10)<<":"<< zsplit*2+(zsplit/10)<<"]"<<endl\
                       <<"set yrange ["<< ysplit-(ysplit/10)<<":"<< ysplit*2+(ysplit/10)<<"]"<<endl\
                       <<"set encoding iso_8859_1"<<endl \
		       <<"set xlabel \"Y-direction [\305]\" font 'Verdana,22' offset 0,-1" <<endl \
                       <<"set ylabel \"X-direction [\305]\" font 'Verdana,22' offset -2,0" <<endl \
                       <<"set xtic  font 'Verdana,22' " <<endl\
                       <<"set ytic  font 'Verdana,22' " <<endl\
                       <<"set cbtic  font 'Verdana,22' " <<endl\
                       <<"set cblabel \"Atom density\" font 'Verdana,22' offset 2,0" <<endl\
                       <<"set format x \"%.0f\";set ytics (\"0\" "<< ysplit<<", \""<<ytick[(ysplit-1)*2+1]/2<<"\" "<<(ysplit+(ysplit*2))/2<<", \""<< ytick[(ysplit-1)*2+1]<<"\" "<< (ysplit*2)<<" )"<<endl\
                       <<"set format y \"%.0f\";set xtics (\"0\" "<< zsplit<<",  \""<< ztick[(zsplit-1)*2+1]<<"\" "<< (zsplit*2)<<" )"<<endl;}
                       
                       if(type=="density")
                       {
	        plotgnu<<"set xrange [0:"<< ztick[(zsplit*2)-1] <<"]"<<endl\
                       <<"set yrange [0:"<< ytick[(ysplit*2)-1] <<"]"<<endl\
                       <<"set xlabel \"Y-direction\" font 'Verdana,22' offset 0,-1" <<endl\
                       <<"set ylabel \"Z-direction\" font 'Verdana,22' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,20' " <<endl\
                       <<"set ytic  font 'Verdana,20' " <<endl\
                       <<"set xtics 5 "<<endl\
                       <<"set ytics 5"<<endl;
		plotgnu1<<"set xrange ["<< zsplit-(zsplit/10)<<":"<< zsplit*2+(zsplit/10)<<"]"<<endl\
		<<"set yrange ["<< ysplit<<":"<< ysplit*2+(ysplit/10)<<"]"<<endl\
                       <<"set xlabel \"Y-direction\" font 'Verdana,22' offset 0,-1" <<endl\
                       <<"set ylabel \"Z-direction\" font 'Verdana,22' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,22' " <<endl\
                       <<"set ytic  font 'Verdana,22' " <<endl\
                       <<"set cbtic  font 'Verdana,22' " <<endl\
                       <<"set cblabel \"Atom density\" font 'Verdana,22' offset 2,0" <<endl\
                       <<"set format x \"%.0f\";set ytics (\"0\" "<< ysplit<<", \""<<ytick[(ysplit-1)*2+1]/2<<"\" "<<(ysplit+(ysplit*2))/2<<", \""<< ytick[(ysplit-1)*2+1]<<"\" "<< (ysplit*2)<<" )"<<endl\
                       <<"set format y \"%.0f\";set xtics (\"0\" "<< zsplit<<",  \""<< ztick[(zsplit-1)*2+1]<<"\" "<< (zsplit*2)<<" )"<<endl\
		       <<"unset cblabel; unset colorbox"<<endl;
		      }
                       }
fclose(xyztick);
plotgnu<<"set terminal postscript solid eps enhanced color font 'Verdana,18'"<<endl\
       <<"set output 'Plot.eps'"<<endl;

       if(type=="density-top")
       {
       plotgnu<<"sp 'extended.out' i 0 matrix w pm3d , 'teter.dat' u ($1)+49.5:($2)+49.5:(1) w p pt 65 ps 2 lc rgb \"black\" lw 3"<<endl;
       }
       else if(type=="density")
	 {
	  
	  plotgnu1<<"set terminal postscript solid eps enhanced color font 'Verdana,18'"<<endl\
          <<"set output 'Plot1.eps'"<<endl;
	  plotgnu1<<"sp 'extended.out' i 0 matrix w pm3d"<<endl;
	 }
plotgnu.close();
plotgnu1.close();
int *densityf, *dev_densityf;
    density=(int *) malloc (sizeof(int)*xsplit*ysplit*zsplit);
    densityf=(int *) malloc (sizeof(int)*xsplit*ysplit*zsplit);
    for(int i=0;i<xsplit;i++)
    {
      for(int j=0;j<ysplit;j++)
      {
        for(int k=0;k<zsplit;k++)
        {
          density[k+j*zsplit+i*ysplit*zsplit]=0;
	  densityf[k+j*zsplit+i*ysplit*zsplit]=0;
        }
      }
    }
///*

///////////////////////////////////////////Initial setup done/////////////////////////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////Special atom selection between 2 distances/////////////////////////////////////////////////////////////////////////
float surf,avgsurf=0.0;
int dirn;
  if(choose_dirn=="z") dirn=2;
  else if(choose_dirn=="x") dirn=0;
  else if(choose_dirn=="y") dirn=1;
  
if(choose_atoms=="yes")
{
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< metal_species<<endl;
  
 
  
  int surfatom;
  for(int i=0 ; i<(nstruct/skip); i++)
  {surfatom=0;surf=0.0;
  for(int j=0; j<count_metal;j++)
  {
    if(METAL[j*3+i*count_metal*3+dirn] > mintop && METAL[j*3+i*count_metal*3+dirn] < maxtop)
    {
    surf+=METAL[j*3+i*count_metal*3+dirn];
    surfatom++;
    }
  }
  surf/=surfatom;
  avgsurf+=surf;
  }
  
  avgsurf/=(nstruct/skip);
  
 cout<<"Average surface atoms position in "<<choose_dirn<<" is "<<avgsurf<<endl;
 cout<<"Average number of surface atoms is "<< surfatom<<endl;
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
      if(strict_dirn == "yes") {covermat<<<dimGride,dimBlocke>>>(dev_A,dev_exch,minbondist,maxbondist,nElt1,(nstruct/skip),dirn,avgsurf,mintop,maxtop);}
      else {covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);}
    }
    else if (cell_type == "monoclinic")
    {
      if(strict_dirn == "yes") {covermat<<<dimGride,dimBlocke>>>(dev_A,dev_exch,minbondist,maxbondist,nElt1,(nstruct/skip),dirn,avgsurf,mintop,maxtop);} //no lattice needed thus use covermat - no problem
      else {covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);}
    }
    
    
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    
    //ofstream exchout;
    //exchout.open("exchout.data");
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
	//exchout<<l<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        //exchout<<endl;
      }
      
      //exchout.close();
     hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 
     
     ofstream coverage;
     coverage.open("coverage.data");
     for(int l=0; l<nstruct/skip; l++)
      {
	coverage <<l<<" "<< float(Aintnum[l]) <<" "<<float(Aintnum[l])/float(surfatom)<<endl;
      }
     coverage.close();
     
}          
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////Special atom selection between 2 distances from a particlar index/////////////////////////////////////////////////////////////////////////

if(choose_atoms_from=="yes")
{
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< check_index<<endl;
  
  float *index_xyz;
  index_xyz = (float *) malloc (sizeof(float)*(nstruct/skip)*3);
  for(int i=0 ; i<(nstruct/skip); i++)
  {
  for(int j=0 ; j<3; j++)
  {
    index_xyz[i*3+j]=0.0;
  }
  }
  
  //careful!! directly we choose metal array!! not good!! but works for now!!
  
  for(int i=0 ; i<(nstruct/skip); i++)
  {
  for(int j=0 ; j<3; j++)
  {
    index_xyz[i*3+j]=METAL[check_index*3+i*count_metal*3+j];
  }
  }
 
  
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,index_xyz,sizeof(float)*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration

    
    dim3 dimBlocke(32,1,32);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,1,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
      covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,check_index,dev_lattice,nElt1,(nstruct/skip));
    }
    else if (cell_type == "monoclinic")
    {
      covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,check_index,dev_lattice,nElt1,(nstruct/skip));
    }
    
    
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    
    ofstream exchout;
    exchout.open("exchout.data");
    
    
    for(int l=0; l<(nstruct/skip); l++)
      {
	//exchout<<l<<" "<<index_xyz[l*3]<<" "<<index_xyz[l*3+1]<<" "<<index_xyz[l*3+2]<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        exchout<<l<<" "<<Aintnum[l]<<" "<<Abulknum[l]<<endl;
      }
      
      exchout.close();
      
     hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 

     
}          
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////Density calculation starts/////////////////////////////////////////////////////////////////////////////////////////////

if(density_plot=="yes")
{

    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_density,sizeof(int)*xsplit*ysplit*zsplit);
    hipMalloc((void **)&dev_xtick, sizeof(float)*xsplit*2);
    hipMalloc((void **)&dev_ytick, sizeof(float)*ysplit*2);
    hipMalloc((void **)&dev_ztick, sizeof(float)*zsplit*2);
    if(plot=="int")
    {
      hipMemcpy(dev_A,A_int,sizeof(float)*(nstruct/skip)*nA_int*3,hipMemcpyHostToDevice);
    }
    else if(plot=="bulk")
    {
      hipMemcpy(dev_A,A_bulk,sizeof(float)*(nstruct/skip)*nA_bulk*3,hipMemcpyHostToDevice);
    }
    else if(plot=="all")
    {
      hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    }
    hipMemcpy(dev_density,density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_xtick,xtick,sizeof(float)*xsplit*2,hipMemcpyHostToDevice);
    hipMemcpy(dev_ytick,ytick,sizeof(float)*ysplit*2,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*zsplit*2,hipMemcpyHostToDevice);
    
    printf("%ld B needed\n",(((nstruct/skip)*nElt1*3+xsplit*ysplit*zsplit+xsplit*2+ysplit*2+zsplit*2)*4));
    dim3 dimBlock(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGrid(((nstruct/skip)+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    if(choose_atoms=="yes")
    {
    density_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,dev_exch,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);
    hipFree(dev_exch);
    }
    else if(choose_atoms_from=="yes")
    {
    density_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,dev_exch,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);
    hipFree(dev_exch);
    }
    else
    {
      cout<<"I am computing"<<endl;
    density_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);      
    }
    //hipMemcpy(density,dev_density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
     hipFree(dev_A);
}
/////////////////////////////////////////////////////Density calculation done//////////////////////////////////////////////////////////////////////////////////






////////////////////////////////////////////////////Dipole orientation computation/////////////////////////////////////////////////////////////////////////////     
    
   if(dipole_orient=="yes" && type=="density-top")
   {
    
    int *h2onum;
  int *h2oneigh;
  float *h2oxyz;
  float *h2oneighdist;
  float *h2odistmat, *dev_h2odistmat;
  
  int OHsamples=20;
  
    h2onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    h2oneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    h2oneighdist = (float *) malloc (sizeof(float)*OHsamples*nElt1*(nstruct/skip));
    h2oxyz = (float *) malloc (sizeof(float)*15*nElt1*(nstruct/skip));  //Oxyz,H1xyz,H2xyz,Hmidpoint,dipole alignment with surface normal
    h2odistmat = (float *) malloc (sizeof(float)*nElt1*nElt2);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      h2onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      h2oneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      h2oneighdist[j*OHsamples+i*nElt1*OHsamples+k]=0.0;
      }
      for(int k =0 ;k<15;k++)
      {
      h2oxyz[j*15+i*nElt1*15+k]=0.0;
      }
    }
  }   
      
      
cout <<"gpu begins OH distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_h2odistmat,sizeof(float)*nElt1*nElt2);
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
//dim3 struct to define elements of the execution configuration

    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,1);
    
     
    //ofstream neigh;
   // neigh.open("H2O.dat");
   
    
for(int i = 0; i<(nstruct/skip); i++)
{
  
  //neigh << i<<endl<<endl;
  
   for(int i1=0;i1<nElt1;i1++)
  {
    for(int j1=0; j1<nElt2; j1++)
    {
      h2odistmat[j1+i1*nElt2]=0.0;
    }
  }
  
  hipMemcpy(dev_h2odistmat,h2odistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOH,nElt1,nElt2,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOH,nElt1,nElt2,dev_lattice,i);      
    }

hipMemcpy(h2odistmat,dev_h2odistmat,sizeof(float)*nElt1*nElt2,hipMemcpyDeviceToHost);
   
  
  for(int i1=0;i1<nElt1;i1++) //O
  {
    int k=0;
    //neigh << i1<<" ";
    for(int j1=0; j1<nElt2; j1++) //H
    {
      //neigh <<j1<<" "<<h2odistmat[j1+i1*nElt2]<<" ";
      if(h2odistmat[j1+i1*nElt2] !=0.0)
      {
	h2onum[i1+i*nElt1]++;
	h2oneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	h2oneighdist[i1*OHsamples+i*nElt1*OHsamples+k] = h2odistmat[j1+i1*nElt2];
	k++;
      }
    }
    //neigh<<endl;
  }

}   
 
 //neigh.close();
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_h2odistmat);
    hipFree(dev_lattice);
    

cout <<"gpu ends"<<endl;
   
cout<<" done !!!"<<endl;    
  
    //sorting needs to be done!!!!

cout<<"Swapping of bonds based on distance "<<endl;

float swapa,swapb;
    for(int i=0;i<(nstruct/skip);i++)
     {
      for(int j=0; j<nElt1; j++)
       {
	 
	for(int swf=0; swf<h2onum[j+i*nElt1]-1;swf++)
	{
	for(int sws=swf+1; sws<h2onum[j+i*nElt1];sws++)
	{
	if(h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf] > h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws] && sws !=swf && h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws] !=0.0 && h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf] !=0.0) 
	{
	   swapa = h2oneigh[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    h2oneigh[j*OHsamples+i*nElt1*OHsamples+swf]=h2oneigh[j*OHsamples+i*nElt1*OHsamples+sws];
	    h2oneigh[j*OHsamples+i*nElt1*OHsamples+sws]=swapa;
	   swapb = h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf]=h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws];
	    h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws]=swapb;
	}
	}
	}	
       }
     }	
   
cout<<" done !!!"<<endl;
//H2onum has number of H around O within 1.2 AA
//swap H1 short OH and H2, long

 //cout<<"Write Hxyz.dat"<<endl;

  //ofstream watercount;
  //watercount.open("Hxyz.dat");
  
  
  float *angdist,*angdistH1,*angdistH2;
  
  angdist = (float *) malloc (sizeof(float)*180);
  angdistH1 = (float *) malloc (sizeof(float)*180);
  angdistH2 = (float *) malloc (sizeof(float)*180);
  
  for(int i=0; i<=180;i++)
  {
    angdist[i]=0.0;
    angdistH1[i]=0.0;
    angdistH2[i]=0.0;
  }
  
  float val;
  
  
  ofstream OHzdist;
  OHzdist.open("OHzdist.dat");
  ofstream dipOHdist;
  dipOHdist.open("dipOHdist.dat");
for(int i=0; i<(nstruct/skip); i++)
{
 // cout<<i<<endl;
 for(int j=0; j<nElt1;j++)
 {
    if(order=="YZX") {
      val=A[j*3+i*nElt1*3+2];    
    }
    if(order=="XYZ") {
      val=A[j*3+i*nElt1*3+1];      
    }
    if(order=="ZXY") {
      val=A[j*3+i*nElt1*3];      
    }
   if(h2onum[j+i*nElt1]==2 && exch[j+i*nElt1]==1 &&val<maxtop && val>mintop)
   //if(h2onum[j+i*nElt1]==2)
   {
        //watercount << i <<" "<<j<<" ";
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+k]= A[j*3+i*nElt1*3+k];
     //watercount << h2oxyz[j*15+i*nElt1*15+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+3+k]= B[h2oneigh[j*OHsamples+i*nElt1*OHsamples]*3+i*nElt2*3+k];
     //watercount << h2oxyz[j*9+i*nElt1*9+3+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+6+k]= B[h2oneigh[j*OHsamples+i*nElt1*OHsamples+1]*3+i*nElt2*3+k];
     //watercount << h2oxyz[j*9+i*nElt1*9+6+k] <<" ";
     }
     OHzdist <<A[j*3+i*nElt1*3+2]-avgsurf<<" "<<B[h2oneigh[j*OHsamples+i*nElt1*OHsamples]*3+i*nElt2*3+2]-avgsurf<<endl;
     OHzdist <<A[j*3+i*nElt1*3+2]-avgsurf<<" "<<B[h2oneigh[j*OHsamples+i*nElt1*OHsamples+1]*3+i*nElt2*3+2]-avgsurf<<endl;
     float chk1,chk2,chk3,chk4,chk5,chk6;
     float a1,b1,c1,d1,e1,f1,g1,h1,i1;
     float latx,laty,latz,latxy,latxz,latyz;
     //check_together(h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],\
                    h2oxyz[j*15+i*nElt1*15+3],h2oxyz[j*15+i*nElt1*15+4],h2oxyz[j*15+i*nElt1*15+5],\
                    h2oxyz[j*15+i*nElt1*15+6],h2oxyz[j*15+i*nElt1*15+7],h2oxyz[j*15+i*nElt1*15+8],\
                    lattice[i*6],lattice[i*6+1],lattice[i*6+2],lattice[i*6+3],lattice[i*9+4],lattice[i*6+5]);
     
     a1=h2oxyz[j*15+i*nElt1*15];
     b1=h2oxyz[j*15+i*nElt1*15+1];
     c1=h2oxyz[j*15+i*nElt1*15+2];
     d1=h2oxyz[j*15+i*nElt1*15+3];
     e1=h2oxyz[j*15+i*nElt1*15+4];
     f1=h2oxyz[j*15+i*nElt1*15+5];
     g1=h2oxyz[j*15+i*nElt1*15+6];
     h1=h2oxyz[j*15+i*nElt1*15+7];
     i1=h2oxyz[j*15+i*nElt1*15+8];
     latx=lattice[i*6];
     laty=lattice[i*6+1];
     latz=lattice[i*6+2];
     latxy=lattice[i*6+3];
     latxz=lattice[i*6+4];
     latyz=lattice[i*6+5];
     
chk1= (a1-d1);
chk2= (b1-e1);
chk3= (c1-f1);
chk4= (a1-g1);
chk5= (b1-h1);
chk6= (c1-i1);

if(cell_type == "orthorhombic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) e1=e1+laty; else e1=e1-laty;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
if(fabs(chk4) > latx/2.0) { if (chk4 >0) g1=g1+latx; else g1=g1-latx;}
if(fabs(chk5) > laty/2.0) { if (chk5 >0) h1=h1+laty; else h1=h1-laty;}
if(fabs(chk6) > latz/2.0) { if (chk6 >0) i1=i1+latz; else i1=i1-latz;}
}

else if(cell_type == "monoclinic")
{
if(fabs(chk2) > laty/2.0) { if (chk2 >0) {e1=e1+laty; d1=d1+latxy;} else {e1=e1-laty; d1=d1-latxy;}}
chk1= (a1-d1);
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
if(fabs(chk5) > laty/2.0) { if (chk5 >0) {h1=h1+laty; g1=g1+latxy;} else {h1=h1-laty; g1=g1-latxy;}}
chk4= (a1-g1);
if(fabs(chk4) > latx/2.0) { if (chk4 >0) g1=g1+latx; else g1=g1-latx;}
if(fabs(chk6) > latz/2.0) { if (chk6 >0) i1=i1+latz; else i1=i1-latz;}
}
     
     h2oxyz[j*15+i*nElt1*15+3]=d1;
     h2oxyz[j*15+i*nElt1*15+4]=e1;
     h2oxyz[j*15+i*nElt1*15+5]=f1;
     h2oxyz[j*15+i*nElt1*15+6]=g1;
     h2oxyz[j*15+i*nElt1*15+7]=h1;
     h2oxyz[j*15+i*nElt1*15+8]=i1;
     
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+9+k]=(h2oxyz[j*15+i*nElt1*15+3+k]+h2oxyz[j*15+i*nElt1*15+6+k])/2.0;
     //watercount << h2oxyz[j*15+i*nElt1*15+9+k] <<" ";
     }
     //////////////////////////////angle between dipole and z axis 0,0,1(specific - be careful if using other directions//////////////////////////////////////////////////
     h2oxyz[j*15+i*nElt1*15+12]=angle(0,0,0,0,0,1,h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],h2oxyz[j*15+i*nElt1*15+9],h2oxyz[j*15+i*nElt1*15+9+1],h2oxyz[j*15+i*nElt1*15+9+2]);
     h2oxyz[j*15+i*nElt1*15+13]=angle(0,0,0,0,0,1,h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],h2oxyz[j*15+i*nElt1*15+3],h2oxyz[j*15+i*nElt1*15+4],h2oxyz[j*15+i*nElt1*15+5]);
     h2oxyz[j*15+i*nElt1*15+14]=angle(0,0,0,0,0,1,h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],h2oxyz[j*15+i*nElt1*15+6],h2oxyz[j*15+i*nElt1*15+7],h2oxyz[j*15+i*nElt1*15+8]);
     //watercount << h2oxyz[j*15+i*nElt1*15+12]*(180.0/3.141);
     angdist[int(ceil(h2oxyz[j*15+i*nElt1*15+12]*(180.0/3.14159)))]++;
     angdistH1[int(ceil(h2oxyz[j*15+i*nElt1*15+13]*(180.0/3.141)))]++;
     angdistH2[int(ceil(h2oxyz[j*15+i*nElt1*15+14]*(180.0/3.141)))]++;
     //watercount<<endl;
     dipOHdist << j<<" "<<h2oxyz[j*15+i*nElt1*15+12]*(180.0/3.14159)<<" "<<h2oxyz[j*15+i*nElt1*15+13]*(180.0/3.141)<<" "<<\
     h2oxyz[j*15+i*nElt1*15+14]*(180.0/3.141)<<endl;
   }
   
  }
}
  //watercount.close();
  OHzdist.close();
  dipOHdist.close();
  float totalangdist=0.0;
  for(int i=0; i<=180;i++)
  {
    totalangdist +=angdist[i];
  }
  ofstream angldist;
  angldist.open("Dipole-orient.dat");
  angldist <<"T num num1 num2"<<endl;
   for(int i=0; i<=180;i++)
  {
    //angldist << i<<" "<<angdist[i]/(totalangdist)<<endl;
    angldist << i<<" "<<angdist[i]/float(nstruct/skip)<<" "<< angdistH1[i]/float(nstruct/skip)<<" "<< angdistH2[i]/float(nstruct/skip)<<endl;
 
  }
  angldist.close();
  free(angdist);
  free(angdistH1);
  free(angdistH2);
  free(h2onum);
  free(h2oneigh);
  free(h2oneighdist);
  free(h2oxyz);
  free(h2odistmat);
   }
    //distribution of hydrogen atoms per chosen range
    int nlines;
    float DOG;
    float *OHzdistmat;
    int *OHzhist;
    system("wc -l < OHzdist.dat > nlines");
    ifstream nline;
    nline.open("nlines");
    nline>>nlines;
    nline.close();
    system("rm nlines");
    OHzdistmat = (float*) malloc (sizeof(float)*nlines);
    ifstream hist;
    hist.open("OHzdist.dat");
    for(int i=0;i<nlines;i++)
    {
            hist >> DOG >>OHzdistmat[i];
    }
    hist.close();
    //cout<<"hello"<<endl;
    float Hmax,Hint;
    float *Htick;
    Hmax=5.0;
    //Hmax = maximum(OHzdistmat,nlines,0);
     //cout<<"okay"<<endl;
    OHzhist=(int *) malloc (sizeof(int)*100);
    Htick=(float*) malloc (sizeof(float)*100*2);
    //cout<<Hmax<<endl;
    Hint = Hmax/100.0;
    
    for(int i=0;i<100;i++)
    {
      OHzhist[i]=0;
      Htick[i*2]=(i*Hint);
      Htick[i*2+1]=((i+1)*Hint);
    }
    
   
    for(int i=0;i<100;i++)
    {
      for(int j=0;j<nlines;j++)
    {
            if(OHzdistmat[j]>=Htick[i*2] && OHzdistmat[j]<Htick[i*2+1])
	    {
	      OHzhist[i]++;
	    }
    }
    }
    float cellvolume;
    
    if(cell_type=="orthorhombic")
    {
      cellvolume = xvec*yvec*Hint;
    }
    else if(cell_type=="monoclinic")
    {
      cellvolume= xvec*xvec*zint*0.866025403;
    }
    ofstream angldist;
  angldist.open("OHzhist.dat");
  angldist <<"Z num"<<endl;
   for(int i=0; i<100;i++)
  {
    angldist << Htick[i*2+1]<<" "<<float(OHzhist[i])/float((nstruct/skip)*cellvolume)<<endl;
  }
  angldist.close();

  free(OHzdistmat);
  free(OHzhist);
  free(Htick);
////////////////////////////////////////////////////end of dipole orientation calculation /////////////////////////////////////////////////////////////////////////////////
    
    
    
    
////////////////////////////////////////////////////atop orientation computation/////////////////////////////////////////////////////////////////////////////     
     
     //I use the names of variables from H bonds but has nothing to do with water molecules - just a quick hack
   if(atop_orient=="yes" && type=="density-top")
   {
   // cout<<"Hungarian Horntail"<<endl;
  int *h2onum;
  int *h2oneigh;
  float *h2oxyz;
  float *h2oneighdist;
  float *h2odistmat, *dev_h2odistmat;
  
  //int OHsamples=count_metal;
    int OHsamples=20;
    h2onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    h2oneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    h2oneighdist = (float *) malloc (sizeof(float)*OHsamples*nElt1*(nstruct/skip));
    h2oxyz = (float *) malloc (sizeof(float)*7*nElt1*(nstruct/skip));  //Oxyz,H1xyz,H2xyz,Hmidpoint,dipole alignment with surface normal
    h2odistmat = (float *) malloc (sizeof(float)*nElt1*count_metal);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      h2onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      h2oneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      h2oneighdist[j*OHsamples+i*nElt1*OHsamples+k]=0.0;
      }
      for(int k =0 ;k<7;k++)
      {
      h2oxyz[j*7+i*nElt1*7+k]=0.0;
      }
    }
  }   
      
      
cout <<"gpu begins OH distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_h2odistmat,sizeof(float)*nElt1*count_metal);
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
//dim3 struct to define elements of the execution configuration

    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(count_metal+dimBlock.y-1)/dimBlock.y,1);
    

   
    
for(int i = 0; i<(nstruct/skip); i++) // goint through every frame
{
  
 // neigh << i<<endl<<endl;
  
   for(int i1=0;i1<nElt1;i1++) // initializing the distance matrix
  {
    for(int j1=0; j1<count_metal; j1++)
    {
      h2odistmat[j1+i1*count_metal]=0.0;
    }
  }
  
  hipMemcpy(dev_h2odistmat,h2odistmat,sizeof(float)*nElt1*count_metal,hipMemcpyHostToDevice);
  //All O atoms within bondistOM distance is selected and stored in the above matrix
  
    if(cell_type == "orthorhombic")
    {
          //Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOM,nElt1,count_metal,dev_lattice,i);
	  Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOM,nElt1,count_metal,dev_lattice,i,dirn);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOM,nElt1,count_metal,dev_lattice,i,dirn);  
	  //Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOM,nElt1,count_metal,dev_lattice,i); 
    }

hipMemcpy(h2odistmat,dev_h2odistmat,sizeof(float)*nElt1*count_metal,hipMemcpyDeviceToHost);
   
   //cout<<count_metal<<endl;
   
  for(int i1=0;i1<nElt1;i1++) //for each O atoms
  {
    int k=0;
    for(int j1=0; j1<count_metal; j1++) // for each Cu atoms
    {
      if(h2odistmat[j1+i1*count_metal] !=0.0)
      {
	h2onum[i1+i*nElt1]++;
	h2oneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	h2oneighdist[i1*OHsamples+i*nElt1*OHsamples+k] = h2odistmat[j1+i1*count_metal];
	k++;
      }
    }
  }

}   //frames done

// neigh.close();
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_h2odistmat);
    hipFree(dev_lattice);
//sorting needs to be done!!!!

cout<<"Swapping of bonds based on distance "<<endl;

float swapa,swapb;
    for(int i=0;i<(nstruct/skip);i++)
     {
      for(int j=0; j<nElt1; j++)
       {
	 
	for(int swf=0; swf<h2onum[j+i*nElt1]-1;swf++)
	{
	for(int sws=swf+1; sws<h2onum[j+i*nElt1];sws++)
	{
	if(h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf] > h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws] && sws !=swf && h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws] !=0.0 && h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf] !=0.0) 
	{
	   swapa = h2oneigh[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    h2oneigh[j*OHsamples+i*nElt1*OHsamples+swf]=h2oneigh[j*OHsamples+i*nElt1*OHsamples+sws];
	    h2oneigh[j*OHsamples+i*nElt1*OHsamples+sws]=swapa;
	   swapb = h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    h2oneighdist[j*OHsamples+i*nElt1*OHsamples+swf]=h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws];
	    h2oneighdist[j*OHsamples+i*nElt1*OHsamples+sws]=swapb;
	}
	}
	}	
       }
     }	
   


   

cout <<"gpu ends"<<endl;
   
cout<<" done !!!"<<endl;    
    

//h2oneigh contain the IDs of all possible Cu neighbour atoms.


  
  
  float *angdist;
  
  angdist = (float *) malloc (sizeof(float)*180);
  
  for(int i=0; i<=180;i++)
  {
    angdist[i]=0.0;
  }
  
  float val;
  ofstream atopjdist;
  atopjdist.open("atopjdist.dat");
for(int i=0; i<(nstruct/skip); i++) // each frame
{
  //cout<<i<<endl;
 for(int j=0; j<nElt1;j++) // for each oxygen
 {
    if(order=="YZX") {
      val=A[j*3+i*nElt1*3+2];    
    }
    if(order=="XYZ") {
      val=A[j*3+i*nElt1*3+1];      
    }
    if(order=="ZXY") {
      val=A[j*3+i*nElt1*3];      
    }
    //cout<<j<<endl;
   if(h2onum[j+i*nElt1]>0 && exch[j+i*nElt1]==1 && val < maxtop && val > mintop)
   {
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*7+i*nElt1*7+k]= A[j*3+i*nElt1*3+k]; // oxygen atom position
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*7+i*nElt1*7+3+k]= METAL[h2oneigh[j*OHsamples+i*nElt1*OHsamples]*3+i*count_metal*3+k];//Cu position
     //watercount << h2oxyz[j*7+i*nElt1*7+3+k] <<" ";
     }
     //check_together(h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2],\
                    h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                    lattice[i*6],lattice[i*6+1],lattice[i*6+2],lattice[i*6+3],lattice[i*9+4],lattice[i*6+5]);
                    
                    
     float chk1,chk2,chk3;
     float a1,b1,c1,d1,e1,f1;
     float latx,laty,latz,latxy,latxz,latyz;
     //check_together(h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],\
                    h2oxyz[j*15+i*nElt1*15+3],h2oxyz[j*15+i*nElt1*15+4],h2oxyz[j*15+i*nElt1*15+5],\
                    h2oxyz[j*15+i*nElt1*15+6],h2oxyz[j*15+i*nElt1*15+7],h2oxyz[j*15+i*nElt1*15+8],\
                    lattice[i*6],lattice[i*6+1],lattice[i*6+2],lattice[i*6+3],lattice[i*9+4],lattice[i*6+5]);
     
     a1=h2oxyz[j*7+i*nElt1*7];
     b1=h2oxyz[j*7+i*nElt1*7+1];
     c1=h2oxyz[j*7+i*nElt1*7+2];
     d1=h2oxyz[j*7+i*nElt1*7+3];
     e1=h2oxyz[j*7+i*nElt1*7+4];
     f1=h2oxyz[j*7+i*nElt1*7+5];

     latx=lattice[i*6];
     laty=lattice[i*6+1];
     latz=lattice[i*6+2];
     latxy=lattice[i*6+3];
     latxz=lattice[i*6+4];
     latyz=lattice[i*6+5];
     
chk1= (a1-d1);
chk2= (b1-e1);
chk3= (c1-f1);


if(cell_type == "orthorhombic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) e1=e1+laty; else e1=e1-laty;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
}

else if(cell_type == "monoclinic")
{
if(fabs(chk2) > laty/2.0) { if (chk2 >0) {e1=e1+laty; d1=d1+latxy;} else {e1=e1-laty; d1=d1-latxy;}}
chk1= (a1-d1);
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
}
     
     h2oxyz[j*7+i*nElt1*7+3]=d1;
     h2oxyz[j*7+i*nElt1*7+4]=e1;
     h2oxyz[j*7+i*nElt1*7+5]=f1;
                    
                    
                    
     //////////////////////////////angle between dipole and z axis (specific - be careful if using other directions//////////////////////////////////////////////////
     //h2oxyz[j*7+i*nElt1*7+6]=angle(h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5]+5.0,\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2]);
       //h2oxyz[j*7+i*nElt1*7+6]=angle(0,0,0,0,0,1,\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2]);
       h2oxyz[j*7+i*nElt1*7+6]=angle(h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                    h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5]+1,\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2]);
     //watercount << h2oxyz[j*7+i*nElt1*7+3] <<" "<< h2oxyz[j*7+i*nElt1*7+3+1] <<" "<< h2oxyz[j*7+i*nElt1*7+3+2] <<" ";
     //watercount << h2oxyz[j*7+i*nElt1*7+6]*(180.0/3.141);
     angdist[int(floor(h2oxyz[j*7+i*nElt1*7+6]*(180.0/3.14159)))]++;
    // watercount<<endl;
     atopjdist << j<<" "<<h2oxyz[j*7+i*nElt1*7+6]*(180.0/3.14159)<<endl;
   }
   
  }
}
  //watercount.close();
  atopjdist.close();
float totalatopdist=0.0;
  for(int i=0; i<=180;i++)
  {
    totalatopdist +=angdist[i];
  }
  ofstream angldist;
  angldist.open("atop-orient.dat");
  angldist <<"T num"<<endl;
   for(int i=0; i<=180;i++)
  {
    //angldist << i<<" "<<angdist[i]/(totalatopdist)<<endl;
    angldist << i<<" "<<angdist[i]/float(nstruct/skip)<<endl;
  }
  angldist.close();
  free(angdist);
  free(h2onum);
  free(h2oneigh);
  free(h2oneighdist);
  free(h2oxyz);
  free(h2odistmat);     
     
     
   }
    
////////////////////////////////////////////////////end of atop orientation calculation /////////////////////////////////////////////////////////////////////////////////
    
    
    
    
    
    
       
    
    
    
    
    
    
    
    if(filter_density == "yes")
    {
    hipMalloc((void **)&dev_densityf,sizeof(int)*xsplit*ysplit*zsplit);
    hipMemcpy(dev_densityf,densityf,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyHostToDevice);
    dim3 dimBlockee(10,10,10);
    dim3 dimGridee((xsplit+dimBlockee.x-1)/dimBlockee.x,(ysplit+dimBlockee.y-1)/dimBlockee.y,(zsplit+dimBlockee.z-1)/dimBlockee.y);
    if(order=="XYZ") {
    density_filter<<<dimGridee,dimBlockee>>>(dev_density, dev_densityf,ysplit,zsplit,xsplit);
    }
    else if(order=="ZXY") {
    density_filter<<<dimGridee,dimBlockee>>>(dev_density, dev_densityf,xsplit,ysplit,zsplit);
    }
    else if(order=="YZX") {
    density_filter<<<dimGridee,dimBlockee>>>(dev_density, dev_densityf,zsplit,xsplit,ysplit);
    }
    hipMemcpy(density,dev_densityf,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
    }
    else
    {
      hipMemcpy(density,dev_density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
    }
FILE *densityfile=fopen("out","wt");



 if(plot =="int" || plot == "bulk")
  {
    int diffdirn;
if(vacuum.at(0)=='z') diffdirn=2;
if(vacuum.at(0)=='y') diffdirn=1;
if(vacuum.at(0)=='x') diffdirn=0;

  float minAz,maxAz,midAz;
  minAz=A_int[diffdirn];
  maxAz=A_int[diffdirn];
  for(int j=0; j<nA_int;j++)
    {
      if(A_int[j*3+diffdirn] > maxAz) maxAz= A_int[j*3+diffdirn];
      if(A_int[j*3+diffdirn] < minAz) minAz= A_int[j*3+diffdirn];
    }
    midAz=(maxAz+minAz)/2.0;
  ofstream diffplot;
  ofstream diffplot1;
  diffplot.open("diffplot");
  diffplot1.open("diffplot1");
  if (type=="density" && scope=="bulk-interface" && plot=="int")
 {
   for(int i=0; i<nstruct/skip;i++)
   {
    for(int j=0; j<nA_int;j++)
    {
      if(A_int[j*3+diffdirn] < midAz)
      {
    diffplot <<A_int[j*3+i*nA_int*3]<<" "<<A_int[j*3+i*nA_int*3+1]<<" "<<A_int[j*3+i*nA_int*3+2]<<endl;
      }
      else 
      {
    diffplot1 <<A_int[j*3+i*nA_int*3]<<" "<<A_int[j*3+i*nA_int*3+1]<<" "<<A_int[j*3+i*nA_int*3+2]<<endl;
      }
    }
   }
 }
 if (type=="density" && scope=="bulk-interface" && plot=="bulk")
 {
   for(int i=0; i<nstruct/skip;i++)
   {
    for(int j=0; j<nA_bulk;j++)
    {
      if(A_bulk[j*3+diffdirn] < midAz)
      {
    diffplot <<A_bulk[j*3+i*nA_bulk*3]<<" "<<A_bulk[j*3+i*nA_bulk*3+1]<<" "<<A_bulk[j*3+i*nA_bulk*3+2]<<endl;
      }
      else 
      {
    diffplot1 <<A_bulk[j*3+i*nA_bulk*3]<<" "<<A_bulk[j*3+i*nA_bulk*3+1]<<" "<<A_bulk[j*3+i*nA_bulk*3+2]<<endl;
      }
    }
   }
 }
 diffplot.close();
 diffplot1.close();  
  }




if(order=="XYZ") {

for(int l=0;l<ysplit;l++)
{
  //fprintf(densityfile,"%d %d\n",zsplit,xsplit);
  fprintf(densityfile,"#%d \n",l);
for(int m=0;m<zsplit;m++)
{
for(int k=0;k<xsplit;k++)
{
    if(type=="density-top") 
  {
fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit])/float(maxbondist-minbondist));
  }
  else
  {
    fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit])/float(nElt1));
  }
}
fprintf(densityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n"); 
}
}
if(order=="ZXY") {

for(int k=0;k<xsplit;k++)
{
  fprintf(densityfile,"#%d \n",k);
for(int l=0;l<ysplit;l++)
{
for(int m=0;m<zsplit;m++)
{
      if(type=="density-top") 
  {
fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit])/float(maxbondist-minbondist));
  }
    else
  {
    fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit])/float(nElt1));
  }
}
fprintf(densityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}
}
if(order=="YZX") {

for(int m=0;m<zsplit;m++)
{
  fprintf(densityfile,"#%d \n",m);
for(int k=0;k<xsplit;k++)
{
for(int l=0;l<ysplit;l++)
{
if(type=="density-top") 
  {
fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit])/float(maxbondist-minbondist));
  }
else
  {
fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit])/float(nElt1));
  }
}
fprintf(densityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}  
}
fclose(densityfile);

hipFree(dev_density);
hipFree(dev_densityf);
if (type=="density" || type=="densityvelocity");
{
hipFree(dev_A);
hipFree(dev_xtick);
hipFree(dev_ytick);
hipFree(dev_ztick);
}

if(write_cube == "yes")
{
  FILE *densityfile=fopen("cube","wt");
  if(order=="XYZ") {

for(int l=0;l<ysplit;l++)
{
for(int m=0;m<zsplit;m++)
{
for(int k=0;k<xsplit;k++)
{
fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
}
}
}
  }
else if(order=="ZXY") {

for(int k=0;k<xsplit;k++)
{
for(int l=0;l<ysplit;l++)
{
for(int m=0;m<zsplit;m++)
{
fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit]));
if (m % 6 == 5) fprintf(densityfile,"\n");
}
fprintf(densityfile,"\n");
}
}
}
else if(order=="YZX") {

for(int k=0;k<xsplit;k++)
{
for(int l=0;l<ysplit;l++)
{
for(int m=0;m<zsplit;m++)
{
fprintf(densityfile,"%f ",float(density[m+l*zsplit+k*ysplit*zsplit]));
if (m % 6 == 5) fprintf(densityfile,"\n");
}
fprintf(densityfile,"\n");
}
}
}

fclose(densityfile);
}

if(type=="density-top" || (type=="density" && scope=="bulk-interface"))
{
  system("paste out out out | tail -n +2 > ext-out ; cat ext-out ext-out ext-out > extended.out ");
}

}
