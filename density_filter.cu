#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




//filter for densities. buggy - donot use

#include "cudatools.cuh"

__global__ void density_filter(int *density, int *densityf, int mainsplit,int sub1split, int sub2split)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  int i1,i2,i3,j1,j2,j3,k1,k2,k3;
  if(rowid < mainsplit && widid < sub1split-1 && colid < sub2split-1 && colid != 0 && widid !=0)
  {
    i1=colid+widid*sub1split+rowid*sub2split*sub1split;
    j1=colid+(widid+1)*sub1split+rowid*sub2split*sub1split+1;
    k1=colid+(widid-1)*sub1split+rowid*sub2split*sub1split-1;
    i2=(colid-1)+widid*sub1split+rowid*sub2split*sub1split;
    j2=(colid-1)+(widid+1)*sub1split+rowid*sub2split*sub1split;
    k2=(colid-1)+(widid-1)*sub1split+rowid*sub2split*sub1split;
    i3=(colid+1)+widid*sub1split+rowid*sub2split*sub1split;
    j3=(colid+1)+(widid+1)*sub1split+rowid*sub2split*sub1split;
    k3=(colid+1)+(widid-1)*sub1split+rowid*sub2split*sub1split;
    densityf[colid+widid*mainsplit+rowid*sub2split*sub1split] = (density[i1]+density[i2]+density[i3]+density[j1]+density[j2]+density[j3]+density[k1]+density[k2]+density[k3])/9.0;
  }
}

