#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void covermatmono(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    //if(rowid ==2 && widid == 1){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z < bondist+0.01 )
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}

__global__ void covermatmono(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct, float *latti)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    //if(rowid ==2 && widid == 1){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z < bondist+0.01 )
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}


__global__ void covermatmono(float *x, float*y, int *a, float minbondist, float maxbondist, int nElt1, int nElt2, int nstruct, float *latti , int dirn, float surf,float mintop, float maxtop  )
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct && x[rowid*3+widid*nElt1*3+dirn] >mintop && x[rowid*3+widid*nElt1*3+dirn] <maxtop && y[colid*3+widid*nElt2*3+dirn] >mintop && y[colid*3+widid*nElt2*3+dirn] <maxtop)
  //if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    //if(rowid ==2 && widid == 1){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z < maxbondist && z > minbondist && x[rowid*3+widid*nElt1*3+dirn] > surf+minbondist && x[rowid*3+widid*nElt1*3+dirn] < surf+maxbondist)
    //if (z < maxbondist && z > minbondist)
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}

__global__ void covermatmono(float *x, float*y, int *a, float minbondist, float maxbondist, int nElt1, int nElt2, int nstruct, float *latti , float mintop, float maxtop,int dirn, float surf)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct && widid < nstruct && y[colid*3+widid*nElt2*3+dirn] >mintop && y[colid*3+widid*nElt2*3+dirn] <maxtop&& x[rowid*3+widid*nElt1*3+dirn] > maxtop)
  //if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    //if(rowid ==2 && widid == 1){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z < maxbondist && z > minbondist && x[rowid*3+widid*nElt1*3+dirn] > surf+minbondist && x[rowid*3+widid*nElt1*3+dirn] < surf+maxbondist)
    //if (z < maxbondist && z > minbondist)
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}


__global__ void covermatmono(float *x, int *a, int nElt1, int nstruct, float minaz, float maxaz, int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  if(rowid < nElt1 &&  widid < nstruct)
  {
        if (x[rowid*3+widid*nElt1*3+dirn] < maxaz &&  x[rowid*3+widid*nElt1*3+dirn] > minaz)
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

 }
}

//residenttime
__global__ void covermatmono(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct,float *latti,int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkrr;
  
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    chkrr=fabs(x[rowid*3+widid*nElt1*3+dirn]-y[colid*3+widid*nElt2*3+dirn]);
    if (chkrr <= bondist)
    {    
      atomicAdd(&a[rowid+widid*nElt1],1);
    }
 }
}

__global__ void covermatmono(float *x, float *y, int *a, float minbondist, float maxbondist, int index,float *latti, int nElt1, int nstruct)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  if(rowid < nElt1 && widid < nstruct)
  //if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[widid*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[widid*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[widid*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    //if(rowid ==2 && widid == 1){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z <= maxbondist && z >= minbondist)
    //if (z < maxbondist && z > minbondist)
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}