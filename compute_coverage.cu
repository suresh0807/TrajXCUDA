//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"
int *Mcover, *Scover;
int *Mcoversum, *Scoversum;
int *dev_cover;


void compute_coverage(void)
{
 Mcover= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
 Scover= (int *) malloc (sizeof(int)*nElt2*(nstruct/skip));
// Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          Mcover[i+l*nElt1]=0;
        }
      for(int i=0; i< nElt2; i++)
        {
          Scover[i+l*nElt2]=0;
        }        
      }

 Mcoversum= (int *) malloc (sizeof(int)*(nstruct/skip));   
 Scoversum= (int *) malloc (sizeof(int)*(nstruct/skip));
 for(int l=0; l<nstruct/skip; l++)
    {
          Mcoversum[l]=0;
	  Scoversum[l]=0;
      }
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_cover,sizeof(int)*nElt1*(nstruct/skip));


//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlock(10,10,10);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands
 
    //For surface metal atoms
    hipMemcpy(dev_cover,Mcover,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_cover,Mbondist,nElt1,nElt1,(nstruct/skip),xvec,yvec,zvec);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_cover,Mbondist,nElt1,nElt1,(nstruct/skip),xvec,yvec,zvec,yxvec);
    }
    hipMemcpy(Mcover,dev_cover,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
    hipFree(dev_B);
    hipFree(dev_cover);
    
    
    
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_cover,sizeof(int)*nElt2*(nstruct/skip));
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    
    dim3 dimBlocks(10,10,10);
    dim3 dimGrids((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);
    
    hipMemcpy(dev_cover,Scover,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGrids,dimBlocks>>>(dev_B,dev_A,dev_cover,bondist,nElt2,nElt1,(nstruct/skip),xvec,yvec,zvec);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrids,dimBlocks>>>(dev_B,dev_A,dev_cover,bondist,nElt2,nElt1,(nstruct/skip),xvec,yvec,zvec,yxvec);
    }
    hipMemcpy(Scover,dev_cover,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_cover);
    
    
     for(int l=0; l<(nstruct/skip); l++)
    {
       for(int i=0; i< nElt1; i++)
         {
	   //cout <<l<<" "<<i<<" "<<Mcover[i+l*nElt1]<<endl;
          if(Mcover[i+l*nElt1] < maxneigh && Mcover[i+l*nElt1] > minneigh)
	  {
	    Mcoversum[l]+=1;
	  }
	}
    }
    
    for(int l=0; l<(nstruct/skip); l++)
    {
       for(int i=0; i< nElt2; i++)
         {
	  // cout <<l<<" "<<i<<" "<<Scover[i+l*nElt2]<<endl;
          if( Scover[i+l*nElt2] > 0)
	  {
	    Scoversum[l]+=1;
	  }
	}
    }
    
    int *sorter = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));

    int surf_atom_counter=0;
    float int_max, int_min;
    
     for(int l=0; l<(nstruct/skip); l++)
    {
       for(int i=0; i< nElt1; i++)
         {
          if(Mcover[i+l*nElt1] < maxneigh && Mcover[i+l*nElt1] > minneigh){sorter[i+l*nElt1] = 0;if(l==0){surf_atom_counter++;}}
          else {sorter[i+l*nElt1]=1;}
          //cout << l<<" "<<i<<" "<< sorter[i+l*nElt1] <<" "<< A[i*3+l*nElt1*3+2]<<endl;        
	}
    }    
    cout <<surf_atom_counter<<endl;
    float *int_maxmin = (float *) malloc (sizeof(float)*surf_atom_counter*3);
    
       int counta=0;
       for(int i=0; i< nElt1; i++)
       {
          if(sorter[i] == 0){int_maxmin[counta] = A[i*3];
	                     int_maxmin[counta+1] = A[i*3+1];
			     int_maxmin[counta+2] = A[i*3+2];
			     counta+=3;
	    // cout <<A[i*3+2]<<endl;
	  }       
       }    
    
    int_max = maximum(int_maxmin,surf_atom_counter,2);
    int_min = minimum(int_maxmin,surf_atom_counter,2);
    
    cout <<"Minimum Z position "<<int_min<<endl;
    cout <<"Maximum Z position "<<int_max<<endl;
    
    
    
// printing the rdf data to be visualized
//    FILE *Ofragplotchk=fopen("O-fragments-chk.data","wt");
// for(int l=0; l<nstruct; l++)
//    {
//      fprintf(Ofragplotchk,"%d\n",l);
//      for(int i=0; i< nElt1; i++)
//        {
//          fprintf(Ofragplotchk,"%d %d\n",i,Ofrag[i+l*nElt1]);
//        }
//      }
        
//    fclose(Ofragplotchk);
    free(Mcover);
    free(Scover);
    ofstream coverplot;
    coverplot.open("percentage-coverage.data");
    //FILE *coverplot=fopen("percentage-coverage.data","wt");
 for(int l=0; l<(nstruct/skip); l++)
    {
          coverplot << l <<" "<< Mcoversum[l] <<" "<< Scoversum[l] <<" "<< (float(Scoversum[l])/float(Mcoversum[l]))*100.0 <<endl;
    }
    coverplot.close();    
    //fclose(coverplot);
  free(Mcoversum);
  free(Scoversum);
}
