#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"

void compute_Zdist(void)
{
cout << type <<" is chosen"<<endl;
cout<<"Zdist: " << nElt1 <<" "<< Elt1 << " atoms are there in each structure"<<endl;

int Dirn,split;
float min,max;

if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
  Dirn = 2;
  min=minz;
  max=maxz;
  split=zsplit;
  
}

else if(set_max_y == "yes")
{
  minz = set_miny;
  maxz = set_maxy;
  Dirn=1;
  min=miny;
  max=maxy;
  split=ysplit;
}
else if(set_max_x == "yes")
{
  minz = set_minx;
  maxz = set_maxx;
  Dirn=0;
  min=minx;
  max=maxx;
  split=xsplit;
}



printf("%f %f\n",min,max);

    zrange=max-min;

    zint = zrange/split;

    ztick=(float*) malloc (sizeof(float)*split*2);
    
    for(int i=0;i<split;i++)
    {
      ztick[i*2]=min+(i*zint);
      ztick[i*2+1]=min+((i+1)*zint);
    }
    

FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<split;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
fclose(xyztick);

    density=(int *) malloc (sizeof(int)*split);

        for(int k=0;k<split;k++)
        {
          density[k]=0;
        }

///*


    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_density,sizeof(int)*split);
    hipMalloc((void **)&dev_ztick, sizeof(float)*split*2);
    hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_density,density,sizeof(int)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*split*2,hipMemcpyHostToDevice);
    
    cout << "Memory for storing structure data: "<<((nstruct/skip)*nElt1*3)*4/float(1000000000)<< " Gbs" << endl;
    cout << "Memory for local storage: "<<(split+(split*2))*4/float(1000000000)<< " Gbs" << endl;   
    
    dim3 dimBlock(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGrid(((nstruct/skip)+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    Xdist_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,(nstruct/skip),nElt1,split,dev_ztick,Dirn);
    hipMemcpy(density,dev_density,sizeof(int)*split,hipMemcpyDeviceToHost);
    
    
    xvec=0;yvec=0;zvec=0;
    for(int i=0; i<(nstruct/skip);i++)
    {
        xvec+=lattice[i*6];
        yvec+=lattice[1+i*6];
        zvec+=lattice[2+i*6];
    }
    
    xvec /= (nstruct/skip);
    yvec /= (nstruct/skip);
    zvec /= (nstruct/skip);
    
    
    
    
    
    
    
    
    float cellvolume;
    
    if(cell_type=="orthorhombic")
    {
      if(set_max_z == "yes") cellvolume = xvec*yvec*zint;
      if(set_max_x == "yes") cellvolume = yvec*zvec*zint;
      if(set_max_y == "yes") cellvolume = xvec*zvec*zint;
    }
    else if(cell_type=="monoclinic")
    {
    if(set_max_z == "yes") cellvolume= xvec*xvec*zint*0.866025403; //*sin(60) for rhombohedral 
    if(set_max_x == "yes") cellvolume= yvec*zvec*zint*0.866025403;
    if(set_max_y == "yes") cellvolume= xvec*zvec*zint*0.866025403;
    }
    
    
FILE *Zdistfile=fopen("zdist.data","wt");
FILE *Zdistdenfile=fopen("zdistden.data","wt");
int zstart = 0;
int zend = split;
//int startfromzero=0;  not used because relative distances get messed up
for(int m=zstart;m<zend;m++)
{
  //if(startfromzero == 0 && float(density[m])/(xvec*yvec*zint*(nstruct/skip)) == 0) {}
  //else {fprintf(Zdistfile,"%f %f \n ",zint*startfromzero, float(density[m])/(xvec*yvec*zint*(nstruct/skip)));startfromzero++;}
  fprintf(Zdistfile,"%f %f \n ",ztick[m*2+1], float(density[m])/(cellvolume*(nstruct/skip)));
  fprintf(Zdistdenfile,"%f %f \n ",ztick[m*2+1], float(density[m]));
}
fclose(Zdistfile);
fclose(Zdistdenfile);
hipFree(dev_density);
hipFree(dev_A);
hipFree(dev_ztick);

}
