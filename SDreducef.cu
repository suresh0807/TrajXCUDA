#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void SDreducef(float *a, float *b, int nElt1, int SD_store, int origins,int skips)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  if(rowid < nElt1 && colid < SD_store)
  {
    for(int i = 0; i < origins/skips; i++)
    {
      //printf("%f \n", a[rowid+(colid*nElt1)+(i*nElt1*SD_store)]); 
      b[rowid+colid*nElt1] += a[rowid+(colid*nElt1)+(i*nElt1*SD_store)];
    }
    b[rowid+colid*nElt1] = b[rowid+colid*nElt1]/((float(origins)/float(skips)));
  }
}

__global__ void SDreducef(float *a, float *b, int nElt1, int SD_store, int nstruct,int skips,int i)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  if(rowid < nElt1 )
  {
    for(int i1 = 0; i1 < nstruct; i1++)
    {
      b[rowid+i*nElt1] += a[rowid+i1*nElt1];
    }
    b[rowid+i*nElt1] = b[rowid+i*nElt1]/float(nstruct);
  }
  


/* extern __shared__ int share[];
 unsigned int tid = threadIdx.x;
 unsigned int rowid = threadIdx.x + blockIdx.x * blockDim.x;
 share[tid] = a[rowid];
 __syncthreads();
 
 for(int i=1; i<blockDim.x ; i *=2)
 {
   if(tid % (2*i) ==0)
   {
     share[tid] += share[tid+i];
   }
   __syncthreads();
 }
 
 if(tid ==0) { b[blockIdx.x] = share[0];}
   if(rowid < nElt1 )
  {
   b[rowid+i*nElt1] = b[rowid+i*nElt1]/float(nstruct);
  }
*/

}

