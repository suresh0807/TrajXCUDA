#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void Hbondmat(float *x, float*y, int *a, int *b, float bondist, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  //float vec[3],vecall;
  float chk;
  float z;
  //int chker;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      //vecall=x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k];
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec; 
	                        //  if(vecall >0){vec[0]=y[colid*3+widid*nElt2*3+k]+ xvec;}
	                        //  else {vec[0]=y[colid*3+widid*nElt2*3+k]- xvec;}}}
      }}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec; 
	//                          if(vecall >0){vec[1]=y[colid*3+widid*nElt2*3+k]+ yvec;}
	//                          else {vec[1]=y[colid*3+widid*nElt2*3+k]- yvec;}}}
      }}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec; 
	//                          if(vecall >0){vec[2]=y[colid*3+widid*nElt2*3+k]+ zvec;}
	//                          else {vec[2]=y[colid*3+widid*nElt2*3+k]- zvec;}}}
      }}
      z+=(chk * chk);
    }
    z=sqrt(z);
   // printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);
    if (z < bondist)
    {
       //chker = a[rowid+widid*nElt1];
        //if(a[rowid+widid*nElt1] == 0) {b[rowid*2+widid*nElt1*2]=colid;b[rowid*2+widid*nElt1*2+1]=colid;}
        
       //b[rowid*2+widid*nElt1*2+1]=colid;
       //if(widid ==4998){chker = a[rowid+widid*nElt1]; printf("%d %d %d %d %f\n",widid,rowid,chker,colid,z);}
       //a[rowid+widid*nElt1]++;
       atomicAdd(&a[rowid+widid*nElt1],1);
       atomicCAS(&b[rowid*2+widid*nElt1*2],-1,colid);
       atomicCAS(&b[rowid*2+widid*nElt1*2+1],-1,colid);
       atomicCAS(&b[rowid*2+widid*nElt1*2+1],b[rowid*2+widid*nElt1*2],colid);
  
       //if(a[rowid+widid*nElt1] != 0) {b[rowid*2+widid*nElt1*2+1]=colid;}
       //if(widid ==4998){printf("%d %d %f\n",rowid,colid,z);}
           //for(int k=0; k< 2 ; k++){
           //{b[rowid*6+widid*nElt1*6+a[rowid+widid*nElt1]*3+k]=y[colid*3+widid*nElt2*3+k];}
           //{ if(a[rowid+widid*nElt1] == 1){b[rowid*6+widid*nElt1*6+k]=vec[k];}
	   //  else if(a[rowid+widid*nElt1] == 2){b[rowid*6+widid*nElt1*6+3+k]=vec[k];} }

	   //                        }
     }
    //     if (z < bondist)
    //{
    // if(widid ==4998){chker = a[rowid+widid*nElt1]; printf("%d %d %d %d %f\n",widid,rowid,chker,colid,z);}
      //chker = a[rowid+widid*nElt1];
      //b[rowid*2+widid*nElt1*2+chker-1]=colid;
    //}
      
    }
}

__global__ void Hbondmat(float *x, float*y, int *a, int *b, float *c,float bondist, int nElt1, int nElt2, int nstruct, float *latti)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    chk=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      else if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      else if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    
    // I cannot save the distances directly because, 
    // I am unable to control the flow, I see a zero sometimes :(
    if (z < bondist)
    {
       atomicAdd(&a[rowid+widid*nElt1],1);
       /*
       atomicCAS(&b[rowid*4+widid*nElt1*4],-1,colid);
       atomicCAS(&b[rowid*4+widid*nElt1*4+1],-1,colid);
       atomicCAS(&b[rowid*4+widid*nElt1*4+2],-1,colid);
       atomicCAS(&b[rowid*4+widid*nElt1*4+3],-1,colid);
       else if(a[rowid+widid*nElt1]==2)
       {
       atomicCAS(&b[rowid*4+widid*nElt1*4+1],b[rowid*4+widid*nElt1*4],colid);
       }
       else if(a[rowid+widid*nElt1]==3)
       {
       atomicCAS(&b[rowid*4+widid*nElt1*4+2],b[rowid*4+widid*nElt1*4],colid);
       }
       else if(a[rowid+widid*nElt1]==4)
       {
       atomicCAS(&b[rowid*4+widid*nElt1*4+3],b[rowid*4+widid*nElt1*4],colid);
       }
       */
     }
    }
}

__global__ void Hbondmat(float *x, float*y, float *c,float bondist, int nElt1, int nElt2, float *latti,int widid)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  //int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 )
  {
    z=0.0;
    chk=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0)      {if(chk > xvec/2) {chk = chk - xvec;}}
      else if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      else if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    
    if (z < bondist)
    {
       c[colid+rowid*nElt2] = z;
     }
    }
}

//Atop orientation
__global__ void Hbondmat(float *x, float*y, float *c,float bondist, int nElt1, int nElt2, float *latti,int widid,int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  //int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 )
  {
    z=0.0;
    chk=0.0;
    chkz=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    chkz=fabs(x[rowid*3+widid*nElt1*3+dirn]-y[colid*3+widid*nElt2*3+dirn]);
    if (chkz <= bondist)
    {
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0)      {if(chk > xvec/2) {chk = chk - xvec;}}
      else if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      else if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    c[colid+rowid*nElt2] = z;
    
     }
    }
}


__global__ void Hbondmat(float *x, float*y, int*exch, float *c,float bondist, int nElt1, int nElt2, float *latti,int widid)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  //int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 &&exch[rowid+widid*nElt1] >0 )
  {
    z=0.0;
    chk=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      else if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      else if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    
    if (z < bondist)
    {
       c[colid+rowid*nElt2] = z;
     }
    }
}