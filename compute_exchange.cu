//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void compute_exchange(void)
{
  
  cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;
  
  int  *exchflip, *dense;
  float *exchsum, *exchdensity;
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  exchflip= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
 
 if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
}
printf("%f %f\n",minz,maxz);

    zrange=maxz-minz;

    zint = zrange/zsplit;

    ztick=(float*) malloc (sizeof(float)*zsplit*2);
    
    for(int i=0;i<zsplit;i++)
    {
      ztick[i*2]=minz+(i*zint);
      ztick[i*2+1]=minz+((i+1)*zint);
    }
    

FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<zsplit;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
fclose(xyztick);

    exchdensity=(float *) malloc (sizeof(float)*zsplit);
    dense = (int *) malloc (sizeof(int)*zsplit);
        for(int k=0;k<zsplit;k++)
        {
          exchdensity[k]=0.0;
	  dense[k]=1;
        }
 
 
// Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
          exchflip[i+l*nElt1]=0;
        }     
      }

 exchsum= (float *) malloc (sizeof(float)*nElt1);   
 
 for(int l=0; l<nElt1; l++)
    {
          exchsum[l]=0.0;
      }
//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));


//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlock(10,10,10);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(count_metal+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands
 
    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),xvec,yvec,zvec);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),xvec,yvec,zvec,yxvec);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_exch);
    

   int startbulk=0, startint=0;
  
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1;}
          exchsum[i]+=exch[i+l*nElt1];
        }     
      }
    
    for(int l=1; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1] != exch[i+l*nElt1-1*nElt1]) {exchflip[i+l*nElt1]=1;} else {exchflip[i+l*nElt1]=0;}
        }     
      }
      int interflip=0, bulkflip=0;
    for(int l=1; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i]==1 && exchflip[i+l*nElt1]==1) {interflip++;} 
          else if (exch[i]==0 && exchflip[i+l*nElt1]==1){bulkflip++;}
        }     
      } 
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i]==1) {startint++;}
          else if (exch[i]==0){startbulk++;}
        }
      cout <<interflip/startint<<"  "<<bulkflip/startbulk<<endl;
    
    cout<<"Its okay"<<endl;
    
        for(int k=0;k<zsplit;k++)
        {
	  for(int i=0; i< nElt1; i++)
	  {
	    if(A[i*3+2] > ztick[k*2] && A[i*3+2] < ztick[k*2+1])
	    {
	     dense[k]++;
             exchdensity[k]+=exchsum[i];
	    }
	  }
	  exchdensity[k]=exchdensity[k]/float(dense[k]);
        } 

      
      
    ofstream exchplot;
    exchplot.open("exchange.data");
 
    
    
    
    
 for(int l=0; l<(nstruct/skip); l++)
    {
      exchplot <<l<<" ";
      for(int j=0; j<nElt1; j++)
      {
      
          exchplot << exch[j+l*nElt1]<<" ";
      }
      exchplot <<endl;
    }
    exchplot.close();    

  /*  exchplot.open("exchange.data");
      for(int k=0; k<zsplit; k++)
      {
          exchplot << ztick[k*2+1] <<" "<<exchdensity[k]<<endl;
      }
  
    /*  for(int k=0; k<nElt1; k++)
      {
          exchplot << A[k*3+2] <<" "<<exchsum[k]<<endl;
      }*/
  //  exchplot.close(); 
  free(exchsum);
  free(exch);
  free(A);
  free(METAL);
  
}
