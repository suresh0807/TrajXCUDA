#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void velocity_calc(float *A,float *B,float *C,int nstruct,int natoms,int xsplit,int ysplit, int zsplit, float *xtick,float *ytick, float *ztick)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  float chkx,chky,chkz,velx,vely,velz,velmag;
  if(rowid < nstruct && colid <natoms)
  {
  chkx=A[colid*3+rowid*natoms*3];
  chky=A[colid*3+(rowid*natoms*3)+1];
  chkz=A[colid*3+(rowid*natoms*3)+2];
  velx=B[colid*3+rowid*natoms*3];
  vely=B[colid*3+(rowid*natoms*3)+1];
  velz=B[colid*3+(rowid*natoms*3)+2];
  velmag=sqrt(velx*velx+vely*vely+velz*velz);  
    for(int k=0;k<xsplit;k++)
    {
      for(int l=0;l<ysplit;l++)
      {
        for(int m=0;m<zsplit;m++)
        {
//if(A[colid*3+rowid*natoms*3] > xtick[k*2] && A[colid*3+rowid*natoms*3] < xtick[k*2+1] && A[colid*3+(rowid*natoms*3)+1] > ytick[l*2] && A[colid*3+(rowid*natoms*3)+1] < ytick[l*2+1] && A[colid*3+(rowid*natoms*3)+2] > ztick[m*2] && A[colid*3+(rowid*natoms*3)+2] < ztick[m*2+1])        
if(chkx > xtick[k*2] && chkx < xtick[k*2+1] && chky > ytick[l*2] && chky < ytick[l*2+1] && chkz > ztick[m*2] && chkz < ztick[m*2+1])
          {
            atomicAdd(&C[m+l*zsplit+k*ysplit*zsplit],velmag);
            break;
          }
        }
      }
    }
  }
}

