#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################





#include "cudatools.cuh"

__global__ void Hbond_calc(float *A,float *Hbond_density,float *angle_density,float *distance_density, float *OHdensity, float *O_Hdensity, float *angdensity, int nstruct,int natoms,int zsplit, float *ztick, float angle, float max_O_O,int Dirn)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  if(rowid < nstruct && colid <natoms)
  {
        for(int m=0;m<zsplit;m++)
        {
        if( A[colid*9+(rowid*natoms*9)+Dirn] >= ztick[m*2] && A[colid*9+(rowid*natoms*9)+Dirn] < ztick[m*2+1])
          {
            atomicAdd(&Hbond_density[m],1.0);
            atomicAdd(&angle_density[m],A[colid*9+(rowid*natoms*9)+3]);
            atomicAdd(&distance_density[m],A[colid*9+(rowid*natoms*9)+4]);
	    atomicAdd(&OHdensity[m],A[colid*9+(rowid*natoms*9)+6]);
            atomicAdd(&O_Hdensity[m],A[colid*9+(rowid*natoms*9)+7]);
	    atomicAdd(&angdensity[m],A[colid*9+(rowid*natoms*9)+8]);
            break;
          }
        }
   }
}


