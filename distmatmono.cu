#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void distmatmono(float *x, float*y, float *a, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec, float rdf_max_rad)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
//  float chk;
  float chkx,chky,chkz;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
 /*   for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }*/
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    index = z * bin / maxval;
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
  }
}

__global__ void distmatmono(float *x, float*y, float *a,int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec, float rdf_max_rad, int whichwater)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
//  float chk;
  float chkx,chky,chkz;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
 /*   for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }*/
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater && exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }

}
}

__global__ void distmatmono(float *x, float*y, float *a,int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float *latti, float rdf_max_rad, int whichwater)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
//  float chk;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    
 /*   for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }*/
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater )//&& exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }

}
}

__global__ void distmatmono(float *x, float*y, float *a,int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float *latti, float rdf_max_rad, int whichwater, int interface)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
//  float chk;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    if(y[colid*3+widid*nElt1*3+2] > x[rowid*3+widid*nElt1*3+2])
    {
 /*   for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }*/
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater)//&& exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    }
}
}


__global__ void distmatmono(float *x, float*y, float *a,int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float *latti, float rdf_max_rad, int whichwater, float mid, int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
//  float chk;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];


    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    //z=sqrt(chk*chk);
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater && x[rowid*3+widid*nElt1*3+dirn] <= mid && y[colid*3+widid*nElt2*3+dirn] >= x[rowid*3+widid*nElt1*3+dirn] )//&& y[colid*3+widid*nElt2*3+2] <= mid)//&& exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if(exch1[rowid+nElt1*widid]==whichwater && x[rowid*3+widid*nElt1*3+dirn] >= mid && y[colid*3+widid*nElt2*3+dirn] <= x[rowid*3+widid*nElt1*3+dirn] )//&& y[colid*3+widid*nElt2*3+2] >= mid)//&& exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
   
}
}


__global__ void distmatmono(float *x, float*y, float *a, int nElt1, int nElt2, int nstruct, float *latti, float cutoff)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz,xvec,yvec,zvec,yxvec;
  float z;
  float index=0;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct && rowid != colid)
  {
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    z=0.0;
    chkx=fabs(x[rowid*4+widid*nElt1*4]-y[colid*4+widid*nElt2*4]);
    chky=fabs(x[rowid*4+widid*nElt1*4+1]-y[colid*4+widid*nElt2*4+1]);
    chkz=fabs(x[rowid*4+widid*nElt1*4+2]-y[colid*4+widid*nElt2*4+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    
    if(z >0 && z < cutoff) {index = (x[rowid*4+widid*nElt1*4+3]*y[colid*4+widid*nElt2*4+3]) / z;}
    
    a[colid+rowid*nElt2+widid*nElt1*nElt2]=index;
    
  }
}