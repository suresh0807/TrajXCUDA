#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


#include "cudatools.cuh"

__global__ void SDreduceHB(float *a, float *b, int SD_store, int initHBnum)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  if(rowid < SD_store )
  {
    for(int i=0;i<initHBnum;i++)
    {
      b[rowid]+=a[i+rowid*initHBnum];
    }
    b[rowid] = b[rowid]/float(initHBnum);
  }
}


