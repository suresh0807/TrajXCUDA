//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void choose_atoms()
{
  
   
  cout << Elt1 <"within "<< bondist <<" \AA of"<< metal_species<<endl;
  
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));

//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    hipFree(dev_exch);
    
    for(int l=0; l<nstruct/skip; l++)
      {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
        }
      }

          
}