#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void Ofragmat(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
   // printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);
    if (z < bondist)
    {
    atomicAdd(&a[rowid+widid*nElt1],1);
    }
  }
}

