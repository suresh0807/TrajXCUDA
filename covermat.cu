#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void covermat(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    //if(rowid ==0){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z < bondist+0.01 )
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}

__global__ void covermat(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct, float *latti)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    //if(rowid ==0){
    //printf(" %d %d %d %f %f \n",rowid,colid,widid,z,bondist);}
    if (z < bondist )
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }

  /*if(y[colid*3+widid*nElt2*3] > x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] > x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] > x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
  else if(y[colid*3+widid*nElt2*3] < x[rowid*3+widid*nElt1*3] && y[colid*3+widid*nElt2*3+1] < x[rowid*3+widid*nElt1*3+1] && y[colid*3+widid*nElt2*3+2] < x[rowid*3+widid*nElt1*3+2] )
  {
   atomicAdd(&a[rowid+widid*nElt1],1);
  }
 */
 }
}


__global__ void covermat(float *x, float*y, int *a, float minbondist, float maxbondist, int nElt1, int nElt2, int nstruct, float *latti, int dirn, float surf,float mintop, float maxtop)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct && x[rowid*3+widid*nElt1*3+dirn] >=mintop && x[rowid*3+widid*nElt1*3+dirn] <=maxtop&& y[colid*3+widid*nElt2*3+dirn] >=mintop && y[colid*3+widid*nElt2*3+dirn] <=maxtop)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    if ( z >= minbondist && z <= maxbondist &&x[rowid*3+widid*nElt1*3+dirn] > surf+minbondist && x[rowid*3+widid*nElt1*3+dirn] < surf+maxbondist)
    {
       atomicAdd(&a[rowid+widid*nElt1],1);
    }
 }
}
__global__ void covermat(float *x, float*y, int *a, float minbondist, float maxbondist, int nElt1, int nElt2, int nstruct, float *latti, float mintop, float maxtop,int dirn, float surf)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct&& y[colid*3+widid*nElt2*3+dirn] >mintop && y[colid*3+widid*nElt2*3+dirn] <maxtop && x[rowid*3+widid*nElt1*3+dirn] > maxtop)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    if (z > minbondist && z < maxbondist && x[rowid*3+widid*nElt1*3+dirn] > surf+minbondist && x[rowid*3+widid*nElt1*3+dirn] < surf+maxbondist)
    {
       atomicAdd(&a[rowid+widid*nElt1],1);
    }
 }
}


__global__ void covermat(float *x, int *a, float minbondist, float maxbondist, int nElt1, int nstruct, int dirn, float surf,float mintop, float maxtop)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  if(rowid < nElt1  && widid < nstruct && x[rowid*3+widid*nElt1*3+dirn] >mintop && x[rowid*3+widid*nElt1*3+dirn] <maxtop)
  {
    if (x[rowid*3+widid*nElt1*3+dirn] > surf+minbondist && x[rowid*3+widid*nElt1*3+dirn] < surf+maxbondist)
    {
       atomicAdd(&a[rowid+widid*nElt1],1);
    }
 }
}



__global__ void covermat(float *x, int *a, int nElt1, int nstruct, float minaz, float maxaz, int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  if(rowid < nElt1 && widid < nstruct)
  {
    if (x[rowid*3+widid*nElt1*3+dirn] < maxaz &&  x[rowid*3+widid*nElt1*3+dirn] > minaz)
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }
 }
}

//residenttime
/*
__global__ void covermat(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct,float *latti,int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    chkz=fabs(x[rowid*3+widid*nElt1*3+dirn]-y[colid*3+widid*nElt2*3+dirn]);
    if (chkz <= bondist)
    {
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[widid*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      else if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      else if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    
    if (z <= bondist+1.0 ) //(1.0 is tentative to allow for atoms between the spheres)
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }
    }
 }
}
*/
__global__ void covermat(float *x, float*y, int *a, float bondist, int nElt1, int nElt2, int nstruct,float *latti,int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkz;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    chkz=fabs(x[rowid*3+widid*nElt1*3+dirn]-y[colid*3+widid*nElt2*3+dirn]);
    if(chkz <= bondist)  { atomicAdd(&a[rowid+widid*nElt1],1);}
  }
}

__global__ void covermat(float *x, float *y, int *a, float minbondist, float maxbondist, int index,float *latti, int nElt1, int nstruct)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  if(rowid < nElt1 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[widid*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    if (z >= minbondist  &&  z <= maxbondist )
    {
      atomicAdd(&a[rowid+widid*nElt1],1);
    }
  }
}
