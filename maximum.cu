#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

float maximum(float *a,int natoms,int stride)
{
  int j;
  float max;
  max=a[stride];
  for(j=1;j<natoms;j++)
  {
    if(a[j*3+stride] > max)
    {
      max=a[j*3+stride];
    }
  }
return max;
}

