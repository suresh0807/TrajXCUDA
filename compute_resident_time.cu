#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


//resident time

#include "cudatools.cuh"

void compute_resident_time(void)
{

  
    
  //Aintnum is used for all cases - note
  
  //lines for exchange matrix-------------------------------------
  //------------------------------------------------------------------------------------------------------------------
cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;

  int *exch1, *exch2;
  int *dev_exch1;
  exch1= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  exch2= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  
 for(int l=0; l<nstruct/skip; l++)
    {
        for(int i=0; i< nElt1; i++)
        {
          exch1[i+l*nElt1]=0;
	  exch2[i+l*nElt1]=0;
        }
        Aintnum[l]=0;
        Abulknum[l]=0;
    }
      
      float GLOBALAminz=A[dircover];
      float GLOBALAmaxz=A[dircover];
      float MIDAz=0.0, MINAz=0.0, MAXAz=0.0;
      
  for(int i=0; i<1; i++)
  {
    for(int j=1; j<nElt1; j++)
    {
    if(A[j*3+i*nElt1*3+dircover] > GLOBALAmaxz)    GLOBALAmaxz=A[j*3+i*nElt1*3+dircover];
    else if(A[j*3+i*nElt1*3+dircover] < GLOBALAminz)    GLOBALAminz=A[j*3+i*nElt1*3+dircover];
    }
  }
  
  MIDAz=(GLOBALAmaxz+GLOBALAminz)/2.0;
  
  MAXAz=MIDAz+(bondist_int_bulk/2.0);
  MINAz=MIDAz-(bondist_int_bulk/2.0);
  float AVGintnum=0.0, AVGbulknum=0.0;
  
  int whichwater;
      if(lifetime_for=="int"){whichwater=1;}
      else if(lifetime_for=="bulk"){whichwater=1;}
      else if(lifetime_for=="all"){whichwater=2;} 
  
  
  if(lifetime_for=="int")
  {
//Allocate memory in GPU device

float *dev_A1;

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A1,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));


//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A1,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    


//dim3 struct to define elements of the execution configuration

    dim3 dimBlockc(10,10,10);
    dim3 dimGridc((nElt1+dimBlockc.x-1)/dimBlockc.x,(count_metal+dimBlockc.y-1)/dimBlockc.y,((nstruct/skip)+dimBlockc.z-1)/dimBlockc.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

  //bondist_int_bulk refers to the basic allowance within which it the correlation is allowed to start..
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic")
    {
    //covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk,nElt1,count_metal,(nstruct/skip),dev_lattice,dircover);
    covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     //covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk,nElt1,count_metal,(nstruct/skip),dev_lattice,dircover);
     covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
  //bondist_int_bulk2 refers to the larger allowance beyong which it is definitely zero...
    hipMemcpy(dev_exch1,exch2,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic")
    {
    //covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk2,nElt1,count_metal,(nstruct/skip),dev_lattice,dircover);
    covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk2,nElt1,count_metal,(nstruct/skip),dev_lattice);
      
    }
    else if (cell_type == "monoclinic")
    {
     //covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk2,nElt1,count_metal,(nstruct/skip),dev_lattice,dircover);
     covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_B,dev_exch1,bondist_int_bulk2,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch2,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
     hipFree(dev_A1);
     hipFree(dev_B);
     hipFree(dev_exch1);
  
// 1 in exch matrix means the atom satisfies the criterium
   
   for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
	  if (exch2[i+l*nElt1]>0) {exch2[i+l*nElt1]=1;}
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch1[i+l*nElt1]==0) Abulknum[l]++;
        }
        AVGintnum+=Aintnum[l];
        AVGbulknum+=Abulknum[l];
    }
AVGintnum/=float(nstruct/skip);
AVGbulknum/=float(nstruct/skip);
cout <<AVGintnum<<" "<<AVGbulknum<<" "<<AVGintnum+AVGbulknum<<endl;
  }
  
  else if(lifetime_for=="bulk")
  {
 float *dev_A1;

    hipMalloc((void **)&dev_A1,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));


//Copy data from host to device

    hipMemcpy(dev_A1,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    


//dim3 struct to define elements of the execution configuration

    dim3 dimBlockc(32,1,32);
    dim3 dimGridc((nElt1+dimBlockc.x-1)/dimBlockc.x,1,((nstruct/skip)+dimBlockc.z-1)/dimBlockc.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

  
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
  
    hipMemcpy(dev_exch1,exch2,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGridc,dimBlockc>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MIDAz-(bondist_int_bulk2/2.0),MIDAz+(bondist_int_bulk2/2.0),dircover);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGridc,dimBlockc>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MIDAz-(bondist_int_bulk2/2.0),MIDAz+(bondist_int_bulk2/2.0),dircover);
    }
    hipMemcpy(exch2,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
     hipFree(dev_A1);
     hipFree(dev_exch1);
    
// 1 in exch matrix means the atom satisfies the criterium
   
   
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
	  if (exch2[i+l*nElt1]>0) {exch2[i+l*nElt1]=1;}
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch1[i+l*nElt1]==0) Abulknum[l]++;
        }
        AVGintnum+=Aintnum[l];
        AVGbulknum+=Abulknum[l];
    }
AVGintnum/=float(nstruct/skip);
AVGbulknum/=float(nstruct/skip);
cout <<AVGintnum<<" "<<AVGbulknum<<" "<<AVGintnum+AVGbulknum<<endl;   
  }
  
  else if (lifetime_for=="all")
  {
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
  }
  
  ///////////////////////////////////////////Special atom selection between 2 distances/////////////////////////////////////////////////////////////////////////

  
  
else if(choose_atoms=="yes")
{
  float surf,avgsurf=0.0;
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< metal_species<<endl;
  

  int dirn;
  if(choose_dirn=="z") dirn=2;
  else if(choose_dirn=="x") dirn=0;
  else if(choose_dirn=="y") dirn=1;
  
  
  int surfatom;
  for(int i=0 ; i<(nstruct/skip); i++)
  {surfatom=0;surf=0.0;
  for(int j=0; j<count_metal;j++)
  {
    if(METAL[j*3+i*count_metal*3+dirn] > mintop && METAL[j*3+i*count_metal*3+dirn] < maxtop)
    {
    surf+=METAL[j*3+i*count_metal*3+dirn];
    surfatom++;
    }
  }
  surf/=surfatom;
  avgsurf+=surf;
  }
  
  avgsurf/=(nstruct/skip);
  
 cout<<"Average surface atoms position in "<<choose_dirn<<" is "<<avgsurf<<endl;
 cout<<"Average number of surface atoms is "<< surfatom<<endl;
  
 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);
    }
    
    
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    hipMemcpy(dev_exch1,exch2,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,bondist_int_bulk2,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,bondist_int_bulk2,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);
    }
    
    
    hipMemcpy(exch2,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_exch1);
    
    //ofstream exchout;
    //exchout.open("exchout.data");
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
	//exchout<<l<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
	  if (exch2[i+l*nElt1]>0) {exch2[i+l*nElt1]=1;}
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch1[i+l*nElt1]==0) Abulknum[l]++;
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        //exchout<<endl;
      }
      
      //exchout.close();
    // hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 
     
}          
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////

//exchange matrix created--------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------
  ///////////////////////////////////////////Special atom selection between 2 distances from a particlar index/////////////////////////////////////////////////////////////////////////

else if(choose_atoms_from=="yes")
{
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< check_index<<endl;
  
  float *index_xyz;
  index_xyz = (float *) malloc (sizeof(float)*(nstruct/skip)*3);
  
  for(int i=0 ; i<(nstruct/skip); i++)
  {
  for(int j=0 ; j<3; j++)
  {
    index_xyz[i*3+j]+=METAL[check_index*3+i*count_metal*3+j];
  }
  }
 
  
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,index_xyz,sizeof(float)*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(32,1,32);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,1,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
      covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,maxbondist,check_index,dev_lattice,nElt1,(nstruct/skip));
    }
    else if (cell_type == "monoclinic")
    {
      covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,maxbondist,check_index,dev_lattice,nElt1,(nstruct/skip));
    }
    
    
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //we are using just dev_exch1 - careful- dont change the below code
    hipMemcpy(dev_exch1,exch2,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
      covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,bondist_int_bulk2,check_index,dev_lattice,nElt1,(nstruct/skip));
    }
    else if (cell_type == "monoclinic")
    {
      covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,bondist_int_bulk2,check_index,dev_lattice,nElt1,(nstruct/skip));
    }
    
    
    hipMemcpy(exch2,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    
    //ofstream exchout;
    //exchout.open("exchout.data");
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
	//exchout<<l<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
	  if (exch2[i+l*nElt1]>0) {exch2[i+l*nElt1]=1;}
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch1[i+l*nElt1]==0) Abulknum[l]++;
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        //exchout<<endl;
      }
      
           //exchout.close();

   
}          
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////

  
////////////////////////////////////LIFETIME-PART///////////////////////////////////////////
  
  
  
  
    total_time = (nstruct/skip)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    num_bins =  int ((nstruct/skip)/((diffuse_time*1000)/timestep));
    num_bins -= 1; 
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    cout <<num_bins <<" number of bins are chosen for the Hydrogen bond analysis"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    origins = SD_store;
    cout <<"This will correlate for "<<SD_store<<" frames in the input trajectory "<<endl;
    int restarts = origins/skips;
    cout <<restarts<<" restarts from each bin will be taken"<<endl;
  
    
       SDsum1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
	   SDsum1[ia]=0.0;
	}
	
	float *FDsum1;
	FDsum1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=1;ia<SD_store-1;ia++)
        {
	   FDsum1[ia]=0.0;
	}
	float *Hexch1, *Hexch2;

	Hexch1 = (float *) malloc (sizeof(float)*nElt1*SD_store*2);
	Hexch2 = (float *) malloc (sizeof(float)*nElt1*SD_store*2);
 //########################################################################################################################################
//########################################################################################################################################
//BINNING AND AVERAGING THE HBAF
//#########################################################################################################################################
//######################################################################################################################################### 
float *Hexch_sized1;// correlation
float *Hexch_sized;// correlation

    for(int i=0; i<num_bins;i++)  //start bin
    {//int avgcounter=0; 
        printf("Cuda start: All lifetimes  %d \n",i);


    
         for(int i1=0;i1<SD_store*2;i1++)
         {
	     for(int k1=0; k1<nElt1; k1++)//shared hydrogen
             {
	       Hexch1[k1+i1*nElt1]=0.0;
	       Hexch2[k1+i1*nElt1]=0.0;
	     }
	 }
	 

int **initHBcol1;// storing the location of the atom in the exch matrix 

initHBcol1 = (int **) malloc (sizeof(int *)*SD_store*2);
     for(int i1=0; i1<SD_store*2; i1++)// for each frame
     {
     initHBcol1[i1] = (int *) malloc (sizeof(int)*Aintnum[i1+i*SD_store]);
     for(int j1=0; j1<Aintnum[i1+i*SD_store]; j1++)// for number of H bonds this frame
     {
      initHBcol1[i1][j1]=0;
     }
     }
     
    if(whichwater != 2)
    {
    for(int i1=0;i1<SD_store*2;i1++)
     {int chker=0;
      for(int j1=0; j1<nElt1; j1++)
       {
	if(exch1[j1+i1*nElt1+i*SD_store*nElt1] == 1 ) 
        {
	  Hexch1[j1+i1*nElt1]=1.0; //acceptor check
	  initHBcol1[i1][chker] = j1+i1*nElt1; chker+=1;
	}
	if(exch2[j1+i1*nElt1+i*SD_store*nElt1] == 1 ) 
        {
	  Hexch2[j1+i1*nElt1]=1.0; 
	}
      }
      //if(i==0){cout<<i1<<" "<<chker<<endl;}
      
     }
    }
	//cout <<"exch matrix done "<<endl;
	
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////
     //////////////////////////////////////////////////////origin//////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////////////////
     for(int j=0; j<SD_store; j=j+skips) //go through restart points or origins
     {      
       //cout <<Aintnum[j+i*SD_store]<<" ";
       //cout<<initHBnum[j+i*SD_store]<<endl;
       if(Aintnum[j+i*SD_store] > 0){
       Hexch_sized1 = (float *) malloc (sizeof(float)*Aintnum[j+i*SD_store]*SD_store);
       Hexch_sized = (float *) malloc (sizeof(float)*Aintnum[j+i*SD_store]*SD_store);
         for(int ia=0;ia<SD_store;ia++)
         {
           for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)////only those present in the first frame in the restart bin
           { 
	        Hexch_sized1[ja+ia*Aintnum[j+i*SD_store]] = Hexch1[initHBcol1[j][ja]+ia*nElt1];
	        Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] = Hexch2[initHBcol1[j][ja]+ia*nElt1];
	   }
	 }
	/* 
	 if(j==2 && i==0)
	 {
	   cout<<endl;
	   for(int ia=0;ia<SD_store;ia++)
           {
           for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)////only those present in the first frame in the restart bin
           { 
	       cout<<Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] <<" ";
	   }
	   cout<<endl;
	   }
	   cout<<"after: "<<endl;
	 }
	 */
	 
	 if(HB_lifestyle=="continuous")
	 {
         for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)///for continuous lifetime
         {
           for(int ia=1;ia<SD_store;ia++)
           {
	     if(Hexch_sized[ja+(ia-1)*Aintnum[j+i*SD_store]] == 0) {Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] = 0;Hexch_sized1[ja+ia*Aintnum[j+i*SD_store]] = 0;} 
	   }
	 }
	 }
	 
	 
	 if(HB_lifestyle=="transient")
	 {
	   //do transient time approximation here
	   
         for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)///for transient intermittent lifetime
         {
           for(int ia=0;ia<SD_store;ia++)
           {
	     if(Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] == 0) 
	     {
	       for(int chk=1; chk<=transtime; chk++)
	       {
		 if(ia + chk >= SD_store){break;}
		 else if(Hexch_sized[ja+(ia+chk)*Aintnum[j+i*SD_store]] == 1){ Hexch_sized[ja+ia*Aintnum[j+i*SD_store]]=1;Hexch_sized1[ja+ia*Aintnum[j+i*SD_store]]=1;break;}
	       }
	    } 
	   }
	 }
	 
	 for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)///for imposing continuous lifetime after transient time
         {
           for(int ia=1;ia<SD_store;ia++)
           {
	     if(Hexch_sized[ja+(ia-1)*Aintnum[j+i*SD_store]] == 0) {Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] = 0;Hexch_sized1[ja+ia*Aintnum[j+i*SD_store]] = 0;} 
	   }
	 }
	 }
      /*
       * 
       * 
       * 
         if(j==2 && i==0)
	 {
	   cout<<endl;
	   for(int ia=0;ia<SD_store;ia++)
           {
           for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)////only those present in the first frame in the restart bin
           { 
	       cout<<Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] <<" ";
	   }
	   cout<<endl;
	   }
	 }
      
      
         */
       //SD=(float *) malloc (sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
      
      
      
      SD1=(float *) malloc (sizeof(float)*Aintnum[j+i*SD_store]*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
	 for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)
	 {
           //SD[ja+ia*initHBnum[j+i*SD_store]]=0.0;
	   SD1[ja+ia*Aintnum[j+i*SD_store]]=0.0;
	 }
	}
       
      
      dim3 dimBlocka(32,1,32);
      dim3 dimGrida((Aintnum[j+i*SD_store]+dimBlocka.x-1)/dimBlocka.x,1,(SD_store+dimBlocka.z-1)/dimBlocka.z);

       
      
      
      float *dev_A1;
            
      hipMalloc((void **)&dev_A,sizeof(float)*Aintnum[j+i*SD_store]*SD_store);
      hipMalloc((void **)&dev_A1,sizeof(float)*Aintnum[j+i*SD_store]*SD_store);
      hipMalloc((void **)&dev_SD1,sizeof(float)*Aintnum[j+i*SD_store]*SD_store);
      
      hipMemcpy(dev_A,Hexch_sized1,sizeof(float)*Aintnum[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
      hipMemcpy(dev_A1,Hexch_sized,sizeof(float)*Aintnum[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
      hipMemcpy(dev_SD1,SD1,sizeof(float)*Aintnum[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);

      HBAF_calc<<<dimGrida,dimBlocka>>>(dev_A,dev_A1,dev_SD1,SD_store,Aintnum[j+i*SD_store],i,j,origins,skips);

      //hipMemcpy(SD1,dev_SD1,sizeof(float)*Aintnum[j+i*SD_store]*SD_store,hipMemcpyDeviceToHost);
      hipFree(dev_A);
      hipFree(dev_A1);
      hipFree(dev_lattice);
      //SDavg=(float *) malloc (sizeof(float)*SD_store);
      SDavg1=(float *) malloc (sizeof(float)*SD_store);
      for(int ja =0; ja<SD_store;ja++)
      {
      //SDavg[ja]=0.0;
      SDavg1[ja]=0.0;
      }
      int fairy=0;
      //hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
      //hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

      dim3 dimBlocks(1024,1,1);
      dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);
      //SDreduce<<<dimGrids,dimBlocks>>>(dev_SD,dev_SDavg,SD_store,initHBnum[j+i*SD_store],fairy);
      //hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
       
      //hipFree(dev_SD);
      //hipFree(dev_SDavg);
      
      hipMalloc((void **)&dev_SDavg1,sizeof(float)*SD_store);
      hipMemcpy(dev_SDavg1,SDavg1,sizeof(float)*SD_store,hipMemcpyHostToDevice);
      SDreduce<<<dimGrids,dimBlocks>>>(dev_SD1,dev_SDavg1,SD_store,Aintnum[j+i*SD_store],fairy);
      hipMemcpy(SDavg1,dev_SDavg1,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
       
      hipFree(dev_SD1);
      hipFree(dev_SDavg1);
      
           
     // SDsum1[0]+=1;
//cout<<Aintnum[j+i*SD_store]<<" "<<SDavg1[0]<<" ";
      for(int ja =0; ja<SD_store;ja++)
      {
      SDsum1[ja]+=SDavg1[ja];
      //SDsum1[ja]+=(SDavg1[ja]/SDavg1[0]);
      }
  
      for(int ja=1; ja<SD_store -1;ja++)
      {
      //FDsum[ja]+=-((SDavg[ja+1]-SDavg[ja-1]) / (((ja+1)*(timestep/1000))-((ja-1)*(timestep/1000))));
      FDsum1[ja]+=-((SDavg1[ja+1]-SDavg1[ja-1]) / (((ja+1)*(timestep/1000))-((ja-1)*(timestep/1000))));      
      }
      
      
      
      
      free(Hexch_sized);
      //free(SDavg);
      //free(SD);
      free(Hexch_sized1);
      free(SDavg1);
      //free(SD1);
      //avgcounter++;
       }
    }//origins over
    

  
  free(initHBcol1);
     
   }

  free(Hexch1);
   free(Hexch2);
  ofstream Hexchplot, Hexchplotac;
  
      Hexchplotac.open("ct.data");
     for(int ja =0; ja<SD_store;ja++)
     {
       //SDsum[ja]/=float(num_bins*restarts);
       SDsum1[ja]/=float(num_bins*restarts);
       //Hexchplotac <<ja*timestep/1000<<" "<<SDsum1[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<SDsum1[ja]/SDsum1[0]<<endl;
       //Hexchplot <<ja*timestep/1000<<" "<<SDsum[ja]<<endl;
      } 
      //Hexchplot.close();
      Hexchplotac.close();
      
    
      //Hexchplot.open("ft-pair.data");
      Hexchplotac.open("ft.data");
     for(int ja =1; ja<SD_store-1;ja++)
      {
       //FDsum[ja]/=num_bins*restarts;
       FDsum1[ja]/=num_bins*restarts;
       //Hexchplot <<ja*timestep/1000<<" "<<FDsum[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<FDsum1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     
     int avg_every = 100;
     float *FD_avg1;
     //FD_avg=(float *) malloc (sizeof(float)*SD_store);
     FD_avg1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
          // FD_avg[ia]=0.0;
	   FD_avg1[ia]=0.0;
	}
     
      for(int ja=(avg_every/2)+1; ja<SD_store -(avg_every/2)-1;ja++)
      {
	for(int ka=ja-(avg_every/2);ka<=ja+(avg_every/2)-1;ka++)
	{
	 // FD_avg[ja]+= FDsum[ka];
	  FD_avg1[ja]+= FDsum1[ka];
	}
	//FD_avg[ja]/=avg_every;
	FD_avg1[ja]/=avg_every;
      }
      
      Hexchplotac.open("ft-avg.data");
      //Hexchplot.open("ft-avg-pair.data");
     for(int ja =(avg_every/2)+1; ja<SD_store -(avg_every/2)-1;ja++)
      {
       //Hexchplot <<ja*timestep/1000<<" "<<FD_avg[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<FD_avg1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     
      
      
      
     float lifetime1;
     float *int_SDsum1;
     float *cum_SDsum1;
     //int_SDsum = (float*) malloc (sizeof(float)*SD_store);
     //cum_SDsum = (float*) malloc (sizeof(float)*SD_store);
     int_SDsum1 = (float*) malloc (sizeof(float)*SD_store);
     cum_SDsum1 = (float*) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
           //int_SDsum[ia]=0.0;
	   //cum_SDsum[ia]=0.0;
	   int_SDsum1[ia]=0.0;
	   cum_SDsum1[ia]=0.0;
	}
     for(int ja =1; ja<SD_store;ja++)
      {
       //int_SDsum[ja]= (((ja*timestep*0.001) - ((ja-1)*timestep*0.001)) * ((SDsum[ja] + SDsum[ja-1])/2.0));
       //cum_SDsum[ja]=cum_SDsum[ja-1]+int_SDsum[ja];
       int_SDsum1[ja]= (((ja*timestep*0.001) - ((ja-1)*timestep*0.001)) * (((SDsum1[ja]/SDsum1[0]) + (SDsum1[ja-1]/SDsum1[0]))/2.0));
       cum_SDsum1[ja]=cum_SDsum1[ja-1]+int_SDsum1[ja];
       if(ja == SD_store-1) {//lifetime = cum_SDsum[ja];
	 lifetime1 = cum_SDsum1[ja];}
      }  
     
     //Hexchplot.open("ct-integrate-pair.data");
     Hexchplotac.open("ct-integrate.data");
     for(int ja =0; ja<SD_store;ja++)
      {
       //Hexchplot <<ja*timestep/1000<<" "<<cum_SDsum[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<cum_SDsum1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     //Hexchplot.open("Hbond-lifetime-pair.data");
     //Hexchplot <<"Lifetime from the integral of c(t) is : "<<lifetime<<" ps"<<endl;
     //Hexchplot.close();
     Hexchplot.open("lifetime.data");
     Hexchplot <<"Lifetime from the integral of c(t) is : "<<lifetime1<<" ps"<<endl;
     Hexchplot.close();
     
 

 
  free(A);
  free(B);
 }
