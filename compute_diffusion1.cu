#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"



void compute_diffusion1()
{
    if(plot=="int")        { nElt1 = nA_int;}
    else if(plot=="bulk")  { nElt1 = nA_bulk;}
    
    
    if(diffuse_direction == "xyz") {xsrt=0;xend=3;xski=1;}
    else if(diffuse_direction == "xy") {xsrt=0;xend=2;xski=1;}
    else if(diffuse_direction == "x") {xsrt=0;xend=1;xski=1;}
    else if(diffuse_direction == "y") {xsrt=1;xend=2;xski=1;}
    else if(diffuse_direction == "z") {xsrt=2;xend=3;xski=1;}
    else if(diffuse_direction == "xz") {xsrt=0;xend=3;xski=2;}
    else if(diffuse_direction == "yz") {xsrt=1;xend=3;xski=1;}
    
    
    total_time = (nstruct)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    cout <<"In each bin "<<SD_store<<" frames are kept"<<endl;
    cout <<nstruct/skips<<" restarts will be taken"<<endl;

    SD=(float *) malloc (sizeof(float)*nElt1*SD_store);

      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SD[k+j*nElt1]=0.0;
	}
      }
      cout<<"bull"<<endl;
    SDsum=(float *) malloc (sizeof(float)*nElt1*(nstruct/skips));

      for(int j=0;j<nstruct/skips;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SDsum[k+j*nElt1]=0.0;
	}
      }
    cout<<"bull"<<endl;
    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_B,sizeof(float)*(nstruct/skip)*nElt2*3);
    hipMalloc((void **)&dev_SDsum,sizeof(float)*nElt1*nstruct/skips);
    hipMalloc((void **)&dev_SD,sizeof(float)*nElt1*SD_store);
   
    if(plot=="int")
    {
      hipMemcpy(dev_A,A_int,sizeof(float)*nstruct*nA_int*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B_int,sizeof(float)*nstruct*nB_int*3,hipMemcpyHostToDevice);
        cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;
    }
    else if(plot=="bulk")
    {
      hipMemcpy(dev_A,A_bulk,sizeof(float)*nstruct*nA_bulk*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B_bulk,sizeof(float)*nstruct*nB_bulk*3,hipMemcpyHostToDevice);
        cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;
    }
    else if(plot=="all")
    {
cout<<"hi "<< (nstruct/skip)*nElt1*3<<endl;
      hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
cout<<"hi"<<endl;
    hipMemcpy(dev_B,B,sizeof(float)*(nstruct/skip)*nElt2*3,hipMemcpyHostToDevice);
        cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;
    }
    
    


    hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store,hipMemcpyHostToDevice);

    for(int i=1; i<SD_store;i++)
    {      
      
      for(int j1=0;j1<nstruct/skips;j1++)
      {
	for(int k1=0; k1<nElt1; k1++)
	{
          SDsum[k1+j1*nElt1]=0.0;
	}
      }
     
      hipMemcpy(dev_SDsum,SDsum,sizeof(float)*nElt1*nstruct/skips,hipMemcpyHostToDevice);

      //cout<<"Ha start"<<endl;

      dim3 dimBlock(32,1,32);
      dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,1,(nstruct+dimBlock.z-1)/dimBlock.z);
      SD_calc<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_SDsum,SD_store,nElt1,i,(nstruct-i)/skips,skips,xsrt,xend,xski);

      //cout<<"Ha end"<<endl;
      
      dim3 dimBlockf(1024,1,1);
      dim3 dimGridf((nElt1+dimBlockf.x-1)/dimBlockf.x,1,1);      
      SDreducef<<<dimGridf,dimBlockf>>>(dev_SDsum,dev_SD,nElt1,SD_store,(nstruct-i)/skips,skips,i);
      
      
      hipMemcpy(SDsum,dev_SDsum,sizeof(float)*nElt1*nstruct/skips,hipMemcpyDeviceToHost);
      //cout<<i<<" step done"<<endl;
    }

    hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store,hipMemcpyDeviceToHost);
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_SDsum);
    
    
        
    SDavg=(float *) malloc (sizeof(float)*SD_store);
    for(int j =0; j<SD_store;j++)
    {
      SDavg[j]=0.0;
    }
    hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
    hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

    dim3 dimBlocks(1024,1,1);
    dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);
    SDreduce<<<dimGrids,dimBlocks>>>(dev_SD,dev_SDavg,SD_store,nElt1);

    hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
    hipFree(dev_SD);
    hipFree(dev_SDavg);
    
// printing the msd data to be visualized

    FILE *MSDplot=fopen("msd.data","wt");
      fprintf(MSDplot,"# Time (ps) MSD (A^(2)) \n");
    for(int i =1; i<SD_store;i++)
    {
      fprintf(MSDplot,"%f %f\n",(float(i)*timestep)/1000.0,SDavg[i]);
    }
    fclose(MSDplot);
   
    FILE *DIFFplot=fopen("diffco.data","wt");
      fprintf(DIFFplot,"# Time (ps) D (A^(2)/fs) \n");
    for(int i =1; i<SD_store;i++)
    {
      fprintf(DIFFplot,"%f %f\n",(float(i)*timestep)/1000.0,SDavg[i]/(6*i*timestep));
    }
    fclose(DIFFplot);


 
}
