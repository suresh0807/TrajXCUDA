//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void compute_coulomb_charge(void)
{


  float *coulomb_distmat;
  float *coulomb_final;
  coulomb_distmat=(float *) malloc (sizeof(float)*nElt1*nElt2*(nstruct/skip));
  coulomb_final=(float *) malloc (sizeof(float)*(nstruct/skip));
  for(int i=0;i<(nstruct/skip);i++)
  {
    coulomb_final[i]=0.0;
  for(int j=0;j<nElt1;j++)
  {
    
  for(int k=0;k<nElt2;k++)
  {

    coulomb_distmat[k+j*nElt2+i*nElt1*nElt2]=0.0;

  }
  }
  }
  float *dev_coulomb_distmat;
  
  
  
  
  hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
  hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*4);
  hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*4);
  hipMalloc((void **)&dev_coulomb_distmat,sizeof(float)*nElt1*nElt2*(nstruct/skip));
    
//Copy data from host to device

  hipMemcpy(dev_A,A_all,sizeof(float)*nElt1*(nstruct/skip)*4,hipMemcpyHostToDevice);
  hipMemcpy(dev_B,B_all,sizeof(float)*nElt2*(nstruct/skip)*4,hipMemcpyHostToDevice);
  hipMemcpy(dev_lattice,lattice,sizeof(float)*6*(nstruct/skip),hipMemcpyHostToDevice);  
  hipMemcpy(dev_coulomb_distmat,coulomb_distmat,sizeof(float)*nElt1*nElt2*(nstruct/skip),hipMemcpyHostToDevice);
  
  cout<<"GPU coulomb matrix computation"<<endl;
  
    dim3 dimBlock(10,10,10);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);
   
    
    if(cell_type == "orthorhombic")
    {
	distmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_coulomb_distmat,nElt1,nElt2,(nstruct/skip),dev_lattice,Coulomb_cutoff);
    }
    else if(cell_type == "monoclinic")
    {
	distmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_coulomb_distmat,nElt1,nElt2,(nstruct/skip),dev_lattice,Coulomb_cutoff);
    }
    
    hipMemcpy(coulomb_distmat,dev_coulomb_distmat,sizeof(float)*nElt1*nElt2*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    cout<<"GPU computation completed: freeing memory"<<endl;
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_coulomb_distmat);
    hipFree(dev_lattice);
    
    for(int i=0;i<(nstruct/skip);i++)
    {
    for(int j=0;j<nElt1;j++)
    {
    for(int k=0;k<nElt2;k++)
    {
    coulomb_final[i]+=coulomb_distmat[k+j*nElt2+i*nElt1*nElt2];
    }
    }
    }
  
  

    ofstream coulomb;
    coulomb.open("coulomb_E.dat");
    for(int i=0;i<(nstruct/skip);i++)
    {
    coulomb<<i<<" "<<coulomb_final[i]/2<<endl;
    }
    coulomb.close();
    
    //free(A_all);
    //free(B_all);
  }





