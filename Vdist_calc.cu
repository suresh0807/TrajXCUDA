#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################





#include "cudatools.cuh"

__global__ void Vdist_calc(float *A,float *B, float *density,float *xdensity,float *ydensity,float *zdensity,int nstruct,int natoms,int zsplit, float *ztick,int Dirn)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  float vel;
  if(rowid < nstruct && colid <natoms)
  {
        for(int m=0;m<zsplit;m++)
        {
        if( A[colid*3+(rowid*natoms*3)+Dirn] >= ztick[m*2] && A[colid*3+(rowid*natoms*3)+Dirn] < ztick[m*2+1])
          {
	    vel= sqrt((B[colid*3+(rowid*natoms*3)]*B[colid*3+(rowid*natoms*3)]) + (B[colid*3+(rowid*natoms*3)+1]*B[colid*3+(rowid*natoms*3)+1]) + (B[colid*3+(rowid*natoms*3)+2]*B[colid*3+(rowid*natoms*3)+2]));
	    atomicAdd(&density[m],vel);
	    vel=fabs(B[colid*3+(rowid*natoms*3)]);
	    atomicAdd(&xdensity[m],vel);
	    vel=fabs(B[colid*3+(rowid*natoms*3)+1]);
	    atomicAdd(&ydensity[m],vel);
	    vel=fabs(B[colid*3+(rowid*natoms*3)+2]);
	    atomicAdd(&zdensity[m],vel);
            break;
          }
        }
   }
}