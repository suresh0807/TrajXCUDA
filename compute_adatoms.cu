#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


///Now can process a very long trajectory also!! Still some optimizations needed in the initial regions

#include "cudatools.cuh"

void compute_adatoms(void)
{

  //lines for exchange matrix------NO implication whatsoever for the Hbonds computation-------------------------------
  //------------------------------------------------------------------------------------------------------------------
cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;

  int *exch1;
  int *dev_exch1;
  exch1= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  // Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
        for(int i=0; i< nElt1; i++)
        {
          exch1[i+l*nElt1]=0;
        }
        Aintnum[l]=0;
        Abulknum[l]=0;
    }
      
      float GLOBALAminz=A[dircover];
      float GLOBALAmaxz=A[dircover];
      float MIDAz=0.0, MINAz=0.0, MAXAz=0.0;
      
  for(int i=0; i<1; i++)
  {
    for(int j=1; j<nElt1; j++)
    {
    if(A[j*3+i*nElt1*3+dircover] > GLOBALAmaxz)    GLOBALAmaxz=A[j*3+i*nElt1*3+dircover];
    else if(A[j*3+i*nElt1*3+dircover] < GLOBALAminz)    GLOBALAminz=A[j*3+i*nElt1*3+dircover];
    }
  }
  
  MIDAz=(GLOBALAmaxz+GLOBALAminz)/2.0;

  MAXAz=MIDAz+(bondist_int_bulk/2.0);
  MINAz=MIDAz-(bondist_int_bulk/2.0);
  
 
   ///////////////////////////////////////////Special atom selection between 2 distances/////////////////////////////////////////////////////////////////////////

  
if(choose_atoms=="yes")
{
  float surf,avgsurf=0.0;
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< metal_species<<endl;
  

  int dirn;
  if(choose_dirn=="z") dirn=2;
  else if(choose_dirn=="x") dirn=0;
  else if(choose_dirn=="y") dirn=1;
  
  
  int surfatom;
  for(int i=0 ; i<(nstruct/skip); i++)
  {surfatom=0;surf=0.0;
  for(int j=0; j<count_metal;j++)
  {
    if(METAL[j*3+i*count_metal*3+dirn] > mintop && METAL[j*3+i*count_metal*3+dirn] < maxtop)
    {
    surf+=METAL[j*3+i*count_metal*3+dirn];
    surfatom++;
    }
  }
  surf/=surfatom;
  avgsurf+=surf;
  }
  
  avgsurf/=(nstruct/skip);
  
 cout<<"Average surface atom position in "<<choose_dirn<<" is "<<avgsurf<<endl;
 cout<<"Average number of surface atoms is "<< surfatom<<endl;
  
 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);

    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,mintop,maxtop,dirn,avgsurf);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch1,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,mintop,maxtop,dirn,avgsurf);
    }
    
    
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    
    //ofstream exchout;
    //exchout.open("exchout.data");
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
	//exchout<<l<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1;}
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        //exchout<<endl;
      }
      
      //exchout.close();
    // hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 
     
  
}          
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////

//exchange matrix created--------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------
 /* debug--------------
 for(int l=0; l<nstruct/skip; l++)
    {
        for(int i=0; i< nElt1; i++)
        {
          if(exch1[i+l*nElt1]==1){cout<<l<<" "<<i<<endl;}
        }
    }
  */
  int *Onum; //number of O atoms within a specific distance (must be greater than 2 if used 3.2) of the H atom.
  int *Hbondneigh; // Ids of the (O) neighbours of the H atom. 
  float *Hbondneighdist; // OH distance for all neighbours.
  
  float *Hdistmat, *dev_Hdistmat;// Distance matrix to capture the O within x \AA of H atom.
  
  int OHsamples=50; /// maximum number of OH bonds considered within bonddist sphere
  
    Onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    Hbondneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    Hbondneighdist = (float *) malloc (sizeof(float)*OHsamples*nElt1*(nstruct/skip));
    Hdistmat = (float *) malloc (sizeof(float)*nElt1*nElt2);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      Onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k]=0.0;
      }
    }
  }   
      
 for(int i=0;i<nElt1;i++)
  {
    for(int j=0; j<nElt2; j++)
    {
      Hdistmat[j+i*nElt2]=0.0;
    }
  }
      
cout <<"gpu begins distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_Hdistmat,sizeof(float)*nElt1*nElt2);
    
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_Hdistmat,Hdistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
    
   //dim3 struct to define elements of the execution configuration


    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,1);
for(int i = 0; i<(nstruct/skip); i++) /// go into each frame
{
   for(int i1=0;i1<nElt1;i1++)///initialize Hdistmat for 1 frame
  {
    for(int j1=0; j1<nElt2; j1++)
    {
      Hdistmat[j1+i1*nElt2]=0.0;
    }
  }
  hipMemcpy(dev_Hdistmat,Hdistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_exch1,dev_Hdistmat,bondist,nElt1,nElt2,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_exch1,dev_Hdistmat,bondist,nElt1,nElt2,dev_lattice,i);      
    }

hipMemcpy(Hdistmat,dev_Hdistmat,sizeof(float)*nElt1*nElt2,hipMemcpyDeviceToHost);
   
  for(int i1=0;i1<nElt1;i1++) //hydrogen
  {
    int k=0;
    for(int j1=0; j1<nElt2; j1++) //oxygen 
    {
      if(Hdistmat[j1+i1*nElt2] !=0.0) // check if an oxygen atom is within the 'bonddist' distance from hydrogen atom
      {
	Onum[i1+i*nElt1]++;
	Hbondneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	Hbondneighdist[i1*OHsamples+i*nElt1*OHsamples+k] = Hdistmat[j1+i1*nElt2];
	k++;
      }
    }
  }

}   
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_Hdistmat);
    hipFree(dev_lattice);
    hipFree(dev_exch1);


//Hbondneigh contain the IDs of four possible oxygen neighbour atoms if the hydrogen chosen is participating in a hydrogen bond.
//Onum has the number of oxygen atoms within a specified distance (see input file).

cout <<"gpu ends"<<endl; 

      
//swapping the indices of nearest oxygens to get the two shortest OH bonds up front, 
//since it usually describes the H bond

cout<<"Swapping of OH bonds based on distance "<<endl;

float swapa,swapb;
    for(int i=0;i<(nstruct/skip);i++)
     {
      for(int j=0; j<nElt1; j++)
       {
	 
	for(int swf=0; swf<Onum[j+i*nElt1]-1;swf++)
	{
	for(int sws=swf+1; sws<Onum[j+i*nElt1];sws++)
	{
	if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf] > Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws] && sws !=swf && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws] !=0.0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf] !=0.0) 
	{
	   swapa = Hbondneigh[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    Hbondneigh[j*OHsamples+i*nElt1*OHsamples+swf]=Hbondneigh[j*OHsamples+i*nElt1*OHsamples+sws];
	    Hbondneigh[j*OHsamples+i*nElt1*OHsamples+sws]=swapa;
	   swapb = Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf]=Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws];
	    Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws]=swapb;
	}
	}
	}	
       }
     }	
   
cout<<" done !!!"<<endl;
   
   
   ofstream Hneigh;
Hneigh.open("Adneigh.data");
Hneigh<<"S H N F FD      S   SD      OHO     ALEX<   MATTI   OO"<<endl;
  for(int i=0;i<nstruct/skip;i++)
  {      
    for(int j=0; j<nElt1; j++)
    {  
      for(int k=0; k<OHsamples;k++)
      {
	if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] !=0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k] != Hbondneighdist[j*OHsamples+i*nElt1*OHsamples])
	{
      Hneigh<<i<<" "<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k]<<endl;
	}
      //Hneigh<<j<<" "<<Onum[j+i*nElt1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+1]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+2]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+2]<<" "<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+3]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+3]<<endl;
      }
    } 
  }  
Hneigh.close();
  
cout<<"Done!!!"<<endl;

int *Batop, *Bbridge, *Bhcp, *Bfcc;
int *Catop, *Cbridge, *Chcp, *Cfcc;

Batop= (int *) malloc (sizeof(int)*nstruct/skip*nElt1);
Bbridge= (int *) malloc (sizeof(int)*nstruct/skip*nElt1);
Bhcp= (int *) malloc (sizeof(int)*nstruct/skip*nElt1);
Bfcc= (int *) malloc (sizeof(int)*nstruct/skip*nElt1);
Catop= (int *) malloc (sizeof(int)*nstruct/skip);
Cbridge= (int *) malloc (sizeof(int)*nstruct/skip);
Chcp= (int *) malloc (sizeof(int)*nstruct/skip);
Cfcc= (int *) malloc (sizeof(int)*nstruct/skip);
for(int i=0;i<nstruct/skip;i++)
  {      
    for(int j=0; j<nElt1; j++)
    {
      Batop[j+i*nElt1]=0;
      Bbridge[j+i*nElt1]=0;
      Bhcp[j+i*nElt1]=0;
      Bfcc[j+i*nElt1]=0;
    }
      Catop[i]=0;
      Cbridge[i]=0;
      Chcp[i]=0;
      Cfcc[i]=0;
  }

int atop=0;
int bridge=0;
int fcc=0;
int hcp=0;
int k=0;
for(int i=0;i<nstruct/skip;i++)
//for(int i=0;i<1;i++)
  {      
    for(int j=0; j<nElt1; j++)
    {  
      float A1=Hbondneighdist[j*OHsamples+i*nElt1*OHsamples];
      float A2=Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+1];
      float A3=Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+2];
      
      if(A1<2.75 && A2<2.75 && A3>2.70&& A1!=0&&A2!=0&&A3!=0) {bridge++; Bbridge[j+i*nElt1]=1;Cbridge[i]++;
	//cout<<"bridge "<<i<<endl;
	cout<<i<<" "<<A1<<" "<<A2<<" "<<A3<<" bridge"<<endl;
	      }
      else if(A2>2.75 && A1!=0 && A2!=0) 
        {
	atop++; Batop[j+i*nElt1]=1;Catop[i]++;
	cout<<i<<" "<<A1<<" "<<A2<<" "<<A3<<" atop"<<endl;
	}
      else if(A1<2.75 && A2<2.75 && A3<2.75&& A1!=0&&A2!=0&&A3!=0) 
      {
	
	for(k=3;k<25;k++)
	{
	  
	  if(Hbondneighdist[k+j*OHsamples+i*nElt1*OHsamples] !=0)
	  {
	    //cout<<A[j*3+i*nElt1*3]<<" "<<A[Hbondneigh[k+j*OHsamples+i*nElt1*OHsamples]*3+i*nElt1*3]<<" "<<A[j*3+i*nElt1*3+1]<<" "<<A[Hbondneigh[k+j*OHsamples+i*nElt1*OHsamples]*3+i*nElt1*3+1]<<endl;
	  if(fabs(A[j*3+i*nElt1*3]-A[Hbondneigh[k+j*OHsamples+i*nElt1*OHsamples]*3+i*nElt1*3])<0.6 && fabs(A[j*3+i*nElt1*3+1]-A[Hbondneigh[k+j*OHsamples+i*nElt1*OHsamples]*3+i*nElt1*3+1])<0.6)
	  {
	    hcp++;Bhcp[j+i*nElt1]=1;Chcp[i]++;cout<<i<<" "<<A1<<" "<<A2<<" "<<A3<<" hcp"<<endl;break;
	  }
	  }
        }
        if(k==25){fcc++; Bfcc[j+i*nElt1]=1;Cfcc[i]++;cout<<i<<" "<<A1<<" "<<A2<<" "<<A3<<" fcc"<<endl;}
      }
      else if(A1!=0&&A2!=0&&A3!=0) 
      {
	cout<<i<<endl;
      }
      //if(exch1[j+i*nElt1]>0) {cout<<i<<endl;}
    }
  }

   cout<<"Atop :"<<atop<<" Bridge :"<<bridge<<" fcc :"<<fcc<<" hcp :"<<hcp<<endl;
  free(exch1);
  free(Hbondneighdist);
  free(Onum);
  free(Hbondneigh);
  free(Hdistmat);
  
  
  compute_lifetime(Bfcc,Cfcc,25);
  system("mv ct.data ct-fcc.data; mv ct-integrate.data ct-fcc-integrate.data; mv ft.data ft-fcc.data; mv ft-avg.data ft-fcc-avg.data; mv lifetime.data lifetime-fcc.data");
  compute_lifetime(Bhcp,Chcp,25);
  system("mv ct.data ct-hcp.data; mv ct-integrate.data ct-hcp-integrate.data; mv ft.data ft-hcp.data; mv ft-avg.data ft-hcp-avg.data; mv lifetime.data lifetime-hcp.data");
  compute_lifetime(Bbridge,Cbridge,5);
  system("mv ct.data ct-bridge.data; mv ct-integrate.data ct-bridge-integrate.data; mv ft.data ft-bridge.data; mv ft-avg.data ft-bridge-avg.data; mv lifetime.data lifetime-bridge.data");
  compute_lifetime(Batop,Catop,5);
  system("mv ct.data ct-atop.data; mv ct-integrate.data ct-atop-integrate.data; mv ft.data ft-atop.data; mv ft-avg.data ft-atop-avg.data; mv lifetime.data lifetime-atop.data");
  
  free(Batop);
  free(Bbridge);
  free(Bfcc);
  free(Bhcp);
  free(Catop);
  free(Cbridge);
  free(Cfcc);
  free(Chcp);
}
