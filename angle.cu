#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"
//central atom then the other two
float angle(float x, float y, float z, float x1, float y1, float z1, float x2, float y2, float z2)
{
  //float pi = 3.14159;
  float AB[3], AC[3];
  float ABdotAC=0.0;
  float lenAB=0.0, lenAC=0.0;
  float angleret;
  AB[0] = (x1-x);
  AB[1] = (y1-y);
  AB[2] = (z1-z);
  AC[0] = (x2-x);
  AC[1] = (y2-y);
  AC[2] = (z2-z);
  for(int k=0;k<3;k++)
  {
  ABdotAC += AB[k]*AC[k];
  }
  for(int k=0;k<3;k++)
  {
    lenAB += square(AB[k]);
    lenAC += square(AC[k]);
  }
  lenAB=sqrt(lenAB);
  lenAC=sqrt(lenAC);
  if(ABdotAC/(lenAB*lenAC) > 1.0 || ABdotAC/(lenAB*lenAC) < -1.0)
  {
    angleret = 3.14159;
  }
  else
  {
  angleret= acos(ABdotAC/(lenAB*lenAC)) ;//* 180/pi;
  }
  return (angleret);
}

float angle(float x1, float y1, float z1, float x2, float y2, float z2,float x3, float y3, float z3, float x4, float y4, float z4)
{
  //float pi = 3.14159;
  float AB[3], AC[3];
  float ABdotAC=0.0;
  float lenAB=0.0, lenAC=0.0;
  float angleret;
  AB[0] = (x2-x1);
  AB[1] = (y2-y1);
  AB[2] = (z2-z1);
  AC[0] = (x4-x3);
  AC[1] = (y4-y3);
  AC[2] = (z4-z3);
  for(int k=0;k<3;k++)
  {
  ABdotAC += AB[k]*AC[k];
  }
  //ABdotAC=fabs(ABdotAC);
  for(int k=0;k<3;k++)
  {
    lenAB += square(AB[k]);
    lenAC += square(AC[k]);
  }
  lenAB=sqrt(lenAB);
  lenAC=sqrt(lenAC);
  if(ABdotAC/(lenAB*lenAC) > 1.0 || ABdotAC/(lenAB*lenAC) < -1.0)
  {
    angleret = 3.14159;
  }
  else
  {
  angleret= acos(ABdotAC/(lenAB*lenAC)) ;//* 180/pi;
  }
  return (angleret);
}