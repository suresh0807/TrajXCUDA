//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"

void compute_Vdist(void)
{
cout << type <<" is chosen"<<endl;
cout<<"Vdist: " << nElt1 <<" "<< Elt1 << " atoms are there in each structure"<<endl;

int Dirn;
float swp;

if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
  Dirn = 2;
}

else if(set_max_y == "yes")
{
  minz = set_miny;
  maxz = set_maxy;
  Dirn=1;
  swp=zvec;
  zvec=yvec;
  yvec=swp;
}
else if(set_max_x == "yes")
{
  minz = set_minx;
  maxz = set_maxx;
  Dirn=0;
  swp=zvec;
  zvec=xvec;
  xvec=swp;
}
printf("%f %f\n",minz,maxz);

    zrange=maxz-minz;

    zint = zrange/zsplit;

    ztick=(float*) malloc (sizeof(float)*zsplit*2);
    
    for(int i=0;i<zsplit;i++)
    {
      ztick[i*2]=minz+(i*zint);
      ztick[i*2+1]=minz+((i+1)*zint);
    }
    
float *Vxdensity, *Vydensity, *Vzdensity;
float *dev_Vxdensity, *dev_Vydensity, *dev_Vzdensity;
    
FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<zsplit;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
fclose(xyztick);

    Vdensity=(float *) malloc (sizeof(float)*zsplit);
    Vxdensity=(float *) malloc (sizeof(float)*zsplit);
    Vydensity=(float *) malloc (sizeof(float)*zsplit);
    Vzdensity=(float *) malloc (sizeof(float)*zsplit);

        for(int k=0;k<zsplit;k++)
        {
          Vdensity[k]=0.0;
	  Vxdensity[k]=0.0;
	  Vydensity[k]=0.0;
	  Vzdensity[k]=0.0;
        }
    density=(int *) malloc (sizeof(int)*zsplit);

        for(int k=0;k<zsplit;k++)
        {
          density[k]=0;
        }
///*


    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_VEL,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_Vdensity,sizeof(float)*zsplit);
    hipMalloc((void **)&dev_Vxdensity,sizeof(float)*zsplit);
    hipMalloc((void **)&dev_Vydensity,sizeof(float)*zsplit);
    hipMalloc((void **)&dev_Vzdensity,sizeof(float)*zsplit);
    hipMalloc((void **)&dev_ztick, sizeof(float)*zsplit*2);
    hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_VEL,VEL,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_Vdensity,Vdensity,sizeof(float)*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_Vxdensity,Vxdensity,sizeof(float)*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_Vydensity,Vydensity,sizeof(float)*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_Vzdensity,Vzdensity,sizeof(float)*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*zsplit*2,hipMemcpyHostToDevice);
    
    cout << "Memory for storing structure data: "<<((nstruct/skip)*nElt1*3)*4/float(1000000000)<< " Gbs" << endl;
    cout << "Memory for local storage: "<<((zsplit*4)+(zsplit*2))*4/float(1000000000)<< " Gbs" << endl;   
    
    //int dirn;
    
    //if(vel_dirn=="all"){dirn=0;}
    //else if(vel_dirn=="x"){dirn=1;}
    //else if(vel_dirn=="y"){dirn=2;}
    //else if(vel_dirn=="z"){dirn=3;}
    
    dim3 dimBlock(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGrid(((nstruct/skip)+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    Vdist_calc<<<dimGrid,dimBlock>>>(dev_A,dev_VEL,dev_Vdensity,dev_Vxdensity,dev_Vydensity,dev_Vzdensity,(nstruct/skip),nElt1,zsplit,dev_ztick,Dirn);
    hipMemcpy(Vdensity,dev_Vdensity,sizeof(float)*zsplit,hipMemcpyDeviceToHost);
    hipMemcpy(Vxdensity,dev_Vxdensity,sizeof(float)*zsplit,hipMemcpyDeviceToHost);
    hipMemcpy(Vydensity,dev_Vydensity,sizeof(float)*zsplit,hipMemcpyDeviceToHost);
    hipMemcpy(Vzdensity,dev_Vzdensity,sizeof(float)*zsplit,hipMemcpyDeviceToHost);
    hipFree(dev_VEL);
    
    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_density,sizeof(int)*zsplit);
    hipMemcpy(dev_density,density,sizeof(int)*zsplit,hipMemcpyHostToDevice);
    
    cout << "Memory for storing structure data: "<<((nstruct/skip)*nElt1*3)*4/float(1000000000)<< " Gbs" << endl;
    cout << "Memory for local storage: "<<(zsplit+(zsplit*2))*4/float(1000000000)<< " Gbs" << endl;   
    
    dim3 dimBlocks(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGrids(((nstruct/skip)+dimBlocks.x-1)/dimBlocks.x,(nElt1+dimBlocks.y-1)/dimBlocks.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    Xdist_calc<<<dimGrids,dimBlocks>>>(dev_A,dev_density,(nstruct/skip),nElt1,zsplit,dev_ztick,Dirn);
    hipMemcpy(density,dev_density,sizeof(int)*zsplit,hipMemcpyDeviceToHost);
    
    
    
FILE *Vdistfile=fopen("vdist.data","wt");
//FILE *Vdistnormfile=fopen("vdistnorm.data","wt");
//FILE *Zdistfile=fopen("zdist.data","wt");
//FILE *Zdist1file=fopen("zdist1.data","wt");
int zstart = 0;
int zend = zsplit;
//int startfromzero=0;  not used because relative distances get messed up
for(int m=zstart;m<zend;m++)
{
  //if(startfromzero == 0 && float(density[m])/(xvec*yvec*zint*(nstruct/skip)) == 0) {}
  //else {fprintf(Zdistfile,"%f %f \n ",zint*startfromzero, float(density[m])/(xvec*yvec*zint*(nstruct/skip)));startfromzero++;}
  //fprintf(Zdistfile,"%f %f \n ",ztick[m*2+1], float(density[m])/(xvec*yvec*zint*(nstruct/skip)));
  //fprintf(Zdist1file,"%f %f \n ",ztick[m*2+1], float(density[m]));
  fprintf(Vdistfile,"%f %f %f %f %f \n ",ztick[m*2+1], Vdensity[m]/float(density[m]),Vxdensity[m]/float(density[m]),Vydensity[m]/float(density[m]),Vzdensity[m]/float(density[m]));
  //fprintf(Vdistnormfile,"%f %f \n ",ztick[m*2+1], float(Vdensity[m])/float(density[m]));
//  fprintf(Zdistfile,"%f %f \n ",ztick[m*2+1], float(density[m])/((nstruct/skip)));
}
//fclose(Zdistfile);
//fclose(Zdist1file);
fclose(Vdistfile);
//fclose(Vdistnormfile);

hipFree(dev_Vdensity);
hipFree(dev_Vxdensity);
hipFree(dev_Vydensity);
hipFree(dev_Vzdensity);
hipFree(dev_density);
hipFree(dev_A);
hipFree(dev_ztick);

}
