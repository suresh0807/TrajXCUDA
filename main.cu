//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


#include "cudatools.cuh"

float Coulomb_cutoff;
float mintop,maxtop;
char c;
string Dummy;
string type;
string order;
int xsplit,ysplit,zsplit;
int skip,skips;
int color,symbol,switchint;
float size, bondist, Mbondist, minbondist,maxbondist;
//char Elt1[2],Elt2[2];
string Elt1,Elt2,dipole_orient,atop_orient;
string set_max_x,set_max_y,set_max_z;
float minx,maxx,minz,maxz,miny,maxy,set_minz,set_maxz,set_minx,set_maxx,set_miny,set_maxy;
int natoms,nstruct;
int *exch, *dev_exch;
float *A, *Aint, *B, *VEL, *METAL ; //Host
float *A_int, *B_int, *A_bulk, *B_bulk;
float *A_all, *B_all;
float *dev_A,*dev_B; 
float *Aavg;
int dircover;
float xvec,yvec,zvec,yxvec;
float *lattice, *dev_lattice;
//char felt[2];
string felt,choose_dirn;
int nElt1,nElt2;
string average;
string unwrap,choose_atoms;
string command,unwrapout;
int SIZER;
int maxneigh, minneigh;
float timestep, diffuse_time, total_time,bondistOH,bondistOM;
int num_bins, SD_store, origins;
float *SD, *SDsum;
float *dev_SD , *dev_SDsum; 
float *SD1, *SDsum1;
float *dev_SD1 , *dev_SDsum1;
float *SDavgf;
float *dev_SDavgf;
float *SDavg;
float *dev_SDavg;
float *SDavg1;
float *dev_SDavg1;
float MSDint;
int max_frames, *dev_MSDchaos, check_index;
string keyword,filter_density,HBlifetime,HB_criterium_set,density_plot;
string vacuum, transient_HB_definition,HB_lifestyle;
string veloc,diffusion_kernel,rdf_between,HB_crossing,strict_dirn,choose_atoms_from;
int count_metal;
string rdf_metal_exclude,diffuse_direction,HB_histograms,write_cube;
float metal_atom_volume, Hbond_angle_dev,bondist_int_bulk,bondist_int_bulk2;
float rdf_max_rad, max_O_O;
string cell_type, metal_species,scope,plot,msd_for,HB_for,diffco_for,rdf_for, vdos_for, include_transient_bonds,lifetime_for;
int nA_int, nB_int, nA_bulk, nB_bulk, *Aintnum, *Bintnum, *Abulknum, *Bbulknum;
int xsrt, xend, xski,start_frame;
float *velocity, *dev_velocity,*Vdensity, *dev_Vdensity;
float *dev_VEL;
int transtime;
string file_path;
string ext_charge;
float  *MSDtics;
int *MSDchaos, MSDsteps;
int *MSDcounter, *dev_MSDcounter;



int main(void)
{
  

  
 cout <<"#2015"<<endl;
 cout <<"################# TrajXCUDA #########################"<<endl;
 cout <<"########## Suresh Kondati Natarajan #################"<<endl;
 cout <<"##### Lehrstuhl fuer theoretische chemie ############"<<endl;
 cout <<"########## Ruhr Universitaet Bochum #################"<<endl<<endl;  
 cout <<"You can do one of the following operations:"<<endl<<endl;
 cout <<"1. Density profile - side view animation"<<endl;
 cout <<"2. Density profile - top view"<<endl;
 cout <<"3. Radial distribution function"<<endl;
 cout <<"4. Velocity profile - side view animation"<<endl;
 cout <<"5. Water fragments counter"<<endl;
 cout <<"6. Percent coverage of species at surface (need to be tested - orthorhombic only)"<<endl;
 cout <<"7. Diffusion coefficient"<<endl;
 cout <<"8. Vibrational density of states"<<endl;
 cout <<"9. Z-distribution"<<endl;
 cout <<"10. Hbonds"<<endl;
 cout <<"11. exchange"<<endl;
 cout <<"12. V-distribution"<<endl<<endl; 
 
 //char lElt1[2],lElt2[2];
 //string lElt1,lElt2;
 skip=1;
 SIZER=100;
 scope="bulk";
 ifstream elts;
 elts.open("input.dat",ios::in);
 while (!elts.eof())
 { 
 elts >> keyword;
 if (keyword.at(0) == '#') {elts.ignore(256,'\n');}
 else
 {
 if(keyword == "compute"){elts >> type;
  if(type=="density") {switchint=1;}
  else if(type=="density-top") {switchint=2;}
  else if(type=="rdf") {switchint=3;}
  else if(type=="densityvelocity"|| type =="densityvelocity-top") {switchint=4;} 
  else if(type=="O-fragments") {switchint=5;}
  else if(type=="percentage-coverage"){switchint=6;}
  else if(type=="diffusion-coefficient"){switchint=7;}
  else if(type=="VDOS"){switchint=8;}
  else if(type=="Z-dist"){switchint=9;}
  else if(type=="Hbonds"){switchint=10;}
  else if(type=="exchange"){switchint=11;}
  else if(type=="V-dist"){switchint=12;}
  else if(type=="orient"){switchint=13;}
  else if(type=="residenttime"){switchint=14;}
  else if(type=="Adatoms"){switchint=15;}
  else if(type=="tetrawater"){switchint=16;}
  else if(type=="coulomb"){switchint=17;}
  else {cout << type <<" is Not a valid value for "<<keyword << endl; exit(0);}}
 else if(keyword == "unwrap_traj"){elts >> unwrap;}
 else if(keyword == "element_1"){elts >> Elt1 ;}
 else if(keyword == "element_2"){elts >> Elt2 ;}
 else if(keyword == "surface_element"){elts >> Elt1;}
 else if(keyword == "adsorbate_element"){elts >> Elt2;}
 else if(keyword == "max_bond_distance"){elts >> bondist;}
 else if(keyword == "make_grid"){elts >> xsplit >> ysplit >> zsplit;}
 else if(keyword == "atoms_between"){elts >> mintop >> maxtop;}
 else if(keyword == "rdf_between"){elts >> rdf_between >> mintop >> maxtop;}
 else if(keyword == "vacuum_direction")
 {
 elts >> vacuum;
 if (vacuum.at(0) == 'z') { dircover=2; if(type == "density" || type =="densityvelocity") {order = "XYZ";} else if (type == "density-top"|| type =="densityvelocity-top") {order = "YZX";} }
 else if (vacuum.at(0) == 'y') { dircover=1; if(type == "density" || type =="densityvelocity") {order = "ZXY";} else if (type == "density-top"|| type =="densityvelocity-top") {order = "XYZ";} }
 else if (vacuum.at(0) == 'x') { dircover=0; if(type == "density" || type =="densityvelocity") {order = "YZX";} else if (type == "density-top"|| type =="densityvelocity-top") {order = "ZXY";} }
 else { cout << " enter x,y or z for 'vacuum_direction' keyword"<< endl;}
 }
 else if(keyword == "rdf_bins"){elts >> bin;}
 else if(keyword == "write_cube"){elts >> write_cube;}
 else if(keyword == "rdf_max_rad"){elts >> rdf_max_rad;} 
 else if(keyword == "equilibrium_metal_bond_distance"){elts >> Mbondist;}
 else if(keyword == "metal_coordination_number"){elts >> maxneigh >>minneigh;}
 else if(keyword == "skip_frames"){elts >> skip;}
 else if(keyword == "msd_skip_restarts" || keyword == "HB_skip_restarts" || keyword == "vdos_skip_restarts"){elts >> skips;}
 else if(keyword == "msd_timestep_fs" || keyword == "HB_timestep_fs"|| keyword == "vdos_timestep_fs"){elts >> timestep;}
 else if(keyword == "msd_lookback_ps" || keyword == "HB_lookback_ps" || keyword == "vdos_lookback_ps"){elts >> diffuse_time;}
 else if(keyword == "average_positions"){elts >> average;}
 else if(keyword == "xfarbe_symbol"){elts >> symbol;}
 else if(keyword == "xfarbe_size"){elts >> size;}
 else if(keyword == "xfarbe_color"){elts >> color;}
 else if(keyword == "read_velocities"){elts >> veloc;}
 else if(keyword == "HBlifetime"){elts >> HBlifetime;}
 else if(keyword == "scope"){elts >> scope;}
 else if(keyword == "choose_atoms"){elts >>Elt1 >>choose_dirn>> strict_dirn>>minbondist >> maxbondist>>metal_species; choose_atoms="yes";}
 else if(keyword == "metal_species"){elts >> metal_species;}
 else if(keyword == "rdf_metal_exclude"){elts >> rdf_metal_exclude >> metal_atom_volume;}
 else if(keyword == "set_maxz"){elts >> set_maxz;}
 else if(keyword == "set_minz"){elts >> set_minz;}
 else if(keyword == "set_maxx"){elts >> set_maxx;}
 else if(keyword == "set_minx"){elts >> set_minx;}
 else if(keyword == "set_maxy"){elts >> set_maxy;}
 else if(keyword == "set_miny"){elts >> set_miny;}
 else if(keyword == "check_index"){elts >> check_index;}
 else if(keyword == "choose_atoms_from"){elts >>Elt1 >>choose_dirn>> strict_dirn>>minbondist >> maxbondist>>check_index; choose_atoms_from="yes";}
 else if(keyword == "set_max_x"){elts >> set_max_x;}
 else if(keyword == "set_max_y"){elts >> set_max_y;}
 else if(keyword == "set_max_z"){elts >> set_max_z;}
 else if(keyword == "max_frames"){elts >> max_frames;}
 else if(keyword == "start_frame"){elts >> start_frame;}
 else if(keyword == "file_path"){elts >> file_path;}
 else if(keyword == "msd_for"){elts >> msd_for;}
 else if(keyword == "transient_HB_definition"){elts >> transient_HB_definition;}
 else if(keyword == "HB_for"){elts >> HB_for; }
 else if(keyword == "lifetime_for"){elts >> lifetime_for;}
 else if(keyword == "unwrapout"){elts >> unwrapout;}
 else if(keyword == "filter_density"){elts >> filter_density;}
 else if(keyword == "max_O_O"){elts >> max_O_O;}
 else if(keyword == "diffusion_kernel"){elts >> diffusion_kernel; if(diffusion_kernel=="z-binning") {elts >> MSDsteps;}}
 else if(keyword == "bondist_int_bulk"){elts >> bondist_int_bulk >> bondist_int_bulk2;}
 else if(keyword == "Hbond_angle_dev"){elts >> Hbond_angle_dev;}
 else if(keyword == "include_transient_bonds"){elts >> include_transient_bonds;}
 else if(keyword == "rdf_for"){elts >> rdf_for;}
 else if(keyword == "diffuse_direction") {elts >> diffuse_direction;}
 else if(keyword == "vdos_for"){elts >> vdos_for;}
 else if(keyword == "atop_orient"){elts >> atop_orient>>bondistOM;}
 else if(keyword == "dipole_orient"){elts >> dipole_orient>>bondistOH;}
 else if(keyword == "HB_crossing"){elts >> HB_crossing;}
 else if(keyword == "plot"){elts >> plot;}
 else if(keyword == "HB_lifestyle"){elts >> HB_lifestyle;  if(HB_lifestyle=="transient"){elts >> transtime;}}
 else if(keyword == "density_plot"){elts >> density_plot;}
 else if(keyword == "cell_type"){elts >> cell_type;}
 else if(keyword == "HB_histograms"){elts >> HB_histograms;}
 else if(keyword == "HB_criterium_set"){elts >> HB_criterium_set;}
 else if(keyword == "coulomb_cutoff"){elts >> Coulomb_cutoff;}
 else if(keyword == "use_external_charges"){elts >> ext_charge;}
 else {cout << "Keyword "<<keyword<<" not supported"<<endl;}
 }
 }
 elts.close();
 //elts.getline(type,SIZER);

//Reads the input file input.dat and based on the string in first line 
//decides what to do!
/* if(type=="density") {switchint=1;}
 else if(type=="density-top") {switchint=2;}
 else if(type=="rdf") {switchint=3;}
 else if(type=="densityvelocity") {switchint=4;} 
 else if(type=="O-fragments") {switchint=5;}
 else if(type=="percentage-coverage"){switchint=6;}
 else if(type=="diffusion-coefficient"){switchint=7;}
 else if(type=="VDOS"){switchint=8;}
 else {cout << "Not a valid keyword" << endl; exit(0);}
*/ 
 switch (switchint)
 {   case 1:     cout <<"Density profile- side view animation chosen:" <<endl <<endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >> zsplit;
		 //elts >> order;
		 //elts >> skip;
		 //elts >> average;
                 //elts.close();
		 read_inputs();
                 cout << nElt1 << " "<< Elt1 <<" atoms are there in each structure"<< endl;
                 if(average=="yes")
                 {
		   density_side_avg_pos();
                 }
                 else
                 {
                  compute_densitygrid();
                 }
		 break;
		 
     case 2:	 cout <<"Density profile- top view chosen:"<<endl<<endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >> zsplit;
		 //elts >> mintop >> maxtop;
		 //elts >> order;
		 //elts >> skip;
		 //elts >> average;    
                 //if(average=="yes")
                 //{
                 //  elts >> symbol;
                 //  elts >> size;
		 //  elts >> color;     
                 //}
                 if(dipole_orient == "yes")
		 {
		   Elt2="H";
		 }
                 cout << "Atom chosen: " << Elt1 <<endl;
                 //elts.close();
		 read_inputs();
		 cout <<nElt1 <<" "<<Elt1 << " atoms are there in each structure"<<endl;
                 if(average=="yes")
                 {
                  density_top_avg_pos();
                 }
                 else
                 {
                  compute_densitygrid();               
                 }
                 
                 free(Aavg);
		 break;
		 
     case 3:     cout<<"Radial distribution function chosen:"<<endl<<endl;
                 //elts >> Elt1 >> Elt2;
		 //elts >> bin;
		 //elts >> skip;
		 //elts.close();
		 read_inputs();
		 cout <<nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 cout <<nElt2<<" "<<Elt2<<" atoms are there in each structure"<<endl;
                 compute_rdf_frames();
		 //compute_rdf();
		 break;
		 
     case 4:	 cout<<"Velocity profile- side view animation chosen:"<<endl<<endl;
                 //elts >> Elt1;
                 //elts >> xsplit >> ysplit>> zsplit;
		 //elts >> order;
		 //elts >> skip;
		 //elts >> average;
                 //cout << "Atom chosen: "<<Elt1<<endl;
                 //elts.close(); 
		 read_inputs();
                 cout << nElt1 <<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 compute_densityvelocitygrid();
		 break;
		 
     case 5:     cout<<"Water fragments counter chosen:"<<endl<<endl;
                 Elt1="O";
                 Elt2="H";
                 //elts>>bondist;
                 //elts.close();
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 cout << nElt2<<" "<<Elt2<<" atoms are there in each structure"<<endl;
                 compute_Ofrag();
		 break;
		 
     case 6:     cout <<"Percentage coverage of "<<Elt2<<" on the chosen surface:"<<endl;;
                 //elts >> Elt1;
                 //elts >> Elt2;
		 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 cout << Elt2 << " atoms are chosen for analysis" << endl;
		 //elts >> Mbondist;
		 //cout << Mbondist<<endl;
		 //elts >> bondist;
		 //elts >> maxneigh;		 
                 //elts.close();
		 read_inputs();
                 cout<<nElt1 << " "<<Elt1<< " atoms are there in each structure" << endl;
                 cout<<nElt2 << " "<<Elt2<< " atoms are there in each structure" << endl;
                 compute_coverage();
		 break;
		 
     case 7:     cout << "You are computing diffusion coefficient:" << endl;
                 //elts >> Elt1;
		 //elts >> Elt2;
		 Elt2 = Elt1;
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 //elts >> timestep; //in fs
		 //elts >> diffuse_time; //in ps
		 //elts >> skips;
		 /*if(unwrap == "yes")
		 {
		 cout << "Unwrapping the trajectory"<<endl;  
		 if(cell_type =="orthorhombic"){system("unwrapper.x ortho");}
		 else if(cell_type =="monoclinic"){system("unwrapper.x mono");}
		 }*/
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 if(diffusion_kernel == "long_time") compute_diffusion1();
                 else if (diffusion_kernel == "binning") compute_diffusion_lifetime();
		 else if (diffusion_kernel == "z-binning") compute_diffusion_z();
		 //else if (diffusion_kernel == "binning") compute_diffusion();
                 cout << "MSD data given in (angstrom)^2 in msd.data"<<endl;
		 cout << "Diffusion coefficient data in (angstrom)^2/fs in diffco.data"<<endl;
		 cout << "However, the best way to find diffusion coefficient is"<<endl
		      << "to plot msd.data in gnuplot and fit a linear plot and "<<endl
		      << "take its slope"<<endl;
		 break;
		 
     case 8:     cout << "You are computing VDOS:" << endl;
                 //elts >> Elt1;
		 //elts >> Elt2;
		 Elt2 = Elt1;
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 //elts >> timestep; //in fs
		 //elts >> diffuse_time; //in ps
		 //elts >> skips;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 compute_VDOS();
		 break;
		 
     case 9:     cout << "You are computing Z-distribution:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 compute_Zdist();
		 break;
		 
     case 10:    cout << "You are computing Hbonds:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
		 cout << nElt2<<" "<<Elt2<<" atoms are there in each structure"<<endl;
		 if(HB_crossing == "yes" && HB_lifestyle!="SSP"&& HB_lifestyle!="switchbond") compute_Hbonds2();
		 else if(HB_crossing == "yes" && HB_lifestyle=="SSP") compute_HbondsSSP();
		 else if(HB_crossing == "yes" && HB_lifestyle=="switchbond") {cout<<"switchbond type chosen"<<endl;compute_HbondsSSP();}
		 else if (HB_crossing == "no") compute_Hbonds1();
		 break;
		 
      case 11:     cout <<"Exchange of "<<Elt1<<" atoms"<<endl;;
                 //elts >> Elt1;
                 //elts >> Elt2;
		 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 //cout << Elt2 << " atoms are chosen for analysis" << endl;
		 //elts >> Mbondist;
		 //cout << Mbondist<<endl;
		 //elts >> bondist;
		 //elts >> maxneigh;		 
                 //elts.close();
		 read_inputs();
                 cout<<nElt1 << " "<<Elt1<< " atoms are there in each structure" << endl;
                 //cout<<nElt2 << " "<<Elt2<< " atoms are there in each structure" << endl;
                 compute_exchange();
		 break;	 
		 
      case 12:     cout << "You are computing Velocity-distribution:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 compute_Vdist();
		 break;
		 
      case 13:     cout << "You are computing water orientation:" << endl;
                 Elt1="O";
                 Elt2="H";
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
                 compute_angledist();
		 break;
		 
      case 14:    cout << "You are computing Resident times:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
		 compute_resident_time();
		 break;	 
		 
      case 15:    cout << "You are computing Adatoms:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
                 cout << Elt1 << " atoms are chosen for analysis" << endl;
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
		 compute_adatoms();
		 break;	
       
       case 16:  cout << "You are computing Tetrahedral parameter of water:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
		 compute_tetrawater();
		 break;
		 
       case 17:  cout << "You are computing Coulomb charge:" << endl;
                 //elts >> Elt1;
		 //elts >> xsplit >> ysplit >>zsplit;
		 //elts >> order;
		 //elts >> skip;
                 //elts >> set_max >> set_maxz >> set_minz;
                 //elts.close();
		 read_inputs();
                 cout << nElt1<<" "<<Elt1<<" atoms are there in each structure"<<endl;
		 compute_coulomb_charge();
		 break;
}
 

hipDeviceReset();
return 0;
}
