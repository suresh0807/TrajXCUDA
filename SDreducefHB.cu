#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void SDreducefHB(float *a, float *b, int initHBnum, int SD_store, int origins, int skips)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
 // int widid=threadIdx.z + blockIdx.z * blockDim.z;
  if(rowid < initHBnum && colid < SD_store)
  {
    for(int i = 0; i < origins/skips; i++)
    {
      //printf("%f \n", a[rowid+(colid*nElt1)+(i*nElt1*SD_store)]); 
      b[rowid+colid*initHBnum] += a[rowid+(colid*initHBnum)+(i*initHBnum*SD_store)];
    }
    b[rowid+colid*initHBnum] = b[rowid+colid*initHBnum]/(float(origins)/float(skips));
  }
}

