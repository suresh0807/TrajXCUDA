#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void reducef(float *a, float *b, int bin, int nstruct, int nElt1)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  if(rowid < bin && colid < nstruct)
  {
    for(int i=0;i<nElt1;i++)
    {
      b[rowid+colid*bin]+=a[rowid+(i*bin)+(colid*nElt1*bin)];
    }
  }
}

