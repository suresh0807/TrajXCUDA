#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


#include "cudatools.cuh"

void density_top_avg_pos()
{

cout <<" Copper average positions - computing"<<endl;

  if(cell_type=="orthorhombic")
  {
     for(int i=0; i< nstruct/skip;i++)
     {
      for(int j=0; j< nElt1; j++)
      {
       if(i>0)
       {
        if(fabs(A[j*3+i*nElt1*3]-A[j*3]) > xvec/2)
        {
	 if(A[j*3+i*nElt1*3]-A[j*3] > 0){ A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]-xvec;}
	 else if (A[j*3+i*nElt1*3]-A[j*3] < 0){ A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]+xvec;}
        }
        if(fabs(A[j*3+i*nElt1*3+1]-A[j*3+1]) > yvec/2)
        {
         if(A[j*3+i*nElt1*3+1]-A[j*3+1] > 0){A[j*3+i*nElt1*3+1]=A[j*3+i*nElt1*3+1]-yvec;}
         else if(A[j*3+i*nElt1*3+1]-A[j*3+1] < 0){A[j*3+i*nElt1*3+1]=A[j*3+i*nElt1*3+1]+yvec;}
        }
        if(fabs(A[j*3+i*nElt1*3+2]-A[j*3+2]) > zvec/2)
        {
         if(A[j*3+i*nElt1*3+2]-A[j*3+2] > 0){ A[j*3+i*nElt1*3+2]=A[j*3+i*nElt1*3+2]-zvec;}
         else if(A[j*3+i*nElt1*3+2]-A[j*3+2] < 0){ A[j*3+i*nElt1*3+2]=A[j*3+i*nElt1*3+2]+zvec;}
        }
       }
       Aavg[j*3]+=A[j*3+i*nElt1*3];
       Aavg[j*3+1]+=A[j*3+i*nElt1*3+1];
       Aavg[j*3+2]+=A[j*3+i*nElt1*3+2];
      }
     }
  }
  else if(cell_type=="monoclinic")
  {
    cout<<xvec<<" "<<yvec<<" "<<zvec<<" "<<yxvec<<endl;
     
    for(int i=0; i< nstruct/skip;i++)
     {
      for(int j=0; j< nElt1; j++)
      {
       if(i>0)
       {
	 if(fabs(A[j*3+i*nElt1*3+1]-A[j*3+1]) > yvec/2)
        {
         if(A[j*3+i*nElt1*3+1]-A[j*3+1] > 0){A[j*3+i*nElt1*3+1]=A[j*3+i*nElt1*3+1]-yvec;A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]-yxvec;}
         else if(A[j*3+i*nElt1*3+1]-A[j*3+1] < 0){A[j*3+i*nElt1*3+1]=A[j*3+i*nElt1*3+1]+yvec;A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]+yxvec;}
        }
        if(fabs(A[j*3+i*nElt1*3]-A[j*3]) > xvec/2)
        {
	 if(A[j*3+i*nElt1*3]-A[j*3] > 0){ A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]-xvec;}
	 else if (A[j*3+i*nElt1*3]-A[j*3] < 0){ A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]+xvec;}
        }
        if(fabs(A[j*3+i*nElt1*3+2]-A[j*3+2]) > zvec/2)
        {
         if(A[j*3+i*nElt1*3+2]-A[j*3+2] > 0){ A[j*3+i*nElt1*3+2]=A[j*3+i*nElt1*3+2]-zvec;}
         else if(A[j*3+i*nElt1*3+2]-A[j*3+2] < 0){ A[j*3+i*nElt1*3+2]=A[j*3+i*nElt1*3+2]+zvec;}
        }
        
        
        if(fabs(A[j*3+i*nElt1*3+1]-A[j*3+1]) > yvec/6)
        {
         if(A[j*3+i*nElt1*3+1]-A[j*3+1] > 0){A[j*3+i*nElt1*3+1]=A[j*3+i*nElt1*3+1]-(yvec/3);A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]-(yxvec/3);}
         else if(A[j*3+i*nElt1*3+1]-A[j*3+1] < 0){A[j*3+i*nElt1*3+1]=A[j*3+i*nElt1*3+1]+(yvec/3);A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]+(yxvec/3);}
        }
        if(fabs(A[j*3+i*nElt1*3]-A[j*3]) > xvec/6)
        {
	 if(A[j*3+i*nElt1*3]-A[j*3] > 0){ A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]-(xvec/3);}
	 else if (A[j*3+i*nElt1*3]-A[j*3] < 0){ A[j*3+i*nElt1*3]=A[j*3+i*nElt1*3]+(xvec/3);}
        }
        if(fabs(A[j*3+i*nElt1*3+2]-A[j*3+2]) > zvec/6)
        {
         if(A[j*3+i*nElt1*3+2]-A[j*3+2] > 0){ A[j*3+i*nElt1*3+2]=A[j*3+i*nElt1*3+2]-(zvec/3);}
         else if(A[j*3+i*nElt1*3+2]-A[j*3+2] < 0){ A[j*3+i*nElt1*3+2]=A[j*3+i*nElt1*3+2]+(zvec/3);}
        }
       }
       Aavg[j*3]+=A[j*3+i*nElt1*3];
       Aavg[j*3+1]+=A[j*3+i*nElt1*3+1];
       Aavg[j*3+2]+=A[j*3+i*nElt1*3+2];
      }
     }
     
    
  }  
      for(int j=0; j< nElt1; j++)
      {     
       Aavg[j*3]=Aavg[j*3]/(nstruct/skip);
       Aavg[j*3+1]=Aavg[j*3+1]/(nstruct/skip);
       Aavg[j*3+2]=Aavg[j*3+2]/(nstruct/skip); 
     //  printf("%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
      }
/*
      for(int j=0; j< nElt1; j++)
      {     
       Aavg[j*3]=A[j*3];
       Aavg[j*3+1]=A[j*3+1];
       Aavg[j*3+2]=A[j*3+2]; 
     //  printf("%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
      }
      */
   /*    for(int j=0; j< nElt1; j++)
      {     
	for(int k=0; k<3;k++)
	{
          if(Aavg[j*3+k] <0) {Aavg[j*3+k]=1-fabs(Aavg[j*3+k]);}
	}
      }*/
cout <<"Writing the average positions"<<endl;  
FILE *test=fopen("avg-pos.dat","wt");
    FILE *testout=fopen("teter.dat","wt");
      /*for(int j=0; j< nElt1; j++)
      {
	if(order=="YZX") {
	if(Aavg[j*3+2] < maxtop && Aavg[j*3+2] > mintop)
	{cout <<"Alles gut "<< order<<endl;
	cout<<nElt1<<endl;
	cout<<zsplit<<endl;
       printf("%d %f %d %f %f \n",symbol,size,color,Aavg[j*3+1]/yvec,Aavg[j*3]/xvec);
	fprintf(testout,"%s \n",nElt1);
	fprintf(testout,"%d \n", zsplit);
       fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,Aavg[j*3+1]/yvec,Aavg[j*3]/xvec);  
	}cout <<"Alles gut "<< order<<endl;
	}
	else if(order=="XYZ") {
	if(Aavg[j*3+1] < maxtop && Aavg[j*3+1] > mintop)
	{
	fprintf(testout,"%s \n",nElt1);
	fprintf(testout,"%d \n", ysplit);  
       fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,Aavg[j*3]/xvec,Aavg[j*3+2]/zvec);
 //printf("%d %f %d %f %f %f\n",symbol,size,color,Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);      
	}
	}
	else if(order=="ZXY") {
	if(Aavg[j*3] < maxtop && Aavg[j*3] > mintop)
	{
  	fprintf(testout,"%s \n",nElt1);
	fprintf(testout,"%d \n", xsplit);
       fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,Aavg[j*3+2]/zvec,Aavg[j*3+1]/yvec);
      // printf("%d %f %d %f %f %f\n",symbol,size,color,Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]); 
	}
	}
      }*/
      	//fprintf(testout,"%d \n",nElt1);
        //fprintf(testout,"%d \n", ysplit);
        //for(int j=0; j< nElt1; j++)
     // {
	//fprintf(test,"%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
      //}
       // fclose(test);
        
      if(order=="YZX") { if(cell_type=="monoclinic") xvec=xvec+yxvec;
      for(int j=0; j< nElt1; j++)
      {
	if(Aavg[j*3+2] <= maxtop && Aavg[j*3+2] >= mintop)
	{  float app = Aavg[j*3+1]*(float(ysplit)/yvec);
	   float bpp = Aavg[j*3]*(float(xsplit)/xvec);
	   //if (app <0) {app = app+1;}
	   //if (bpp <0) {bpp = bpp+1;}
	   //if (app >1) {app = app-1;}
	   //if (bpp >1) {bpp = bpp-1;}
	 // fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,app,bpp);
	 fprintf(testout,"%f %f \n",(app),(bpp));
	  fprintf(test,"%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
	}
      }
      }
      else if(order=="XYZ") {
	for(int j=0; j< nElt1; j++)
      {
	if(Aavg[j*3+1] < maxtop && Aavg[j*3+1] > mintop)
	{  float app = Aavg[j*3]/xvec;
	   float bpp = Aavg[j*3+2]/zvec;
	   if (app <0) {app = app+1;}
	   if (bpp <0) {bpp = bpp+1;}
	   if (app >1) {app = app-1;}
	   if (bpp >1) {bpp = bpp-1;}
	  //fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,app,bpp);
	  fprintf(testout,"%f %f \n",(app*xsplit)+xsplit,(bpp*zsplit)+zsplit);
	  fprintf(test,"%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
	}
      }
      }
           else if(order=="ZXY") {
	for(int j=0; j< nElt1; j++)
      {
	if(Aavg[j*3] < maxtop && Aavg[j*3] > mintop)
	{  float app = Aavg[j*3+2]/zvec;
	   float bpp = Aavg[j*3+1]/yvec;
	   if (app <0) {app = app+1;}
	   if (bpp <0) {bpp = bpp+1;}
	   if (app >1) {app = app-1;}
	   if (bpp >1) {bpp = bpp-1;}
	  //fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,app,bpp);
	  fprintf(testout,"%f %f \n",(app*zsplit)+zsplit,(bpp*ysplit)+ysplit);
	  fprintf(test,"%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
	}
      }
      }
    fclose(testout);
    fclose(test);

    


    
    
        
cout <<"Alles gut"<<endl;
}