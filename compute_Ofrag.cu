//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


//both Elt1 and Elt2 are O atoms. The problem here is that O-O distances are only used to find H bonds,
//which can be very wrong. Check compute_hbonds instead.

#include "cudatools.cuh"
int *Ofrag;
int *Ofragsum;
int *dev_Ofrag;



void compute_Ofrag(void)
{
 Ofrag= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
// Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          Ofrag[i+l*nElt1]=0;
        }
      }
 Ofragsum= (int *) malloc (sizeof(int)*4*(nstruct/skip));   
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< 4; i++)
        {
          Ofragsum[i+l*4]=0;
        }
      }
      

//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*nstruct/skip*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*nstruct/skip*3);
    hipMalloc((void **)&dev_Ofrag,sizeof(int)*nElt1*nstruct);
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*nstruct/skip*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*nstruct/skip*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_Ofrag,Ofrag,sizeof(int)*nElt1*nstruct/skip,hipMemcpyHostToDevice);
//dim3 struct to define elements of the execution configuration


    dim3 dimBlock(10,10,10);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    Ofragmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_Ofrag,bondist,nElt1,nElt2,nstruct/skip,xvec,yvec,zvec);

    hipMemcpy(Ofrag,dev_Ofrag,sizeof(int)*nElt1*nstruct/skip,hipMemcpyDeviceToHost);
    
    hipFree(dev_A);
    hipFree(dev_B);
    
 for(int l=0; l<nstruct/skip; l++)
    {
       for(int i=0; i< nElt1; i++)
         {
          if(Ofrag[i+l*nElt1] ==2)
	  {
	    Ofragsum[l*4]+=1;
	  }
	  else if(Ofrag[i+l*nElt1] ==1)
	  {
	    Ofragsum[(l*4)+1]+=1;
	  }
	  else if(Ofrag[i+l*nElt1] ==0)
	  {
	    Ofragsum[(l*4)+2]+=1;
	  }
	  else 
	  {
	    Ofragsum[(l*4)+3]+=1;
	  }	   
	}
     }
    
    
    

//    FILE *Ofragplotchk=fopen("O-fragments-chk.data","wt");
// for(int l=0; l<nstruct; l++)
//    {
//      fprintf(Ofragplotchk,"%d\n",l);
//      for(int i=0; i< nElt1; i++)
//        {
//          fprintf(Ofragplotchk,"%d %d\n",i,Ofrag[i+l*nElt1]);
//        }
//      }
        
//    fclose(Ofragplotchk);
    free(Ofrag);
    
    FILE *Ofragplot=fopen("O-fragments.data","wt");
    fprintf(Ofragplot,"#frame h2o oh o h3o\n");
 for(int l=0; l<nstruct/skip; l++)
    {
          fprintf(Ofragplot,"%d %d %d %d %d %d %d\n",l,Ofragsum[l*4],Ofragsum[1+l*4],Ofragsum[2+l*4],Ofragsum[3+l*4],Ofragsum[l*4]+Ofragsum[1+l*4]+Ofragsum[2+l*4]+Ofragsum[3+l*4],(Ofragsum[l*4]+Ofragsum[1+l*4]+Ofragsum[2+l*4]+Ofragsum[3+l*4])*2);
       }
        //frame h2o oh o h3o
    fclose(Ofragplot);

  free(Ofragsum);
}
