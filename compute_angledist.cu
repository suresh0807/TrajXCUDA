#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void compute_angledist(void)
{

  
  //lines for exchange matrix-----------------------------------------------------------------------------------------
  //------------------------------------------------------------------------------------------------------------------
  if(choose_atoms=="yes")
{
cout << Elt1 <<" within "<< bondist <<" angstrom of "<< metal_species<<endl;
  
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));

//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
        }
      }
      
     hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 
}      
  
//exchange matrix created--------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------
  
  
  
  int *h2onum;
  int *h2oneigh;
  float *h2oxyz;
  
  float *h2odistmat, *dev_h2odistmat;
  
  int OHsamples=4;
  
    h2onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    h2oneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    h2oxyz = (float *) malloc (sizeof(float)*13*nElt1*(nstruct/skip));  //Oxyz,H1xyz,H2xyz,Hmidpoint,dipole alignment with surface normal
    h2odistmat = (float *) malloc (sizeof(float)*nElt1*nElt2);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      h2onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      h2oneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      }
      for(int k =0 ;k<9;k++)
      {
      h2oxyz[j*9+i*nElt1*9+k]=0.0;
      }
    }
  }   
      
      
cout <<"gpu begins OH distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_h2odistmat,sizeof(float)*nElt1*nElt2);
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
//dim3 struct to define elements of the execution configuration

    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,1);
    
    bondist=1.10; // water molecule only if OH distance is within this value
    //ofstream neigh;
    //neigh.open("H2O.dat");
    
for(int i = 0; i<(nstruct/skip); i++)
{
  
 // neigh << i<<endl<<endl;
  
   for(int i1=0;i1<nElt1;i1++)
  {
    for(int j1=0; j1<nElt2; j1++)
    {
      h2odistmat[j1+i1*nElt2]=0.0;
    }
  }
  
  hipMemcpy(dev_h2odistmat,h2odistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondist,nElt1,nElt2,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondist,nElt1,nElt2,dev_lattice,i);      
    }

hipMemcpy(h2odistmat,dev_h2odistmat,sizeof(float)*nElt1*nElt2,hipMemcpyDeviceToHost);
   
   
   
  for(int i1=0;i1<nElt1;i1++)
  {
    int k=0;
   // neigh << i1<<" ";
    for(int j1=0; j1<nElt2; j1++)
    {
     // neigh <<j1<<" "<<h2odistmat[j1+i1*nElt2]<<" ";
      if(h2odistmat[j1+i1*nElt2] !=0.0)
      {
	h2onum[i1+i*nElt1]++;
	h2oneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	k++;
      }
    }
    //neigh<<endl;
  }

}   
 
 //neigh.close();
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_h2odistmat);
    hipFree(dev_lattice);
    

cout <<"gpu ends"<<endl;
   
cout<<" done !!!"<<endl;    
    
    
    
//Hbondneigh contain the IDs of four possible oxygen neighbour atoms if the hydrogen chosen is participating in a hydrogen bond.
//Onum has the number of oxygen atoms within a specified distance (see input file).

 cout<<"Write Hxyz.dat"<<endl;

  ofstream watercount;
  watercount.open("Hxyz.dat");
  
  
  int *angdist;
  
  angdist = (int *) malloc (sizeof(int)*180);
  
  for(int i=0; i<=180;i++)
  {
    angdist[i]=0;
  }
  
  
for(int i=0; i<(nstruct/skip); i++)
{
 // cout<<i<<endl;
 for(int j=0; j<nElt1;j++)
 {
     
   if(h2onum[j+i*nElt1]==2 && exch[j+i*nElt1]==1)
   //if(h2onum[j+i*nElt1]==2)
   {
        watercount << i <<" "<<j<<" ";
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*9+i*nElt1*9+k]= A[j*3+i*nElt1*3+k];
     watercount << h2oxyz[j*9+i*nElt1*9+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*9+i*nElt1*9+3+k]= B[h2oneigh[j*OHsamples+i*nElt1*OHsamples]*3+i*nElt2*3+k];
     //watercount << h2oxyz[j*9+i*nElt1*9+3+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*9+i*nElt1*9+6+k]= B[h2oneigh[j*OHsamples+i*nElt1*OHsamples+1]*3+i*nElt2*3+k];
     //watercount << h2oxyz[j*9+i*nElt1*9+6+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*9+i*nElt1*9+9+k]=(h2oxyz[j*9+i*nElt1*9+3+k]+h2oxyz[j*9+i*nElt1*9+6+k])/2.0;
     //watercount << h2oxyz[j*9+i*nElt1*9+9+k] <<" ";
     }
     
     h2oxyz[j*9+i*nElt1*9+12]=angle(0,0,0,0,0,1,h2oxyz[j*9+i*nElt1*9],h2oxyz[j*9+i*nElt1*9+1],h2oxyz[j*9+i*nElt1*9+2],h2oxyz[j*9+i*nElt1*9+9],h2oxyz[j*9+i*nElt1*9+9+1],h2oxyz[j*9+i*nElt1*9+9+2]);
     watercount << h2oxyz[j*9+i*nElt1*9+12]*(180.0/3.14159);
     angdist[int(ceil(h2oxyz[j*9+i*nElt1*9+12]*(180.0/3.14159)))]++;
     watercount<<endl;
   }
   
  }
}
  watercount.close();
  

  ofstream angldist;
  angldist.open("Dipole-orient.dat");
  
   for(int i=0; i<=180;i++)
  {
    angldist << i<<" "<<angdist[i]<<endl;
  }
  angldist.close();
hipFree(dev_exch);
  
}
