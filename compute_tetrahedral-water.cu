#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


#include "cudatools.cuh"

void compute_tetrawater(void)
{


  float *Onum; //tetrahedral parameter for each O atom
  int *Hbondneigh; // Ids of the O neighbours 
  float *Hbondneighdist; // OO distance for all neighbours.
  int *Oneigh;// Ids of 4 O neighbours
  float *Hdistmat, *dev_Hdistmat;// Distance matrix to capture the O within x \AA of O atom.
  
  int OHsamples=20; /// maximum number of OH bonds considered within bonddist sphere
  
    Onum = (float *) malloc (sizeof(float)*nElt1*(nstruct/skip));
    Hbondneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    Oneigh = (int *) malloc (sizeof(int)*4*nElt1*(nstruct/skip));
    Hbondneighdist = (float *) malloc (sizeof(float)*OHsamples*nElt1*(nstruct/skip));
    Hdistmat = (float *) malloc (sizeof(float)*nElt1*nElt1);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      Onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k]=0.0;
      }
    }
  }   
      
 for(int i=0;i<nElt1;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      Hdistmat[j+i*nElt1]=0.0;
    }
  }
      
cout <<"gpu begins distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_Hdistmat,sizeof(float)*nElt1*nElt1);
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_Hdistmat,Hdistmat,sizeof(float)*nElt1*nElt1,hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);

   //dim3 struct to define elements of the execution configuration


    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
for(int i = 0; i<(nstruct/skip); i++) /// go into each frame
{
  
   for(int i1=0;i1<nElt1;i1++)///initialize Hdistmat for 1 frame
   {
    for(int j1=0; j1<nElt1; j1++)
    {
      Hdistmat[j1+i1*nElt1]=0.0;
    }
   }
  
    hipMemcpy(dev_Hdistmat,Hdistmat,sizeof(float)*nElt1*nElt1,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_Hdistmat,bondist,nElt1,nElt1,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_Hdistmat,bondist,nElt1,nElt1,dev_lattice,i);      
    }

hipMemcpy(Hdistmat,dev_Hdistmat,sizeof(float)*nElt1*nElt1,hipMemcpyDeviceToHost);
   
  for(int i1=0;i1<nElt1;i1++) //oxygen1
  {
    int k=0;
    for(int j1=0; j1<nElt1; j1++) //oxygen2 
    { 
      if(Hdistmat[j1+i1*nElt1] !=0.0) // check if an oxygen atom is within the 'bonddist' distance from another O atom
      {
	Onum[i1+i*nElt1]++;
	Hbondneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	Hbondneighdist[i1*OHsamples+i*nElt1*OHsamples+k] = Hdistmat[j1+i1*nElt1];
	k++;
      }
    }
  }

}   
    
  
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_Hdistmat);
    hipFree(dev_lattice);


//Hbondneigh contain the IDs of 20 possible oxygen neighbour atoms.
//Onum has the number of oxygen atoms within a specified distance (see input file).

cout <<"gpu ends"<<endl; 

      
//swapping the indices of nearest oxygens to get the two shortest OH bonds up front, 
//since it usually describes the H bond

cout<<"Swapping of OO bonds based on distance "<<endl;

float swapa,swapb;
    for(int i=0;i<(nstruct/skip);i++)
     {
      for(int j=0; j<nElt1; j++)
       {
	 //if(i==0){cout<<j<<" ";}
	for(int swf=0; swf<Onum[j+i*nElt1]-1;swf++)
	{
	for(int sws=swf+1; sws<Onum[j+i*nElt1];sws++)
	{
	if(Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf] > Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws] && sws !=swf && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws] !=0.0 && Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf] !=0.0) 
	{
	   swapa = Hbondneigh[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    Hbondneigh[j*OHsamples+i*nElt1*OHsamples+swf]=Hbondneigh[j*OHsamples+i*nElt1*OHsamples+sws];
	    Hbondneigh[j*OHsamples+i*nElt1*OHsamples+sws]=swapa;
	    swapb = Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf]; 
	    Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+swf]=Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws];
	    Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+sws]=swapb;
	}
	}
	}
	for(int k =0 ;k<20;k++)
        {
	//if(i==0){cout<<Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k]<<" "<<Hbondneighdist[j*OHsamples+i*nElt1*OHsamples+k]<<"  ";}
	}
	//if(i==0){cout<<endl;}
       }
       
     }	
   
cout<<" done !!!"<<endl;
   
   
  free(Hbondneighdist);
  free(Hdistmat);
  free(Onum);
  
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      for(int k =0 ;k<4;k++)
      {
      Oneigh[j*4+i*nElt1*4+k]=Hbondneigh[j*OHsamples+i*nElt1*OHsamples+k];
      }
    }
  }
  
  
  
  free(Hbondneigh);
  
  float angle_final;
  float *OneighXYZ, *angXYZ, *Otetra;
  OneighXYZ = (float *) malloc (sizeof(float)*5*3);
  Otetra = (float *) malloc (sizeof(float)*(nstruct/skip)*nElt1);
  angXYZ = (float *) malloc (sizeof(float)*6);
  for (int i=0;i<15;i++)
  {
    OneighXYZ[i]=0;
  }
  for (int i=0;i<6;i++)
  {
    angXYZ[i]=0;
  }
  
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      angle_final=0.0;
      Otetra[j+i*nElt1]=0;
      for(int ss=0; ss<3;ss++)
      {
	OneighXYZ[ss]=A[j*3+i*nElt1*3+ss];
      }
      for(int k =0 ;k<4;k++)
      {
	for(int ss=0; ss<3;ss++)
        {
	  OneighXYZ[3+k*3+ss]=A[Oneigh[j*4+i*nElt1*4+k]*3+i*nElt1*3+ss];
	}
      }
      angXYZ[0]=angle(OneighXYZ[0],OneighXYZ[1],OneighXYZ[2],OneighXYZ[3],OneighXYZ[4],OneighXYZ[5],\
	OneighXYZ[6],OneighXYZ[7],OneighXYZ[8]);
      angXYZ[1]=angle(OneighXYZ[0],OneighXYZ[1],OneighXYZ[2],OneighXYZ[3],OneighXYZ[4],OneighXYZ[5],\
	OneighXYZ[9],OneighXYZ[10],OneighXYZ[11]);
      angXYZ[2]=angle(OneighXYZ[0],OneighXYZ[1],OneighXYZ[2],OneighXYZ[3],OneighXYZ[4],OneighXYZ[5],\
	OneighXYZ[12],OneighXYZ[13],OneighXYZ[14]);
      angXYZ[3]=angle(OneighXYZ[0],OneighXYZ[1],OneighXYZ[2],OneighXYZ[6],OneighXYZ[7],OneighXYZ[8],\
	OneighXYZ[9],OneighXYZ[10],OneighXYZ[11]);
      angXYZ[4]=angle(OneighXYZ[0],OneighXYZ[1],OneighXYZ[2],OneighXYZ[6],OneighXYZ[7],OneighXYZ[8],\
	OneighXYZ[12],OneighXYZ[13],OneighXYZ[14]);
      angXYZ[5]=angle(OneighXYZ[0],OneighXYZ[1],OneighXYZ[2],OneighXYZ[9],OneighXYZ[10],OneighXYZ[11],\
	OneighXYZ[12],OneighXYZ[13],OneighXYZ[14]);
      for(int ss=0;ss<6;ss++)
      {
      angle_final= angle_final + square(cos(angXYZ[ss])+(1.0/3.0));
      }
      Otetra[j+i*nElt1]= 1.0- ( (3.0/8.0) * angle_final );
      //if(i==0 && j==0) {cout<<angXYZ[0]<<" "<<cos(angXYZ[0])<<" "<<angle_final<<" "<<Otetra[j+i*nElt1]<<endl;}
    }
  }
  
 /* 
  for(int i=0;i<1;i++)
  {
    for(int j=0;j<nElt1;j++)
    {
      cout<<j<<" "<<Oneigh[j*4+i*nElt1*4+0]<<" "<<Oneigh[j*4+i*nElt1*4+1]<<" "<<\
      Oneigh[j*4+i*nElt1*4+2]<<" "<<Oneigh[j*4+i*nElt1*4+3]<<" "<<Otetra[j+i*nElt1]<<endl;
    }
  }
  */
 
 
  free(OneighXYZ);
  free(Oneigh);
  free(angXYZ);
  

int Dirn,split;
float min,max;

if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
  Dirn = 2;
  min=minz;
  max=maxz;
  split=zsplit;
  
}

else if(set_max_y == "yes")
{
  minz = set_miny;
  maxz = set_maxy;
  Dirn=1;
  min=miny;
  max=maxy;
  split=ysplit;
}
else if(set_max_x == "yes")
{
  minz = set_minx;
  maxz = set_maxx;
  Dirn=0;
  min=minx;
  max=maxx;
  split=xsplit;
}



printf("%f %f\n",min,max);

    zrange=max-min;

    zint = zrange/split;

    ztick=(float*) malloc (sizeof(float)*split*2);
    
    for(int i=0;i<split;i++)
    {
      ztick[i*2]=min+(i*zint);
      ztick[i*2+1]=min+((i+1)*zint);
    }
    

FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<split;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
fclose(xyztick);

float *tetradensity;
    density=(int *) malloc (sizeof(int)*split);
tetradensity=(float *) malloc (sizeof(float)*split);
        for(int k=0;k<split;k++)
        {
          density[k]=0;
	  tetradensity[k]=0;
        }

///*

    float *dev_tetra, *dev_tetradensity;
    
    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_tetra,sizeof(float)*(nstruct/skip)*nElt1);
    hipMalloc((void **)&dev_density,sizeof(int)*split);
    hipMalloc((void **)&dev_tetradensity,sizeof(float)*split);
    hipMalloc((void **)&dev_ztick, sizeof(float)*split*2);
    hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_tetra,Otetra,sizeof(float)*(nstruct/skip)*nElt1,hipMemcpyHostToDevice);
    hipMemcpy(dev_density,density,sizeof(int)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_tetradensity,tetradensity,sizeof(float)*split,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*split*2,hipMemcpyHostToDevice);
    
    cout << "Memory for storing structure data: "<<((nstruct/skip)*nElt1*3)*4/float(1000000000)<< " Gbs" << endl;
    cout << "Memory for local storage: "<<(split+(split*2))*4/float(1000000000)<< " Gbs" << endl;   
    
    dim3 dimBlockr(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGridr(((nstruct/skip)+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    Xdist_calc<<<dimGridr,dimBlockr>>>(dev_A,dev_tetra,dev_tetradensity,dev_density,(nstruct/skip),nElt1,split,dev_ztick,Dirn);
    hipMemcpy(density,dev_density,sizeof(int)*split,hipMemcpyDeviceToHost);
    hipMemcpy(tetradensity,dev_tetradensity,sizeof(float)*split,hipMemcpyDeviceToHost);
    
    
    
FILE *Zdistfile=fopen("tetradist.data","wt");
int zstart = 0;
int zend = split;
//int startfromzero=0;  not used because relative distances get messed up
for(int m=zstart;m<zend;m++)
{
  //if(startfromzero == 0 && float(density[m])/(xvec*yvec*zint*(nstruct/skip)) == 0) {}
  //else {fprintf(Zdistfile,"%f %f \n ",zint*startfromzero, float(density[m])/(xvec*yvec*zint*(nstruct/skip)));startfromzero++;}
  fprintf(Zdistfile,"%f %f \n ",ztick[m*2+1], float(tetradensity[m]/density[m]));
}
fclose(Zdistfile);
hipFree(dev_density);
hipFree(dev_tetradensity);
hipFree(dev_A);
hipFree(dev_tetra);
hipFree(dev_ztick);
  
   free(Otetra);
   free(tetradensity);
}
