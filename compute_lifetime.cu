#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################


//life time used by compute_lifetime.cu

#include "cudatools.cuh"

void compute_lifetime(int *A, int *B, float time)
{

  //A is the 0/1 file and B has the number of species to be correlated
   
////////////////////////////////////LIFETIME-PART///////////////////////////////////////////
  
    diffuse_time=time;
  
  
    total_time = (nstruct/skip)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    num_bins =  int ((nstruct/skip)/((diffuse_time*1000)/timestep));
    num_bins -= 1; 
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    cout <<num_bins <<" number of bins are chosen for the Hydrogen bond analysis"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    origins = SD_store;
    cout <<"This will correlate for "<<SD_store<<" frames in the input trajectory "<<endl;
    int restarts = origins/skips;
    cout <<restarts<<" restarts from each bin will be taken"<<endl;
  
    
       SDsum1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
	   SDsum1[ia]=0.0;
	}
	
	float *FDsum1;
	FDsum1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=1;ia<SD_store-1;ia++)
        {
	   FDsum1[ia]=0.0;
	}
	float *Hexch1;

	Hexch1 = (float *) malloc (sizeof(float)*nElt1*SD_store*2);
 //########################################################################################################################################
//########################################################################################################################################
//BINNING AND AVERAGING THE HBAF
//#########################################################################################################################################
//######################################################################################################################################### 
float *Hexch_sized1;// correlation
float *Hexch_sized;// correlation

    for(int i=0; i<num_bins;i++)  //start bin
    {//int avgcounter=0; 
        printf("Cuda start: All lifetimes  %d \n",i);


    
         for(int i1=0;i1<SD_store*2;i1++)
         {
	     for(int k1=0; k1<nElt1; k1++)//shared hydrogen
             {
	       Hexch1[k1+i1*nElt1]=0.0;
	     }
	 }
	 

int **initHBcol1;// storing the location of the atom in the exch matrix 

initHBcol1 = (int **) malloc (sizeof(int *)*SD_store*2);
     for(int i1=0; i1<SD_store*2; i1++)// for each frame
     {
     initHBcol1[i1] = (int *) malloc (sizeof(int)*B[i1+i*SD_store]);
     for(int j1=0; j1<B[i1+i*SD_store]; j1++)// for number of H bonds this frame
     {
      initHBcol1[i1][j1]=0;
     }
     }
     
   
    for(int i1=0;i1<SD_store*2;i1++)
     {int chker=0;
      for(int j1=0; j1<nElt1; j1++)
       {
	if(A[j1+i1*nElt1+i*SD_store*nElt1] == 1 ) 
        {
	  Hexch1[j1+i1*nElt1]=1.0; //acceptor check
	  initHBcol1[i1][chker] = j1+i1*nElt1; chker+=1;
	}
      }
     }
    
	//cout <<"exch matrix done "<<endl;
	
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////
     //////////////////////////////////////////////////////origin//////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////////////////
     for(int j=0; j<SD_store; j=j+skips) //go through restart points or origins
     { 
       if(B[j+i*SD_store] > 0){
       Hexch_sized1 = (float *) malloc (sizeof(float)*B[j+i*SD_store]*SD_store);
       Hexch_sized = (float *) malloc (sizeof(float)*B[j+i*SD_store]*SD_store);
         for(int ia=0;ia<SD_store;ia++)
         {
           for(int ja=0; ja<B[j+i*SD_store]; ja++)////only those present in the first frame in the restart bin
           { 
	        Hexch_sized1[ja+ia*B[j+i*SD_store]] = Hexch1[initHBcol1[j][ja]+ia*nElt1];
	        Hexch_sized[ja+ia*B[j+i*SD_store]] = Hexch1[initHBcol1[j][ja]+ia*nElt1];
	   }
	 }
	/* 
	 if(j==2 && i==0)
	 {
	   cout<<endl;
	   for(int ia=0;ia<SD_store;ia++)
           {
           for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)////only those present in the first frame in the restart bin
           { 
	       cout<<Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] <<" ";
	   }
	   cout<<endl;
	   }
	   cout<<"after: "<<endl;
	 }
	 */
	 
	 if(HB_lifestyle=="continuous")
	 {
         for(int ja=0; ja<B[j+i*SD_store]; ja++)///for continuous lifetime
         {
           for(int ia=1;ia<SD_store;ia++)
           {
	     if(Hexch_sized[ja+(ia-1)*B[j+i*SD_store]] == 0) {Hexch_sized[ja+ia*B[j+i*SD_store]] = 0;Hexch_sized1[ja+ia*B[j+i*SD_store]] = 0;} 
	   }
	 }
	 }
	 
	 
	 if(HB_lifestyle=="transient")
	 {
	   //do transient time approximation here
	   
         for(int ja=0; ja<B[j+i*SD_store]; ja++)///for transient intermittent lifetime
         {
           for(int ia=0;ia<SD_store;ia++)
           {
	     if(Hexch_sized[ja+ia*B[j+i*SD_store]] == 0) 
	     {
	       for(int chk=1; chk<=transtime; chk++)
	       {
		 if(ia + chk >= SD_store){break;}
		 else if(Hexch_sized[ja+(ia+chk)*B[j+i*SD_store]] == 1){ Hexch_sized[ja+ia*B[j+i*SD_store]]=1;Hexch_sized1[ja+ia*B[j+i*SD_store]]=1;break;}
	       }
	    } 
	   }
	 }
	 
	 for(int ja=0; ja<B[j+i*SD_store]; ja++)///for imposing continuous lifetime after transient time
         {
           for(int ia=1;ia<SD_store;ia++)
           {
	     if(Hexch_sized[ja+(ia-1)*B[j+i*SD_store]] == 0) {Hexch_sized[ja+ia*B[j+i*SD_store]] = 0;Hexch_sized1[ja+ia*B[j+i*SD_store]] = 0;} 
	   }
	 }
	 }
      /*
       * 
       * 
       * 
         if(j==2 && i==0)
	 {
	   cout<<endl;
	   for(int ia=0;ia<SD_store;ia++)
           {
           for(int ja=0; ja<Aintnum[j+i*SD_store]; ja++)////only those present in the first frame in the restart bin
           { 
	       cout<<Hexch_sized[ja+ia*Aintnum[j+i*SD_store]] <<" ";
	   }
	   cout<<endl;
	   }
	 }
      
      
         */
       //SD=(float *) malloc (sizeof(float)*initHBnum[j+i*SD_store]*SD_store);
      
      
      
      SD1=(float *) malloc (sizeof(float)*B[j+i*SD_store]*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
	 for(int ja=0; ja<B[j+i*SD_store]; ja++)
	 {
           //SD[ja+ia*initHBnum[j+i*SD_store]]=0.0;
	   SD1[ja+ia*B[j+i*SD_store]]=0.0;
	 }
	}
       
      
      dim3 dimBlocka(32,1,32);
      dim3 dimGrida((B[j+i*SD_store]+dimBlocka.x-1)/dimBlocka.x,1,(SD_store+dimBlocka.z-1)/dimBlocka.z);

       
      
      
      float *dev_A1;
            
      hipMalloc((void **)&dev_A,sizeof(float)*B[j+i*SD_store]*SD_store);
      hipMalloc((void **)&dev_A1,sizeof(float)*B[j+i*SD_store]*SD_store);
      hipMalloc((void **)&dev_SD1,sizeof(float)*B[j+i*SD_store]*SD_store);
      
      hipMemcpy(dev_A,Hexch_sized1,sizeof(float)*B[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
      hipMemcpy(dev_A1,Hexch_sized,sizeof(float)*B[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);
      hipMemcpy(dev_SD1,SD1,sizeof(float)*B[j+i*SD_store]*SD_store,hipMemcpyHostToDevice);

      HBAF_calc<<<dimGrida,dimBlocka>>>(dev_A,dev_A1,dev_SD1,SD_store,B[j+i*SD_store],i,j,origins,skips);

      //hipMemcpy(SD1,dev_SD1,sizeof(float)*Aintnum[j+i*SD_store]*SD_store,hipMemcpyDeviceToHost);
      hipFree(dev_A);
      hipFree(dev_A1);
      hipFree(dev_lattice);
      //SDavg=(float *) malloc (sizeof(float)*SD_store);
      SDavg1=(float *) malloc (sizeof(float)*SD_store);
      for(int ja =0; ja<SD_store;ja++)
      {
      //SDavg[ja]=0.0;
      SDavg1[ja]=0.0;
      }
      int fairy=0;
      //hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
      //hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

      dim3 dimBlocks(1024,1,1);
      dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);
      //SDreduce<<<dimGrids,dimBlocks>>>(dev_SD,dev_SDavg,SD_store,initHBnum[j+i*SD_store],fairy);
      //hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
       
      //hipFree(dev_SD);
      //hipFree(dev_SDavg);
      
      hipMalloc((void **)&dev_SDavg1,sizeof(float)*SD_store);
      hipMemcpy(dev_SDavg1,SDavg1,sizeof(float)*SD_store,hipMemcpyHostToDevice);
      SDreduce<<<dimGrids,dimBlocks>>>(dev_SD1,dev_SDavg1,SD_store,B[j+i*SD_store],fairy);
      hipMemcpy(SDavg1,dev_SDavg1,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
       
      hipFree(dev_SD1);
      hipFree(dev_SDavg1);
      
           
     // SDsum1[0]+=1;
//cout<<Aintnum[j+i*SD_store]<<" "<<SDavg1[0]<<" ";
      for(int ja =0; ja<SD_store;ja++)
      {
      SDsum1[ja]+=SDavg1[ja];
      //SDsum1[ja]+=(SDavg1[ja]/SDavg1[0]);
      }
  
      for(int ja=1; ja<SD_store -1;ja++)
      {
      //FDsum[ja]+=-((SDavg[ja+1]-SDavg[ja-1]) / (((ja+1)*(timestep/1000))-((ja-1)*(timestep/1000))));
      FDsum1[ja]+=-((SDavg1[ja+1]-SDavg1[ja-1]) / (((ja+1)*(timestep/1000))-((ja-1)*(timestep/1000))));      
      }
      
      
      
      
      free(Hexch_sized);
      //free(SDavg);
      //free(SD);
      free(Hexch_sized1);
      free(SDavg1);
      //free(SD1);
      //avgcounter++;
       }
    }//origins over
    

  
  free(initHBcol1);
     
   }

  free(Hexch1);
  ofstream Hexchplot, Hexchplotac;
  
      Hexchplotac.open("ct.data");
     for(int ja =0; ja<SD_store;ja++)
     {
       //SDsum[ja]/=float(num_bins*restarts);
       SDsum1[ja]/=float(num_bins*restarts);
       //Hexchplotac <<ja*timestep/1000<<" "<<SDsum1[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<SDsum1[ja]/SDsum1[0]<<endl;
       //Hexchplot <<ja*timestep/1000<<" "<<SDsum[ja]<<endl;
      } 
      //Hexchplot.close();
      Hexchplotac.close();
      
    
      //Hexchplot.open("ft-pair.data");
      Hexchplotac.open("ft.data");
     for(int ja =1; ja<SD_store-1;ja++)
      {
       //FDsum[ja]/=num_bins*restarts;
       FDsum1[ja]/=num_bins*restarts;
       //Hexchplot <<ja*timestep/1000<<" "<<FDsum[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<FDsum1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     
     int avg_every = 100;
     float *FD_avg1;
     //FD_avg=(float *) malloc (sizeof(float)*SD_store);
     FD_avg1=(float *) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
          // FD_avg[ia]=0.0;
	   FD_avg1[ia]=0.0;
	}
     
      for(int ja=(avg_every/2)+1; ja<SD_store -(avg_every/2)-1;ja++)
      {
	for(int ka=ja-(avg_every/2);ka<=ja+(avg_every/2)-1;ka++)
	{
	 // FD_avg[ja]+= FDsum[ka];
	  FD_avg1[ja]+= FDsum1[ka];
	}
	//FD_avg[ja]/=avg_every;
	FD_avg1[ja]/=avg_every;
      }
      
      Hexchplotac.open("ft-avg.data");
      //Hexchplot.open("ft-avg-pair.data");
     for(int ja =(avg_every/2)+1; ja<SD_store -(avg_every/2)-1;ja++)
      {
       //Hexchplot <<ja*timestep/1000<<" "<<FD_avg[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<FD_avg1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     
      
      
      
     float lifetime1;
     float *int_SDsum1;
     float *cum_SDsum1;
     //int_SDsum = (float*) malloc (sizeof(float)*SD_store);
     //cum_SDsum = (float*) malloc (sizeof(float)*SD_store);
     int_SDsum1 = (float*) malloc (sizeof(float)*SD_store);
     cum_SDsum1 = (float*) malloc (sizeof(float)*SD_store);
        for(int ia=0;ia<SD_store;ia++)
        {
           //int_SDsum[ia]=0.0;
	   //cum_SDsum[ia]=0.0;
	   int_SDsum1[ia]=0.0;
	   cum_SDsum1[ia]=0.0;
	}
     for(int ja =1; ja<SD_store;ja++)
      {
       //int_SDsum[ja]= (((ja*timestep*0.001) - ((ja-1)*timestep*0.001)) * ((SDsum[ja] + SDsum[ja-1])/2.0));
       //cum_SDsum[ja]=cum_SDsum[ja-1]+int_SDsum[ja];
       int_SDsum1[ja]= (((ja*timestep*0.001) - ((ja-1)*timestep*0.001)) * (((SDsum1[ja]/SDsum1[0]) + (SDsum1[ja-1]/SDsum1[0]))/2.0));
       cum_SDsum1[ja]=cum_SDsum1[ja-1]+int_SDsum1[ja];
       if(ja == SD_store-1) {//lifetime = cum_SDsum[ja];
	 lifetime1 = cum_SDsum1[ja];}
      }  
     
     //Hexchplot.open("ct-integrate-pair.data");
     Hexchplotac.open("ct-integrate.data");
     for(int ja =0; ja<SD_store;ja++)
      {
       //Hexchplot <<ja*timestep/1000<<" "<<cum_SDsum[ja]<<endl;
       Hexchplotac <<ja*timestep/1000<<" "<<cum_SDsum1[ja]<<endl;
      }  
      //Hexchplot.close();
      Hexchplotac.close();
     //Hexchplot.open("Hbond-lifetime-pair.data");
     //Hexchplot <<"Lifetime from the integral of c(t) is : "<<lifetime<<" ps"<<endl;
     //Hexchplot.close();
     Hexchplot.open("lifetime.data");
     Hexchplot <<"Lifetime from the integral of c(t) is : "<<lifetime1<<" ps"<<endl;
     Hexchplot.close();
     
 

}
