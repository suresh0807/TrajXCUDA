#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"

__global__ void VAF_calc(float *A,float *B,float *C,int *D,int SD_store,int nElt1, int i, int j,int origins,int skips,int whichwater)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  //int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  widid < origins/skips)
  {

    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]*B[rowid*3+(j*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k];
      z+=chk;
    }
//     printf("%f \n",C[rowid+(j*nElt1)+(widid*nElt1*SD_store)]);
    //if(D[rowid+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater && D[rowid+(j*nElt1)+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater)
    if(D[rowid+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater)
    {
    C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ;
    }
    else if(whichwater==2)
    {
     C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ; 
    }
  }
}

