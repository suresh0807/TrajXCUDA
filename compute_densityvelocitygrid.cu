//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"

void compute_densityvelocitygrid()
{
    if(plot=="int")        { nElt1 = nA_int;}
    else if(plot=="bulk")  { nElt1 = nA_bulk;}
    
cout << type <<" is chosen"<<endl;
cout<<"densityvelocitygrid: " << nElt1 << Elt1 << " atoms are there in each structure"<<endl;
    printf("%f %f %f %f %f %f\n",minx,maxx,miny,maxy,minz,maxz);
    
    if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
}
    if(set_max_x == "yes")
{
  minx = set_minx;
  maxx = set_maxx;
}
    if(set_max_y == "yes")
{
  miny = set_miny;
  maxy = set_maxy;
}
    xrange=maxx-minx;
    yrange=maxy-miny;
    zrange=maxz-minz;

    xint = xrange/xsplit;
    yint = yrange/ysplit;
    zint = zrange/zsplit;

    xtick=(float*) malloc (sizeof(float)*xsplit*2);
    ytick=(float*) malloc (sizeof(float)*ysplit*2);
    ztick=(float*) malloc (sizeof(float)*zsplit*2);
    for(int i=0;i<xsplit;i++)
    {
      xtick[i*2]=minx+(i*xint);
      xtick[i*2+1]=minx+((i+1)*xint);
    }
    for(int i=0;i<ysplit;i++)
    {
      ytick[i*2]=miny+(i*yint);
      ytick[i*2+1]=miny+((i+1)*yint);
    }
    for(int i=0;i<zsplit;i++)
    {
      ztick[i*2]=minz+(i*zint);
      ztick[i*2+1]=minz+((i+1)*zint);
    }

FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<xsplit;i++)
    {
      fprintf(xyztick,"%f %f \n",xtick[i*2],xtick[i*2+1]);
    }
      fprintf(xyztick," \n");
    for(int i=0;i<ysplit;i++)
    {
      fprintf(xyztick,"%f %f \n",ytick[i*2],ytick[i*2+1]);    
    }
      fprintf(xyztick," \n");
    for(int i=0;i<zsplit;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
fclose(xyztick);

    density=(int *) malloc (sizeof(int)*xsplit*ysplit*zsplit);
    for(int i=0;i<xsplit;i++)
    {
      for(int j=0;j<ysplit;j++)
      {
        for(int k=0;k<zsplit;k++)
        {
          density[k+j*zsplit+i*ysplit*zsplit]=0;
        }
      }
    }
    
    velocity=(float *) malloc (sizeof(float)*xsplit*ysplit*zsplit);
    for(int i=0;i<xsplit;i++)
    {
      for(int j=0;j<ysplit;j++)
      {
        for(int k=0;k<zsplit;k++)
        {
          velocity[k+j*zsplit+i*ysplit*zsplit]=0.0;
        }
      }
    }
    
    
///*
    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_density,sizeof(int)*xsplit*ysplit*zsplit);
    hipMalloc((void **)&dev_xtick, sizeof(float)*xsplit*2);
    hipMalloc((void **)&dev_ytick, sizeof(float)*ysplit*2);
    hipMalloc((void **)&dev_ztick, sizeof(float)*zsplit*2);
    if(plot=="int")
    {
      hipMemcpy(dev_A,A_int,sizeof(float)*(nstruct/skip)*nA_int*3,hipMemcpyHostToDevice);
    }
    else if(plot=="bulk")
    {
      hipMemcpy(dev_A,A_bulk,sizeof(float)*(nstruct/skip)*nA_bulk*3,hipMemcpyHostToDevice);
    }
    else if(plot=="all")
    {
      hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    }
    hipMemcpy(dev_density,density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_xtick,xtick,sizeof(float)*xsplit*2,hipMemcpyHostToDevice);
    hipMemcpy(dev_ytick,ytick,sizeof(float)*ysplit*2,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*zsplit*2,hipMemcpyHostToDevice);
    
    printf("%ld B needed\n",(((nstruct/skip)*nElt1*3+xsplit*ysplit*zsplit+xsplit*2+ysplit*2+zsplit*2)*4));
    dim3 dimBlock(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGrid(((nstruct/skip)+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    density_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);
    hipMemcpy(density,dev_density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
    hipFree(dev_density);
    
    
    
    hipMalloc((void **)&dev_VEL,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_velocity,sizeof(float)*xsplit*ysplit*zsplit);
    hipMemcpy(dev_VEL,VEL,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_velocity,velocity,sizeof(float)*xsplit*ysplit*zsplit,hipMemcpyHostToDevice);
    printf("%ld B needed\n",(((nstruct/skip)*nElt1*3+xsplit*ysplit*zsplit+xsplit*2+ysplit*2+zsplit*2)*4));
    velocity_calc<<<dimGrid,dimBlock>>>(dev_A,dev_VEL,dev_velocity,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);
    hipMemcpy(velocity,dev_velocity,sizeof(float)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
    hipFree(dev_velocity);
    hipFree(dev_VEL);
    hipFree(dev_A);
    hipFree(dev_xtick);
    hipFree(dev_ytick);
    hipFree(dev_ztick);
    
    
FILE *densityfile=fopen("out","wt");
FILE *velocityfile=fopen("velout","wt");

if(order=="XYZ") {

for(int l=0;l<ysplit;l++)
{
  //fprintf(densityfile,"%d %d\n",zsplit,xsplit);
  fprintf(densityfile,"#%d \n",l);
  fprintf(velocityfile,"#%d \n",l);
for(int m=0;m<zsplit;m++)
{
for(int k=0;k<xsplit;k++)
{
    fprintf(velocityfile,"%f ",velocity[m+l*zsplit+k*ysplit*zsplit]/(density[m+l*zsplit+k*ysplit*zsplit]+0.0001));
    fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
}
fprintf(densityfile,"\n");
fprintf(velocityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}
}
if(order=="ZXY") {

for(int k=0;k<xsplit;k++)
{
  fprintf(densityfile,"#%d \n",k);
  fprintf(velocityfile,"#%d \n",k);
for(int l=0;l<ysplit;l++)
{
for(int m=0;m<zsplit;m++)
{
    fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
    fprintf(velocityfile,"%f ",velocity[m+l*zsplit+k*ysplit*zsplit]/(density[m+l*zsplit+k*ysplit*zsplit]+0.0001));
}
fprintf(densityfile,"\n");
fprintf(velocityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}
}
if(order=="YZX") {

for(int m=0;m<zsplit;m++)
{
  fprintf(densityfile,"#%d \n",m);
  fprintf(velocityfile,"#%d \n",m);
for(int k=0;k<xsplit;k++)
{
for(int l=0;l<ysplit;l++)
{
    fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
    fprintf(velocityfile,"%f ",velocity[m+l*zsplit+k*ysplit*zsplit]/(density[m+l*zsplit+k*ysplit*zsplit]+0.0001));
}
fprintf(densityfile,"\n");
fprintf(velocityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}  
}
fclose(densityfile);
fclose(velocityfile);



}
