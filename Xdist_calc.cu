#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################





#include "cudatools.cuh"

__global__ void Xdist_calc(float *A,int *density,int nstruct,int natoms,int zsplit, float *ztick, int Dirn)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  if(rowid < nstruct && colid <natoms)
  {
        for(int m=0;m<zsplit;m++)
        {
        if( A[colid*3+(rowid*natoms*3)+Dirn] >= ztick[m*2] && A[colid*3+(rowid*natoms*3)+Dirn] < ztick[m*2+1])
          {
            atomicAdd(&density[m],1);
            break;
  //          density[m+l*zsplit+k*ysplit*zsplit]+=1;
          }
        }
   }
}


__global__ void Xdist_calc(float *A, float *B,float *tetradensity, int *density,int nstruct,int natoms,int zsplit, float *ztick, int Dirn)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  if(rowid < nstruct && colid <natoms)
  {
        for(int m=0;m<zsplit;m++)
        {
        if( A[colid*3+(rowid*natoms*3)+Dirn] >= ztick[m*2] && A[colid*3+(rowid*natoms*3)+Dirn] < ztick[m*2+1])
          {
	    atomicAdd(&density[m],1);
            atomicAdd(&tetradensity[m],B[colid+rowid*natoms]);
            break;
  //          density[m+l*zsplit+k*ysplit*zsplit]+=1;
          }
        }
   }
}