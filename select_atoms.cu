#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"



void select_atoms(string whichmat)
{
  
  if(whichmat != "box")
  {
    
    
    cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;
  
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
// Initialize distance matrix and histogram matrix
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

      float GLOBALAminz=A[dircover];
      float GLOBALAmaxz=A[dircover];
      float MIDAz=0.0, MINAz=0.0, MAXAz=0.0;
      
  for(int i=0; i<1; i++)
  {
    for(int j=1; j<nElt1; j++)
    {
    if(A[j*3+i*nElt1*3+dircover] > GLOBALAmaxz)    GLOBALAmaxz=A[j*3+i*nElt1*3+dircover];
    else if(A[j*3+i*nElt1*3+dircover] < GLOBALAminz)    GLOBALAminz=A[j*3+i*nElt1*3+dircover];
    }
  }
  
  MIDAz=(GLOBALAmaxz+GLOBALAminz)/2.0;
  
  MAXAz=MIDAz+(bondist/2.0);
  MINAz=MIDAz-(bondist/2.0);
  
  
  if(msd_for=="int")
  {
//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));

//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
      //covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    //hipFree(dev_exch);
    for(int l=0; l<nstruct/skip; l++)
      {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
        }
      }
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    
      ofstream exchplot;
      }
  
   
  
  else if(msd_for=="bulk")
  {
    //Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    
//Copy data from host to device

    
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(32,1,32);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,1,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_exch,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_exch,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
      //covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A);
    //hipFree(dev_exch);
      for(int l=0; l<nstruct/skip; l++)
      {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Abulknum[l]++;}
          else if (exch[i+l*nElt1]==0) Aintnum[l]++;
        }
      }
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    ofstream exchplot;
 

  }
  
  else if (msd_for=="all")
  {
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    
    MSDchaos=(int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    for(int l=0; l<(nstruct/skip);l++)
    {
    for(int i=0;i<nElt1;i++)
    {
      MSDchaos[i+l*nElt1]=0;
    }
    }
    
    //MSDsteps=ceil((GLOBALAmaxz-GLOBALAminz))/2;
    //MSDsteps=10;
    cout<<"The cell will be divided into "<<MSDsteps<<" slices along vacuum direction for MSD computation"<<endl;
    MSDint = (GLOBALAmaxz-GLOBALAminz)/MSDsteps;
    cout<<"Each slice will be "<<MSDint<<" Angstrom long"<<endl;
    cout <<GLOBALAminz<< " "<<GLOBALAmaxz<< " "<<(GLOBALAmaxz-GLOBALAminz)<<" "<<MSDsteps<<endl;
    MSDtics=(float*) malloc (sizeof(float)*MSDsteps*2);
    for(int i=0;i<MSDsteps;i++)
    {
      MSDtics[i*2] = GLOBALAminz +(i*MSDint);
      MSDtics[i*2+1] = GLOBALAminz +((i+1)*MSDint);
    }
    
    for(int l=0; l<(nstruct/skip);l++)
    {
    for(int i=0; i< nElt1;i++)
    {
      for(int ii=0;ii<MSDsteps;ii++)
      {
      if(A[i*3+dircover+l*nElt1*3] > MSDtics[ii*2] && A[i*3+dircover+l*nElt1*3] <= MSDtics[ii*2+1])
      {
	MSDchaos[i+l*nElt1]=ii;
      }
      }
      //cout<<i<<" "<<MSDchaos[i]<<endl;
    }
    }
   

   
  }
    
 }
 
   else if(whichmat == "box")
  {
     exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
    }

    if(set_max_z == "yes")
    {
     minz = set_minz;
     maxz = set_maxz; 
    }
    if(set_max_x == "yes")
    {
     minx = set_minx;
     maxx = set_maxx;
    }
    if(set_max_y == "yes")
    {
     miny = set_miny;
     maxy = set_maxy;
    }
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));

//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(32,1,32);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,1,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    
    boxmat<<<dimGride,dimBlocke>>>(dev_A,dev_exch,nElt1,(nstruct/skip),maxx,minx,maxy,miny,maxz,minz);
   
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A);
    //hipFree(dev_exch);
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1;}
        }     
      }
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    ofstream exchplot;
    exchplot.open("exchange.data");

 for(int l=0; l<(nstruct/skip); l++)
    {
      exchplot <<l<<" ";
      for(int j=0; j<nElt1; j++)
      {
                exchplot << exch[j+l*nElt1]<<" ";
      }
      exchplot <<endl;
    }
    exchplot.close();    

 //ofstream intnum;
 //intnum.open("intnum.data");
 
      for(int l=0; l<nstruct/skip; l++)
      {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
        }
      //for(int i=0; i< nElt2; i++)
      //  {
      //    if (exch2[i+l*nElt2]>0) {exch2[i+l*nElt2]=1; Bintnum[l]++;}
      //    else if (exch2[i+l*nElt2]==0) Bbulknum[l]++;
      //  }
      //intnum << l<<" "<<Aintnum[l]<<endl;
      }
  //intnum.close();
  } 

}
