#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"


__global__ void boxmat(float *x, int *a, int nElt1, int nstruct,float maxx, float minx, float maxy,float miny, float maxz, float minz)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  
  if(rowid < nElt1 && widid < nstruct)
  {

   if( x[rowid*3+widid*nElt1*3] < maxx && x[rowid*3+widid*nElt1*3] > minx &&  x[rowid*3+widid*nElt1*3+1] < maxy && x[rowid*3+widid*nElt1*3+1] > miny && x[rowid*3+widid*nElt1*3+2] < maxz && x[rowid*3+widid*nElt1*3+2] > minz )
   {
     atomicAdd(&a[rowid+widid*nElt1],1);
   }
   
   }
}