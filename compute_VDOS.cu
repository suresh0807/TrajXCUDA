#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"



void compute_VDOS()
{
  
  cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;
  
  int *exch;
  int *dev_exch;
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
 
// Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
      }

//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));

//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlock(10,10,10);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(count_metal+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    //hipFree(dev_exch);
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1;}
        }     
      }
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
  /*  
ofstream exchplot;
exchplot.open("exchange.data");

 for(int l=0; l<(nstruct/skip); l++)
    {
      exchplot <<l<<" ";
      for(int j=0; j<nElt1; j++)
      {
      
          exchplot << exch[j+l*nElt1]<<" ";
      }
      exchplot <<endl;
    }
    exchplot.close();    

  /*  exchplot.open("exchange.data");
      for(int k=0; k<zsplit; k++)
      {
          exchplot << ztick[k*2+1] <<" "<<exchdensity[k]<<endl;
      }
  
      for(int k=0; k<nElt1; k++)
      {
          exchplot << A[k*3+2] <<" "<<exchsum[k]<<endl;
      }*/
  //  exchplot.close(); 
///////////////////////////////////////////Special atom selection between 2 distances/////////////////////////////////////////////////////////////////////////

  
  if(choose_atoms=="yes")
{
  float surf,avgsurf=0.0;
int dirn;
  if(choose_dirn=="z") dirn=2;
  else if(choose_dirn=="x") dirn=0;
  else if(choose_dirn=="y") dirn=1;
  
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< metal_species<<endl;
  
 
  
  int surfatom;
  for(int i=0 ; i<(nstruct/skip); i++)
  {surfatom=0;surf=0.0;
  for(int j=0; j<count_metal;j++)
  {
    if(METAL[j*3+i*count_metal*3+dirn] > mintop && METAL[j*3+i*count_metal*3+dirn] < maxtop)
    {
    surf+=METAL[j*3+i*count_metal*3+dirn];
    surfatom++;
    }
  }
  surf/=surfatom;
  avgsurf+=surf;
  }
  
  avgsurf/=(nstruct/skip);
  
 cout<<"Average surface atoms position in "<<choose_dirn<<" is "<<avgsurf<<endl;
 cout<<"Average number of surface atoms is "<< surfatom<<endl;
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
      if(strict_dirn == "yes") {covermat<<<dimGride,dimBlocke>>>(dev_A,dev_exch,minbondist,maxbondist,nElt1,(nstruct/skip),dirn,avgsurf,mintop,maxtop);}
      else {covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);}
    }
    else if (cell_type == "monoclinic")
    {
      if(strict_dirn == "yes") {covermat<<<dimGride,dimBlocke>>>(dev_A,dev_exch,minbondist,maxbondist,nElt1,(nstruct/skip),dirn,avgsurf,mintop,maxtop);} //no lattice needed thus use covermat - no problem
      else {covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);}
    }
    
    
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    
    //ofstream exchout;
    //exchout.open("exchout.data");
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
	//exchout<<l<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        //exchout<<endl;
      }
      
      //exchout.close();
     hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 
     
    
}          
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////

   cout <<"hola"<<endl;
  
    total_time = (nstruct)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    num_bins =  int ((nstruct)/((diffuse_time*1000)/timestep));
    num_bins -= 1; 
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    cout <<num_bins <<" number of bins are chosen for the VDOS analysis"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    origins = SD_store;
    cout <<"This will correlate for "<<SD_store<<"frames in the input trajectory "<<endl;
    int restarts = origins/skips;
    cout <<restarts<<" restarts from each bin will be taken"<<endl;
    if(Elt1=="all")
    {
    nElt1=natoms;
    nElt2=natoms;
    }
    SD=(float *) malloc (sizeof(float)*nElt1*SD_store*origins/skips);
    for(int i=0;i<origins/skips;i++)
    {
      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SD[k+j*nElt1+i*nElt1*SD_store]=0.0;
	}
      }
    }
        SDsum=(float *) malloc (sizeof(float)*nElt1*SD_store*origins/skips);
    for(int i=0;i<origins/skips;i++)
    {
      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SDsum[k+j*nElt1+i*nElt1*SD_store]=0.0;
	}
      }
    }
///*


    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*nstruct*nElt1*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nstruct*nElt1*3);
    hipMalloc((void **)&dev_SD,sizeof(float)*nElt1*SD_store*origins/skips);
    hipMemcpy(dev_A,VEL,sizeof(float)*nstruct*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,VEL,sizeof(float)*nstruct*nElt1*3,hipMemcpyHostToDevice);    
    
    cout << "Memory required to store velocity information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local VAF storage: "<<(nElt1*SD_store*restarts)*4/float(1000000000)<< " Gbs" << endl;


    dim3 dimBlocka(32,1,32);
    dim3 dimGrida((nElt1+dimBlocka.x-1)/dimBlocka.x,1,((origins/skips)+dimBlocka.z-1)/dimBlocka.z);

    //dim3 dimBlock(10,10,10);
    //dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(SD_store+dimBlock.y-1)/dimBlock.y,((origins/skips)+dimBlock.z-1)/dimBlock.z);    
    
  //    hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store*origins,hipMemcpyHostToDevice);    

    for(int i=0; i<num_bins;i++)
    //for(int i=0; i<num_bins;i++)
    {
             for(int i1=0;i1<origins/skips;i1++)
              {
               for(int j1=0;j1<SD_store;j1++)
                {
	         for(int k1=0; k1<nElt1; k1++)
	          {
                    SD[k1+j1*nElt1+i1*nElt1*SD_store]=0.0;
	          }
                }
              }
      int whichwater;
      if(msd_for=="int"){whichwater=1;}
      else if(msd_for=="bulk"){whichwater=0;}
      else if(msd_for=="all"){whichwater=2;}
      if(choose_atoms=="yes"){whichwater=1;}
hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyHostToDevice);
      for(int j=0; j<SD_store; j++)
      {
	//int j=1;
      VAF_calc<<<dimGrida,dimBlocka>>>(dev_A,dev_B,dev_SD,dev_exch,SD_store,nElt1,i,j,origins,skips,whichwater);
      }
hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyDeviceToHost);
    for(int i1=0;i1<origins/skips;i1++)
    {
      for(int j1=0;j1<SD_store;j1++)
      {
	for(int k1=0; k1<nElt1; k1++)
	{
          SDsum[k1+j1*nElt1+i1*nElt1*SD_store]+=SD[k1+j1*nElt1+i1*nElt1*SD_store];
	}
      }
    }
     cout << (i+1)*diffuse_time << " pico seconds done"<<endl;  
    }
    
//      hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store*origins,hipMemcpyDeviceToHost);



    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_SD);
     
    SDavgf=(float *) malloc (sizeof(float)*SD_store*nElt1);
    for(int i=0; i<SD_store; i++)
    {
      for(int j =0; j<nElt1;j++)
      {
        SDavgf[j+i*nElt1]=0.0;
      }
    }
    hipMalloc((void **)&dev_SDsum,sizeof(float)*nElt1*SD_store*origins/skips);
    hipMalloc((void **)&dev_SDavgf,sizeof(float)*SD_store*nElt1);
    hipMemcpy(dev_SDavgf,SDavgf,sizeof(float)*SD_store*nElt1,hipMemcpyHostToDevice);
    hipMemcpy(dev_SDsum,SDsum,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyHostToDevice);
    
    dim3 dimBlockf(32,32,1);
    dim3 dimGridf((nElt1+dimBlockf.x-1)/dimBlockf.x,(SD_store+dimBlockf.y-1)/dimBlockf.y,1);
    cout<<"start reduce "<< origins/skips <<" "<<nElt1<< " "<<SD_store<<endl;
    
    SDreducef<<<dimGridf,dimBlockf>>>(dev_SDsum,dev_SDavgf,nElt1,SD_store,origins,skips);
    
    hipMemcpy(SDavgf,dev_SDavgf,sizeof(float)*SD_store*nElt1,hipMemcpyDeviceToHost);
    hipFree(dev_SDsum);

    
    SDavg=(float *) malloc (sizeof(float)*SD_store);
    for(int j =0; j<SD_store;j++)
    {
      SDavg[j]=0.0;
    }
    hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
    hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

    dim3 dimBlocks(1024,1,1);
    dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);
    SDreduce<<<dimGrids,dimBlocks>>>(dev_SDavgf,dev_SDavg,SD_store,nElt1);

    hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
    
    hipFree(dev_SDavgf);
    hipFree(dev_SDavg);
    
       printf("Padding with zeros and gaussian smoothing... from timteatro\n");
 int padd = 2;
    fftw_complex *VAFavg=(fftw_complex *) fftw_malloc (sizeof(fftw_complex)*SD_store*padd);
    for(int j =0; j<SD_store*padd;j++)
    {
      VAFavg[j]=0.0;
    }    
    for(int j =0; j<SD_store;j++)
    {
      VAFavg[j]=SDavg[j];
    }    
     for(int j =0; j<SD_store;j++)
    {
    //  cout<<j<<" "<<SDavg[j]<<" "<<creal(VAFavg[j])<<endl;
    }
      for(int j =1; j<  SD_store*padd;j++)
    {
      VAFavg[j] /= VAFavg[0];
    }  

    VAFavg[0]=1.0;
    float sigma = float(SD_store) / 2.50;
    
    for(int j =0; j<SD_store*padd;j++)
    {
      VAFavg[j] *=exp( -j * j/(2*sigma*sigma))/(sigma *2.506628274631000);
    } 

     for(int j =1; j<SD_store*padd;j++)
    {
      VAFavg[j] /= VAFavg[0];
    }

    VAFavg[0]=1.0;
// printing the rdf data to be visualized

    fftw_complex    norm = 1.0f;
    fftw_complex   *dft_out;
    fftw_plan       dft_plan;
    
    dft_out = (fftw_complex *) fftw_malloc(SD_store * padd * sizeof(fftw_complex));
    
   dft_plan = fftw_plan_dft_1d(SD_store*padd, VAFavg, dft_out, FFTW_FORWARD, FFTW_ESTIMATE);
   printf("#   done.\n# Executing FFT\n");
   fftw_execute(dft_plan);

   norm = 0.00f;
   for (int m = 0; m <= (SD_store * padd) / 2; m++)
   {
      if (creal(dft_out[m] * conj(dft_out[m])) > creal(norm))
      {
         norm = dft_out[m] * conj(dft_out[m]);
      }
   }
      norm = 1/norm;
   
   
      
    FILE *VAFplot=fopen("vaf.data","wt");
      fprintf(VAFplot,"# Time (ps) VAF (A^(2)/ps^(2)) \n");
    for(int i =0; i<SD_store*padd;i++)
    {

      fprintf(VAFplot,"%f %f\n",(float(i)*timestep)/1000.0,VAFavg[i]);
    }
    fclose(VAFplot);
    
    
    FILE *VDOSplot=fopen("vdos.data","wt");
      fprintf(VDOSplot,"# Time (ps) VAF (A^(2)/ps^(2)) \n");
    for(int i =0; i<(SD_store*padd)/2;i++)
    {

      fprintf(VDOSplot, "%17.9E %17.9E %17.9E %17.9E\n",
              i / (2.99792458E10 * (diffuse_time) * padd  * 1E-12),
              creal(dft_out[i]),
              cimag(dft_out[i]),
              norm * dft_out[i] * conj(dft_out[i]));
    }
    fclose(VDOSplot);    
    
    
}
