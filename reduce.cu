#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void reduce(float *a, float *b, int bin, int nstruct)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  if(rowid < bin )
  {
    for(int i=0;i<nstruct;i++)
    {
      b[rowid]+=a[rowid+(i*bin)];
    }
  }
}

