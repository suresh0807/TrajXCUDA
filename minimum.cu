#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

float minimum(float *a,int natoms,int stride)
{
  int j;
  float min;
  min=a[stride];
  for(j=1;j<natoms;j++)
  {
    if(a[j*3+stride] < min)
    {
      min=a[j*3+stride];
    }
  }
return min;
}

