#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"
float *rdf;
float *dev_rdf;
int bin;
float bin_rad;
float pi;
float *rdfavgf;
float rho;
float *dev_rdfavgf;
float *rdfavg;
float *dev_rdfavg;


void compute_rdf(void)
{
 /*
  if(multiply=="yes")
  {
  float *Aext, *Bext;
  
  nElt1ext= nElt1*4;
  nElt2ext= nElt2*4;
  
  Aext= (float *) malloc (sizeof(float)*nElt1*3*(nstruct/skip));
  Bext= (float *) malloc (sizeof(float)*nElt2*3*(nstruct/skip));
  
  for(int i=0; i<(nstruct/skip); i++)
  {
    for(int j=0;j<nElt1;j++)
    {
      Aext[j*3+ i*nElt1ext*3]=A[j*3+ i*nElt1*3]
    }
    for(int j=nElt1;j<nElt1ext/3;j++)
    {
      Aext[j*3+ i*nElt1ext*3]=A[(j-nElt1)*3+ i*nElt1*3]+lattice[(j-nElt1)*6+i*nElt1*6];
    }
    for(int j=nElt1;j<nElt1ext/2;j++)
    {
      Aext[j*3+ i*nElt1ext*3]  =A[(j-(nElt1*2))*3+ i*nElt1*3]  +lattice[(j-(nElt1*2))*6+i*nElt1*6+3];
      Aext[j*3+ i*nElt1ext*3+1]=A[(j-(nElt1*2))*3+ i*nElt1*3+1]+lattice[(j-(nElt1*2))*6+i*nElt1*6+3+1];
    }
    for(int j=nElt1*2;j<nElt1ext;j++)
    {
      Aext[j*3+ i*nElt1ext*3]  =A[(j-(nElt1*3))*3+ i*nElt1*3]  +lattice[(j-(nElt1*3))*6+i*nElt1*6]  +lattice[(j-(nElt1*3))*6+i*nElt1*6+3];
      Aext[j*3+ i*nElt1ext*3+1]=A[(j-(nElt1*3))*3+ i*nElt1*3+1]+lattice[(j-(nElt1*3))*6+i*nElt1*6+1];
    }
  }
  
  }
  */
  int whichwater;
  int *exch1, *exch2;
  int *dev_exch1, *dev_exch2;
  float *dev_A1, *dev_A2;
  
  exch1= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  exch2= (int *) malloc (sizeof(int)*nElt2*(nstruct/skip));

  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Bintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Bbulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
// Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch1[i+l*nElt1]=0;
        }
      for(int i=0; i< nElt2; i++)
        {
          exch2[i+l*nElt2]=0;
        }
        Abulknum[l]=0;
        Bbulknum[l]=0;
        Aintnum[l]=0;
        Bintnum[l]=0;
      }
  
      
      float GLOBALAminz=A[dircover];
      float GLOBALAmaxz=A[dircover];
      float MIDAz=0.0, MINAz=0.0, MAXAz=0.0;
      
  for(int i=0; i<1; i++)
  {
    for(int j=1; j<nElt1; j++)
    {
    if(A[j*3+i*nElt1*3+dircover] > GLOBALAmaxz)    GLOBALAmaxz=A[j*3+i*nElt1*3+dircover];
    else if(A[j*3+i*nElt1*3+dircover] < GLOBALAminz)    GLOBALAminz=A[j*3+i*nElt1*3+dircover];
    }
  }
  
  MIDAz=(GLOBALAmaxz+GLOBALAminz)/2.0;
  
  MAXAz=MIDAz+(bondist/2.0);
  MINAz=MIDAz-(bondist/2.0);
     
  if(rdf_between == "yes") {MAXAz=maxtop; MINAz=mintop; MIDAz=(MAXAz+MINAz)/2.0;bondist=MAXAz-MINAz;}
  
  hipMalloc((void **)&dev_exch1,sizeof(int)*nElt1*(nstruct/skip));
  hipMalloc((void **)&dev_exch2,sizeof(int)*nElt2*(nstruct/skip));
  hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
  hipMemcpy(dev_exch2,exch2,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyHostToDevice);
  hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
  hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
  
  
  if(rdf_for=="int")
  {
    
  cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;

//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A1,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A2,sizeof(float)*nElt2*(nstruct/skip)*3);
    

//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A1,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);


//dim3 struct to define elements of the execution configuration

    dim3 dimBlock(10,10,10);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(count_metal+dimBlock.y-1)/dimBlock.y,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

  
    
    
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGrid,dimBlock>>>(dev_A1,dev_B,dev_exch1,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid,dimBlock>>>(dev_A1,dev_B,dev_exch1,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A1);

    dim3 dimBlock2(10,10,10);
    dim3 dimGrid2((nElt2+dimBlock2.x-1)/dimBlock2.x,(count_metal+dimBlock2.y-1)/dimBlock2.y,((nstruct/skip)+dimBlock2.z-1)/dimBlock2.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    hipMemcpy(dev_A2,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGrid2,dimBlock2>>>(dev_A2,dev_B,dev_exch2,bondist,nElt2,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid2,dimBlock2>>>(dev_A2,dev_B,dev_exch2,bondist,nElt2,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch2,dev_exch2,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A2);
     hipFree(dev_B);
    //hipFree(dev_exch);
     
     
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch1[i+l*nElt1]==0) Abulknum[l]++;
        }
      for(int i=0; i< nElt2; i++)
        {
          if (exch2[i+l*nElt2]>0) {exch2[i+l*nElt2]=1; Bintnum[l]++;}
          else if (exch2[i+l*nElt2]==0) Bbulknum[l]++;
        }
      }
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_exch2,exch2,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyHostToDevice);

    
      whichwater=1;
  }
  
    
  else if(rdf_for=="bulk")
  {
    
//Allocate memory in GPU device


    hipMalloc((void **)&dev_A1,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A2,sizeof(float)*nElt2*(nstruct/skip)*3);

    
//Copy data from host to device

    hipMemcpy(dev_A1,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    
    //dim3 struct to define elements of the execution configuration

    dim3 dimBlock(32,1,32);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,1,((nstruct/skip)+dimBlock.z-1)/dimBlock.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

  
    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGrid,dimBlock>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid,dimBlock>>>(dev_A1,dev_exch1,nElt1,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    hipMemcpy(exch1,dev_exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A1);

     hipMemcpy(dev_A2,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
     
    dim3 dimBlock2(32,1,32);
    dim3 dimGrid2((nElt2+dimBlock2.x-1)/dimBlock2.x,1,((nstruct/skip)+dimBlock2.z-1)/dimBlock2.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    if (cell_type == "orthorhombic")
    {
    covermat<<<dimGrid2,dimBlock2>>>(dev_A2,dev_exch2,nElt2,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGrid2,dimBlock2>>>(dev_A2,dev_exch2,nElt2,(nstruct/skip),MINAz,MAXAz,dircover);
    }
    hipMemcpy(exch2,dev_exch2,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A2);
    //hipFree(dev_exch);
     
    
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch1[i+l*nElt1]>0) {exch1[i+l*nElt1]=1; Abulknum[l]++;}
          else if (exch1[i+l*nElt1]==0) Aintnum[l]++;
        }
      for(int i=0; i< nElt2; i++)
        {
          if (exch2[i+l*nElt2]>0) {exch2[i+l*nElt2]=1; Bbulknum[l]++;}
          else if (exch2[i+l*nElt2]==0) Bintnum[l]++;
        }
      }
    hipMemcpy(dev_exch1,exch1,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_exch2,exch2,sizeof(int)*nElt2*(nstruct/skip),hipMemcpyHostToDevice);

      
      whichwater=1;
    
  }

  
  if(rdf_for=="all"){whichwater=2;}
  
pi=3.14159;
    rdf= (float *) malloc (sizeof(float)*nElt1*bin*(nstruct/skip));
// Initialize distance matrix and histogram matrix
 for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
      {
        for(int j=0; j< bin; j++)
        {
          rdf[j+i*bin+l*nElt1*bin]=0.0;
        }
      }
    }

//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_rdf,sizeof(float)*nElt1*bin*(nstruct/skip));
    double evvalo = double(((nElt1+nElt2)/1000.0*(nstruct/skip)/1000.0*3)*4/1000.0);
    cout << evvalo  \
         << "Gbs of memory needed to store coordinates" <<endl;
             evvalo = double(nElt1/1000.0)*double(nstruct/skip)*double(bin/1000.0)*(4.0/1000.0);
    std::cout << evvalo  \
         << "Gbs of memory needed to store distance matrix" <<endl;	 
    
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_rdf,rdf,sizeof(float)*nElt1*bin*(nstruct/skip),hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration
      


    //bin_rad=(xvec/2)/float(bin);
    bin_rad = rdf_max_rad/float(bin);
    dim3 dimBlockrdf(10,10,10);
    dim3 dimGridrdf((nElt1+dimBlockrdf.x-1)/dimBlockrdf.x,(nElt2+dimBlockrdf.y-1)/dimBlockrdf.y,((nstruct/skip)+dimBlockrdf.z-1)/dimBlockrdf.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands
    
    
    
 //rdf kernels are different for int, bulk and all. For all, usual rdf is done and it makes no sense to compute this for interfacial systems.
 //For bulk, a mid point is defined and a box is cut out from the middle and for all atoms
 //in the box, every other atom within the bondist is correlarted.
 //For int, a small region from the surface, effectively the first layer is chosen.
 //and a hemisphere is sampled for each interface, giving the final rdf.
    if(cell_type == "orthorhombic")
    {
      if(rdf_for =="int")
      {
	distmat<<<dimGridrdf,dimBlockrdf>>>(dev_A,dev_B,dev_rdf,dev_exch1,dev_exch2,bin_rad,bin,nElt1,nElt2,(nstruct/skip),dev_lattice,rdf_max_rad,whichwater,MIDAz,dircover);
      }
      else
    distmat<<<dimGridrdf,dimBlockrdf>>>(dev_A,dev_B,dev_rdf,dev_exch1,dev_exch2,bin_rad,bin,nElt1,nElt2,(nstruct/skip),dev_lattice,rdf_max_rad,whichwater);
    }
    else if(cell_type == "monoclinic")
    {
      if(rdf_for =="int")
      {
	distmatmono<<<dimGridrdf,dimBlockrdf>>>(dev_A,dev_B,dev_rdf,dev_exch1,dev_exch2,bin_rad,bin,nElt1,nElt2,(nstruct/skip),dev_lattice,rdf_max_rad,whichwater,MIDAz,dircover);
      }
      else
    distmatmono<<<dimGridrdf,dimBlockrdf>>>(dev_A,dev_B,dev_rdf,dev_exch1,dev_exch2,bin_rad,bin,nElt1,nElt2,(nstruct/skip),dev_lattice,rdf_max_rad,whichwater);
    }
    
    hipMemcpy(rdf,dev_rdf,sizeof(float)*nElt1*bin*(nstruct/skip),hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_exch1);
    hipFree(dev_exch2);
    hipFree(dev_lattice);
    
    
    rdfavgf=(float *) malloc (sizeof(float)*bin*(nstruct/skip));
    for(int i=0; i<(nstruct/skip); i++)
    {
      for(int j =0; j<bin;j++)
      {
        rdfavgf[j+i*bin]=0.0;
      }
    }
    hipMalloc((void **)&dev_rdfavgf,sizeof(float)*bin*(nstruct/skip));
    hipMemcpy(dev_rdfavgf,rdfavgf,sizeof(float)*bin*(nstruct/skip),hipMemcpyHostToDevice);

    dim3 dimBlockf(32,32,1);
    dim3 dimGridf((bin+dimBlockf.x-1)/dimBlockf.x,((nstruct/skip)+dimBlockf.y-1)/dimBlockf.y,1);
    reducef<<<dimGridf,dimBlockf>>>(dev_rdf,dev_rdfavgf,bin,(nstruct/skip),nElt1);
    hipMemcpy(rdfavgf,dev_rdfavgf,sizeof(float)*bin*(nstruct/skip),hipMemcpyDeviceToHost);
    hipFree(dev_rdf);

    rdfavg=(float *) malloc (sizeof(float)*bin);
    for(int j =0; j<bin;j++)
    {
      rdfavg[j]=0.0;
    }
    hipMalloc((void **)&dev_rdfavg,sizeof(float)*bin);
    hipMemcpy(dev_rdfavg,rdfavg,sizeof(float)*bin,hipMemcpyHostToDevice);

    dim3 dimBlocks(1024,1,1);
    dim3 dimGrids((bin+dimBlocks.x-1)/dimBlocks.x,1,1);
    reduce<<<dimGrids,dimBlocks>>>(dev_rdfavgf,dev_rdfavg,bin,(nstruct/skip));

    hipMemcpy(rdfavg,dev_rdfavg,sizeof(float)*bin,hipMemcpyDeviceToHost);
    hipFree(dev_rdfavgf);
    hipFree(dev_rdfavg);

    for(int k=0; k< bin; k++)
        {
          cout<<rdfavg[k]<<" ";
        }
        cout<<endl;
    
    xvec=0;yvec=0;zvec=0;
    for(int i=0; i<(nstruct/skip);i++)
    {
        xvec+=lattice[i*6];
        yvec+=lattice[1+i*6];
        zvec+=lattice[2+i*6];
    }
    
    xvec /= (nstruct/skip);
    yvec /= (nstruct/skip);
    zvec /= (nstruct/skip);
    
    float AVGAintnum=0.0, AVGAbulknum=0.0;
    float AVGBintnum=0.0, AVGBbulknum=0.0;
    for(int l=0; l<nstruct/skip; l++)
    {
        AVGAintnum+=Aintnum[l];
        AVGAbulknum+=Abulknum[l];
	AVGBintnum+=Bintnum[l];
        AVGBbulknum+=Bbulknum[l];
    }
AVGAintnum/=float(nstruct/skip);
AVGAbulknum/=float(nstruct/skip);
AVGBintnum/=float(nstruct/skip);
AVGBbulknum/=float(nstruct/skip);
cout <<AVGAintnum<<" "<<AVGAbulknum<<" "<<AVGBintnum<<" "<<AVGBbulknum<<endl;
// printing the rdf data to be visualized
    rho = 0.0;
    
    float cellvolume,intvolume,bulkvolume;
    
    if(cell_type=="orthorhombic")
    {
      cellvolume = xvec*yvec*zvec;
    if(vacuum.at(0) =='z') {intvolume = xvec*yvec*(MIDAz-GLOBALAminz);bulkvolume = xvec*yvec*bondist;}
    else if(vacuum.at(0) =='x') {intvolume = zvec*yvec*(MIDAz-GLOBALAminz);bulkvolume = zvec*yvec*bondist;}
    else if(vacuum.at(0) =='y') {intvolume = xvec*zvec*(MIDAz-GLOBALAminz);bulkvolume = xvec*zvec*bondist;}
    //intvolume = xvec*yvec*(MIDAz-GLOBALAminz);
    //intvolume = xvec*yvec*bondist; // effectively considered as only one interface made of two hemispheres from each interfacial region 
    //bulkvolume = xvec*yvec*bondist;
    }
    else if(cell_type=="monoclinic")
    {
    cellvolume= xvec*xvec*zvec*0.866025403; //*sin(60) for rhombohedral 
    intvolume = xvec*xvec*(MIDAz-GLOBALAminz)*0.866025403;
    //intvolume=xvec*xvec*bondist*0.866025403;
    bulkvolume=xvec*xvec*bondist*0.866025403;
    }
    
    
    if (Elt1 != rdf_metal_exclude && Elt2 != rdf_metal_exclude && rdf_for == "all")
    {
      rho = (nElt1*nElt2)/(cellvolume - (float(count_metal) * metal_atom_volume));
      cout<<(nElt1*nElt2)<<" "<<cellvolume<<" "<<(float(count_metal) * metal_atom_volume)<<" "<<rho<<endl;
    }
    else if (Elt1 != rdf_metal_exclude && Elt2 != rdf_metal_exclude && rdf_for == "int")
    {
      for(int l=0;l<(nstruct/skip);l++)
      {
      //rho += (Aintnum[l]*Bintnum[l])/(intvolume); // since, two interfacial regions
      rho += (Aintnum[l]*nElt2/2.0)/(intvolume);
      }
      rho /=float(nstruct/skip);
      rho/=2.0; // since only the surface area of hemisphere taken for normalizing
      cout<<(AVGAintnum*AVGBintnum)/intvolume<<" "<<rho<<endl;
    }
    else if (Elt1 != rdf_metal_exclude && Elt2 != rdf_metal_exclude && rdf_for == "bulk")
    {
      for(int l=0;l<(nstruct/skip);l++)
      {  
      rho += (Abulknum[l]*Bbulknum[l])/bulkvolume;
      }
      rho /=float(nstruct/skip);
      cout<<(AVGAbulknum*AVGBbulknum)<<" "<< bulkvolume <<" "<<(AVGAbulknum*AVGBbulknum)/bulkvolume<<" "<<rho<<endl;
    }
    else  rho=(nElt1*nElt2)/cellvolume ;
    
    FILE *rdfplot=fopen("rdf.data","wt");
    for(int i =1; i<bin;i++)
    {
      fprintf(rdfplot,"%f %f\n",((bin_rad*(i+1))+(bin_rad*(i)))/2,((rdfavg[i])/(nstruct/skip))/(rho*4*pi*(square(bin_rad*(i+1)))*bin_rad));
      //fprintf(rdfplot,"%f %f\n",((bin_rad*(i+1))+(bin_rad*(i)))/2,((rdfavg[i])/(nstruct/skip)));
      //cout<<i<<" "<<rho<<" "<<4*pi*(square(bin_rad*(i+1)))*bin_rad<<" "<<rho*4*pi*(square(bin_rad*(i+1)))*bin_rad<<endl;
    }
    fclose(rdfplot);
  
}
