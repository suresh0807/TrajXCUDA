#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"
//float minx,maxx,minz,maxz,miny,maxy;
//float xrange,yrange,zrange;
//float xint,yint,zint;
//float *xtick,*ytick,*ztick,*dev_xtick,*dev_ytick,*dev_ztick;
//int *density, *dev_density;
void compute_densitygrid1()
{
  
    ////////////////////Initial setup////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  
    if(plot=="int")        { nElt1 = nA_int;}
    else if(plot=="bulk")  { nElt1 = nA_bulk;}
    
cout << type <<" is chosen"<<endl;
cout<<"densitygrid: " << nElt1 << Elt1 << " atoms are there in each structure"<<endl;
    printf("%f %f %f %f %f %f\n",minx,maxx,miny,maxy,minz,maxz);
    
    if(set_max_z == "yes")
{
  minz = set_minz;
  maxz = set_maxz;
}
    if(set_max_x == "yes")
{
  minx = set_minx;
  maxx = set_maxx;
}
    if(set_max_y == "yes")
{
  miny = set_miny;
  maxy = set_maxy;
}
    xrange=maxx-minx;
    yrange=maxy-miny;
    zrange=maxz-minz;

    xint = xrange/xsplit;
    yint = yrange/ysplit;
    zint = zrange/zsplit;

    xtick=(float*) malloc (sizeof(float)*xsplit*2);
    ytick=(float*) malloc (sizeof(float)*ysplit*2);
    ztick=(float*) malloc (sizeof(float)*zsplit*2);
    for(int i=0;i<xsplit;i++)
    {
      xtick[i*2]=minx+(i*xint);
      xtick[i*2+1]=minx+((i+1)*xint);
    }
    for(int i=0;i<ysplit;i++)
    {
      ytick[i*2]=miny+(i*yint);
      ytick[i*2+1]=miny+((i+1)*yint);
    }
    for(int i=0;i<zsplit;i++)
    {
      ztick[i*2]=minz+(i*zint);
      ztick[i*2+1]=minz+((i+1)*zint);
    }
    
    ofstream plotgnu;
    plotgnu.open("plot.gnu");
    
  if(type=="density-top") 
  {
    if(order=="YZX") {
      ztick[0]=mintop;
      ztick[1]=maxtop;      
    }
     if(order=="XYZ") {
      ytick[0]=mintop;
      ytick[1]=maxtop;      
    }
    if(order=="ZXY") {
      xtick[0]=mintop;
      xtick[1]=maxtop;      
    }
    plotgnu.precision(4);    
    
    plotgnu<<"set view map"<<endl\

<<"set palette defined (0  0.0 0.0 0.5, 1  0.0 0.0 1.0, \
                     2  0.0 0.5 1.0, \
                     3  0.0 1.0 1.0, \
                     4  0.5 1.0 0.5, \
                     5  1.0 1.0 0.0, \
                     6  1.0 0.5 0.0, \
                     7  1.0 0.0 0.0, \
                     8  0.5 0.0 0.0 )"<<endl\

<<"set pm3d interpolate 20,20"<<endl\
//<<"set size ratio @@ratio@@"<<endl\

<<"unset key"<<endl\
<<"set contour base"<<endl\
<<"set cntrparam bspline"<<endl\
<<"set cntrparam levels 5"<<endl;
  }

  FILE *xyztick = fopen("xyztick.dat","wt");
    for(int i=0;i<xsplit;i++)
    {
      fprintf(xyztick,"%f %f \n",xtick[i*2],xtick[i*2+1]);
    }
      fprintf(xyztick," \n");
    for(int i=0;i<ysplit;i++)
    {
      fprintf(xyztick,"%f %f \n",ytick[i*2],ytick[i*2+1]);    
    }
      fprintf(xyztick," \n");
    for(int i=0;i<zsplit;i++)
    {
      fprintf(xyztick,"%f %f \n",ztick[i*2],ztick[i*2+1]);
    }
    fprintf(xyztick," \n");
    if(order=="YZX")  {fprintf(xyztick,"%f %f %f %f %f\n",xtick[0],xtick[(xsplit-1)*2+1],ytick[0],ytick[(ysplit-1)*2+1],xtick[(xsplit-1)*2+1]/ytick[(ysplit-1)*2+1]); 
                       plotgnu<<"set size ratio "<<xtick[(xsplit-1)*2+1]/ytick[(ysplit-1)*2+1]<<endl\
                       <<"set xrange ["<< xsplit-(xsplit/10)<<":"<< xsplit*2+(xsplit/10)<<"]"<<endl\
                       <<"set yrange ["<< ysplit-(ysplit/10)<<":"<< ysplit*2+(ysplit/10)<<"]"<<endl\
                       <<"set xlabel \"Y-direction\" font 'Verdana,18' offset 0,-1" <<endl\
                       <<"set ylabel \"X-direction\" font 'Verdana,18' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,18' " <<endl\
                       <<"set ytic  font 'Verdana,18' " <<endl\
                        <<"set cbtic  font 'Verdana,18' " <<endl\
                       <<"set cblabel \"Number of atoms\" font 'Verdana,18' offset 2,0" <<endl\
                       <<"set format x \"%2.1f\";set ytics (\"0.00\" "<< xsplit<<", \""<<xtick[(xsplit-1)*2+1]/2<<"\" "<<(xsplit+(xsplit*2))/2<<", \""<< xtick[(xsplit-1)*2+1]<<"\" "<< (xsplit*2)<<" )"<<endl\
                       <<"set format y \"%2.1f\";set xtics (\"0.00\" "<< ysplit<<", \""<<ytick[(ysplit-1)*2+1]/2<<"\" "<<(ysplit+(ysplit*2))/2<<", \""<< ytick[(ysplit-1)*2+1]<<"\" "<< (ysplit*2)<<" )"<<endl;}
                       
    else if(order=="XYZ") { fprintf(xyztick,"%f %f \n",ztick[0],ztick[(zsplit-1)*2+1],xtick[0],xtick[(xsplit-1)*2+1],ztick[(zsplit-1)*2+1]/xtick[(xsplit-1)*2+1]);
                       plotgnu<<"set size ratio "<<ztick[(zsplit-1)*2+1]/xtick[(xsplit-1)*2+1]<<endl\
                       <<"set xrange ["<< xsplit-(xsplit/10)<<":"<< xsplit*2+(xsplit/10)<<"]"<<endl\
                       <<"set zrange ["<< zsplit-(zsplit/10)<<":"<< zsplit*2+(zsplit/10)<<"]"<<endl\
                       <<"set xlabel \"Z-direction\" font 'Verdana,18' offset 0,-1" <<endl\
                       <<"set zlabel \"X-direction\" font 'Verdana,18' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,18' " <<endl\
                       <<"set ytic  font 'Verdana,18' " <<endl\
                        <<"set cbtic  font 'Verdana,18' " <<endl\
                       <<"set cblabel \"Number of atoms\" font 'Verdana,18' offset 2,0" <<endl\
                       <<"set format x \"%2.1f\";set ytics (\"0.00\" "<< zsplit<<", \""<<ztick[(zsplit-1)*2+1]/2<<"\" "<<(zsplit+(zsplit*2))/2<<", \""<< ztick[(zsplit-1)*2+1]<<"\" "<< (zsplit*2)<<" )"<<endl\
                       <<"set format y \"%2.1f\";set xtics (\"0.00\" "<< xsplit<<", \""<<xtick[(xsplit-1)*2+1]/2<<"\" "<<(xsplit+(xsplit*2))/2<<", \""<< xtick[(xsplit-1)*2+1]<<"\" "<< (xsplit*2)<<" )"<<endl;}
                       
    else if(order=="ZXY") { fprintf(xyztick,"%f %f \n",ytick[0],ytick[(ysplit-1)*2+1],ztick[0],ztick[(zsplit-1)*2+1],ytick[(ysplit-1)*2+1]/ztick[(zsplit-1)*2+1]);
                       plotgnu<<"set size ratio "<<ytick[(ysplit-1)*2+1]/ztick[(zsplit-1)*2+1]<<endl\
                       <<"set zrange ["<< zsplit-(zsplit/10)<<":"<< zsplit*2+(zsplit/10)<<"]"<<endl\
                       <<"set yrange ["<< ysplit-(ysplit/10)<<":"<< ysplit*2+(ysplit/10)<<"]"<<endl\
                       <<"set zlabel \"Y-direction\" font 'Verdana,18' offset 0,-1" <<endl\
                       <<"set ylabel \"Z-direction\" font 'Verdana,18' offset -2,0" <<endl\
                       <<"set xtic  font 'Verdana,18' " <<endl\
                       <<"set ytic  font 'Verdana,18' " <<endl\
                       <<"set cbtic  font 'Verdana,18' " <<endl\
                       <<"set cblabel \"Number of atoms\" font 'Verdana,18' offset 2,0" <<endl\
                       <<"set format x \"%2.1f\";set ytics (\"0.00\" "<< ysplit<<", \""<<ytick[(ysplit-1)*2+1]/2<<"\" "<<(ysplit+(ysplit*2))/2<<", \""<< ytick[(ysplit-1)*2+1]<<"\" "<< (ysplit*2)<<" )"<<endl\
                       <<"set format y \"%2.1f\";set xtics (\"0.00\" "<< zsplit<<", \""<<ztick[(zsplit-1)*2+1]/2<<"\" "<<(zsplit+(zsplit*2))/2<<", \""<< ztick[(zsplit-1)*2+1]<<"\" "<< (zsplit*2)<<" )"<<endl;}
fclose(xyztick);
plotgnu<<"set terminal postscript solid eps enhanced color font 'Verdana,18'"<<endl\
       <<"set output 'Plot.eps'"<<endl\
       <<"sp 'extended.out' i 0 matrix w pm3d , 'teter.dat' u 1:2:(1) w p pt 6 ps 5.5 lc rgb \"white\""<<endl;
plotgnu.close();
int *densityf, *dev_densityf;
    density=(int *) malloc (sizeof(int)*xsplit*ysplit*zsplit);
    densityf=(int *) malloc (sizeof(int)*xsplit*ysplit*zsplit);
    for(int i=0;i<xsplit;i++)
    {
      for(int j=0;j<ysplit;j++)
      {
        for(int k=0;k<zsplit;k++)
        {
          density[k+j*zsplit+i*ysplit*zsplit]=0;
	  densityf[k+j*zsplit+i*ysplit*zsplit]=0;
        }
      }
    }
///*

///////////////////////////////////////////Initial setup done/////////////////////////////////////////////////////////////////////////////////////////////////


///////////////////////////////////////////Special atom selection between 2 distances/////////////////////////////////////////////////////////////////////////

if(choose_atoms=="yes")
{
cout << Elt1 <<" within "<< minbondist << " and " << maxbondist<<" angstrom of "<< metal_species<<endl;
  

  int dirn;
  if(choose_dirn=="z") dirn=2;
  else if(choose_dirn=="x") dirn=0;
  else if(choose_dirn=="y") dirn=1;
  
  float surf,avgsurf=0.0;
  int surfatom;
  for(int i=0 ; i<(nstruct/skip); i++)
  {surfatom=0;surf=0.0;
  for(int j=0; j<count_metal;j++)
  {
    if(METAL[j*3+i*count_metal*3+dirn] > mintop && METAL[j*3+i*count_metal*3+dirn] < maxtop)
    {
    surf+=METAL[j*3+i*count_metal*3+dirn];
    surfatom++;
    }
  }
  surf/=surfatom;
  avgsurf+=surf;
  }
  
  avgsurf/=(nstruct/skip);
  
 cout<<"Average surface atoms position in "<<choose_dirn<<" is "<<avgsurf<<endl;
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
  
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

 
//Allocate memory in GPU device

    
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);
    }
    else if (cell_type == "monoclinic")
    {
    covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,minbondist,maxbondist,nElt1,count_metal,(nstruct/skip),dev_lattice,dirn,avgsurf,mintop,maxtop);
    }
    
    
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    
    
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    
    //ofstream exchout;
    //exchout.open("exchout.data");
    
    
    for(int l=0; l<nstruct/skip; l++)
      {
	//exchout<<l<<endl;
      for(int i=0; i< nElt1; i++)
        {
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
	  //exchout<<i<<" "<<exch[i+l*nElt1]<<endl;
        }
        //exchout<<endl;
      }
      
      //exchout.close();
     hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice); 
     
     ofstream coverage;
     coverage.open("coverage.data");
     for(int l=0; l<nstruct/skip; l++)
      {
	coverage << l<< " "<< Aintnum[l]/surfatom<<endl;
      }
     coverage.close();
     
}      
   
///////////////////////////////////////////////special selection done - stored in exch///////////////////////////////////////////////////////////////////////////////////



///////////////////////////////////////////////////Density calculation starts/////////////////////////////////////////////////////////////////////////////////////////////

if(density_plot=="yes")
{

    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*(nstruct/skip)*nElt1*3);
    hipMalloc((void **)&dev_density,sizeof(int)*xsplit*ysplit*zsplit);
    hipMalloc((void **)&dev_xtick, sizeof(float)*xsplit*2);
    hipMalloc((void **)&dev_ytick, sizeof(float)*ysplit*2);
    hipMalloc((void **)&dev_ztick, sizeof(float)*zsplit*2);
    if(plot=="int")
    {
      hipMemcpy(dev_A,A_int,sizeof(float)*(nstruct/skip)*nA_int*3,hipMemcpyHostToDevice);
    }
    else if(plot=="bulk")
    {
      hipMemcpy(dev_A,A_bulk,sizeof(float)*(nstruct/skip)*nA_bulk*3,hipMemcpyHostToDevice);
    }
    else if(plot=="all")
    {
      hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    }
    hipMemcpy(dev_density,density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyHostToDevice);
    hipMemcpy(dev_xtick,xtick,sizeof(float)*xsplit*2,hipMemcpyHostToDevice);
    hipMemcpy(dev_ytick,ytick,sizeof(float)*ysplit*2,hipMemcpyHostToDevice);
    hipMemcpy(dev_ztick,ztick,sizeof(float)*zsplit*2,hipMemcpyHostToDevice);
    
    printf("%ld B needed\n",(((nstruct/skip)*nElt1*3+xsplit*ysplit*zsplit+xsplit*2+ysplit*2+zsplit*2)*4));
    dim3 dimBlock(32,32,1);
    //dim3 dimBlock(1,1,1);
    dim3 dimGrid(((nstruct/skip)+dimBlock.x-1)/dimBlock.x,(nElt1+dimBlock.y-1)/dimBlock.y,1);
    //dim3 dimGrid(nstruct,nElt1,1);
    if(choose_atoms=="yes")
    {
    density_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,dev_exch,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);
    hipFree(dev_exch);
    }
    else
    {
    density_calc<<<dimGrid,dimBlock>>>(dev_A,dev_density,(nstruct/skip),nElt1,xsplit,ysplit,zsplit,dev_xtick,dev_ytick,dev_ztick);      
    }
    //hipMemcpy(density,dev_density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
     hipFree(dev_A);
}
/////////////////////////////////////////////////////Density calculation done//////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////Dipole orientation computation/////////////////////////////////////////////////////////////////////////////     
     
     
   if(dipole_orient=="yes" && type=="density-top")
   {
    
    int *h2onum;
  int *h2oneigh;
  float *h2oxyz;
  
  float *h2odistmat, *dev_h2odistmat;
  
  int OHsamples=4;
  
    h2onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    h2oneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    h2oxyz = (float *) malloc (sizeof(float)*15*nElt1*(nstruct/skip));  //Oxyz,H1xyz,H2xyz,Hmidpoint,dipole alignment with surface normal
    h2odistmat = (float *) malloc (sizeof(float)*nElt1*nElt2);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      h2onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      h2oneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      }
      for(int k =0 ;k<15;k++)
      {
      h2oxyz[j*15+i*nElt1*15+k]=0.0;
      }
    }
  }   
      
      
cout <<"gpu begins OH distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nElt2*(nstruct/skip)*3);
    hipMalloc((void **)&dev_h2odistmat,sizeof(float)*nElt1*nElt2);
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*nElt2*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
//dim3 struct to define elements of the execution configuration

    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(nElt2+dimBlock.y-1)/dimBlock.y,1);
    
     
    //ofstream neigh;
   // neigh.open("H2O.dat");
   
    
for(int i = 0; i<(nstruct/skip); i++)
{
  
  //neigh << i<<endl<<endl;
  
   for(int i1=0;i1<nElt1;i1++)
  {
    for(int j1=0; j1<nElt2; j1++)
    {
      h2odistmat[j1+i1*nElt2]=0.0;
    }
  }
  
  hipMemcpy(dev_h2odistmat,h2odistmat,sizeof(float)*nElt1*nElt2,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOH,nElt1,nElt2,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOH,nElt1,nElt2,dev_lattice,i);      
    }

hipMemcpy(h2odistmat,dev_h2odistmat,sizeof(float)*nElt1*nElt2,hipMemcpyDeviceToHost);
   
  
  for(int i1=0;i1<nElt1;i1++)
  {
    int k=0;
    //neigh << i1<<" ";
    for(int j1=0; j1<nElt2; j1++)
    {
      //neigh <<j1<<" "<<h2odistmat[j1+i1*nElt2]<<" ";
      if(h2odistmat[j1+i1*nElt2] !=0.0)
      {
	h2onum[i1+i*nElt1]++;
	h2oneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	k++;
      }
    }
    //neigh<<endl;
  }

}   
 
 //neigh.close();
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_h2odistmat);
    hipFree(dev_lattice);
    

cout <<"gpu ends"<<endl;
   
cout<<" done !!!"<<endl;    
  
    
//Hbondneigh contain the IDs of four possible oxygen neighbour atoms if the hydrogen chosen is participating in a hydrogen bond.
//Onum has the number of oxygen atoms within a specified distance (see input file).

 //cout<<"Write Hxyz.dat"<<endl;

  //ofstream watercount;
  //watercount.open("Hxyz.dat");
  
  
  float *angdist,*angdistH1,*angdistH2;
  
  angdist = (float *) malloc (sizeof(float)*180);
  angdistH1 = (float *) malloc (sizeof(float)*180);
  angdistH2 = (float *) malloc (sizeof(float)*180);
  for(int i=0; i<=180;i++)
  {
    angdist[i]=0.0;
    angdistH1[i]=0.0;
    angdistH2[i]=0.0;
  }
  
  float val;
for(int i=0; i<(nstruct/skip); i++)
{
 // cout<<i<<endl;
 for(int j=0; j<nElt1;j++)
 {
    if(order=="YZX") {
      val=A[j*3+i*nElt1*3+2];    
    }
    if(order=="XYZ") {
      val=A[j*3+i*nElt1*3+1];      
    }
    if(order=="ZXY") {
      val=A[j*3+i*nElt1*3];      
    }
   if(h2onum[j+i*nElt1]==2 && exch[j+i*nElt1]==1 &&val<maxtop && val>mintop)
   //if(h2onum[j+i*nElt1]==2)
   {
        //watercount << i <<" "<<j<<" ";
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+k]= A[j*3+i*nElt1*3+k];
     //watercount << h2oxyz[j*13+i*nElt1*13+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+3+k]= B[h2oneigh[j*OHsamples+i*nElt1*OHsamples]*3+i*nElt2*3+k];
     //watercount << h2oxyz[j*9+i*nElt1*9+3+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+6+k]= B[h2oneigh[j*OHsamples+i*nElt1*OHsamples+1]*3+i*nElt2*3+k];
     //watercount << h2oxyz[j*9+i*nElt1*9+6+k] <<" ";
     }
     
     float chk1,chk2,chk3,chk4,chk5,chk6;
     float a1,b1,c1,d1,e1,f1,g1,h1,i1;
     float latx,laty,latz,latxy,latxz,latyz;
     //check_together(h2oxyz[j*13+i*nElt1*13],h2oxyz[j*13+i*nElt1*13+1],h2oxyz[j*13+i*nElt1*13+2],\
                    h2oxyz[j*13+i*nElt1*13+3],h2oxyz[j*13+i*nElt1*13+4],h2oxyz[j*13+i*nElt1*13+5],\
                    h2oxyz[j*13+i*nElt1*13+6],h2oxyz[j*13+i*nElt1*13+7],h2oxyz[j*13+i*nElt1*13+8],\
                    lattice[i*6],lattice[i*6+1],lattice[i*6+2],lattice[i*6+3],lattice[i*9+4],lattice[i*6+5]);
     
     a1=h2oxyz[j*15+i*nElt1*15];
     b1=h2oxyz[j*15+i*nElt1*15+1];
     c1=h2oxyz[j*15+i*nElt1*15+2];
     d1=h2oxyz[j*15+i*nElt1*15+3];
     e1=h2oxyz[j*15+i*nElt1*15+4];
     f1=h2oxyz[j*15+i*nElt1*15+5];
     g1=h2oxyz[j*15+i*nElt1*15+6];
     h1=h2oxyz[j*15+i*nElt1*15+7];
     i1=h2oxyz[j*15+i*nElt1*15+8];
     latx=lattice[i*6];
     laty=lattice[i*6+1];
     latz=lattice[i*6+2];
     latxy=lattice[i*6+3];
     latxz=lattice[i*6+4];
     latyz=lattice[i*6+5];
     
chk1= (a1-d1);
chk2= (b1-e1);
chk3= (c1-f1);
chk4= (a1-g1);
chk5= (b1-h1);
chk6= (c1-i1);

if(cell_type == "orthorhombic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) e1=e1+laty; else e1=e1-laty;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
if(fabs(chk4) > latx/2.0) { if (chk4 >0) g1=g1+latx; else g1=g1-latx;}
if(fabs(chk5) > laty/2.0) { if (chk5 >0) h1=h1+laty; else h1=h1-laty;}
if(fabs(chk6) > latz/2.0) { if (chk6 >0) i1=i1+latz; else i1=i1-latz;}
}

else if(cell_type == "monoclinic")
{
if(fabs(chk2) > laty/2.0) { if (chk2 >0) {e1=e1+laty; d1=d1+latxy;} else {e1=e1-laty; d1=d1-latxy;}}
chk1= (a1-d1);
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
if(fabs(chk5) > laty/2.0) { if (chk5 >0) {h1=h1+laty; g1=g1+latxy;} else {h1=h1-laty; g1=g1-latxy;}}
chk4= (a1-g1);
if(fabs(chk4) > latx/2.0) { if (chk4 >0) g1=g1+latx; else g1=g1-latx;}
if(fabs(chk6) > latz/2.0) { if (chk6 >0) i1=i1+latz; else i1=i1-latz;}
}
     
     h2oxyz[j*15+i*nElt1*15+3]=d1;
     h2oxyz[j*15+i*nElt1*15+4]=e1;
     h2oxyz[j*15+i*nElt1*15+5]=f1;
     h2oxyz[j*15+i*nElt1*15+6]=g1;
     h2oxyz[j*15+i*nElt1*15+7]=h1;
     h2oxyz[j*15+i*nElt1*15+8]=i1;
     
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*15+i*nElt1*15+9+k]=(h2oxyz[j*15+i*nElt1*15+3+k]+h2oxyz[j*15+i*nElt1*15+6+k])/2.0;
     //watercount << h2oxyz[j*13+i*nElt1*13+9+k] <<" ";
     }
     //////////////////////////////angle between dipole and z axis 0, 0, 1(specific - be careful if using other directions//////////////////////////////////////////////////
     h2oxyz[j*15+i*nElt1*15+12]=angle(0,0,0,0,0,1,h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],h2oxyz[j*15+i*nElt1*15+9],h2oxyz[j*15+i*nElt1*15+9+1],h2oxyz[j*15+i*nElt1*15+9+2]);
     h2oxyz[j*15+i*nElt1*15+13]=angle(0,0,0,0,0,1,h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],h2oxyz[j*15+i*nElt1*15+3],h2oxyz[j*15+i*nElt1*15+9+4],h2oxyz[j*15+i*nElt1*15+9+5]);
     h2oxyz[j*15+i*nElt1*15+14]=angle(0,0,0,0,0,1,h2oxyz[j*15+i*nElt1*15],h2oxyz[j*15+i*nElt1*15+1],h2oxyz[j*15+i*nElt1*15+2],h2oxyz[j*15+i*nElt1*15+6],h2oxyz[j*15+i*nElt1*15+9+7],h2oxyz[j*15+i*nElt1*15+9+8]);
     //watercount << h2oxyz[j*13+i*nElt1*13+12]*(180.0/3.141);
     angdist[int(ceil(h2oxyz[j*15+i*nElt1*15+12]*(180.0/3.141)))]++;
     angdistH1[int(ceil(h2oxyz[j*15+i*nElt1*15+13]*(180.0/3.141)))]++;
     angdistH2[int(ceil(h2oxyz[j*15+i*nElt1*15+14]*(180.0/3.141)))]++;
     //watercount<<endl;
   }
   
  }
}
  //watercount.close();
  

  ofstream angldist;
  angldist.open("Dipole-orient.dat");
  angldist <<"T num num1 num2"<<endl;
   for(int i=0; i<=180;i++)
  {
    angldist << i<<" "<<angdist[i]/(nstruct/skip)<<" "<< angdistH1[i]/(nstruct/skip)<<""<< angdistH2[i]/(nstruct/skip)<<endl;
  }
  angldist.close();
  free(angdist);
  free(angdistH1);
  free(angdistH2);
  free(h2onum);
  free(h2oneigh);
  free(h2oxyz);
  free(h2odistmat);
   }
    
////////////////////////////////////////////////////end of dipole orientation calculation /////////////////////////////////////////////////////////////////////////////////
    
    
    
    
////////////////////////////////////////////////////atop orientation computation/////////////////////////////////////////////////////////////////////////////     
     
     
   if(atop_orient=="yes" && type=="density-top")
   {
    
  int *h2onum;
  int *h2oneigh;
  float *h2oxyz;
  
  float *h2odistmat, *dev_h2odistmat;
  
  int OHsamples=count_metal;
  
    h2onum = (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
    h2oneigh = (int *) malloc (sizeof(int)*OHsamples*nElt1*(nstruct/skip));
    h2oxyz = (float *) malloc (sizeof(float)*7*nElt1*(nstruct/skip));  //Oxyz,H1xyz,H2xyz,Hmidpoint,dipole alignment with surface normal
    h2odistmat = (float *) malloc (sizeof(float)*nElt1*count_metal);
    
  for(int i=0;i<nstruct/skip;i++)
  {
    for(int j=0; j<nElt1; j++)
    {
      h2onum[j+i*nElt1] =0;
      for(int k =0 ;k<OHsamples;k++)
      {
      h2oneigh[j*OHsamples+i*nElt1*OHsamples+k]=0;
      }
      for(int k =0 ;k<7;k++)
      {
      h2oxyz[j*7+i*nElt1*7+k]=0.0;
      }
    }
  }   
      
      
cout <<"gpu begins OH distance computation"<<endl;
//Allocate memory in GPU device

    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_h2odistmat,sizeof(float)*nElt1*count_metal);
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));
//Copy data from host to device

    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
//dim3 struct to define elements of the execution configuration

    dim3 dimBlock(32,32,1);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(count_metal+dimBlock.y-1)/dimBlock.y,1);
    
     // water molecule only if Cu-O distance is within this value
   // ofstream neigh;
   // neigh.open("H2O.dat");
   
    
for(int i = 0; i<(nstruct/skip); i++)
{
  
 // neigh << i<<endl<<endl;
  
   for(int i1=0;i1<nElt1;i1++)
  {
    for(int j1=0; j1<count_metal; j1++)
    {
      h2odistmat[j1+i1*count_metal]=0.0;
    }
  }
  
  hipMemcpy(dev_h2odistmat,h2odistmat,sizeof(float)*nElt1*count_metal,hipMemcpyHostToDevice);
//Cuda kernal execution for distance matrix with CUDA timing API commands
    if(cell_type == "orthorhombic")
    {
          Hbondmat<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOM,nElt1,count_metal,dev_lattice,i);
    }
    else if(cell_type == "monoclinic")
    {
          Hbondmatmono<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_h2odistmat,bondistOM,nElt1,count_metal,dev_lattice,i);      
    }

hipMemcpy(h2odistmat,dev_h2odistmat,sizeof(float)*nElt1*count_metal,hipMemcpyDeviceToHost);
   
   //cout<<count_metal<<endl;
   
  for(int i1=0;i1<nElt1;i1++)
  {
    int k=0;
   // neigh << i1<<" ";
    for(int j1=0; j1<count_metal; j1++)
    {
      //neigh <<j1<<" "<<h2odistmat[j1+i1*count_metal]<<" ";
      if(h2odistmat[j1+i1*count_metal] !=0.0)
      {
	h2onum[i1+i*nElt1]++;
	h2oneigh[i1*OHsamples+i*nElt1*OHsamples+k] = j1;
	k++;
      }
    }
   // neigh<<endl;
  }

}   
 
// neigh.close();
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_h2odistmat);
    hipFree(dev_lattice);
    

cout <<"gpu ends"<<endl;
   
cout<<" done !!!"<<endl;    
    

//Hbondneigh contain the IDs of four possible oxygen neighbour atoms if the hydrogen chosen is participating in a hydrogen bond.
//Onum has the number of oxygen atoms within a specified distance (see input file).

 //cout<<"Write Hxyz.dat"<<endl;

 // ofstream watercount;
 // watercount.open("Hxyz.dat");
  
  
  float *angdist;
  
  angdist = (float *) malloc (sizeof(float)*180);
  
  for(int i=0; i<=180;i++)
  {
    angdist[i]=0.0;
  }
  
  float val;
for(int i=0; i<(nstruct/skip); i++)
{
  //cout<<i<<endl;
 for(int j=0; j<nElt1;j++)
 {
    if(order=="YZX") {
      val=A[j*3+i*nElt1*3+2];    
    }
    if(order=="XYZ") {
      val=A[j*3+i*nElt1*3+1];      
    }
    if(order=="ZXY") {
      val=A[j*3+i*nElt1*3];      
    }
    //cout<<j<<endl;
   if(h2onum[j+i*nElt1]>0 && exch[j+i*nElt1]==1 && val < maxtop && val > mintop)
   {
     //watercount << i <<" "<<j<<" ";
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*7+i*nElt1*7+k]= A[j*3+i*nElt1*3+k];
     //watercount << h2oxyz[j*7+i*nElt1*7+k] <<" ";
     }
     for(int k=0; k<3;k++)
     {
     h2oxyz[j*7+i*nElt1*7+3+k]= METAL[h2oneigh[j*OHsamples+i*nElt1*OHsamples]*3+i*count_metal*3+k];
     //watercount << h2oxyz[j*7+i*nElt1*7+3+k] <<" ";
     }
     //check_together(h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2],\
                    h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                    lattice[i*6],lattice[i*6+1],lattice[i*6+2],lattice[i*6+3],lattice[i*9+4],lattice[i*6+5]);
                    
                    
     float chk1,chk2,chk3;
     float a1,b1,c1,d1,e1,f1;
     float latx,laty,latz,latxy,latxz,latyz;
     //check_together(h2oxyz[j*13+i*nElt1*13],h2oxyz[j*13+i*nElt1*13+1],h2oxyz[j*13+i*nElt1*13+2],\
                    h2oxyz[j*13+i*nElt1*13+3],h2oxyz[j*13+i*nElt1*13+4],h2oxyz[j*13+i*nElt1*13+5],\
                    h2oxyz[j*13+i*nElt1*13+6],h2oxyz[j*13+i*nElt1*13+7],h2oxyz[j*13+i*nElt1*13+8],\
                    lattice[i*6],lattice[i*6+1],lattice[i*6+2],lattice[i*6+3],lattice[i*9+4],lattice[i*6+5]);
     
     a1=h2oxyz[j*7+i*nElt1*7];
     b1=h2oxyz[j*7+i*nElt1*7+1];
     c1=h2oxyz[j*7+i*nElt1*7+2];
     d1=h2oxyz[j*7+i*nElt1*7+3];
     e1=h2oxyz[j*7+i*nElt1*7+4];
     f1=h2oxyz[j*7+i*nElt1*7+5];

     latx=lattice[i*6];
     laty=lattice[i*6+1];
     latz=lattice[i*6+2];
     latxy=lattice[i*6+3];
     latxz=lattice[i*6+4];
     latyz=lattice[i*6+5];
     
chk1= (a1-d1);
chk2= (b1-e1);
chk3= (c1-f1);


if(cell_type == "orthorhombic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) e1=e1+laty; else e1=e1-laty;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
}

else if(cell_type == "monoclinic")
{
if(fabs(chk2) > laty/2.0) { if (chk2 >0) {e1=e1+laty; d1=d1+latxy;} else {e1=e1-laty; d1=d1-latxy;}}
chk1= (a1-d1);
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d1=d1+latx; else d1=d1-latx;}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f1=f1+latz; else f1=f1-latz;}
}
     
     h2oxyz[j*7+i*nElt1*7+3]=d1;
     h2oxyz[j*7+i*nElt1*7+4]=e1;
     h2oxyz[j*7+i*nElt1*7+5]=f1;
                    
                    
                    
     //////////////////////////////angle between dipole and z axis (specific - be careful if using other directions//////////////////////////////////////////////////
     //h2oxyz[j*7+i*nElt1*7+6]=angle(h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5]+5.0,\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2]);
       h2oxyz[j*7+i*nElt1*7+6]=angle(0,0,0,0,0,1,\
                                   h2oxyz[j*7+i*nElt1*7+3],h2oxyz[j*7+i*nElt1*7+4],h2oxyz[j*7+i*nElt1*7+5],\
                                   h2oxyz[j*7+i*nElt1*7],h2oxyz[j*7+i*nElt1*7+1],h2oxyz[j*7+i*nElt1*7+2]);
     //watercount << h2oxyz[j*7+i*nElt1*7+3] <<" "<< h2oxyz[j*7+i*nElt1*7+3+1] <<" "<< h2oxyz[j*7+i*nElt1*7+3+2] <<" ";
     //watercount << h2oxyz[j*7+i*nElt1*7+6]*(180.0/3.141);
     angdist[int(floor(h2oxyz[j*7+i*nElt1*7+6]*(180.0/3.141)))]++;
    // watercount<<endl;
   }
   
  }
}
  //watercount.close();
  

  ofstream angldist;
  angldist.open("atop-orient.dat");
  angldist <<"T num"<<endl;
   for(int i=0; i<=180;i++)
  {
    angldist << i<<" "<<angdist[i]/(nstruct/skip)<<endl;
  }
  angldist.close();
  free(angdist);
  free(h2onum);
  free(h2oneigh);
  free(h2oxyz);
  free(h2odistmat);     
     
     
   }
    
////////////////////////////////////////////////////end of atop orientation calculation /////////////////////////////////////////////////////////////////////////////////
    
    
    
    
    
    
       
    
    
    
    
    
    
    
    if(filter_density == "yes")
    {
    hipMalloc((void **)&dev_densityf,sizeof(int)*xsplit*ysplit*zsplit);
    hipMemcpy(dev_densityf,densityf,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyHostToDevice);
    dim3 dimBlockee(10,10,10);
    dim3 dimGridee((xsplit+dimBlockee.x-1)/dimBlockee.x,(ysplit+dimBlockee.y-1)/dimBlockee.y,(zsplit+dimBlockee.z-1)/dimBlockee.y);
    if(order=="XYZ") {
    density_filter<<<dimGridee,dimBlockee>>>(dev_density, dev_densityf,ysplit,zsplit,xsplit);
    }
    else if(order=="ZXY") {
    density_filter<<<dimGridee,dimBlockee>>>(dev_density, dev_densityf,xsplit,ysplit,zsplit);
    }
    else if(order=="YZX") {
    density_filter<<<dimGridee,dimBlockee>>>(dev_density, dev_densityf,zsplit,xsplit,ysplit);
    }
    hipMemcpy(density,dev_densityf,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
    }
    else
    {
      hipMemcpy(density,dev_density,sizeof(int)*xsplit*ysplit*zsplit,hipMemcpyDeviceToHost);
    }
FILE *densityfile=fopen("out","wt");

if(order=="XYZ") {
  
for(int l=0;l<ysplit;l++)
{
  //fprintf(densityfile,"%d %d\n",zsplit,xsplit);
  fprintf(densityfile,"#%d \n",l);
for(int m=0;m<zsplit;m++)
{
for(int k=0;k<xsplit;k++)
{
    if(type=="density-top") 
  {
fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
  }
  else
  {
    fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]/nElt1);
  }
}
fprintf(densityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}
}
if(order=="ZXY") {

for(int k=0;k<xsplit;k++)
{
  fprintf(densityfile,"#%d \n",k);
for(int l=0;l<ysplit;l++)
{
for(int m=0;m<zsplit;m++)
{
      if(type=="density-top") 
  {
fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
  }
    else
  {
    fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]/nElt1);
  }
}
fprintf(densityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}
}
if(order=="YZX") {

for(int m=0;m<zsplit;m++)
{
  fprintf(densityfile,"#%d \n",m);
for(int k=0;k<xsplit;k++)
{
for(int l=0;l<ysplit;l++)
{
if(type=="density-top") 
  {
fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]);
  }
else
  {
fprintf(densityfile,"%d ",density[m+l*zsplit+k*ysplit*zsplit]/nElt1);
  }
}
fprintf(densityfile,"\n");
}
//fprintf(densityfile,"5\n");
//fprintf(densityfile,"1 10.\n");
//fprintf(densityfile,"2 15.\n");
//fprintf(densityfile,"3 20.\n");
//fprintf(densityfile,"4 25.\n");
//fprintf(densityfile,"5 30.\n");
//fprintf(densityfile,"6 \n");
}  
}
fclose(densityfile);

hipFree(dev_density);
hipFree(dev_densityf);
if (type=="density" || type=="densityvelocity");
{
hipFree(dev_A);
hipFree(dev_xtick);
hipFree(dev_ytick);
hipFree(dev_ztick);
}

if(type=="density-top")
{
  system("paste out out out | tail -n +2 > ext-out ; cat ext-out ext-out ext-out > extended.out ");
}

}
