#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################



#include "cudatools.cuh"

__global__ void HBAF_calc(float *A,float *B,float *C,int SD_store,int initHBnum, int i, int j,int origins,int skips)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  //int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float z;
  if(rowid < initHBnum   && widid < SD_store)
  {

      z=A[rowid]*B[rowid+(widid*initHBnum)];
      
//     printf("%f \n",C[rowid+(j*nElt1)+(widid*nElt1*SD_store)]);
//    if(D[rowid+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater && D[rowid+(j*nElt1)+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater)
//    {
      C[rowid+widid*initHBnum] += z ;
      //atomicAdd(C[rowid+widid*initHBnum],z);
//    }
//    else if(whichwater==2)
 //   {
//     C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ; 
//    }
  }
}
