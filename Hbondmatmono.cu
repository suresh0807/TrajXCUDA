#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void Hbondmatmono(float *x, float*y, int *a, int *b, float bondist, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    if (z < bondist)
    {
       atomicAdd(&a[rowid+widid*nElt1],1);
       atomicCAS(&b[rowid*2+widid*nElt1*2],-1,colid);
       atomicCAS(&b[rowid*2+widid*nElt1*2+1],-1,colid);
       atomicCAS(&b[rowid*2+widid*nElt1*2+1],b[rowid*2+widid*nElt1*2],colid);
     }
    }
}

__global__ void Hbondmatmono(float *x, float*y, int *a, int *b, float bondist, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec, float *latti)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chkx,chky,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    if (z < bondist)
    {
       atomicAdd(&a[rowid+widid*nElt1],1);
       atomicCAS(&b[rowid*2+widid*nElt1*2],-1,colid);
       atomicCAS(&b[rowid*2+widid*nElt1*2+1],-1,colid);
       atomicCAS(&b[rowid*2+widid*nElt1*2+1],b[rowid*2+widid*nElt1*2],colid);
     }
    }
}

__global__ void Hbondmatmono(float *x, float*y, float *c,float bondist, int nElt1, int nElt2, float *latti,int widid)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  //int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float xvec,yvec,zvec,yxvec;
  float chkx,chky,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 )
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    if (z < bondist)
    {
       c[colid+rowid*nElt2] = z;
    }
   }
}

__global__ void Hbondmatmono(float *x, float*y, float *c,float bondist, int nElt1, int nElt2, float *latti,int widid,int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  //int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float xvec,yvec,zvec,yxvec,chkrr;
  float chkx,chky,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 )
  {
    z=0.0;chkrr=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkrr=fabs(x[rowid*3+widid*nElt1*3+dirn]-y[colid*3+widid*nElt2*3+dirn]);
    if (chkrr <= bondist)
    {
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
    
    c[colid+rowid*nElt2] = z;
        }
   }
}


__global__ void Hbondmatmono(float *x, float*y, int*exch, float *c,float bondist, int nElt1, int nElt2, float *latti,int widid)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  //int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float xvec,yvec,zvec,yxvec;
  float chkx,chky,chkz;
  float z;
  if(rowid < nElt1 && colid < nElt2 &&exch[rowid+widid*nElt1]>0)
  {
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    yxvec=latti[3+widid*6];
    chkx=fabs(x[rowid*3+widid*nElt1*3]-y[colid*3+widid*nElt2*3]);
    chky=fabs(x[rowid*3+widid*nElt1*3+1]-y[colid*3+widid*nElt2*3+1]);
    chkz=fabs(x[rowid*3+widid*nElt1*3+2]-y[colid*3+widid*nElt2*3+2]);
    if(chky > yvec/2) {chky = chky - yvec; chkx = chkx - yxvec;}
    if(chkx > xvec/2) {chkx = chkx - xvec;}
    if(chkz > zvec/2) {chkz = chkz - zvec;}
    z=sqrt((chkx*chkx) + (chky*chky) + (chkz*chkz));
   
    if (z < bondist)
    {
       c[colid+rowid*nElt2] = z;
     }
   }
}