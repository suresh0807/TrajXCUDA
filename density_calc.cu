#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




//the matrix fills Z-(vacuum) axis first and then Y and then x at last.

#include "cudatools.cuh"

__global__ void density_calc(float *A,int *density,int nstruct,int natoms,int xsplit,int ysplit, int zsplit, float *xtick,float *ytick, float *ztick)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  if(rowid < nstruct && colid <natoms)
  {
    for(int k=0;k<xsplit;k++)
    {
      for(int l=0;l<ysplit;l++)
      {
        for(int m=0;m<zsplit;m++)
        {
        if(A[colid*3+rowid*natoms*3] >= xtick[k*2] && A[colid*3+rowid*natoms*3] < xtick[k*2+1] && A[colid*3+(rowid*natoms*3)+1] >= ytick[l*2] && A[colid*3+(rowid*natoms*3)+1] < ytick[l*2+1] && A[colid*3+(rowid*natoms*3)+2] >= ztick[m*2] && A[colid*3+(rowid*natoms*3)+2] < ztick[m*2+1])
          {
            atomicAdd(&density[m+l*zsplit+k*ysplit*zsplit],1);
	    break;
          }
           
        }
         
      }
      
    }
    }
}

__global__ void density_calc(float *A,int *density,int *exch,int nstruct,int natoms,int xsplit,int ysplit, int zsplit, float *xtick,float *ytick, float *ztick)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  if(rowid < nstruct && colid <natoms)
  {
    for(int k=0;k<xsplit;k++)
    {
      for(int l=0;l<ysplit;l++)
      {
        for(int m=0;m<zsplit;m++)
        {
        if(A[colid*3+rowid*natoms*3] >= xtick[k*2] && A[colid*3+rowid*natoms*3] < xtick[k*2+1] && A[colid*3+(rowid*natoms*3)+1] >= ytick[l*2] && A[colid*3+(rowid*natoms*3)+1] < ytick[l*2+1] && A[colid*3+(rowid*natoms*3)+2] >= ztick[m*2] && A[colid*3+(rowid*natoms*3)+2] < ztick[m*2+1] && exch[colid+rowid*natoms]==1)
          {
            atomicAdd(&density[m+l*zsplit+k*ysplit*zsplit],1);
	    break;
          }
           
        }
         
      }
      
    }
    }
}