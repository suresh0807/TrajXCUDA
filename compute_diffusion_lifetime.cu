#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"




void compute_diffusion_lifetime()
{
  
    select_atoms(msd_for);
  
/*
  cout << "metal "<< metal_species<<" counts "<<count_metal<<endl;
  
  int *exch;
  int *dev_exch;
  exch= (int *) malloc (sizeof(int)*nElt1*(nstruct/skip));
  Aintnum = (int *) malloc (sizeof(int)*(nstruct/skip));
  Abulknum = (int *) malloc (sizeof(int)*(nstruct/skip));
// Initialize distance matrix and histogram matrix
  for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          exch[i+l*nElt1]=0;
        }     
        Abulknum[l]=0;
        Aintnum[l]=0;
      }

//Allocate memory in GPU device

    hipMalloc((void **)&dev_B,sizeof(float)*count_metal*(nstruct/skip)*3);
    hipMalloc((void **)&dev_A,sizeof(float)*nElt1*(nstruct/skip)*3);
    hipMalloc((void **)&dev_exch,sizeof(int)*nElt1*(nstruct/skip));
    hipMalloc((void **)&dev_lattice,sizeof(float)*6*(nstruct/skip));

//Copy data from host to device

    hipMemcpy(dev_B,METAL,sizeof(float)*count_metal*(nstruct/skip)*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_A,A,sizeof(float)*nElt1*(nstruct/skip)*3,hipMemcpyHostToDevice);
    

//dim3 struct to define elements of the execution configuration


    dim3 dimBlocke(10,10,10);
    dim3 dimGride((nElt1+dimBlocke.x-1)/dimBlocke.x,(count_metal+dimBlocke.y-1)/dimBlocke.y,((nstruct/skip)+dimBlocke.z-1)/dimBlocke.z);

//Cuda kernal execution for distance matrix with CUDA timing API commands

    //For surface metal atoms
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    hipMemcpy(dev_lattice,lattice,sizeof(int)*6*(nstruct/skip),hipMemcpyHostToDevice);
    if (cell_type == "orthorhombic") 
    {
    covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    else if (cell_type == "monoclinic")
    {
     covermatmono<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
      //covermat<<<dimGride,dimBlocke>>>(dev_A,dev_B,dev_exch,bondist,nElt1,count_metal,(nstruct/skip),dev_lattice);
    }
    hipMemcpy(exch,dev_exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyDeviceToHost);
    //For substrate atoms
     hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_lattice);
    //hipFree(dev_exch);
    for(int l=0; l<nstruct/skip; l++)
    {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1;}
        }     
      }
    hipMemcpy(dev_exch,exch,sizeof(int)*nElt1*(nstruct/skip),hipMemcpyHostToDevice);
    ofstream exchplot;
    exchplot.open("exchange.data");

 for(int l=0; l<(nstruct/skip); l++)
    {
      exchplot <<l<<" ";
      for(int j=0; j<nElt1; j++)
      {
      
          exchplot << exch[j+l*nElt1]<<" ";
      }
      exchplot <<endl;
    }
    exchplot.close();    

 //ofstream intnum;
 //intnum.open("intnum.data");
 
      for(int l=0; l<nstruct/skip; l++)
      {
      for(int i=0; i< nElt1; i++)
        {
          if (exch[i+l*nElt1]>0) {exch[i+l*nElt1]=1; Aintnum[l]++;}
          else if (exch[i+l*nElt1]==0) Abulknum[l]++;
        }
      //for(int i=0; i< nElt2; i++)
      //  {
      //    if (exch2[i+l*nElt2]>0) {exch2[i+l*nElt2]=1; Bintnum[l]++;}
      //    else if (exch2[i+l*nElt2]==0) Bbulknum[l]++;
      //  }
      //intnum << l<<" "<<Aintnum[l]<<endl;
      }
  //intnum.close();
  */
///////////////////////unwrap after exch matrix - sensible///////////////
//##########################################################################  

if( unwrap == "yes")
{

cout << "Unwrapping the trajectory begins"<<endl;
  float lr[3];

for(int i =0; i<(nstruct/skip); i++)
{
 for(int j =0 ; j<nElt1; j++)
 {
  for(int k =0; k<3;k++)
  {
   Aint[j*3+i*nElt1*3+k]=0.0;  
  }
  if(i==0)
  {
   for(int k =0; k<3;k++)
   {
    Aint[j*3+i*nElt1*3+k]=A[j*3+i*nElt1*3+k];
   }
  }
 }
}

if(cell_type == "orthorhombic")
 {
  for(int i=1; i<(nstruct/skip); i++)
  {//cout<<lnElt1<<endl;
  //cout<<"hi"<<endl;
      for(int j=0;j<nElt1;j++)
      {//cout<<Elt1<<" ";
          for(int k=0 ;k<3;k++)
          {
          lr[k] = A[j*3+i*nElt1*3+k] - A[j*3+(i-1)*nElt1*3+k];
          if(abs(lr[k]) > lattice[k+i*6]/2.0)
          {
               //cout<<"I am working"<<endl;
              if(lr[k] > 0)
              {
              lr[k] = abs(lr[k]) - lattice[k+i*6];
              Aint[j*3+i*nElt1*3+k] = Aint[j*3+(i-1)*nElt1*3+k] - abs(lr[k]); 
              }
              else
              {
              lr[k] = abs(lr[k]) - lattice[k+i*6];
              Aint[j*3+i*nElt1*3+k] = Aint[j*3+(i-1)*nElt1*3+k] + abs(lr[k]); 
              }     
          }
           else
           {
              Aint[j*3+i*nElt1*3+k] = Aint[j*3+(i-1)*nElt1*3+k] + lr[k]; 
           } //cout<<Aint[j*3+i*lnElt1*3+k]<<" ";
          }//cout<<endl;
      }
  }
 }
 else if(cell_type == "monoclinic")
 {
  for(int i=1; i<(nstruct/skip); i++)
  {
      for(int j=0;j<nElt1;j++)
      {        
        lr[0]=A[j*3+i*nElt1*3]-A[j*3+(i-1)*nElt1*3];
        lr[1]=A[j*3+i*nElt1*3+1]-A[j*3+(i-1)*nElt1*3+1];
        lr[2]=A[j*3+i*nElt1*3+2]-A[j*3+(i-1)*nElt1*3+2];
	
	if(abs(lr[1]) > lattice[1+i*6]/2.0)
        {
        if(lr[1] > 0)
        {
            lr[1] = abs(lr[1]) - lattice[1+i*6];
            Aint[j*3+i*nElt1*3+1] = Aint[j*3+(i-1)*nElt1*3+1] - abs(lr[1]);
            lr[0] = abs(lr[0]) - lattice[3+i*6];
            Aint[j*3+i*nElt1*3+0] = Aint[j*3+(i-1)*nElt1*3+0] - abs(lr[0]);
	   // Aint[j*3+i*lnElt1*3+0] = Aint[j*3+i*lnElt1*3+0] - lattice[3+i*6];
        }
        else
        {
            lr[1] = abs(lr[1]) - lattice[1+i*6];
            Aint[j*3+i*nElt1*3+1] = Aint[j*3+(i-1)*nElt1*3+1] + abs(lr[1]);
            lr[0] = abs(lr[0]) - lattice[3+i*6];
            Aint[j*3+i*nElt1*3+0] = Aint[j*3+(i-1)*nElt1*3+0] + abs(lr[0]);
	    //Aint[j*3+i*lnElt1*3+0] = Aint[j*3+i*lnElt1*3+0] + lattice[3+i*6];
        }
        }
        else
        {
        Aint[j*3+i*nElt1*3+1] = Aint[j*3+(i-1)*nElt1*3+1] + lr[1];
        }
	
	
        for(int k=0;k<3;k=k+2)
        {
        if(abs(lr[k]) > lattice[k+i*6]/2.0)
        {
            if(lr[k] > 0)
            {
                lr[k]= abs(lr[k]) - lattice[k+i*6];
                Aint[j*3+i*nElt1*3+k] = Aint[j*3+(i-1)*nElt1*3+k] - abs(lr[k]);
            }
            else
            {
                lr[k]= abs(lr[k]) - lattice[k+i*6];
                Aint[j*3+i*nElt1*3+k] = Aint[j*3+(i-1)*nElt1*3+k] + abs(lr[k]);
            }
        }
        else
        {
            Aint[j*3+i*nElt1*3+k] = Aint[j*3+(i-1)*nElt1*3+k] + lr[k];
        }
        }
      }
  }
 }


for(int i =0; i<(nstruct/skip); i++)
{
for(int j =0 ; j<nElt1; j++)
{
for(int k =0; k<3;k++)
{
A[j*3+i*nElt1*3+k]=Aint[j*3+i*nElt1*3+k];
}
}
}

if(unwrapout=="yes")
{

ofstream unwrapped;
unwrapped.open("input_unwrapped.xyz");

for(int i =0; i<(nstruct/skip); i++)
{
unwrapped<<nElt1<<endl;
unwrapped<<endl;
for(int j =0 ; j<nElt1; j++)
{unwrapped<<Elt1 <<" ";
for(int k =0; k<3;k++)
{
unwrapped<<A[j*3+i*nElt1*3+k]<<" ";
}
unwrapped<<endl;
}
}
unwrapped.close();
}

free(Aint);
}
//#########################################################################
/////////////////////////////unwrapper finished////////////////////////////
  
  
  
    float intnum=0.0,bulknum=0.0;
    for(int l=0; l<nstruct/skip; l++)
    {
      intnum+=Aintnum[l];
      bulknum+=Abulknum[l];
    }
    intnum/=(nstruct/skip);
    bulknum/=(nstruct/skip);
  
    cout << intnum << " "<< bulknum<<endl;
    int DOF;
    if(diffuse_direction == "xyz") {xsrt=0;xend=3;xski=1;DOF=6;}
    else if(diffuse_direction == "xy") {xsrt=0;xend=2;xski=1;DOF=4;}
    else if(diffuse_direction == "x") {xsrt=0;xend=1;xski=1;DOF=2;}
    else if(diffuse_direction == "y") {xsrt=1;xend=2;xski=1;DOF=2;}
    else if(diffuse_direction == "z") {xsrt=2;xend=3;xski=1;DOF=2;}
    else if(diffuse_direction == "xz") {xsrt=0;xend=3;xski=2;DOF=4;}
    else if(diffuse_direction == "yz") {xsrt=1;xend=3;xski=1;DOF=4;}
 

    total_time = (nstruct)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    num_bins =  int ((nstruct )/((diffuse_time*1000)/timestep));
    num_bins -= 1; 
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    cout <<num_bins <<" number of bins are chosen for the MSD analysis"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    origins = SD_store;
    cout <<"This will correlate for "<<SD_store<<" frames in the input trajectory "<<endl;
    cout <<origins/skips<<" restarts from each bin will be taken"<<endl;


    SD=(float *) malloc (sizeof(float)*nElt1*SD_store*origins/skips);
    for(int i=0;i<origins/skips;i++)
    {
      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SD[k+j*nElt1+i*nElt1*SD_store]=0.0;
	}
      }
    }
        SDsum=(float *) malloc (sizeof(float)*nElt1*SD_store*origins/skips);
    for(int i=0;i<origins/skips;i++)
    {
      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SDsum[k+j*nElt1+i*nElt1*SD_store]=0.0;
	}
      }
    }
///*

    
    
    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*nstruct*nElt1*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nstruct*nElt2*3);
    hipMalloc((void **)&dev_SD,sizeof(float)*nElt1*SD_store*origins/skips);
    hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B,sizeof(float)*(nstruct/skip)*nElt2*3,hipMemcpyHostToDevice);
    cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;


    
   // cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
   // cout << "Memory for local MSD storage: "<<nElt1*SD_store*origins/skips*4/float(1000000000)<< " Gbs" << endl;


    dim3 dimBlock(32,1,32);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,1,((origins/skips)+dimBlock.z-1)/dimBlock.z);

    //dim3 dimBlock(10,10,10);
    //dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(SD_store+dimBlock.y-1)/dimBlock.y,((origins/skips)+dimBlock.z-1)/dimBlock.z);    
    
  //    hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store*origins,hipMemcpyHostToDevice);    

int *numc, *dev_numc;
numc = (int *) malloc (sizeof(int)*SD_store*origins/skips);
hipMalloc((void **)&dev_numc,sizeof(int)*SD_store*origins/skips);
    
int *inout, *dev_inout;
inout = (int *) malloc (sizeof(int)*nElt1*origins/skips);
hipMalloc((void **)&dev_inout,sizeof(int)*nElt1*origins/skips);

    for(int i=0; i<num_bins;i++)
    //for(int i=0; i<num_bins;i++)
    {
             for(int i1=0;i1<origins/skips;i1++)
              {
               for(int j1=0;j1<SD_store;j1++)
                {
		  numc[j1+i1*SD_store]=0;
	         for(int k1=0; k1<nElt1; k1++)
	          {
                    SD[k1+j1*nElt1+i1*nElt1*SD_store]=0.0;
		    inout[k1+i1*nElt1]=1;
	          }
                }
              }
   
      	      int whichwater;
      if(msd_for=="int"||msd_for=="box"||msd_for=="bulk"){whichwater=1;}
      else if(msd_for=="all"){whichwater=2;}
hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyHostToDevice);
hipMemcpy(dev_numc,numc,sizeof(int)*SD_store*origins/skips,hipMemcpyHostToDevice);
hipMemcpy(dev_inout,inout, sizeof(int)*nElt1*(origins/skips),hipMemcpyHostToDevice);
int chase=1;
      for(int j=1; j<SD_store; j++)
      {
      //hipMemcpy(dev_inout,inout, sizeof(int)*nElt1*(origins/skips),hipMemcpyHostToDevice);
      SD_calc<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_SD,dev_exch,dev_numc,dev_inout,SD_store,nElt1,i,j,origins,skips,xsrt,xend,xski,whichwater,chase);
      //SD_calc<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_SD,dev_exch,dev_numc,dev_inout,SD_store,nElt1,i,j,origins,skips,xsrt,xend,xski,whichwater);
      //hipMemcpy(inout,dev_inout, sizeof(int)*nElt1*(origins/skips),hipMemcpyHostToDevice);
      }
hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyDeviceToHost);
hipMemcpy(numc,dev_numc,sizeof(int)*SD_store*origins/skips,hipMemcpyDeviceToHost);

    for(int i1=0;i1<origins/skips;i1++)
    {
     for(int j1=1;j1<SD_store;j1++)
     {
      for(int k1=0; k1<nElt1; k1++)
      {
       SD[k1+j1*nElt1+i1*nElt1*SD_store]/=float(numc[j1+i1*SD_store]);
      }
     }
    }
    
    for(int i1=0;i1<origins/skips;i1++)
    {
      for(int j1=0;j1<SD_store;j1++)
      {
	for(int k1=0; k1<nElt1; k1++)
	{
          SDsum[k1+j1*nElt1+i1*nElt1*SD_store]+=SD[k1+j1*nElt1+i1*nElt1*SD_store];
	}
      }
    }
     cout << (i+1)*diffuse_time << " pico seconds done"<<endl;  
 }
    
//      hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store*origins,hipMemcpyDeviceToHost);



    hipFree(dev_A);
    hipFree(dev_exch);
    hipFree(dev_B);
    hipFree(dev_SD);
    hipFree(dev_numc);
    hipFree(dev_inout);
     
    SDavgf=(float *) malloc (sizeof(float)*SD_store*nElt1);
    for(int i=0; i<SD_store; i++)
    {
      for(int j =0; j<nElt1;j++)
      {
        SDavgf[j+i*nElt1]=0.0;
      }
    }
    hipMalloc((void **)&dev_SDsum,sizeof(float)*nElt1*SD_store*origins/skips);
    hipMalloc((void **)&dev_SDavgf,sizeof(float)*SD_store*nElt1);
    hipMemcpy(dev_SDavgf,SDavgf,sizeof(float)*SD_store*nElt1,hipMemcpyHostToDevice);
    hipMemcpy(dev_SDsum,SDsum,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyHostToDevice);
    
    dim3 dimBlockf(32,32,1);
    dim3 dimGridf((nElt1+dimBlockf.x-1)/dimBlockf.x,(SD_store+dimBlockf.y-1)/dimBlockf.y,1);
    cout<<"start reduce "<< origins/skips <<" "<<nElt1<< " "<<SD_store<<endl;
    
    SDreducef<<<dimGridf,dimBlockf>>>(dev_SDsum,dev_SDavgf,nElt1,SD_store,origins,skips);
    
    hipMemcpy(SDavgf,dev_SDavgf,sizeof(float)*SD_store*nElt1,hipMemcpyDeviceToHost);
    hipFree(dev_SDsum);

    int fairy=0;
    SDavg=(float *) malloc (sizeof(float)*SD_store);
    for(int j =0; j<SD_store;j++)
    {
      SDavg[j]=0.0;
    }
    hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
    hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

    dim3 dimBlocks(1024,1,1);
    dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);

    SDreduce<<<dimGrids,dimBlocks>>>(dev_SDavgf,dev_SDavg,SD_store,nElt1,fairy);
    hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
    
    hipFree(dev_SDavgf);
    hipFree(dev_SDavg);
    
 /*
    if(msd_for == "int")
    {
     for(int i =1; i<SD_store;i++)
    {
      SDavg[i]=(SDavg[i]*nElt1)/ intnum;
    }
    }
    else if(msd_for == "bulk")
    {
     for(int i =1; i<SD_store;i++)
    {
      SDavg[i]=(SDavg[i]*nElt1)/ bulknum;
    }
    }      
   */ 
// printing the msd data to be visualized

    FILE *MSDplot=fopen("msd.data","wt");
      fprintf(MSDplot,"# Time (ps) MSD (A^(2)) \n");
    for(int i =1; i<SD_store;i++)
    {
      fprintf(MSDplot,"%f %f\n",(float(i)*timestep)/1000.0,SDavg[i]/num_bins);
    }
    fclose(MSDplot);
   
    FILE *DIFFplot=fopen("diffco.data","wt");
      fprintf(DIFFplot,"# Time (ps) D (10^-5 cm^2/s) \n");
    for(int i =1; i<SD_store;i++)
    {
      fprintf(DIFFplot,"%f %f\n",(float(i)*timestep)/1000.0,(SDavg[i]/(num_bins*DOF*i*timestep))*10000);
    }
    fclose(DIFFplot);


 
}
