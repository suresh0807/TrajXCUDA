#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void check_together(float a, float b, float c, float d, float e, float f, float g, float h, float i,float latx,float laty,float latz,float latxy,float latxz,float latyz)
{

float chk1,chk2,chk3,chk4,chk5,chk6;

chk1= (a-d);
chk2= (b-e);
chk3= (c-f);
chk4= (a-g);
chk5= (b-h);
chk6= (c-i);


if( fabs(chk1) > latx/2.0 || fabs(chk4) > latx/2.0 || fabs(chk2) > laty/2.0 || fabs(chk5) > laty/2.0 ||\
 fabs(chk3) > latz/2.0 || fabs(chk6) >latz/2.0))
{
if(cell_type == "orthorhombic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d=d+latx; else d=d-latx}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) e=e+laty; else e=e-laty}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f=f+latz; else f=f-latz}
if(fabs(chk4) > latx/2.0) { if (chk4 >0) g=g+latx; else g=g-latx}
if(fabs(chk5) > laty/2.0) { if (chk5 >0) h=h+laty; else h=h-laty}
if(fabs(chk6) > latz/2.0) { if (chk6 >0) i=i+latz; else i=i-latz}
}

else if(cell_type == "monoclinic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d=d+latx; else d=d-latx}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) {e=e+laty; d=d+latxy;} else {e=e-laty; d=d-latxy;}}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f=f+latz; else f=f-latz}
if(fabs(chk4) > latx/2.0) { if (chk4 >0) g=g+latx; else g=g-latx}
if(fabs(chk5) > laty/2.0) { if (chk5 >0) {h=h+laty; g=g+latxy;} else {h=h-laty; g=g-latxy;}}
if(fabs(chk6) > latz/2.0) { if (chk6 >0) i=i+latz; else i=i-latz}
}
}


}


void check_together(float a, float b, float c, float d, float e, float f, float latx,float laty,float latz,float latxy,float latxz,float latyz)
{

float chk1,chk2,chk3;

chk1= (a-d);
chk2= (b-e);
chk3= (c-f);


if( fabs(chk1) > latx/2.0 || fabs(chk2) > laty/2.0 ||  fabs(chk3) > latz/2.0)
{
if(cell_type == "orthorhombic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d=d+latx; else d=d-latx}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) e=e+laty; else e=e-laty}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f=f+latz; else f=f-latz}
}

else if(cell_type == "monoclinic")
{
if(fabs(chk1) > latx/2.0) { if (chk1 >0) d=d+latx; else d=d-latx}
if(fabs(chk2) > laty/2.0) { if (chk2 >0) {e=e+laty; d=d+latxy;} else {e=e-laty; d=d-latxy;}}
if(fabs(chk3) > latz/2.0) { if (chk3 >0) f=f+latz; else f=f-latz}
}
}


}