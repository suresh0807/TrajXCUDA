#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

void density_side_avg_pos()
{


     for(int i=0; i< nstruct/skip;i=i++)
     {
      for(int j=0; j< nElt1; j++)
      {
       if(i>0)
       {
        if(fabs(A[j*3+i*nElt1*3]-A[j*3]) > xvec/2)
        {
         A[j*3+i*nElt1*3]=-A[j*3+i*nElt1*3]+xvec;
        }
        if(fabs(A[j*3+i*nElt1*3+1]-A[j*3+1]) > yvec/2)
        {
         A[j*3+i*nElt1*3+1]=-A[j*3+i*nElt1*3+1]+yvec;
        }
        if(fabs(A[j*3+i*nElt1*3+2]-A[j*3+2]) > zvec/2)
        {
         A[j*3+i*nElt1*3+2]=-A[j*3+i*nElt1*3+2]+zvec;
        }
       }
       Aavg[j*3]+=A[j*3+i*nElt1*3];
       Aavg[j*3+1]+=A[j*3+i*nElt1*3+1];
       Aavg[j*3+2]+=A[j*3+i*nElt1*3+2];
      }
     }
      for(int j=0; j< nElt1; j++)
      {     
       Aavg[j*3]=Aavg[j*3]/(nstruct/skip);
       Aavg[j*3+1]=Aavg[j*3+1]/(nstruct/skip);
       Aavg[j*3+2]=Aavg[j*3+2]/(nstruct/skip); 
     //  printf("%f %f %f \n",Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);
      }
     nstruct=1;
    
    FILE *testout=fopen("teter.dat","wt");
      for(int j=0; j< nElt1; j++)
      {
	if(order=="YZX") {
	fprintf(testout,"%s \n",nElt1);
	fprintf(testout,"%d \n", zsplit);
       fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,Aavg[j*3+1]/yvec,Aavg[j*3]/xvec);       
	}
	if(order=="XYZ") {
	fprintf(testout,"%s \n",nElt1);
	fprintf(testout,"%d \n", ysplit);  
       fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,Aavg[j*3]/xvec,Aavg[j*3+2]/zvec);
 //printf("%d %f %d %f %f %f\n",symbol,size,color,Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]);      
	}
	if(order=="ZXY") {
  	fprintf(testout,"%s \n",nElt1);
	fprintf(testout,"%d \n", xsplit);
       fprintf(testout,"%d %f %d %f %f \n",symbol,size,color,Aavg[j*3+2]/zvec,Aavg[j*3+1]/yvec);
      // printf("%d %f %d %f %f %f\n",symbol,size,color,Aavg[j*3],Aavg[j*3+1],Aavg[j*3+2]); 
	}
      }    
    fclose(testout);

}