#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"




void compute_diffusion()
{
    if(plot=="int")        { nElt1 = nA_int;}
    else if(plot=="bulk")  { nElt1 = nA_bulk;}


    if(diffuse_direction == "xyz") {xsrt=0;xend=3;xski=1;}
    else if(diffuse_direction == "xy") {xsrt=0;xend=2;xski=1;}
    else if(diffuse_direction == "x") {xsrt=0;xend=1;xski=1;}
    else if(diffuse_direction == "y") {xsrt=1;xend=2;xski=1;}
    else if(diffuse_direction == "z") {xsrt=2;xend=3;xski=1;}
    else if(diffuse_direction == "xz") {xsrt=0;xend=3;xski=2;}
    else if(diffuse_direction == "yz") {xsrt=1;xend=3;xski=1;}
 


    total_time = (nstruct)*timestep;
    cout << "Each frame resolves to "<<timestep<< " fs evolution"<<endl;
    cout << "Total simulation time : " << total_time/1000000 <<" ns"<< endl;
    num_bins =  int ((nstruct )/((diffuse_time*1000)/timestep));
    num_bins -= 1; 
    cout << "The analysis will be done for every "<< diffuse_time <<" ps"<<endl;
    cout <<num_bins <<" number of bins are chosen for the MSD analysis"<<endl;
    SD_store = int((diffuse_time*1000)/timestep);
    origins = SD_store;
    cout <<"This will correlate for "<<SD_store<<" frames in the input trajectory "<<endl;
    cout <<origins/skips<<" restarts from each bin will be taken"<<endl;


    SD=(float *) malloc (sizeof(float)*nElt1*SD_store*origins/skips);
    for(int i=0;i<origins/skips;i++)
    {
      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SD[k+j*nElt1+i*nElt1*SD_store]=0.0;
	}
      }
    }
        SDsum=(float *) malloc (sizeof(float)*nElt1*SD_store*origins/skips);
    for(int i=0;i<origins/skips;i++)
    {
      for(int j=0;j<SD_store;j++)
      {
	for(int k=0; k<nElt1; k++)
	{
          SDsum[k+j*nElt1+i*nElt1*SD_store]=0.0;
	}
      }
    }
///*

    
    
    printf("Start of cuda calculation\n");
    hipMalloc((void **)&dev_A,sizeof(float)*nstruct*nElt1*3);
    hipMalloc((void **)&dev_B,sizeof(float)*nstruct*nElt2*3);
    hipMalloc((void **)&dev_SD,sizeof(float)*nElt1*SD_store*origins/skips);
    if(plot=="int")
    {
      hipMemcpy(dev_A,A_int,sizeof(float)*nstruct*nA_int*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B_int,sizeof(float)*nstruct*nB_int*3,hipMemcpyHostToDevice);
        cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;
    }
    else if(plot=="bulk")
    {
      hipMemcpy(dev_A,A_bulk,sizeof(float)*nstruct*nA_bulk*3,hipMemcpyHostToDevice);
    hipMemcpy(dev_B,B_bulk,sizeof(float)*nstruct*nB_bulk*3,hipMemcpyHostToDevice);
        cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;
    }
    else if(plot=="all")
    {
//cout<<"hi "<< (nstruct/skip)*nElt1*3<<endl;
      hipMemcpy(dev_A,A,sizeof(float)*(nstruct/skip)*nElt1*3,hipMemcpyHostToDevice);
//cout<<"hi"<<endl;
    hipMemcpy(dev_B,B,sizeof(float)*(nstruct/skip)*nElt2*3,hipMemcpyHostToDevice);
        cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
    cout << "Memory for local MSD storage: "<<((nElt1*nstruct/skips)+(nElt1*SD_store))*4/float(1000000000)<< " Gbs" << endl;
    }

    
   // cout << "Memory required to store coordinate information: "<<nstruct*nElt1*3*2*4/float(1000000000)<< " Gbs " << endl;
   // cout << "Memory for local MSD storage: "<<nElt1*SD_store*origins/skips*4/float(1000000000)<< " Gbs" << endl;


    dim3 dimBlock(32,1,32);
    dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,1,((origins/skips)+dimBlock.z-1)/dimBlock.z);

    //dim3 dimBlock(10,10,10);
    //dim3 dimGrid((nElt1+dimBlock.x-1)/dimBlock.x,(SD_store+dimBlock.y-1)/dimBlock.y,((origins/skips)+dimBlock.z-1)/dimBlock.z);    
    
  //    hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store*origins,hipMemcpyHostToDevice);    

    for(int i=0; i<num_bins;i++)
    //for(int i=0; i<num_bins;i++)
    {
             for(int i1=0;i1<origins/skips;i1++)
              {
               for(int j1=0;j1<SD_store;j1++)
                {
	         for(int k1=0; k1<nElt1; k1++)
	          {
                    SD[k1+j1*nElt1+i1*nElt1*SD_store]=0.0;
	          }
                }
              }
      
hipMemcpy(dev_SD,SD,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyHostToDevice);
      for(int j=1; j<SD_store; j++)
      {
	//int j=1;
      SD_calc<<<dimGrid,dimBlock>>>(dev_A,dev_B,dev_SD,SD_store,nElt1,i,j,origins,skips,xsrt,xend,xski);
      }
hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyDeviceToHost);
    for(int i1=0;i1<origins/skips;i1++)
    {
      for(int j1=0;j1<SD_store;j1++)
      {
	for(int k1=0; k1<nElt1; k1++)
	{
          SDsum[k1+j1*nElt1+i1*nElt1*SD_store]+=SD[k1+j1*nElt1+i1*nElt1*SD_store];
	}
      }
    }
     cout << (i+1)*diffuse_time << " pico seconds done"<<endl;  
    }
    
//      hipMemcpy(SD,dev_SD,sizeof(float)*nElt1*SD_store*origins,hipMemcpyDeviceToHost);



    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_SD);
     
    SDavgf=(float *) malloc (sizeof(float)*SD_store*nElt1);
    for(int i=0; i<SD_store; i++)
    {
      for(int j =0; j<nElt1;j++)
      {
        SDavgf[j+i*nElt1]=0.0;
      }
    }
    hipMalloc((void **)&dev_SDsum,sizeof(float)*nElt1*SD_store*origins/skips);
    hipMalloc((void **)&dev_SDavgf,sizeof(float)*SD_store*nElt1);
    hipMemcpy(dev_SDavgf,SDavgf,sizeof(float)*SD_store*nElt1,hipMemcpyHostToDevice);
    hipMemcpy(dev_SDsum,SDsum,sizeof(float)*nElt1*SD_store*origins/skips,hipMemcpyHostToDevice);
    
    dim3 dimBlockf(32,32,1);
    dim3 dimGridf((nElt1+dimBlockf.x-1)/dimBlockf.x,(SD_store+dimBlockf.y-1)/dimBlockf.y,1);
    cout<<"start reduce "<< origins/skips <<" "<<nElt1<< " "<<SD_store<<endl;
    
    SDreducef<<<dimGridf,dimBlockf>>>(dev_SDsum,dev_SDavgf,nElt1,SD_store,origins,skips);
    
    hipMemcpy(SDavgf,dev_SDavgf,sizeof(float)*SD_store*nElt1,hipMemcpyDeviceToHost);
    hipFree(dev_SDsum);

    
    SDavg=(float *) malloc (sizeof(float)*SD_store);
    for(int j =0; j<SD_store;j++)
    {
      SDavg[j]=0.0;
    }
    hipMalloc((void **)&dev_SDavg,sizeof(float)*SD_store);
    hipMemcpy(dev_SDavg,SDavg,sizeof(float)*SD_store,hipMemcpyHostToDevice);

    dim3 dimBlocks(1024,1,1);
    dim3 dimGrids((SD_store+dimBlocks.x-1)/dimBlocks.x,1,1);
    SDreduce<<<dimGrids,dimBlocks>>>(dev_SDavgf,dev_SDavg,SD_store,nElt1);

    hipMemcpy(SDavg,dev_SDavg,sizeof(float)*SD_store,hipMemcpyDeviceToHost);
    
    hipFree(dev_SDavgf);
    hipFree(dev_SDavg);
    
// printing the rdf data to be visualized

    FILE *MSDplot=fopen("msd.data","wt");
      fprintf(MSDplot,"# Time (ps) MSD (A^(2)) \n");
    for(int i =1; i<SD_store;i++)
    {
      fprintf(MSDplot,"%f %f\n",(float(i)*timestep)/1000.0,SDavg[i]/num_bins);
    }
    fclose(MSDplot);
   
    FILE *DIFFplot=fopen("diffco.data","wt");
      fprintf(DIFFplot,"# Time (ps) D (A^(2)/fs) \n");
    for(int i =1; i<SD_store;i++)
    {
      fprintf(DIFFplot,"%f %f\n",(float(i)*timestep)/1000.0,SDavg[i]/(num_bins*6*i*timestep));
    }
    fclose(DIFFplot);


 
}
