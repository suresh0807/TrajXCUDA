#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void SD_calc(float *A,float *B,float *C,int SD_store,int nElt1, int i, int j,int origins,int skips, int xsrt,int xend, int xski)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  //int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  widid < origins/skips)
  {

    z=0.0;
    for(int k=xsrt; k< xend ; k=k+xski)
    //for(int k=0; k< 3 ; k=k+1)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(j*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
//     printf("%f \n",C[rowid+(j*nElt1)+(widid*nElt1*SD_store)]);

    C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ;


  }
  
  
 /* int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  rowid <SD_store && widid < origins/skips)
  {

    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(colid*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
//     printf("%f \n",C[rowid+(j*nElt1)+(widid*nElt1*SD_store)]);

    C[rowid+(colid*nElt1)+widid*nElt1*SD_store] += z ;


  }*/
  
}


__global__ void SD_calc(float *A,float *B,float *C,int *D, int *E, int *F, int SD_store,int nElt1, int i, int j,int origins,int skips, int xsrt,int xend, int xski,int whichwater)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  //int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  widid < origins/skips)
  {
    if(D[rowid+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater && D[rowid+(j*nElt1)+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater && F[rowid+widid*nElt1] == 1)
    {
    z=0.0;
    for(int k=xsrt; k< xend ; k=k+xski)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(j*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
    C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ;
    atomicAdd(&E[j+widid*SD_store],1);
    } 
    
    else if(whichwater==2)
    {
    z=0.0;
    for(int k=xsrt; k< xend ; k=k+xski)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(j*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
     C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ; 
     atomicAdd(&E[j+widid*SD_store],1); // counting nelt1 in different time origins
    }
    
    else
    {
     F[rowid+widid*nElt1] = 0; 
    }
  }
}

 __global__ void SD_calc(float *A,float *B,float *C,int *D, int *E, int *F, int SD_store,int nElt1, int i, int j,int origins,int skips, int xsrt,int xend, int xski,int whichwater,int chase)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  //int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  widid < origins/skips)
  {
    if(D[rowid+(widid*skips*nElt1)+(i*nElt1*SD_store)] ==whichwater && F[rowid+widid*nElt1] == 1)
    {
    z=0.0;
    for(int k=xsrt; k< xend ; k=k+xski)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(j*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
    C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ;
    atomicAdd(&E[j+widid*SD_store],1);
    } 
    
    else if(whichwater==2)
    {
    z=0.0;
    for(int k=xsrt; k< xend ; k=k+xski)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(j*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
     C[rowid+(j*nElt1)+widid*nElt1*SD_store] += z ; 
     atomicAdd(&E[j+widid*SD_store],1); // counting nelt1 in different time origins
    }
    
    else
    {
     F[rowid+widid*nElt1] = 0; 
    }
  } 
 /* int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int colid=threadIdx.y + blockIdx.y * blockDim.y;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  rowid <SD_store && widid < origins/skips)
  {

    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(A[rowid*3+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]-B[rowid*3+(colid*nElt1*3)+(widid*skips*nElt1*3)+(i*nElt1*3*SD_store)+k]);
      z+=(chk * chk);
    }
//     printf("%f \n",C[rowid+(j*nElt1)+(widid*nElt1*SD_store)]);

    C[rowid+(colid*nElt1)+widid*nElt1*SD_store] += z ;


  }*/
  
}










__global__ void SD_calc(float *A,float *B,float *C,int SD_store,int nElt1, int i,int nstruct,int skips, int xsrt,int xend, int xski)
{
  int rowid=threadIdx.x + blockIdx.x * blockDim.x;
  int widid=threadIdx.z + blockIdx.z * blockDim.z;
  float chk,z;
  if(rowid < nElt1  &&  widid < nstruct)
  {
    z=0.0;
    for(int k=xsrt; k< xend ; k=k+xski)
    //for(int k=0; k< 3 ; k=k+1)
    {
      chk=fabs(A[rowid*3+(widid*nElt1*3)+k]-B[rowid*3+(widid*nElt1*3)+(i*nElt1*3)+k]);
      z+=(chk * chk);
    }
//     printf("%f \n",C[rowid+(j*nElt1)+(widid*nElt1*SD_store)]);

    C[rowid+widid*nElt1] = z ;


  }
}


