#include "hip/hip_runtime.h"
//################# TrajXCUDA #########################
//######### Suresh Kondati Natarajan ##################
//##### Lehrstuhl fuer theoretische chemie ############
//######## Ruhr Universitaet Bochum ###################




#include "cudatools.cuh"

__global__ void distmat(float *x, float*y, float *a, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec, float rdf_max_rad)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    index = z * bin / maxval;
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
  }
}


__global__ void distmat(float *x, float*y, float *a, int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float xvec, float yvec, float zvec, float yxvec, float rdf_max_rad, int whichwater)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater && exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
  }
}

// Fork bulk region --- exch2 is not needed.
__global__ void distmat(float *x, float*y, float *a, int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float *latti, float rdf_max_rad, int whichwater)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater)// && exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
  }
}

__global__ void distmat(float *x, float*y, float *a, int *exch1, int *exch2, float bin_rad, int bin, int nElt1, int nElt2, int nstruct, float *latti, float rdf_max_rad, int whichwater, float mid, int dirn)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  float maxval;
  int index;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct)
  {
    //maxval = xvec/2;
    maxval = rdf_max_rad;
    z=0.0;
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];

     for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*3+widid*nElt1*3+k]-y[colid*3+widid*nElt2*3+k]);
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      z+=(chk * chk);
    }
    z=sqrt(z);
    index = z * bin / maxval;
    if(exch1[rowid+nElt1*widid]==whichwater && x[rowid*3+widid*nElt1*3+dirn] <= mid  && y[colid*3+widid*nElt2*3+dirn] >= x[rowid*3+widid*nElt1*3+dirn] )//&& y[colid*3+widid*nElt2*3+2] <= mid)// && exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if(exch1[rowid+nElt1*widid]==whichwater && x[rowid*3+widid*nElt1*3+dirn] >= mid  && y[colid*3+widid*nElt2*3+dirn] <= x[rowid*3+widid*nElt1*3+dirn])// && y[colid*3+widid*nElt2*3+2] >=mid && exch2[colid+nElt2*widid]==whichwater)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    else if (whichwater ==2)
    {
    for(int loop=index;loop<bin;loop++)
    {
      if(loop == index) {atomicAdd(&a[index+rowid*bin+widid*nElt1*bin],1.0);break;}
    }
    }
    
  }
}

__global__ void distmat(float *x, float*y, float *a, int nElt1, int nElt2, int nstruct, float *latti, float cutoff)
{
  int rowid=threadIdx.x + blockIdx.x*blockDim.x;
  int colid=threadIdx.y + blockIdx.y*blockDim.y;
  int widid=threadIdx.z + blockIdx.z*blockDim.z;
  float chk,xvec,yvec,zvec;
  float z;
  float index=0.0;
  if(rowid < nElt1 && colid < nElt2 && widid < nstruct && rowid != colid)
  {
    xvec=latti[widid*6];
    yvec=latti[1+widid*6];
    zvec=latti[2+widid*6];
    z=0.0;
    for(int k=0; k< 3 ; k++)
    {
      chk=fabs(x[rowid*4+widid*nElt1*4+k]-y[colid*4+widid*nElt2*4+k]);
      //if(rowid==1){printf("%f %f",x[rowid*4+widid*nElt1*4+k],y[colid*4+widid*nElt2*4+k]);}
      if(k==0) {if(chk > xvec/2) {chk = chk - xvec;}}
      if(k==1) {if(chk > yvec/2) {chk = chk - yvec;}}
      if(k==2) {if(chk > zvec/2) {chk = chk - zvec;}}
      //if(rowid==1){printf("%f %f",z,index);}
      z+=(chk * chk);
    }
    z=sqrt(z);
     if(z >0 && z < cutoff){index = (x[rowid*4+widid*nElt1*4+3]*y[colid*4+widid*nElt2*4+3]) / (z*1.88973);}
    if(rowid==1){printf("%f %f ",z,index);}
    a[colid+rowid*nElt2+widid*nElt1*nElt2]=index;
    
  }
}